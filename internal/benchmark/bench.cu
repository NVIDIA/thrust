#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/pair.h>
#include <thrust/sort.h>
#include <thrust/reduce.h>
#include <thrust/scan.h>
#include <thrust/detail/config.h>

#if THRUST_CPP_DIALECT >= 2011
#include <thrust/random.h>
#include <thrust/shuffle.h>

#include <random>
#endif

#include <algorithm>
#include <numeric>

#include <map>
#include <string>
#include <exception>

#include <iostream>

#include <cassert>
#include <cstdlib>    // For `atoi`.
#include <climits>    // For CHAR_BIT.
#include <cmath>      // For `sqrt` and `abs`.

#include <stdint.h>   // For `intN_t`.

#include "random.h"
#include "timer.h"

#if defined(HAVE_TBB)
  #include "tbb_algos.h"
#endif

#if THRUST_DEVICE_SYSTEM == THRUST_DEVICE_SYSTEM_CUDA
  #include <thrust/system_error.h>      // For `thrust::system_error`
  #include <thrust/system/cuda/error.h> // For `thrust::cuda_category`
#endif

// We don't use THRUST_PP_STRINGIZE and THRUST_PP_CAT because they are new, and
// we want this benchmark to be backwards-compatible to older versions of Thrust.
#define PP_STRINGIZE_(expr) #expr
#define PP_STRINGIZE(expr)  PP_STRINGIZE_(expr)

#define PP_CAT(a, b) a ## b

// We don't use THRUST_NOEXCEPT because it's new, and we want this benchmark to
// be backwards-compatible to older versions of Thrust.
#if THRUST_CPP_DIALECT >= 2011
  #define NOEXCEPT noexcept
#else
  #define NOEXCEPT throw()
#endif

///////////////////////////////////////////////////////////////////////////////

template <typename T>
struct squared_difference
{
private:
  T const average;

public:
  __host__ __device__
  squared_difference(squared_difference const& rhs) : average(rhs.average) {}

  __host__ __device__
  squared_difference(T average_) : average(average_) {}

  __host__ __device__
  T operator()(T x) const
  {
    return (x - average) * (x - average);
  }
};

template <typename T>
struct value_and_count
{
  T           value;
  uint64_t count;

  __host__ __device__
  value_and_count(value_and_count const& other)
    : value(other.value), count(other.count) {}

  __host__ __device__
  value_and_count(T const& value_)
    : value(value_), count(1) {}

  __host__ __device__
  value_and_count(T const& value_, uint64_t count_)
    : value(value_), count(count_) {}

  __host__ __device__
  value_and_count& operator=(value_and_count const& other)
  {
    value = other.value;
    count = other.count;
    return *this;
  }

  __host__ __device__
  value_and_count& operator=(T const& value_)
  {
    value = value_;
    count = 1;
    return *this;
  }
};

template <typename T, typename ReduceOp>
struct counting_op
{
private:
  ReduceOp reduce;

public:
  __host__ __device__
  counting_op() : reduce() {}

  __host__ __device__
  counting_op(counting_op const& other) : reduce(other.reduce) {}

  __host__ __device__
  counting_op(ReduceOp const& reduce_) : reduce(reduce_) {}

  __host__ __device__
  value_and_count<T> operator()(
      value_and_count<T> const& x
    , T const&                  y
    ) const
  {
    return value_and_count<T>(reduce(x.value, y), x.count + 1);
  }

  __host__ __device__
  value_and_count<T> operator()(
      value_and_count<T> const& x
    , value_and_count<T> const& y
    ) const
  {
    return value_and_count<T>(reduce(x.value, y.value), x.count + y.count);
  }
};

template <typename InputIt, typename T>
T arithmetic_mean(InputIt first, InputIt last, T init)
{
  value_and_count<T> init_vc(init, 0);

  counting_op<T, thrust::plus<T> > reduce_vc;

  value_and_count<T> vc
    = thrust::reduce(first, last, init_vc, reduce_vc);

  return vc.value / vc.count;
}

template <typename InputIt>
typename thrust::iterator_traits<InputIt>::value_type
arithmetic_mean(InputIt first, InputIt last)
{
  typedef typename thrust::iterator_traits<InputIt>::value_type T;
  return arithmetic_mean(first, last, T());
}

template <typename InputIt, typename T>
T sample_standard_deviation(InputIt first, InputIt last, T average)
{
  value_and_count<T> init_vc(T(), 0);

  counting_op<T, thrust::plus<T> > reduce_vc;

  squared_difference<T> transform(average);

  value_and_count<T> vc
    = thrust::transform_reduce(first, last, transform, init_vc, reduce_vc);

  return std::sqrt(vc.value / T(vc.count - 1));
}

///////////////////////////////////////////////////////////////////////////////

// Formulas for propagation of uncertainty from:
//
//   https://en.wikipedia.org/wiki/Propagation_of_uncertainty#Example_formulas
//
// Even though it's Wikipedia, I trust it as I helped write that table.
//
// XXX Replace with a proper reference.

// Compute the propagated uncertainty from the multiplication of two uncertain
// values, `A +/- A_unc` and `B +/- B_unc`. Given `f = AB` or `f = A/B`, where
// `A != 0` and `B != 0`, the uncertainty in `f` is approximately:
//
//   f_unc = abs(f) * sqrt((A_unc / A) ^ 2 + (B_unc / B) ^ 2)
//
template <typename T>
__host__ __device__
T uncertainty_multiplicative(
    T const& f
  , T const& A, T const& A_unc
  , T const& B, T const& B_unc
    )
{
  return std::abs(f)
       * std::sqrt((A_unc / A) * (A_unc / A) + (B_unc / B) * (B_unc / B));
}

// Compute the propagated uncertainty from addition of two uncertain values,
// `A +/- A_unc` and `B +/- B_unc`. Given `f = cA + dB` (where `c` and `d` are
// certain constants), the uncertainty in `f` is approximately:
//
//   f_unc = sqrt(c ^ 2 * A_unc ^ 2 + d ^ 2 * B_unc ^ 2)
//
template <typename T>
__host__ __device__
T uncertainty_additive(
    T const& c, T const& A_unc
  , T const& d, T const& B_unc
    )
{
  return std::sqrt((c * c * A_unc * A_unc) + (d * d * B_unc * B_unc));
}

///////////////////////////////////////////////////////////////////////////////

// Return the significant digit of `x`. The result is the number of digits
// after the decimal place to round to (negative numbers indicate rounding
// before the decimal place)
template <typename T>
int find_significant_digit(T x)
{
  if (x == T(0)) return T(0);
  return -int(std::floor(std::log10(std::abs(x))));
}

// Round `x` to `ndigits` after the decimal place (Python-style).
template <typename T, typename N>
T round_to_precision(T x, N ndigits)
{
  double m = (x < 0.0) ? -1.0 : 1.0;
  double pwr = std::pow(T(10.0), ndigits);
  return (std::floor(x * m * pwr + 0.5) / pwr) * m;
}

///////////////////////////////////////////////////////////////////////////////

void print_experiment_header()
{ // {{{
  std::cout << "Thrust Version"
    << ","  << "Algorithm"
    << ","  << "Element Type"
    << ","  << "Element Size"
    << ","  << "Elements per Trial"
    << ","  << "Total Input Size"
    << ","  << "STL Trials"
    << ","  << "STL Average Walltime"
    << ","  << "STL Walltime Uncertainty"
    << ","  << "STL Average Throughput"
    << ","  << "STL Throughput Uncertainty"
    << ","  << "Thrust Trials"
    << ","  << "Thrust Average Walltime"
    << ","  << "Thrust Walltime Uncertainty"
    << ","  << "Thrust Average Throughput"
    << ","  << "Thrust Throughput Uncertainty"
    #if defined(HAVE_TBB)
    << ","  << "TBB Trials"
    << ","  << "TBB Average Walltime"
    << ","  << "TBB Walltime Uncertainty"
    << ","  << "TBB Average Throughput"
    << ","  << "TBB Throughput Uncertainty"
    #endif
    << std::endl;

  std::cout << ""                // Thrust Version.
    << ","  << ""                // Algorithm.
    << ","  << ""                // Element Type.
    << ","  << "bits/element"    // Element Size.
    << ","  << "elements"        // Elements per Trial.
    << ","  << "MiBs"            // Total Input Size.
    << ","  << "trials"          // STL Trials.
    << ","  << "secs"            // STL Average Walltime.
    << ","  << "secs"            // STL Walltime Uncertainty.
    << ","  << "elements/sec"    // STL Average Throughput.
    << ","  << "elements/sec"    // STL Throughput Uncertainty.
    << ","  << "trials"          // Thrust Trials.
    << ","  << "secs"            // Thrust Average Walltime.
    << ","  << "secs"            // Thrust Walltime Uncertainty.
    << ","  << "elements/sec"    // Thrust Average Throughput.
    << ","  << "elements/sec"    // Thrust Throughput Uncertainty.
    #if defined(HAVE_TBB)
    << ","  << "trials"          // TBB Trials.
    << ","  << "secs"            // TBB Average Walltime.
    << ","  << "secs"            // TBB Walltime Uncertainty.
    << ","  << "elements/sec"    // TBB Average Throughput.
    << ","  << "elements/sec"    // TBB Throughput Uncertainty.
    #endif
    << std::endl;
} // }}}

///////////////////////////////////////////////////////////////////////////////

struct experiment_results
{
  double const average_time; // Arithmetic mean of trial times in seconds.
  double const stdev_time;   // Sample standard deviation of trial times.

  experiment_results(double average_time_, double stdev_time_)
    : average_time(average_time_), stdev_time(stdev_time_) {}
};

///////////////////////////////////////////////////////////////////////////////

template <
    template <typename> class Test
  , typename                  ElementMetaType // Has an embedded typedef `type,
                                              // and a static method `name` that
                                              // returns a char const*.
  , uint64_t                  Elements
  , uint64_t                  BaselineTrials
  , uint64_t                  RegularTrials
>
struct experiment_driver
{
  typedef typename ElementMetaType::type element_type;

  static char const* const test_name;
  static char const* const element_type_name; // Element type name as a string.

  static uint64_t const elements;             // # of elements per trial.
  static uint64_t const element_size;         // Size of each element in bits.
  static double   const input_size;           // `elements` * `element_size` in MiB.
  static uint64_t const baseline_trials;      // # of baseline trials per experiment.
  static uint64_t const regular_trials;       // # of regular trials per experiment.

  static void run_experiment()
  { // {{{
    experiment_results stl    = std_experiment();
    experiment_results thrust = thrust_experiment();
    #if defined(HAVE_TBB)
    experiment_results tbb    = tbb_experiment();
    #endif

    double stl_average_walltime    = stl.average_time;
    double thrust_average_walltime = thrust.average_time;
    #if defined(HAVE_TBB)
    double tbb_average_walltime    = tbb.average_time;
    #endif

    double stl_average_throughput    = elements / stl.average_time;
    double thrust_average_throughput = elements / thrust.average_time;
    #if defined(HAVE_TBB)
    double tbb_average_throughput    = elements / tbb.average_time;
    #endif

    double stl_walltime_uncertainty    = stl.stdev_time;
    double thrust_walltime_uncertainty = thrust.stdev_time;
    #if defined(HAVE_TBB)
    double tbb_walltime_uncertainty    = tbb.stdev_time;
    #endif

    double stl_throughput_uncertainty    = uncertainty_multiplicative(
        stl_average_throughput
      , double(elements), 0.0
      , stl_average_walltime, stl_walltime_uncertainty
    );
    double thrust_throughput_uncertainty = uncertainty_multiplicative(
        thrust_average_throughput
      , double(elements), 0.0
      , thrust_average_walltime, thrust_walltime_uncertainty
    );

    #if defined(HAVE_TBB)
    double tbb_throughput_uncertainty    = uncertainty_multiplicative(
        tbb_average_throughput
      , double(elements), 0.0
      , tbb_average_walltime, tbb_walltime_uncertainty
    );
    #endif

    // Round the average walltime and walltime uncertainty to the
    // significant figure of the walltime uncertainty.
    int stl_walltime_precision = std::max(
        find_significant_digit(stl.average_time)
      , find_significant_digit(stl.stdev_time)
    );
    int thrust_walltime_precision = std::max(
        find_significant_digit(thrust.average_time)
      , find_significant_digit(thrust.stdev_time)
    );
    #if defined(HAVE_TBB)
    int tbb_walltime_precision = std::max(
        find_significant_digit(tbb.average_time)
      , find_significant_digit(tbb.stdev_time)
    );
    #endif

    stl_average_walltime = round_to_precision(
        stl_average_walltime, stl_walltime_precision
    );
    thrust_average_walltime = round_to_precision(
        thrust_average_walltime, thrust_walltime_precision
    );
    #if defined(HAVE_TBB)
    tbb_average_walltime = round_to_precision(
        tbb_average_walltime, tbb_walltime_precision
    );
    #endif

    stl_walltime_uncertainty = round_to_precision(
        stl_walltime_uncertainty, stl_walltime_precision
    );
    thrust_walltime_uncertainty = round_to_precision(
        thrust_walltime_uncertainty, thrust_walltime_precision
    );
    #if defined(HAVE_TBB)
    tbb_walltime_uncertainty = round_to_precision(
        tbb_walltime_uncertainty, tbb_walltime_precision
    );
    #endif

    // Round the average throughput and throughput uncertainty to the
    // significant figure of the throughput uncertainty.
    int stl_throughput_precision = std::max(
        find_significant_digit(stl_average_throughput)
      , find_significant_digit(stl_throughput_uncertainty)
    );
    int thrust_throughput_precision = std::max(
        find_significant_digit(thrust_average_throughput)
      , find_significant_digit(thrust_throughput_uncertainty)
    );
    #if defined(HAVE_TBB)
    int tbb_throughput_precision = std::max(
        find_significant_digit(tbb_average_throughput)
      , find_significant_digit(tbb_throughput_uncertainty)
    );
    #endif

    stl_average_throughput = round_to_precision(
        stl_average_throughput, stl_throughput_precision
    );
    thrust_average_throughput = round_to_precision(
        thrust_average_throughput, thrust_throughput_precision
    );
    #if defined(HAVE_TBB)
    tbb_average_throughput = round_to_precision(
        tbb_average_throughput, tbb_throughput_precision
    );
    #endif

    stl_throughput_uncertainty = round_to_precision(
        stl_throughput_uncertainty, stl_throughput_precision
    );
    thrust_throughput_uncertainty = round_to_precision(
        thrust_throughput_uncertainty, thrust_throughput_precision
    );
    #if defined(HAVE_TBB)
    tbb_throughput_uncertainty = round_to_precision(
        tbb_throughput_uncertainty, tbb_throughput_precision
    );
    #endif

    std::cout << THRUST_VERSION                // Thrust Version.
      << ","  << test_name                     // Algorithm.
      << ","  << element_type_name             // Element Type.
      << ","  << element_size                  // Element Size.
      << ","  << elements                      // Elements per Trial.
      << ","  << input_size                    // Total Input Size.
      << ","  << baseline_trials               // STL Trials.
      << ","  << stl_average_walltime          // STL Average Walltime.
      << ","  << stl_walltime_uncertainty      // STL Walltime Uncertainty.
      << ","  << stl_average_throughput        // STL Average Throughput.
      << ","  << stl_throughput_uncertainty    // STL Throughput Uncertainty.
      << ","  << regular_trials                // Thrust Trials.
      << ","  << thrust_average_walltime       // Thrust Average Walltime.
      << ","  << thrust_walltime_uncertainty   // Thrust Walltime Uncertainty.
      << ","  << thrust_average_throughput     // Thrust Average Throughput.
      << ","  << thrust_throughput_uncertainty // Thrust Throughput Uncertainty.
      #if defined(HAVE_TBB)
      << ","  << regular_trials                // TBB Trials.
      << ","  << tbb_average_walltime          // TBB Average Walltime.
      << ","  << tbb_walltime_uncertainty      // TBB Walltime Uncertainty.
      << ","  << tbb_average_throughput        // TBB Average Throughput.
      << ","  << tbb_throughput_uncertainty    // TBB Throughput Uncertainty.
      #endif
      << std::endl;
  } // }}}

private:
  static experiment_results std_experiment()
  {
    return experiment<typename Test<element_type>::std_trial>();
  }

  static experiment_results thrust_experiment()
  {
    return experiment<typename Test<element_type>::thrust_trial>();
  }

  #if defined(HAVE_TBB)
  static experiment_results tbb_experiment()
  {
    return experiment<typename Test<element_type>::tbb_trial>();
  }
  #endif

  template <typename Trial>
  static experiment_results experiment()
  { // {{{
    Trial trial;

    // Allocate storage and generate random input for the warmup trial.
    trial.setup(elements);

    // Warmup trial.
    trial();

    uint64_t const trials
      = trial.is_baseline() ? baseline_trials : regular_trials;

    std::vector<double> times;
    times.reserve(trials);

    for (uint64_t t = 0; t < trials; ++t)
    {
      // Generate random input for next trial.
      trial.setup(elements);

      steady_timer e;

      // Benchmark.
      e.start();
      trial();
      e.stop();

      times.push_back(e.seconds_elapsed());
    }

    double average_time
      = arithmetic_mean(times.begin(), times.end());

    double stdev_time
      = sample_standard_deviation(times.begin(), times.end(), average_time);

    return experiment_results(average_time, stdev_time);
  } // }}}
};

template <
    template <typename> class Test
  , typename                  ElementMetaType
  , uint64_t                  Elements
  , uint64_t                  BaselineTrials
  , uint64_t                  RegularTrials
>
char const* const
experiment_driver<
  Test, ElementMetaType, Elements, BaselineTrials, RegularTrials
>::test_name
  = Test<typename ElementMetaType::type>::test_name();

template <
    template <typename> class Test
  , typename                  ElementMetaType
  , uint64_t                  Elements
  , uint64_t                  BaselineTrials
  , uint64_t                  RegularTrials
>
char const* const
experiment_driver<
  Test, ElementMetaType, Elements, BaselineTrials, RegularTrials
>::element_type_name
  = ElementMetaType::name();

template <
    template <typename> class Test
  , typename                  ElementMetaType
  , uint64_t                  Elements
  , uint64_t                  BaselineTrials
  , uint64_t                  RegularTrials
>
uint64_t const
experiment_driver<
  Test, ElementMetaType, Elements, BaselineTrials, RegularTrials
>::element_size
  = CHAR_BIT * sizeof(typename ElementMetaType::type);

template <
    template <typename> class Test
  , typename                  ElementMetaType
  , uint64_t                  Elements
  , uint64_t                  BaselineTrials
  , uint64_t                  RegularTrials
>
uint64_t const
experiment_driver<
  Test, ElementMetaType, Elements, BaselineTrials, RegularTrials
>::elements
  = Elements;

template <
    template <typename> class Test
  , typename                  ElementMetaType
  , uint64_t                  Elements
  , uint64_t                  BaselineTrials
  , uint64_t                  RegularTrials
>
double const
experiment_driver<
  Test, ElementMetaType, Elements, BaselineTrials, RegularTrials
>::input_size
  = double( Elements /* [elements] */
          * sizeof(typename ElementMetaType::type) /* [bytes/element] */
          )
  / double(1024 * 1024 /* [bytes/MiB] */);

template <
    template <typename> class Test
  , typename                  ElementMetaType
  , uint64_t                  Elements
  , uint64_t                  BaselineTrials
  , uint64_t                  RegularTrials
>
uint64_t const
experiment_driver<
  Test, ElementMetaType, Elements, BaselineTrials, RegularTrials
>::baseline_trials
  = BaselineTrials;

template <
    template <typename> class Test
  , typename                  ElementMetaType
  , uint64_t                  Elements
  , uint64_t                  BaselineTrials
  , uint64_t                  RegularTrials
>
uint64_t const
experiment_driver<
  Test, ElementMetaType, Elements, BaselineTrials, RegularTrials
>::regular_trials
  = RegularTrials;

///////////////////////////////////////////////////////////////////////////////

// Never create variables, pointers or references of any of the `*_trial_base`
// classes. They are purely mixin base classes and do not have vtables and
// virtual destructors. Using them for polymorphism instead of composition will
// probably cause slicing.

struct baseline_trial {};
struct regular_trial {};

template <typename TrialKind = regular_trial>
struct trial_base;

template <>
struct trial_base<baseline_trial>
{
  static bool is_baseline() { return true; }
};

template <>
struct trial_base<regular_trial>
{
  static bool is_baseline() { return false; }
};

template <typename Container, typename TrialKind = regular_trial>
struct inplace_trial_base : trial_base<TrialKind>
{
  Container input;

  void setup(uint64_t elements)
  {
    input.resize(elements);

    randomize(input);
  }
};

template <typename Container, typename TrialKind = regular_trial>
struct copy_trial_base : trial_base<TrialKind>
{
  Container input;
  Container output;

  void setup(uint64_t elements)
  {
    input.resize(elements);
    output.resize(elements);

    randomize(input);
  }
};

#if THRUST_CPP_DIALECT >= 2011
template <typename Container, typename TrialKind = regular_trial>
struct shuffle_trial_base : trial_base<TrialKind>
{
  Container input;

  void setup(uint64_t elements)
  {
    input.resize(elements);

    randomize(input);
  }
};
#endif

///////////////////////////////////////////////////////////////////////////////

template <typename T>
struct reduce_tester
{
  static char const* test_name() { return "reduce"; }

  struct std_trial : inplace_trial_base<std::vector<T>, baseline_trial>
  {
    void operator()()
    {
      if (std::accumulate(this->input.begin(), this->input.end(), T(0)) == 0)
        // Prevent optimizer from removing body.
        std::cout << "xyz";
    }
  };

  struct thrust_trial : inplace_trial_base<thrust::device_vector<T> >
  {
    void operator()()
    {
      thrust::reduce(this->input.begin(), this->input.end());
    }
  };

  #if defined(HAVE_TBB)
  struct tbb_trial : inplace_trial_base<std::vector<T> >
  {
    void operator()()
    {
      tbb_reduce(this->input);
    }
  };
  #endif
};

template <typename T>
struct sort_tester
{
  static char const* test_name() { return "sort"; }

  struct std_trial : inplace_trial_base<std::vector<T>, baseline_trial>
  {
    void operator()()
    {
      std::sort(this->input.begin(), this->input.end());
    }
  };

  struct thrust_trial : inplace_trial_base<thrust::device_vector<T> >
  {
    void operator()()
    {
      thrust::sort(this->input.begin(), this->input.end());
      #if THRUST_DEVICE_SYSTEM == THRUST_DEVICE_SYSTEM_CUDA
        hipError_t err = hipDeviceSynchronize();
        if (err != hipSuccess)
          throw thrust::error_code(err, thrust::cuda_category());
      #endif
    }
  };

  #if defined(HAVE_TBB)
  struct tbb_trial : inplace_trial_base<std::vector<T> >
  {
    void operator()()
    {
      tbb_sort(this->input);
    }
  }
  #endif
};


template <typename T>
struct transform_inplace_tester
{
  static char const* test_name() { return "transform_inplace"; }

  struct std_trial : inplace_trial_base<std::vector<T>, baseline_trial>
  {
    void operator()()
    {
      std::transform(
          this->input.begin(), this->input.end(), this->input.begin()
        , thrust::negate<T>()
      );
    }
  };

  struct thrust_trial : inplace_trial_base<thrust::device_vector<T> >
  {
    void operator()()
    {
      thrust::transform(
          this->input.begin(), this->input.end(), this->input.begin()
        , thrust::negate<T>()
      );
      #if THRUST_DEVICE_SYSTEM == THRUST_DEVICE_SYSTEM_CUDA
        hipError_t err = hipDeviceSynchronize();
        if (err != hipSuccess)
          throw thrust::error_code(err, thrust::cuda_category());
      #endif
    }
  };

  #if defined(HAVE_TBB)
  struct tbb_trial : inplace_trial_base<std::vector<T> >
  {
    void operator()()
    {
      tbb_transform(this->input);
    }
  };
  #endif
};

template <typename T>
struct inclusive_scan_inplace_tester
{
  static char const* test_name() { return "inclusive_scan_inplace"; }

  struct std_trial : inplace_trial_base<std::vector<T>, baseline_trial>
  {
    void operator()()
    {
      std::partial_sum(
          this->input.begin(), this->input.end(), this->input.begin()
      );
    }
  };

  struct thrust_trial : inplace_trial_base<thrust::device_vector<T> >
  {
    void operator()()
    {
      thrust::inclusive_scan(
          this->input.begin(), this->input.end(), this->input.begin()
      );
      #if THRUST_DEVICE_SYSTEM == THRUST_DEVICE_SYSTEM_CUDA
        hipError_t err = hipDeviceSynchronize();
        if (err != hipSuccess)
          throw thrust::error_code(err, thrust::cuda_category());
      #endif
    }
  };

  #if defined(HAVE_TBB)
  struct tbb_trial : inplace_trial_base<std::vector<T> >
  {
    void operator()()
    {
      tbb_scan(this->input);
    }
  };
  #endif
};

template <typename T>
struct copy_tester
{
  static char const* test_name() { return "copy"; }

  struct std_trial : copy_trial_base<std::vector<T> >
  {
    void operator()()
    {
      std::copy(this->input.begin(), this->input.end(), this->output.begin());
    }
  };

  struct thrust_trial : copy_trial_base<thrust::device_vector<T> >
  {
    void operator()()
    {
      thrust::copy(this->input.begin(), this->input.end(), this->input.begin());
      #if THRUST_DEVICE_SYSTEM == THRUST_DEVICE_SYSTEM_CUDA
        hipError_t err = hipDeviceSynchronize();
        if (err != hipSuccess)
          throw thrust::error_code(err, thrust::cuda_category());
      #endif
    }
  };

  #if defined(HAVE_TBB)
  struct tbb_trial : copy_trial_base<std::vector<T> >
  {
    void operator()()
    {
      tbb_copy(this->input, this->output);
    }
  };
  #endif
};

#if THRUST_CPP_DIALECT >= 2011
template <typename T>
struct shuffle_tester
{
  static char const* test_name() { return "shuffle"; }

  struct std_trial : shuffle_trial_base<std::vector<T>, baseline_trial>
  {
    std::default_random_engine g;
    void operator()()
    {
      std::shuffle(this->input.begin(), this->input.end(), this->g);
    }
  };

  struct thrust_trial : shuffle_trial_base<thrust::device_vector<T> >
  {
    thrust::default_random_engine g;
    void operator()()
    {
      thrust::shuffle(this->input.begin(), this->input.end(), this->g);
      #if THRUST_DEVICE_SYSTEM == THRUST_DEVICE_SYSTEM_CUDA
        hipError_t err = hipDeviceSynchronize();
        if (err != hipSuccess)
          throw thrust::error_code(err, thrust::cuda_category());
      #endif
    }
  };
};
#endif

///////////////////////////////////////////////////////////////////////////////

template <
    typename ElementMetaType
  , uint64_t Elements
  , uint64_t BaselineTrials
  , uint64_t RegularTrials
>
void run_core_primitives_experiments_for_type()
{
  experiment_driver<
      reduce_tester
    , ElementMetaType
    , Elements / sizeof(typename ElementMetaType::type)
    , BaselineTrials
    , RegularTrials
  >::run_experiment();

  experiment_driver<
    transform_inplace_tester
    , ElementMetaType
    , Elements / sizeof(typename ElementMetaType::type)
    , BaselineTrials
    , RegularTrials
  >::run_experiment();

  experiment_driver<
      inclusive_scan_inplace_tester
    , ElementMetaType
    , Elements / sizeof(typename ElementMetaType::type)
    , BaselineTrials
    , RegularTrials
  >::run_experiment();

  experiment_driver<
      sort_tester
    , ElementMetaType
//    , Elements / sizeof(typename ElementMetaType::type)
    , (Elements >> 6) // Sorting is more sensitive to element count than
                      // memory footprint.
    , BaselineTrials
    , RegularTrials
  >::run_experiment();

  experiment_driver<
      copy_tester
    , ElementMetaType
    , Elements / sizeof(typename ElementMetaType::type)
    , BaselineTrials
    , RegularTrials
  >::run_experiment();

#if THRUST_CPP_DIALECT >= 2011
  experiment_driver<
      shuffle_tester
    , ElementMetaType
    , Elements / sizeof(typename ElementMetaType::type)
    , BaselineTrials
    , RegularTrials
  >::run_experiment();
#endif
}

///////////////////////////////////////////////////////////////////////////////

#define DEFINE_ELEMENT_META_TYPE(T)                       \
  struct PP_CAT(T, _meta)                                 \
  {                                                       \
    typedef T type;                                       \
                                                          \
    static char const* name() { return PP_STRINGIZE(T); } \
  };                                                      \
  /**/

DEFINE_ELEMENT_META_TYPE(char);
DEFINE_ELEMENT_META_TYPE(int);
DEFINE_ELEMENT_META_TYPE(int8_t);
DEFINE_ELEMENT_META_TYPE(int16_t);
DEFINE_ELEMENT_META_TYPE(int32_t);
DEFINE_ELEMENT_META_TYPE(int64_t);
DEFINE_ELEMENT_META_TYPE(float);
DEFINE_ELEMENT_META_TYPE(double);

///////////////////////////////////////////////////////////////////////////////

template <
    uint64_t Elements
  , uint64_t BaselineTrials
  , uint64_t RegularTrials
>
void run_core_primitives_experiments()
{
  run_core_primitives_experiments_for_type<
    char_meta,    Elements, BaselineTrials, RegularTrials
  >();
  run_core_primitives_experiments_for_type<
    int_meta,     Elements, BaselineTrials, RegularTrials
  >();
  run_core_primitives_experiments_for_type<
    int8_t_meta,  Elements, BaselineTrials, RegularTrials
  >();
  run_core_primitives_experiments_for_type<
    int16_t_meta, Elements, BaselineTrials, RegularTrials
  >();
  run_core_primitives_experiments_for_type<
    int32_t_meta, Elements, BaselineTrials, RegularTrials
  >();
  run_core_primitives_experiments_for_type<
    int64_t_meta, Elements, BaselineTrials, RegularTrials
  >();
  run_core_primitives_experiments_for_type<
    float_meta,   Elements, BaselineTrials, RegularTrials
  >();
  run_core_primitives_experiments_for_type<
    double_meta,  Elements, BaselineTrials, RegularTrials
  >();
}

///////////////////////////////////////////////////////////////////////////////

// XXX Use `std::string_view` when possible.
std::vector<std::string> split(std::string const& str, std::string const& delim)
{
  std::vector<std::string> tokens;
  std::string::size_type prev = 0, pos = 0;
  do
  {
    pos = str.find(delim, prev);
    if (pos == std::string::npos) pos = str.length();
    std::string token = str.substr(prev, pos - prev);
    if (!token.empty()) tokens.push_back(token);
    prev = pos + delim.length();
  }
  while (pos < str.length() && prev < str.length());
  return tokens;
}

///////////////////////////////////////////////////////////////////////////////

struct command_line_option_error : std::exception
{
  virtual ~command_line_option_error() NOEXCEPT {}
  virtual const char* what() const NOEXCEPT = 0;
};

struct only_one_option_allowed : command_line_option_error
{
  // Construct a new `only_one_option_allowed` exception. `key` is the
  // option name and `[first, last)` is a sequence of
  // `std::pair<std::string const, std::string>`s (the values).
  template <typename InputIt>
  only_one_option_allowed(std::string const& key, InputIt first, InputIt last)
    : message()
  {
    message  = "Only one `--";
    message += key;
    message += "` option is allowed, but multiple were received: ";

    for (; first != last; ++first)
    {
      message += "`";
      message += (*first).second;
      message += "` ";
    }

    // Remove the trailing space added by the last iteration of the above loop.
    message.erase(message.size() - 1, 1);

    message += ".";
  }

  virtual ~only_one_option_allowed() NOEXCEPT {}

  virtual const char* what() const NOEXCEPT
  {
    return message.c_str();
  }

private:
  std::string message;
};

struct required_option_missing : command_line_option_error
{
  // Construct a new `requirement_option_missing` exception. `key` is the
  // option name.
  required_option_missing(std::string const& key)
    : message()
  {
    message  = "`--";
    message += key;
    message += "` option is required.";
  }

  virtual ~required_option_missing() NOEXCEPT {}

  virtual const char* what() const NOEXCEPT
  {
    return message.c_str();
  }

private:
  std::string message;
};

struct command_line_processor
{
  typedef std::vector<std::string> positional_options_type;

  typedef std::multimap<std::string, std::string> keyword_options_type;

  typedef std::pair<
    keyword_options_type::const_iterator
  , keyword_options_type::const_iterator
  > keyword_option_values;

  command_line_processor(int argc, char** argv)
    : pos_args(), kw_args()
  { // {{{
    for (int i = 1; i < argc; ++i)
    {
      std::string arg(argv[i]);

      // Look for --key or --key=value options.
      if (arg.substr(0, 2) == "--")
      {
        std::string::size_type n = arg.find('=', 2);

        keyword_options_type::value_type key_value;

        if (n == std::string::npos) // --key
          kw_args.insert(keyword_options_type::value_type(
            arg.substr(2), ""
          ));
        else                        // --key=value
          kw_args.insert(keyword_options_type::value_type(
            arg.substr(2, n - 2), arg.substr(n + 1)
          ));

        kw_args.insert(key_value);
      }
      else // Assume it's positional.
        pos_args.push_back(arg);
    }
  } // }}}

  // Return the value for option `key`.
  //
  // Throws:
  // * `only_one_option_allowed` if there is more than one value for `key`.
  // * `required_option_missing` if there is no value for `key`.
  std::string operator()(std::string const& key) const
  {
    keyword_option_values v = kw_args.equal_range(key);

    keyword_options_type::difference_type d = std::distance(v.first, v.second);

    if      (1 < d)  // Too many options.
      throw only_one_option_allowed(key, v.first, v.second);
    else if (0 == d) // No option.
      throw required_option_missing(key);

    return (*v.first).second;
  }

  // Return the value for option `key`, or `dflt` if `key` has no value.
  //
  // Throws: `only_one_option_allowed` if there is more than one value for `key`.
  std::string operator()(std::string const& key, std::string const& dflt) const
  {
    keyword_option_values v = kw_args.equal_range(key);

    keyword_options_type::difference_type d = std::distance(v.first, v.second);

    if (1 < d)  // Too many options.
      throw only_one_option_allowed(key, v.first, v.second);

    if (0 == d) // No option.
      return dflt;
    else        // 1 option.
      return (*v.first).second;
  }

  // Returns `true` if the option `key` was specified at least once.
  bool has(std::string const& key) const
  {
    return kw_args.count(key) > 0;
  }

private:
  positional_options_type pos_args;
  keyword_options_type    kw_args;
};

///////////////////////////////////////////////////////////////////////////////

int main(int argc, char** argv)
{
  command_line_processor clp(argc, argv);

  #if defined(HAVE_TBB)
  tbb::task_scheduler_init init;

  test_tbb();
  #endif

  #if THRUST_DEVICE_SYSTEM == THRUST_DEVICE_SYSTEM_CUDA
    // Set the CUDA device to use for the benchmark - `0` by default.

    int device = std::atoi(clp("device", "0").c_str());
    // `std::atoi` returns 0 if the conversion fails.

    hipSetDevice(device);
  #endif

  if (!clp.has("no-header"))
    print_experiment_header();

                                          /* Elements |       Trials       */
                                          /*          | Baseline | Regular */
//run_core_primitives_experiments< 1LLU << 21LLU      , 4        , 16      >();
//run_core_primitives_experiments< 1LLU << 22LLU      , 4        , 16      >();
//run_core_primitives_experiments< 1LLU << 23LLU      , 4        , 16      >();
//run_core_primitives_experiments< 1LLU << 24LLU      , 4        , 16      >();
//run_core_primitives_experiments< 1LLU << 25LLU      , 4        , 16      >();
  run_core_primitives_experiments< 1LLU << 26LLU      , 4        , 16      >();
  run_core_primitives_experiments< 1LLU << 27LLU      , 4        , 16      >();
//run_core_primitives_experiments< 1LLU << 28LLU      , 4        , 16      >();
//run_core_primitives_experiments< 1LLU << 29LLU      , 4        , 16      >();

  return 0;
}

// TODO: Add different input sizes and half precision
