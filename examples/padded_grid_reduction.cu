#include "hip/hip_runtime.h"
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/random.h>
#include <thrust/extrema.h>
#include <cmath>
#include <iomanip>
#include <float.h>

// This example computes the minimum and maximum values
// over a padded grid.  The padded values are not considered
// during the reduction operation.


// transform a tuple (int,value) into a tuple (bool,value,value)
// where the bool is true for valid grid values and false for 
// values in the padded region of the grid
template <typename IndexType, typename ValueType>
struct transform_tuple : 
    public thrust::unary_function< thrust::tuple<IndexType,ValueType>, 
                                   thrust::tuple<bool,ValueType,ValueType> >
{
  typedef typename thrust::tuple<IndexType,ValueType>      InputTuple;
  typedef typename thrust::tuple<bool,ValueType,ValueType> OutputTuple;

  IndexType n, N;

  transform_tuple(IndexType n, IndexType N) : n(n), N(N) {}

  __host__ __device__
    OutputTuple operator()(const InputTuple& t) const
    { 
      bool is_valid = (thrust::get<0>(t) % N) < n;
      return OutputTuple(is_valid, thrust::get<1>(t), thrust::get<1>(t));
    }
};


// reduce two tuples (bool,value,value) into a single tuple such that output
// contains the smallest and largest *valid* values.
template <typename IndexType, typename ValueType>
struct reduce_tuple :
    public thrust::binary_function< thrust::tuple<bool,ValueType,ValueType>,
                                    thrust::tuple<bool,ValueType,ValueType>,
                                    thrust::tuple<bool,ValueType,ValueType> >
{
  typedef typename thrust::tuple<bool,ValueType,ValueType> Tuple;

  __host__ __device__
    Tuple operator()(const Tuple& t0, const Tuple& t1) const
    { 
      if(thrust::get<0>(t0) && thrust::get<0>(t1)) // both valid
        return Tuple(true, 
            thrust::min(thrust::get<1>(t0), thrust::get<1>(t1)),
            thrust::max(thrust::get<2>(t0), thrust::get<2>(t1)));
      else if (thrust::get<0>(t0))
        return t0;
      else if (thrust::get<0>(t1))
        return t1;
      else
        return t1; // if neither is valid then it doesn't matter what we return
    }
};

int main(void)
{
  int M = 10;  // number of rows
  int n = 11;  // number of columns excluding padding
  int N = 16;  // number of columns including padding

  thrust::default_random_engine rng(12345);
  thrust::uniform_real_distribution<float> dist(0.0f, 1.0f);

  thrust::device_vector<float> data(M * N, -1);

  // initialize valid values in grid
  for(int i = 0; i < M; i++)
    for(int j = 0; j < n; j++)
      data[i * N + j] = dist(rng);

  // print full grid
  std::cout << "padded grid" << std::endl;
  std::cout << std::fixed << std::setprecision(4);
  for(int i = 0; i < M; i++)
  {
    std::cout << " ";
    for(int j = 0; j < N; j++)
    {
      std::cout << data[i * N + j] << " ";
    }   
    std::cout << "\n";
  }
  std::cout << "\n";

  // compute min & max over valid region of the 2d grid
  typedef thrust::tuple<bool, float, float> result_type;

  result_type                 init(true, FLT_MAX, -FLT_MAX); // initial value
  transform_tuple<int,float>  unary_op(n, N);                // transformation operator
  reduce_tuple<int,float>     binary_op;                     // reduction operator

  auto begin = thrust::make_zip_iterator(thrust::make_tuple(thrust::counting_iterator<int>(0), data.begin()));
  result_type result = thrust::transform_reduce(begin, begin + data.size(), unary_op, init, binary_op);

  std::cout << "minimum value: " << thrust::get<1>(result) << std::endl;
  std::cout << "maximum value: " << thrust::get<2>(result) << std::endl;

  return 0;
}

