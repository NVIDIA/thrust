#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/binary_search.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/random.h>

#include <iostream>
#include <iomanip>

// define a 2d float vector
typedef thrust::tuple<float,float> vec2;

// return a random vec2 in [0,1)^2
vec2 make_random_vec2(void)
{
  static thrust::default_random_engine rng;
  static thrust::uniform_real_distribution<float> u01(0.0f, 1.0f);
  float x = u01(rng);
  float y = u01(rng);
  return vec2(x,y);
}

// hash a point in the unit square to the index of
// the grid bucket that contains it
struct point_to_bucket_index : public thrust::unary_function<vec2,unsigned int>
{
  unsigned int width;  // buckets in the x dimension (grid spacing = 1/width)
  unsigned int height; // buckets in the y dimension (grid spacing = 1/height)

  __host__ __device__
  point_to_bucket_index(unsigned int width, unsigned int height)
    : width(width), height(height) {}

  __host__ __device__
  unsigned int operator()(const vec2& v) const
  {
    // find the raster indices of p's bucket
    unsigned int x = static_cast<unsigned int>(thrust::get<0>(v)) * width;
    unsigned int y = static_cast<unsigned int>(thrust::get<1>(v)) * height;

    // return the bucket's linear index
    return y * width + x;
  }

};

int main(void)
{
  const size_t N = 1000000;

  // allocate some random points in the unit square on the host
  thrust::host_vector<vec2> h_points(N);
  thrust::generate(h_points.begin(), h_points.end(), make_random_vec2);

  // transfer to device
  thrust::device_vector<vec2> points = h_points;

  // allocate storage for a 2D grid
  // of dimensions w x h
  unsigned int w = 200, h = 100;

  // the grid data structure keeps a range per grid bucket:
  // each bucket_begin[i] indexes the first element of bucket i's list of points
  // each bucket_end[i] indexes one past the last element of bucket i's list of points
  thrust::device_vector<unsigned int> bucket_begin(w*h);
  thrust::device_vector<unsigned int> bucket_end(w*h);

  // allocate storage for each point's bucket index
  thrust::device_vector<unsigned int> bucket_indices(N);

  // transform the points to their bucket indices
  thrust::transform(points.begin(),
                    points.end(),
                    bucket_indices.begin(),
                    point_to_bucket_index(w,h));

  // sort the points by their bucket index
  thrust::sort_by_key(bucket_indices.begin(),
                      bucket_indices.end(),
                      points.begin());

  // find the beginning of each bucket's list of points
  thrust::counting_iterator<unsigned int> search_begin(0);
  thrust::lower_bound(bucket_indices.begin(),
                      bucket_indices.end(),
                      search_begin,
                      search_begin + w*h,
                      bucket_begin.begin());

  // find the end of each bucket's list of points
  thrust::upper_bound(bucket_indices.begin(),
                      bucket_indices.end(),
                      search_begin,
                      search_begin + w*h,
                      bucket_end.begin());

  // write out bucket (150, 50)'s list of points
  unsigned int bucket_idx = 50 * w + 150;
  std::cout << "bucket (150, 50)'s list of points:" << std::endl;
  std::cout << std::fixed << std::setprecision(6);
  for(unsigned int point_idx = bucket_begin[bucket_idx];
      point_idx != bucket_end[bucket_idx];
      ++point_idx)
  {
    vec2 p = points[point_idx];
    std::cout << "(" << thrust::get<0>(p) << "," << thrust::get<1>(p) << ")" << std::endl;
  }

  return 0;
}

