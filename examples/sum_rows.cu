#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/random.h>
#include <iostream>

// convert a linear index to a row index
template <typename T>
struct linear_index_to_row_index : public thrust::unary_function<T,T>
{
  T C; // number of columns
  
  __host__ __device__
  linear_index_to_row_index(T C) : C(C) {}

  __host__ __device__
  T operator()(T i)
  {
    return i / C;
  }
};

int main(void)
{
  int R = 5;     // number of rows
  int C = 8;     // number of columns
  thrust::default_random_engine rng;
  thrust::uniform_int_distribution<int> dist(10, 99);

  // initialize data
  thrust::device_vector<int> array(R * C);
  for (size_t i = 0; i < array.size(); i++)
    array[i] = dist(rng);
  
  // allocate storage for row sums and indices
  thrust::device_vector<int> row_sums(R);
  thrust::device_vector<int> row_indices(R);
  
  // compute row sums by summing values with equal row indices
  auto begin = thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(C));
  thrust::reduce_by_key
    (begin,
     begin + (R*C),
     array.begin(),
     row_indices.begin(),
     row_sums.begin(),
     thrust::equal_to<int>(),
     thrust::plus<int>());

  // print data 
  for(int i = 0; i < R; i++)
  {
    std::cout << "[ ";
    for(int j = 0; j < C; j++)
      std::cout << array[i * C + j] << " ";
    std::cout << "] = " << row_sums[i] << "\n";
  }

  return 0;
}

