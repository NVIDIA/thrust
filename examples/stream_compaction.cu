#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/count.h>
#include <thrust/remove.h>
#include <iostream>
#include <iterator>
#include <string>

// this functor returns true if the argument is odd, and false otherwise
template <typename T>
struct is_odd : public thrust::unary_function<T,bool>
{
    __host__ __device__
    bool operator()(T x)
    {
        return x % 2;
    }
};


template <typename Iterator>
void print_range(const std::string& name, Iterator first, Iterator last)
{
    typedef typename std::iterator_traits<Iterator>::value_type T;

    std::cout << name << ": ";
    thrust::copy(first, last, std::ostream_iterator<T>(std::cout, " "));  
    std::cout << "\n";
}

int main(void)
{
    // input size
    size_t N = 10;

    // define some types
    typedef thrust::device_vector<int> Vector;

    // allocate storage for array
    Vector values(N);

    // initialize array to [0, 1, 2, ... ]
    thrust::sequence(values.begin(), values.end());
    
    print_range("values", values.begin(), values.end());

    // allocate output storage, here we conservatively assume all values will be copied
    Vector output(values.size());

    // copy odd numbers to separate array
    auto output_end = thrust::copy_if(values.begin(), values.end(), output.begin(), is_odd<int>());

    print_range("output", output.begin(), output_end);

    // another approach is to count the number of values that will 
    // be copied, and allocate an array of the right size
    size_t N_odd = thrust::count_if(values.begin(), values.end(), is_odd<int>());
    
    Vector small_output(N_odd);
    
    thrust::copy_if(values.begin(), values.end(), small_output.begin(), is_odd<int>());
    
    print_range("small_output", small_output.begin(), small_output.end());

    // we can also compact sequences with the remove functions, which do the opposite of copy
    auto values_end = thrust::remove_if(values.begin(), values.end(), is_odd<int>());

    // since the values after values_end are garbage, we'll resize the vector
    values.resize(values_end - values.begin());

    print_range("values", values.begin(), values.end());

    return 0;
}

