#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/merge.h>
#include <thrust/set_operations.h>
#include <thrust/iterator/discard_iterator.h>
#include <iostream>

// This example illustrates use of the set operation algorithms
//  - merge
//  - set_union
//  - set_intersection
//  - set_difference
//  - set_symmetric_difference
//
// In this context a "set" is simply a sequence of sorted values,
// allowing the standard set operations to be performed more efficiently
// than on unsorted data.  Since the output of a set operation is a valid
// set (i.e. a sorted sequence) it is possible to apply the set operations
// in a nested fashion to compute arbitrary set expressions.
//
// Set operation usage notes:
//   - The output set size is variable (except for thrust::merge),
//     so the return value is important.
//   - Generally one would conservatively allocate storage for the output
//     and then resize or shrink an output container as necessary.
//     Alternatively, one can compute the exact output size by
//     outputting to a discard_iterator.  This approach is more computationally
//     expensive (approximately 2x), but conserves memory capacity.
//     Refer to the SetIntersectionSize function for implementation details.
//   - Sets are allowed to have duplicate elements, which are carried
//     through to the output in a algorithm-specific manner.  Refer
//     to the full documentation for precise semantics.


// helper routine
template <typename String, typename Vector>
void print(const String& s, const Vector& v)
{
  std::cout << s << " [";
  for(size_t i = 0; i < v.size(); i++)
    std::cout << " " << v[i];
  std::cout << " ]\n";
}

template <typename Vector>
void Merge(const Vector& A, const Vector& B)
{
  // merged output is always exactly A.size() + B.size()
  Vector C(A.size() + B.size());

  thrust::merge(A.begin(), A.end(), B.begin(), B.end(), C.begin());

  print("Merge(A,B)", C);
}

template <typename Vector>
void SetUnion(const Vector& A, const Vector& B)
{
  // union output is at most A.size() + B.size()
  Vector C(A.size() + B.size());

  // set_union returns an iterator C_end denoting the end of input
  typename Vector::iterator C_end;
  
  C_end = thrust::set_union(A.begin(), A.end(), B.begin(), B.end(), C.begin());
  
  // shrink C to exactly fit output
  C.erase(C_end, C.end());

  print("Union(A,B)", C);
}

template <typename Vector>
void SetIntersection(const Vector& A, const Vector& B)
{
  // intersection output is at most min(A.size(), B.size())
  Vector C(thrust::min(A.size(), B.size()));

  // set_union returns an iterator C_end denoting the end of input
  typename Vector::iterator C_end;
  
  C_end = thrust::set_intersection(A.begin(), A.end(), B.begin(), B.end(), C.begin());
  
  // shrink C to exactly fit output
  C.erase(C_end, C.end());

  print("Intersection(A,B)", C);
}

template <typename Vector>
void SetDifference(const Vector& A, const Vector& B)
{
  // difference output is at most A.size()
  Vector C(A.size());

  // set_union returns an iterator C_end denoting the end of input
  typename Vector::iterator C_end;
  
  C_end = thrust::set_difference(A.begin(), A.end(), B.begin(), B.end(), C.begin());
  
  // shrink C to exactly fit output
  C.erase(C_end, C.end());

  print("Difference(A,B)", C);
}

template <typename Vector>
void SetSymmetricDifference(const Vector& A, const Vector& B)
{
  // symmetric difference output is at most A.size() + B.size()
  Vector C(A.size() + B.size());

  // set_union returns an iterator C_end denoting the end of input
  typename Vector::iterator C_end;
  
  C_end = thrust::set_symmetric_difference(A.begin(), A.end(), B.begin(), B.end(), C.begin());
  
  // shrink C to exactly fit output
  C.erase(C_end, C.end());

  print("SymmetricDifference(A,B)", C);
}

template <typename Vector>
void SetIntersectionSize(const Vector& A, const Vector& B)
{
  // computes the exact size of the intersection without allocating output
  thrust::discard_iterator<> C_begin, C_end;

  C_end = thrust::set_intersection(A.begin(), A.end(), B.begin(), B.end(), C_begin);

  std::cout << "SetIntersectionSize(A,B) " << (C_end - C_begin) << std::endl;
}


int main(void)
{
  int a[] = {0,2,4,5,6,8,9};
  int b[] = {0,1,2,3,5,7,8};

  thrust::device_vector<int> A(a, a + sizeof(a) / sizeof(int));
  thrust::device_vector<int> B(b, b + sizeof(b) / sizeof(int));

  print("Set A", A);
  print("Set B", B);

  Merge(A,B);
  SetUnion(A,B);
  SetIntersection(A,B);
  SetDifference(A,B);
  SetSymmetricDifference(A,B);

  SetIntersectionSize(A,B);

  return 0;
}

