#include <thrust/iterator/iterator_traits.h>
#include <thrust/device_vector.h>
#include <thrust/for_each.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <iostream>

// This example demonstrates how to build a minimal custom
// Thrust backend by intercepting for_each's dispatch.

// We begin by defining a "system tag", which distinguishes our novel
// backend from other Thrust backends.
// We'll derive my_system from thrust::device_execution_policy to inherit
// the functionality of the default device backend.
// Note that we pass the name of our tag as a template parameter
// to thrust::device_execution_policy
struct my_system : thrust::device_execution_policy<my_system> {};

// Next, we'll create a novel version of for_each which only
// applies to iterators "tagged" with my_system.
// Our version of for_each will print a message and then call
// the regular version of for_each.

// The first parameter to our for_each is my_system. This allows
// Thrust to locate it when dispatching thrust::for_each on iterators
// tagged with my_system. The following parameters are as normal.
template<typename Iterator, typename Function>
  Iterator for_each(my_system, 
                    Iterator first, Iterator last,
                    Function f)
{
  // Our version of for_each was invoked because first and last are tagged with my_tag

  // output a message
  std::cout << "Hello, world from for_each(my_system)!" << std::endl;

  // to call the normal version of for_each, we need to "retag" the iterator
  // arguments with device_system_tag using the retag function. It's safe to
  // retag the iterators with device_system_tag because my_system is related by
  // convertibility.
  thrust::for_each(thrust::retag<thrust::device_system_tag>(first),
                   thrust::retag<thrust::device_system_tag>(last),
                   f);

  return last;
}

int main()
{
  // Create a device_vector, whose iterators are tagged with device_system_tag
  thrust::device_vector<int> vec(1);

  // To ensure that our version of for_each is invoked during dispatch, we
  // retag vec's iterators with my_system. It's safe to retag the iterators with
  // my_system because my_system's base class can convert to device_system_tag
  thrust::for_each(thrust::retag<my_system>(vec.begin()),
                   thrust::retag<my_system>(vec.end()),
                   thrust::identity<int>());

  // Other algorithms that Thrust implements with thrust::for_each will also
  // cause our version of for_each to be invoked when their iterator arguments
  // are tagged with my_system. Because we did not define a specialized version of
  // transform, Thrust dispatches the version it knows for device_system_tag,
  // which my_system inherits.
  thrust::transform(thrust::retag<my_system>(vec.begin()),
                    thrust::retag<my_system>(vec.end()),
                    thrust::retag<my_system>(vec.begin()),
                    thrust::identity<int>());

  // Iterators without my_system are handled normally.
  thrust::for_each(vec.begin(), vec.end(), thrust::identity<int>());

  return 0;
}

