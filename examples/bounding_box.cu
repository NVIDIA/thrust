#include "hip/hip_runtime.h"
#include <thrust/transform_reduce.h>
#include <thrust/device_vector.h>
#include <thrust/pair.h>
#include <thrust/random.h>
#include <thrust/extrema.h>

// This example shows how to compute a bounding box
// for a set of points in two dimensions.

struct point2d
{
  float x, y;
  
  __host__ __device__
  point2d() : x(0), y(0) {}
  
  __host__ __device__
  point2d(float _x, float _y) : x(_x), y(_y) {}
};

// bounding box type
struct bbox
{
  // construct an empty box
  __host__ __device__
  bbox() {}

  // construct a box from a single point
  __host__ __device__
  bbox(const point2d &point)
    : lower_left(point), upper_right(point)
  {}

  // construct a box from a pair of points
  __host__ __device__
  bbox(const point2d &ll, const point2d &ur)
    : lower_left(ll), upper_right(ur)
  {}

  point2d lower_left, upper_right;
};

// reduce a pair of bounding boxes (a,b) to a bounding box containing a and b
struct bbox_reduction : public thrust::binary_function<bbox,bbox,bbox>
{
  __host__ __device__
  bbox operator()(bbox a, bbox b)
  {
    // lower left corner
    point2d ll(thrust::min(a.lower_left.x, b.lower_left.x), thrust::min(a.lower_left.y, b.lower_left.y));
    
    // upper right corner
    point2d ur(thrust::max(a.upper_right.x, b.upper_right.x), thrust::max(a.upper_right.y, b.upper_right.y));
    
    return bbox(ll, ur);
  }
};

int main(void)
{
  const size_t N = 40;
  thrust::default_random_engine rng;
  thrust::uniform_real_distribution<float> u01(0.0f, 1.0f);
  
  // allocate storage for points
  thrust::device_vector<point2d> points(N);
  
  // generate some random points in the unit square
  for(size_t i = 0; i < N; i++)
  {
      float x = u01(rng);
      float y = u01(rng);
      points[i] = point2d(x,y);
  }
  
  // initial bounding box contains first point
  bbox init = bbox(points[0], points[0]);
  
  // binary reduction operation
  bbox_reduction binary_op;
  
  // compute the bounding box for the point set
  bbox result = thrust::reduce(points.begin(), points.end(), init, binary_op);
  
  // print output
  std::cout << "bounding box " << std::fixed;
  std::cout << "(" << result.lower_left.x  << "," << result.lower_left.y  << ") ";
  std::cout << "(" << result.upper_right.x << "," << result.upper_right.y << ")" << std::endl;
  
  return 0;
}
