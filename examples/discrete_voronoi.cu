#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/tuple.h>
#include <thrust/extrema.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/random.h>

#include <iostream>
#include <iomanip>
#include <fstream>
#include <cmath>

#include "include/timer.h"

// Compute an approximate Voronoi Diagram with a Jump Flooding Algorithm (JFA)
//
// References
//   http://en.wikipedia.org/wiki/Voronoi_diagram
//   http://www.comp.nus.edu.sg/~tants/jfa.html
//   http://www.utdallas.edu/~guodongrong/Papers/Dissertation.pdf
//
// Thanks to David Coeurjolly for contributing this example



// minFunctor
// Tuple  = <seeds,seeds + k,seeds + m*k, seeds - k, 
//           seeds - m*k, seeds+ k+m*k,seeds + k-m*k,
//           seeds- k+m*k,seeds - k+m*k, i>
struct minFunctor
{
  int m, n, k;
  
  __host__ __device__
  minFunctor(int m, int n, int k)
    : m(m), n(n), k(k) {}


  //To decide I have to change my current Voronoi site
  __host__ __device__
      int minVoro(int x_i, int y_i, int p, int q)
      {    
          if (q == m*n)
              return p;

          // coordinates of points p and q
          int y_q =  q / m;
          int x_q =  q - y_q * m;
          int y_p =  p / m;
          int x_p =  p - y_p * m;
        
          // squared distances
          int d_iq = (x_i-x_q) * (x_i-x_q) + (y_i-y_q) * (y_i-y_q);
          int d_ip = (x_i-x_p) * (x_i-x_p) + (y_i-y_p) * (y_i-y_p);

          if (d_iq < d_ip)
              return q;  // q is closer
          else
              return p;
      }

  //For each point p+{-k,0,k}, we keep the Site with minimum distance
  template <typename Tuple>
  __host__ __device__
  int operator()(const Tuple &t)
  {
      //Current point and site
      int i = thrust::get<9>(t);
      int v = thrust::get<0>(t);

      //Current point coordinates
      int y = i / m;    
      int x = i - y * m;

      if (x >= k)
      {
          v = minVoro(x, y, v, thrust::get<3>(t));

          if (y >= k)
              v = minVoro(x, y, v, thrust::get<8>(t));

          if (y + k < n)
              v = minVoro(x, y, v, thrust::get<7>(t));
      }

      if (x + k < m)
      { 
          v = minVoro(x, y, v, thrust::get<1>(t));

          if (y >= k)
              v = minVoro(x, y, v, thrust::get<6>(t));
          if (y + k < n)
              v = minVoro(x, y, v, thrust::get<5>(t));
      }

      if (y >= k)
          v = minVoro(x, y, v, thrust::get<4>(t));
      if (y + k < n)
          v = minVoro(x, y, v, thrust::get<2>(t));

      //global return
      return v;
  }
};



// print an M-by-N array
template <typename T>
void print(int m, int n, const thrust::device_vector<T>& d_data)
{
    thrust::host_vector<T> h_data = d_data;

    for(int i = 0; i < m; i++)
    {
        for(int j = 0; j < n; j++)
            std::cout << std::setw(4) << h_data[i * n + j] << " ";
        std::cout << "\n";
    }
}


void generate_random_sites(thrust::host_vector<int> &t, int Nb, int m, int n)
{
  thrust::default_random_engine rng;
  thrust::uniform_int_distribution<int> dist(0, m * n - 1);

  for(int k = 0; k < Nb; k++)
  {
      int index = dist(rng);
      t[index] = index + 1;
  }
}

//Export the tab to PGM image format
void vector_to_pgm(thrust::host_vector<int> &t, int m, int n, const char *out)
{
    assert(static_cast<int>(t.size()) == m * n &&
           "Vector size does not match image dims.");

    std::fstream f(out, std::fstream::out);
    f << "P2\n";
    f << m << " " << n << "\n";
    f << "253\n";

    //Hash function to map values to [0,255]
    auto to_grey_level = [](int in_value) -> int
    {
        return (71 * in_value) % 253;
    };

    for (int value : t)
    {
      f << to_grey_level(value) << " ";
    }
    f << "\n";
    f.close();
}

/************Main Jfa loop********************/
// Perform a jump with step k
void jfa(thrust::device_vector<int>& in,thrust::device_vector<int>& out, unsigned int k, int m, int n)
{
   thrust::transform(
        thrust::make_zip_iterator(
            thrust::make_tuple(in.begin(), 
                               in.begin() + k, 
                               in.begin() + m*k, 
                               in.begin() - k, 
                               in.begin() - m*k, 
                               in.begin() + k+m*k,
                               in.begin() + k-m*k,
                               in.begin() - k+m*k,
                               in.begin() - k-m*k,
                               thrust::counting_iterator<int>(0))),
        thrust::make_zip_iterator(
            thrust::make_tuple(in.begin(), 
				    		   in.begin() + k, 
                               in.begin() + m*k, 
                               in.begin() - k, 
                               in.begin() - m*k, 
                               in.begin() + k+m*k,
                               in.begin() + k-m*k,
                               in.begin() - k+m*k,
                               in.begin() - k-m*k,
                               thrust::counting_iterator<int>(0)))+ n*m,
        out.begin(),
        minFunctor(m,n,k));
}
/********************************************/

void display_time(timer& t)
{
  std::cout << "  ( "<< 1e3 * t.elapsed() << "ms )" << std::endl;
}

int main(void)
{
  int m = 2048; // number of rows
  int n = 2048; // number of columns  
  int s = 1000; // number of sites
  
  timer t;
 
  //Host vector to encode a 2D image
  std::cout << "[Inititialize " << m << "x" << n << " Image]" << std::endl;
  t.restart();
  thrust::host_vector<int> seeds_host(m*n, m*n);
  generate_random_sites(seeds_host,s,m,n);
  display_time(t);
  
  std::cout<<"[Copy to Device]" << std::endl;
  t.restart();
  thrust::device_vector<int> seeds = seeds_host;
  thrust::device_vector<int> temp(seeds);
  display_time(t);

  //JFA+1  : before entering the log(n) loop, we perform a jump with k=1
  std::cout<<"[JFA stepping]" << std::endl;
  t.restart();
  jfa(seeds,temp,1,m,n);
  seeds.swap(temp);
 
  //JFA : main loop with k=n/2, n/4, ..., 1
  for(int k = thrust::max(m,n) / 2; k > 0; k /= 2)
  {
    jfa(seeds,temp,k,m,n);
    seeds.swap(temp);
  }

  display_time(t);
  std::cout <<"  ( " <<  seeds.size() / (1e6 * t.elapsed()) << " MPixel/s ) " << std::endl;
  
  std::cout << "[Device to Host Copy]" << std::endl;
  t.restart();
  seeds_host = seeds;
  display_time(t);
  
  std::cout << "[PGM Export]" << std::endl;
  t.restart();
  vector_to_pgm(seeds_host, m, n, "discrete_voronoi.pgm");
  display_time(t);

  return 0;
}

