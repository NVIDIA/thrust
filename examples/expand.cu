#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/gather.h>
#include <thrust/scan.h>
#include <thrust/fill.h>
#include <thrust/copy.h>

#include <iterator>
#include <iostream>

// This example demonstrates how to expand an input sequence by 
// replicating each element a variable number of times. For example,
//
//   expand([2,2,2],[A,B,C]) -> [A,A,B,B,C,C]
//   expand([3,0,1],[A,B,C]) -> [A,A,A,C]
//   expand([1,3,2],[A,B,C]) -> [A,B,B,B,C,C]
//
// The element counts are assumed to be non-negative integers

template <typename InputIterator1,
          typename InputIterator2,
          typename OutputIterator>
OutputIterator expand(InputIterator1 first1,
                      InputIterator1 last1,
                      InputIterator2 first2,
                      OutputIterator output)
{
  typedef typename thrust::iterator_difference<InputIterator1>::type difference_type;
  
  difference_type input_size  = thrust::distance(first1, last1);
  difference_type output_size = thrust::reduce(first1, last1);

  // scan the counts to obtain output offsets for each input element
  thrust::device_vector<difference_type> output_offsets(input_size, 0);
  thrust::exclusive_scan(first1, last1, output_offsets.begin()); 

  // scatter the nonzero counts into their corresponding output positions
  thrust::device_vector<difference_type> output_indices(output_size, 0);
  thrust::scatter_if
    (thrust::counting_iterator<difference_type>(0),
     thrust::counting_iterator<difference_type>(input_size),
     output_offsets.begin(),
     first1,
     output_indices.begin());

  // compute max-scan over the output indices, filling in the holes
  thrust::inclusive_scan
    (output_indices.begin(),
     output_indices.end(),
     output_indices.begin(),
     thrust::maximum<difference_type>());

  // gather input values according to index array (output = first2[output_indices])
  thrust::gather(output_indices.begin(),
                 output_indices.end(),
                 first2,
                 output);

  // return output + output_size
  thrust::advance(output, output_size);
  return output;
}

template <typename Vector>
void print(const std::string& s, const Vector& v)
{
  typedef typename Vector::value_type T;

  std::cout << s;
  thrust::copy(v.begin(), v.end(), std::ostream_iterator<T>(std::cout, " "));
  std::cout << std::endl;
}

int main(void)
{
  int counts[] = {3,5,2,0,1,3,4,2,4};
  int values[] = {1,2,3,4,5,6,7,8,9};

  size_t input_size  = sizeof(counts) / sizeof(int);
  size_t output_size = thrust::reduce(counts, counts + input_size);

  // copy inputs to device
  thrust::device_vector<int> d_counts(counts, counts + input_size);
  thrust::device_vector<int> d_values(values, values + input_size);
  thrust::device_vector<int> d_output(output_size);

  // expand values according to counts
  expand(d_counts.begin(), d_counts.end(),
         d_values.begin(),
         d_output.begin());

  std::cout << "Expanding values according to counts" << std::endl;
  print(" counts ", d_counts);
  print(" values ", d_values);
  print(" output ", d_output);

  return 0;
}
