#include <thrust/device_vector.h>
#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/execution_policy.h>
#include <iostream>


// This example demonstrates the use of a view: a non-owning wrapper for an
// iterator range which presents a container-like interface to the user.
//
// For example, a view of a device_vector's data can be helpful when we wish to
// access that data from a device function. Even though device_vectors are not
// accessible from device functions, the range_view class allows us to access
// and manipulate its data as if we were manipulating a real container.

template<class Iterator>
class range_view
{
public:
  typedef Iterator iterator;
  typedef typename thrust::iterator_traits<iterator>::value_type value_type;
  typedef typename thrust::iterator_traits<iterator>::pointer pointer;
  typedef typename thrust::iterator_traits<iterator>::difference_type difference_type;
  typedef typename thrust::iterator_traits<iterator>::reference reference;

private:
  const iterator first;
  const iterator last;


public:
  __host__ __device__
  range_view(Iterator first, Iterator last)
      : first(first), last(last) {}
  __host__ __device__
  ~range_view() {}

  __host__ __device__
  difference_type size() const { return thrust::distance(first, last); }


  __host__ __device__
  reference operator[](difference_type n)
  {
    return *(first + n);
  }
  __host__ __device__
  const reference operator[](difference_type n) const
  {
    return *(first + n);
  }

  __host__ __device__
  iterator begin() 
  {
    return first;
  }
  __host__ __device__
  const iterator cbegin() const
  {
    return first;
  }
  __host__ __device__
  iterator end() 
  {
    return last;
  }
  __host__ __device__
  const iterator cend() const
  {
    return last;
  }


  __host__ __device__
  thrust::reverse_iterator<iterator> rbegin()
  {
    return thrust::reverse_iterator<iterator>(end());
  }
  __host__ __device__
  const thrust::reverse_iterator<const iterator> crbegin() const 
  {
    return thrust::reverse_iterator<const iterator>(cend());
  }
  __host__ __device__
  thrust::reverse_iterator<iterator> rend()
  {
    return thrust::reverse_iterator<iterator>(begin());
  }
  __host__ __device__
  const thrust::reverse_iterator<const iterator> crend() const 
  {
    return thrust::reverse_iterator<const iterator>(cbegin());
  }
  __host__ __device__
  reference front() 
  {
    return *begin();
  }
  __host__ __device__
  const reference front()  const
  {
    return *cbegin();
  }

  __host__ __device__
  reference back() 
  {
    return *end();
  }
  __host__ __device__
  const reference back()  const
  {
    return *cend();
  }

  __host__ __device__
  bool empty() const 
  {
    return size() == 0;
  }

};

// This helper function creates a range_view from iterator and the number of
// elements
template <class Iterator, class Size>
range_view<Iterator>
__host__ __device__
make_range_view(Iterator first, Size n)
{
  return range_view<Iterator>(first, first+n);
}

// This helper function creates a range_view from a pair of iterators
template <class Iterator>
range_view<Iterator>
__host__ __device__
make_range_view(Iterator first, Iterator last)
{
  return range_view<Iterator>(first, last);
}

// This helper function creates a range_view from a Vector
template <class Vector>
range_view<typename Vector::iterator>
__host__
make_range_view(Vector& v)
{
  return range_view<typename Vector::iterator>(v.begin(), v.end());
}


// This saxpy functor stores view of X, Y, Z array, and accesses them in
// vector-like way
template<class View1, class View2, class View3>
struct saxpy_functor : public thrust::unary_function<int,void>
{
  const float a;
  View1 x;
  View2 y;
  View3 z;

  __host__ __device__
  saxpy_functor(float _a, View1 _x, View2 _y, View3 _z)
      : a(_a), x(_x), y(_y), z(_z)
  {
  }

  __host__ __device__ 
  void operator()(int i) 
  {
    z[i] = a * x[i] + y[i];
  }
};

// saxpy function, which can either be called form host or device
// The views are passed by value
template<class View1, class View2, class View3>
__host__ __device__
void saxpy(float A, View1 X, View2 Y, View3 Z)
{
  // Z = A * X + Y
  const int size = static_cast<int>(X.size());
  thrust::for_each(thrust::device,
      thrust::make_counting_iterator(0),
      thrust::make_counting_iterator(size),
      saxpy_functor<View1,View2,View3>(A,X,Y,Z));
}

struct f1 : public thrust::unary_function<float,float>
{
  __host__ __device__
  float operator()(float x) const
  {
    return x*3;
  }
};

int main()
{
  using std::cout;
  using std::endl;

  // initialize host arrays
  float x[4] = {1.0, 1.0, 1.0, 1.0};
  float y[4] = {1.0, 2.0, 3.0, 4.0};
  float z[4] = {0.0};

  thrust::device_vector<float> X(x, x + 4);
  thrust::device_vector<float> Y(y, y + 4);
  thrust::device_vector<float> Z(z, z + 4);

  saxpy(
      2.0, 

      // make a range view of a pair of transform_iterators
      make_range_view(thrust::make_transform_iterator(X.cbegin(), f1()),
                      thrust::make_transform_iterator(X.cend(), f1())),

      // range view of normal_iterators
      make_range_view(Y.begin(), thrust::distance(Y.begin(), Y.end())),

      // range view of naked pointers
      make_range_view(Z.data().get(), 4));

  // print values from original device_vector<float> Z 
  // to ensure that range view was mapped to this vector
  for (std::size_t i = 0, n = Z.size(); i < n; ++i)
  {
    cout << "z[" << i << "]= " << Z[i] << endl;
  }


  return 0;
}

