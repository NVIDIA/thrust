#include <thrust/device_vector.h>
#include <thrust/execution_policy.h> // For thrust::device
#include <thrust/reduce.h>
#include <thrust/sequence.h>

#include <hip/hip_runtime.h>

#include <iostream>

// This example shows how to execute a Thrust device algorithm on an explicit
// CUDA stream. The simple program below fills a vector with the numbers
// [0, 1000) (thrust::sequence) and then performs a scan operation
// (thrust::inclusive_scan) on them. Both algorithms are executed on the same
// custom CUDA stream using the CUDA execution policies.
//
// Thrust provides two execution policies that accept CUDA streams that differ
// in when/if they synchronize the stream:
// 1. thrust::cuda::par.on(stream)
//      - `stream` will *always* be synchronized before an algorithm returns.
//      - This is the default `thrust::device` policy when compiling with the
//        CUDA device backend.
// 2. thrust::cuda::par_nosync.on(stream)
//      - `stream` will only be synchronized when necessary for correctness
//        (e.g., returning a result from `thrust::reduce`). This is a hint that
//        may be ignored by an algorithm's implementation.

int main()
{
  thrust::device_vector<int> d_vec(1000);

  // Create the stream:
  hipStream_t custom_stream;
  hipError_t err = hipStreamCreate(&custom_stream);
  if (err != hipSuccess)
  {
    std::cerr << "Error creating stream: " << hipGetErrorString(err) << "\n";
    return 1;
  }

  // Construct a new `nosync` execution policy with the custom stream
  auto nosync_exec_policy = thrust::cuda::par_nosync.on(custom_stream);

  // Fill the vector with sequential data.
  // This will execute using the custom stream and the stream will *not* be
  // synchronized before the function returns, meaning asynchronous work may
  // still be executing after returning and the contents of `d_vec` are
  // undefined. Synchronization is not needed here because the following
  // `inclusive_scan` is executed on the same stream and is therefore guaranteed
  // to be ordered after the `sequence`
  thrust::sequence(nosync_exec_policy, d_vec.begin(), d_vec.end());

  // Construct a new *synchronous* execution policy with the same custom stream
  auto sync_exec_policy = thrust::cuda::par.on(custom_stream);

  // Compute in-place inclusive sum scan of data in the vector.
  // This also executes in the custom stream, but the execution policy ensures
  // the stream is synchronized before the algorithm returns. This guarantees
  // there is no pending asynchronous work and the contents of `d_vec` are
  // immediately accessible.
  thrust::inclusive_scan(sync_exec_policy,
                         d_vec.cbegin(),
                         d_vec.cend(),
                         d_vec.begin());

  // This access is only valid because the stream has been synchronized
  int sum = d_vec.back();

  // Free the stream:
  err = hipStreamDestroy(custom_stream);
  if (err != hipSuccess)
  {
    std::cerr << "Error destroying stream: " << hipGetErrorString(err) << "\n";
    return 1;
  }

  // Print the sum:
  std::cout << "sum is " << sum << std::endl;

  return 0;
}
