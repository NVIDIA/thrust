#include "hip/hip_runtime.h"
#include <thrust/copy.h>
#include <thrust/sort.h>
#include <thrust/memory.h>
#include <thrust/system/cuda/memory.h>

#include <new> // for std::bad_alloc
#include <cassert>
#include <iostream>
#include <iterator>

// This example demonstrates how to intercept calls to malloc
// and free to implement a fallback for hipMalloc.
// When hipMalloc fails to allocate device memory the fallback_allocator
// attempts to allocate pinned host memory and then map the host buffer 
// into the device address space.  The fallback_allocator enables
// the GPU to process data sets that are larger than the device
// memory, albeit with a significantly reduced performance.


// initialize some unsorted data
__global__
void kernel(int * d_ptr, size_t N)
{
  size_t thread_id = blockDim.x * blockIdx.x + threadIdx.x;
  size_t grid_size = blockDim.x * gridDim.x;

  for (size_t i = thread_id; i < N; i += grid_size)
    d_ptr[i] = i % 1024;
}


// derive a simple allocator from cuda::dispatchable for using pinned host memory as a functional fallback
struct fallback_allocator : thrust::cuda::dispatchable<fallback_allocator> {};


// overload malloc on fallback_allocator to implement our special malloc 
// its job to is allocate host memory as a functional fallback when hipMalloc fails
void *malloc(fallback_allocator, std::size_t n)
{
  void *result = 0;

  // attempt to allocate device memory
  if(hipMalloc(&result, n) == hipSuccess)
  {
    std::cout << "  allocated " << n << " bytes of device memory" << std::endl;
  }
  else
  {
    // attempt to allocate pinned host memory
    void *h_ptr = 0;
    if(hipHostMalloc(&h_ptr, n) == hipSuccess)
    {
      // attempt to map host pointer into device memory space
      if(hipHostGetDevicePointer(&result, h_ptr, 0) == hipSuccess)
      {
        std::cout << "  allocated " << n << " bytes of pinned host memory (fallback successful)" << std::endl;
      }
      else
      {
        // attempt to deallocate buffer
        std::cout << "  failed to map host memory into device address space (fallback failed)" << std::endl;
        hipError_t error = hipHostFree(h_ptr);
        if(error)
        {
          throw thrust::system_error(error, thrust::cuda_category(), "hipHostFree failed");
        }

        result = 0;
      }
    }
    else
    {
      std::cout << "  failed to allocate " << n << " bytes of memory (fallback failed)" << std::endl;
    }
  }

  return result;
}


// overload free on fallback_allocator to implement our special free 
// its job to is inspect where the pointer lives and free it appropriately
template<typename Pointer>
void free(fallback_allocator, Pointer ptr)
{
  void *raw_ptr = thrust::raw_pointer_cast(ptr);

  // determine where memory resides
  hipPointerAttribute_t	attributes;

  if(hipPointerGetAttributes(&attributes, raw_ptr) == hipSuccess)
  {
    // free the memory in the appropriate way
    if(attributes.memoryType == hipMemoryTypeHost)
    {
      hipHostFree(raw_ptr);
    }
    else
    {
      hipFree(raw_ptr);
    }
  }
}


int main(void)
{
  // check whether device supports mapped host memory
  int device;
  hipGetDevice(&device);
  hipDeviceProp_t properties;
  hipGetDeviceProperties(&properties, device);

  fallback_allocator alloc;

  if(!properties.canMapHostMemory)
  {
    std::cout << "Device #" << device 
              << " [" << properties.name << "] does not support memory mapping" << std::endl;
    return 0;
  }
  else
  {
    std::cout << "Testing fallback_allocator on device #" << device 
              << " [" << properties.name << "] with " 
              << properties.totalGlobalMem << " bytes of device memory" << std::endl;
  }

  try
  {
    size_t one_million = 1 << 20;
    size_t one_billion = 1 << 30;

    for(size_t n = one_million; n < one_billion; n *= 2)
    {
      // TODO ideally we'd use the fallback_allocator in the vector too
      //thrust::cuda::vector<int, fallback_allocator> d_vec(n);

      std::cout << "attempting to sort " << n << " values" << std::endl;

      // use our special malloc to allocate
      int *raw_ptr = (int *) malloc(alloc, n * sizeof(int));

      if(raw_ptr)
      {
        kernel<<<100,256>>>(raw_ptr, n); // generate unsorted values

        thrust::cuda::pointer<int> begin = thrust::cuda::pointer<int>(raw_ptr);
        thrust::cuda::pointer<int> end   = begin + n;

        // sort the data using our special allocator
        // if temporary memory is required during the sort,
        // our versions of malloc & free will be called
        thrust::sort(alloc, begin, end);

        free(alloc, raw_ptr);
      }
    }
  }
  catch(std::bad_alloc)
  {
    std::cout << "Caught std::bad_alloc" << std::endl;
  }

  return 0;
}

