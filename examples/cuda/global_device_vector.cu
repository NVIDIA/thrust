#include <thrust/detail/config.h>
#include <thrust/device_vector.h>

// If you create a global `thrust::device_vector` with the default allocator,
// you'll get an error during program termination when the memory of the vector
// is freed, as the CUDA runtime cannot be used during program termination.
//
// To get around this, you can create your own allocator which ignores
// deallocation failures that occur because the CUDA runtime is shut down.

extern "C" hipError_t cudaFreeIgnoreShutdown(void* ptr) {
  hipError_t const err = hipFree(ptr);
  if (hipSuccess == err || hipErrorDeinitialized == err)
    return hipSuccess;
  return err; 
}

typedef thrust::system::cuda::detail::cuda_memory_resource<
  hipMalloc, 
  cudaFreeIgnoreShutdown,
  thrust::cuda::pointer<void>
> device_ignore_shutdown_memory_resource;

#if THRUST_CPP_DIALECT >= 2011
  template <typename T>
  using device_ignore_shutdown_allocator = 
    thrust::mr::stateless_resource_allocator<
      T,
      thrust::device_ptr_memory_resource<device_ignore_shutdown_memory_resource>
    >;
    
  thrust::device_vector<double, device_ignore_shutdown_allocator<double>> d;
#else
  thrust::device_vector<
    double, 
    thrust::mr::stateless_resource_allocator<
      double,
      thrust::device_ptr_memory_resource<device_ignore_shutdown_memory_resource>
    > 
  > d;
#endif

int main() {
  d.resize(25);
}

