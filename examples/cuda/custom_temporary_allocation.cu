#include <thrust/system/cuda/vector.h>
#include <thrust/system/cuda/execution_policy.h>
#include <thrust/host_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/pair.h>
#include <cstdlib>
#include <iostream>
#include <sstream>
#include <map>
#include <cassert>

// This example demonstrates how to control how Thrust allocates temporary
// storage during algorithms such as thrust::sort. The idea will be to create a
// simple cache of allocations to search when temporary storage is requested.
// If a hit is found in the cache, we quickly return the cached allocation
// instead of resorting to the more expensive thrust::cuda::malloc.

// Note: Thrust now has its own caching allocator layer; if you just need a
// caching allocator, you ought to use that. This example is still useful
// as a demonstration of how to use a Thrust custom allocator.

// Note: this implementation cached_allocator is not thread-safe. If multiple
// (host) threads use the same cached_allocator then they should gain exclusive
// access to the allocator before accessing its methods.

struct not_my_pointer
{
  not_my_pointer(void* p)
    : message()
  {
    std::stringstream s;
    s << "Pointer `" << p << "` was not allocated by this allocator.";
    message = s.str();
  }

  virtual ~not_my_pointer() {}

  virtual const char* what() const
  {
    return message.c_str();
  }

private:
  std::string message;
};

// A simple allocator for caching hipMalloc allocations.
struct cached_allocator
{
  typedef char value_type;

  cached_allocator() {}

  ~cached_allocator()
  {
    free_all();
  }

  char *allocate(std::ptrdiff_t num_bytes)
  {
    std::cout << "cached_allocator::allocate(): num_bytes == "
              << num_bytes
              << std::endl;

    char *result = 0;

    // Search the cache for a free block.
    free_blocks_type::iterator free_block = free_blocks.find(num_bytes);

    if (free_block != free_blocks.end())
    {
      std::cout << "cached_allocator::allocate(): found a free block"
                << std::endl;

      result = free_block->second;

      // Erase from the `free_blocks` map.
      free_blocks.erase(free_block);
    }
    else
    {
      // No allocation of the right size exists, so create a new one with
      // `thrust::cuda::malloc`.
      try
      {
        std::cout << "cached_allocator::allocate(): allocating new block"
                  << std::endl;

        // Allocate memory and convert the resulting `thrust::cuda::pointer` to
        // a raw pointer.
        result = thrust::cuda::malloc<char>(num_bytes).get();
      }
      catch (std::runtime_error&)
      {
        throw;
      }
    }

    // Insert the allocated pointer into the `allocated_blocks` map.
    allocated_blocks.insert(std::make_pair(result, num_bytes));

    return result;
  }

  void deallocate(char *ptr, size_t)
  {
    std::cout << "cached_allocator::deallocate(): ptr == "
              << reinterpret_cast<void*>(ptr) << std::endl;

    // Erase the allocated block from the allocated blocks map.
    allocated_blocks_type::iterator iter = allocated_blocks.find(ptr);

    if (iter == allocated_blocks.end())
      throw not_my_pointer(reinterpret_cast<void*>(ptr));

    std::ptrdiff_t num_bytes = iter->second;
    allocated_blocks.erase(iter);

    // Insert the block into the free blocks map.
    free_blocks.insert(std::make_pair(num_bytes, ptr));
  }

private:
  typedef std::multimap<std::ptrdiff_t, char*> free_blocks_type;
  typedef std::map<char*, std::ptrdiff_t>      allocated_blocks_type;

  free_blocks_type      free_blocks;
  allocated_blocks_type allocated_blocks;

  void free_all()
  {
    std::cout << "cached_allocator::free_all()" << std::endl;

    // Deallocate all outstanding blocks in both lists.
    for ( free_blocks_type::iterator i = free_blocks.begin()
        ; i != free_blocks.end()
        ; ++i)
    {
      // Transform the pointer to cuda::pointer before calling cuda::free.
      thrust::cuda::free(thrust::cuda::pointer<char>(i->second));
    }

    for( allocated_blocks_type::iterator i = allocated_blocks.begin()
       ; i != allocated_blocks.end()
       ; ++i)
    {
      // Transform the pointer to cuda::pointer before calling cuda::free.
      thrust::cuda::free(thrust::cuda::pointer<char>(i->first));
    }
  }
};

int main()
{
  std::size_t num_elements = 32768;

  thrust::host_vector<int> h_input(num_elements);

  // Generate random input.
  thrust::generate(h_input.begin(), h_input.end(), rand);

  thrust::cuda::vector<int> d_input = h_input;
  thrust::cuda::vector<int> d_result(num_elements);

  std::size_t num_trials = 5;

  cached_allocator alloc;

  for (std::size_t i = 0; i < num_trials; ++i)
  {
    d_result = d_input;

    // Pass alloc through cuda::par as the first parameter to sort
    // to cause allocations to be handled by alloc during sort.
    thrust::sort(thrust::cuda::par(alloc), d_result.begin(), d_result.end());

    // Ensure the result is sorted.
    assert(thrust::is_sorted(d_result.begin(), d_result.end()));
  }

  return 0;
}

