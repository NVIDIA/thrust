#include "hip/hip_runtime.h"
#include <thrust/detail/config.h>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/system/cuda/execution_policy.h>
#include <cassert>

#if THRUST_CPP_DIALECT >= 2011
#include <future>
#endif

// This example demonstrates two ways to achieve algorithm invocations that are asynchronous with
// the calling thread.
//
// The first method wraps a call to thrust::reduce inside a __global__ function. Since __global__ function
// launches are asynchronous with the launching thread, this achieves asynchrony. The result of the reduction
// is stored to a pointer to CUDA global memory. The calling thread waits for the result of the reduction to 
// be ready by synchronizing with the CUDA stream on which the __global__ function is launched.
//
// The second method uses the C++11 library function, std::async, to create concurrency. The lambda function
// given to std::async returns the result of thrust::reduce to a std::future. The calling thread can use the
// std::future to wait for the result of the reduction. This method requires a compiler which supports
// C++11-capable language and library constructs.

template<typename Iterator, typename T, typename BinaryOperation, typename Pointer>
__global__ void reduce_kernel(Iterator first, Iterator last, T init, BinaryOperation binary_op, Pointer result)
{
  *result = thrust::reduce(thrust::cuda::par, first, last, init, binary_op);
}

int main()
{
  size_t n = 1 << 20;
  thrust::device_vector<unsigned int> data(n, 1);
  thrust::device_vector<unsigned int> result(1, 0);

  // method 1: call thrust::reduce from an asynchronous CUDA kernel launch

  // create a CUDA stream 
  hipStream_t s;
  hipStreamCreate(&s);

  // launch a CUDA kernel with only 1 thread on our stream
  reduce_kernel<<<1,1,0,s>>>(data.begin(), data.end(), 0, thrust::plus<int>(), result.data());

  // wait for the stream to finish
  hipStreamSynchronize(s);

  // our result should be ready
  assert(result[0] == n);

  hipStreamDestroy(s);

  // reset the result
  result[0] = 0;

#if THRUST_CPP_DIALECT >= 2011
  // method 2: use std::async to create asynchrony

  // copy all the algorithm parameters
  auto begin        = data.begin();
  auto end          = data.end();
  unsigned int init = 0;
  auto binary_op    = thrust::plus<unsigned int>();

  // std::async captures the algorithm parameters by value
  // use std::launch::async to ensure the creation of a new thread
  std::future<unsigned int> future_result = std::async(std::launch::async, [=]
  {
    return thrust::reduce(begin, end, init, binary_op);
  });

  // wait on the result and check that it is correct
  assert(future_result.get() == n);
#endif

  return 0;
}

