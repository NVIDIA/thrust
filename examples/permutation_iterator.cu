#include <thrust/iterator/permutation_iterator.h>
#include <thrust/reduce.h>
#include <thrust/device_vector.h>
#include <iostream>

// this example fuses a gather operation with a reduction for
// greater efficiency than separate gather() and reduce() calls

int main(void)
{
    // gather locations
    thrust::device_vector<int> map(4);
    map[0] = 3;
    map[1] = 1;
    map[2] = 0;
    map[3] = 5;

    // array to gather from
    thrust::device_vector<int> source(6);
    source[0] = 10;
    source[1] = 20;
    source[2] = 30;
    source[3] = 40;
    source[4] = 50;
    source[5] = 60;

    // fuse gather with reduction: 
    //   sum = source[map[0]] + source[map[1]] + ...
    auto begin = thrust::make_permutation_iterator(source.begin(), map.begin());
    auto end = thrust::make_permutation_iterator(source.begin(), map.end());
    int sum = thrust::reduce(begin, end);

    // print sum
    std::cout << "sum is " << sum << std::endl;

    return 0;
}
