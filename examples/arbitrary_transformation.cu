#include <thrust/for_each.h>
#include <thrust/device_vector.h>
#include <thrust/iterator/zip_iterator.h>
#include <iostream>

// This example shows how to implement an arbitrary transformation of
// the form output[i] = F(first[i], second[i], third[i], ... ).
// In this example, we use a function with 3 inputs and 1 output.
//
// Iterators for all four vectors (3 inputs + 1 output) are "zipped"
// into a single sequence of tuples with the zip_iterator.
//  
// The arbitrary_functor receives a tuple that contains four elements,
// which are references to values in each of the four sequences. When we
// access the tuple 't' with the get() function,
//      get<0>(t) returns a reference to A[i],
//      get<1>(t) returns a reference to B[i],
//      get<2>(t) returns a reference to C[i],
//      get<3>(t) returns a reference to D[i].
//
// In this example, we can implement the transformation,
//      D[i] = A[i] + B[i] * C[i];
// by invoking arbitrary_functor() on each of the tuples using for_each.
//
// Note that we could extend this example to implement functions with an
// arbitrary number of input arguments by zipping more sequence together.
// With the same approach we can have multiple *output* sequences, if we 
// wanted to implement something like
//      D[i] = A[i] + B[i] * C[i];
//      E[i] = A[i] + B[i] + C[i];
//
// The possibilities are endless! :)

struct arbitrary_functor
{
    template <typename Tuple>
    __host__ __device__
    void operator()(Tuple t)
    {
        // D[i] = A[i] + B[i] * C[i];
        thrust::get<3>(t) = thrust::get<0>(t) + thrust::get<1>(t) * thrust::get<2>(t);
    }
};


int main(void)
{
    // allocate storage
    thrust::device_vector<float> A(5);
    thrust::device_vector<float> B(5);
    thrust::device_vector<float> C(5);
    thrust::device_vector<float> D(5);

    // initialize input vectors
    A[0] = 3;  B[0] = 6;  C[0] = 2; 
    A[1] = 4;  B[1] = 7;  C[1] = 5; 
    A[2] = 0;  B[2] = 2;  C[2] = 7; 
    A[3] = 8;  B[3] = 1;  C[3] = 4; 
    A[4] = 2;  B[4] = 8;  C[4] = 3; 

    // apply the transformation
    auto begin = thrust::make_zip_iterator(thrust::make_tuple(A.begin(), B.begin(), C.begin(), D.begin()));
    auto end = thrust::make_zip_iterator(thrust::make_tuple(A.end(),   B.end(),   C.end(),   D.end()));
    thrust::for_each(begin, end, arbitrary_functor());

    // print the output
    for(int i = 0; i < 5; i++)
        std::cout << A[i] << " + " << B[i] << " * " << C[i] << " = " << D[i] << std::endl;
}

