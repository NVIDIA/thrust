#include "hip/hip_runtime.h"
#include <thrust/random.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/functional.h>
#include <thrust/transform_reduce.h>

#include <iostream>
#include <iomanip>
#include <cmath>

// we could vary M & N to find the perf sweet spot

__host__ __device__
unsigned int hash(unsigned int a)
{
    a = (a+0x7ed55d16) + (a<<12);
    a = (a^0xc761c23c) ^ (a>>19);
    a = (a+0x165667b1) + (a<<5);
    a = (a+0xd3a2646c) ^ (a<<9);
    a = (a+0xfd7046c5) + (a<<3);
    a = (a^0xb55a4f09) ^ (a>>16);
    return a;
}

struct estimate_pi : public thrust::unary_function<unsigned int,float>
{
  __host__ __device__
  float operator()(unsigned int thread_id)
  {
    float sum = 0;
    unsigned int N = 10000; // samples per thread

    unsigned int seed = hash(thread_id);

    // seed a random number generator
    thrust::default_random_engine rng(seed);

    // create a mapping from random numbers to [0,1)
    thrust::uniform_real_distribution<float> u01(0,1);

    // take N samples in a quarter circle
    for(unsigned int i = 0; i < N; ++i)
    {
      // draw a sample from the unit square
      float x = u01(rng);
      float y = u01(rng);

      // measure distance from the origin
      float dist = sqrtf(x*x + y*y);

      // add 1.0f if (u0,u1) is inside the quarter circle
      if(dist <= 1.0f)
        sum += 1.0f;
    }

    // multiply by 4 to get the area of the whole circle
    sum *= 4.0f;

    // divide by N
    return sum / static_cast<float>(N);
  }
};

int main(void)
{
  // use 30K independent seeds
  int M = 30000;

  float estimate = thrust::transform_reduce(thrust::counting_iterator<int>(0),
                                            thrust::counting_iterator<int>(M),
                                            estimate_pi(),
                                            0.0f,
                                            thrust::plus<float>());
  estimate /= static_cast<float>(M);

  std::cout << std::setprecision(3);
  std::cout << "pi is approximately " << estimate << std::endl;

  return 0;
}

