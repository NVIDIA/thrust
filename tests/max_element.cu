#include <unittest/unittest.h>
#include <thrust/extrema.h>
#include <thrust/iterator/retag.h>

template <class Vector>
void TestMaxElementSimple(void)
{
    typedef typename Vector::value_type T;

    Vector data(6);
    data[0] = 3;
    data[1] = 5;
    data[2] = 1;
    data[3] = 2;
    data[4] = 5;
    data[5] = 1;

    ASSERT_EQUAL( *thrust::max_element(data.begin(), data.end()), 5);
    ASSERT_EQUAL( thrust::max_element(data.begin(), data.end()) - data.begin(), 1);
    
    ASSERT_EQUAL( *thrust::max_element(data.begin(), data.end(), thrust::greater<T>()), 1);
    ASSERT_EQUAL( thrust::max_element(data.begin(), data.end(), thrust::greater<T>()) - data.begin(), 2);
}
DECLARE_VECTOR_UNITTEST(TestMaxElementSimple);

template<typename T>
void TestMaxElement(const size_t n)
{
    thrust::host_vector<T> h_data = unittest::random_samples<T>(n);
    thrust::device_vector<T> d_data = h_data;

    typename thrust::host_vector<T>::iterator   h_max = thrust::max_element(h_data.begin(), h_data.end());
    typename thrust::device_vector<T>::iterator d_max = thrust::max_element(d_data.begin(), d_data.end());

    ASSERT_EQUAL(h_max - h_data.begin(), d_max - d_data.begin());
    
    typename thrust::host_vector<T>::iterator   h_min = thrust::max_element(h_data.begin(), h_data.end(), thrust::greater<T>());
    typename thrust::device_vector<T>::iterator d_min = thrust::max_element(d_data.begin(), d_data.end(), thrust::greater<T>());

    ASSERT_EQUAL(h_min - h_data.begin(), d_min - d_data.begin());
}
DECLARE_VARIABLE_UNITTEST(TestMaxElement);


template<typename ForwardIterator>
ForwardIterator max_element(my_system &system, ForwardIterator first, ForwardIterator)
{
    system.validate_dispatch();
    return first;
}

void TestMaxElementDispatchExplicit()
{
    thrust::device_vector<int> vec(1);

    my_system sys(0);
    thrust::max_element(sys, vec.begin(), vec.end());

    ASSERT_EQUAL(true, sys.is_valid());
}
DECLARE_UNITTEST(TestMaxElementDispatchExplicit);


template<typename ForwardIterator>
ForwardIterator max_element(my_tag, ForwardIterator first, ForwardIterator)
{
    *first = 13;
    return first;
}

void TestMaxElementDispatchImplicit()
{
    thrust::device_vector<int> vec(1);

    thrust::max_element(thrust::retag<my_tag>(vec.begin()),
                        thrust::retag<my_tag>(vec.end()));

    ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestMaxElementDispatchImplicit);

