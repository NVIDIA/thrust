#include "hip/hip_runtime.h"
#include <unittest/testframework.h>
#include <thrust/system/cuda/memory.h>
#include <hip/hip_runtime.h>
#include "testframework.h"

__global__ void dummy_kernel() {}

bool binary_exists_for_current_device()
{
  // check against the dummy_kernel
  // if we're unable to get the attributes, then
  // we didn't compile a binary compatible with the current device
  hipFuncAttributes attr;
  hipError_t error = hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(dummy_kernel));
  return error == hipSuccess;
}

void list_devices(void)
{
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  if(deviceCount == 0)
  {
    std::cout << "There is no device supporting CUDA" << std::endl;
  }
  
  int selected_device;
  hipGetDevice(&selected_device);
  
  for (int dev = 0; dev < deviceCount; ++dev)
  {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    
    if(dev == 0)
    {
      if(deviceProp.major == 9999 && deviceProp.minor == 9999)
        std::cout << "There is no device supporting CUDA." << std::endl;
      else if(deviceCount == 1)
        std::cout << "There is 1 device supporting CUDA" << std:: endl;
      else
        std::cout << "There are " << deviceCount <<  " devices supporting CUDA" << std:: endl;
    }
    
    std::cout << "\nDevice " << dev << ": \"" << deviceProp.name << "\"";
    if(dev == selected_device)
      std::cout << "  [SELECTED]";
    std::cout << std::endl;
    
    std::cout << "  Major revision number:                         " << deviceProp.major << std::endl;
    std::cout << "  Minor revision number:                         " << deviceProp.minor << std::endl;
    std::cout << "  Total amount of global memory:                 " << deviceProp.totalGlobalMem << " bytes" << std::endl;
  }
  std::cout << std::endl;
}

// provide next, which c++03 doesn't have
template<typename Iterator> Iterator my_next(Iterator iter)
{
  return ++iter;
}


std::vector<int> CUDATestDriver::target_devices(const ArgumentMap &kwargs)
{
  std::vector<int> result;
  
  // by default, test all devices in the system (device id -1)
  int device_id = kwargs.count("device") ? atoi(kwargs.find("device")->second.c_str()) : -1;
  
  if(device_id < 0)
  {
    // target all devices in the system
    int count = 0;
    hipGetDeviceCount(&count);
    
    result.resize(count);
    // XXX iota is not available in c++03
    for(int i = 0; i < count; ++i)
      result[i] = i;
  }
  else
  {
    // target the specified device
    result = std::vector<int>(1,device_id);
  }
  
  return result;
}

bool CUDATestDriver::check_cuda_error(bool concise)
{
  hipError_t error = hipGetLastError();
  if(error)
  {
    if(!concise)
    {
      std::cout << "[ERROR] CUDA Error detected before running tests: [";
      std::cout << std::string(hipGetErrorString(error));
      std::cout << "]" << std::endl;
    }
  } 

  return error;
}

bool CUDATestDriver::post_test_sanity_check(const UnitTest &test, bool concise)
{
  hipError_t error = hipGetLastError();
  if(error && error != hipErrorOutOfMemory)
  {
    if(!concise)
    {
      std::cout << "\t[ERROR] CUDA Error detected after running " << test.name << ": [";
      std::cout << std::string(hipGetErrorString(error));
      std::cout << "]" << std::endl;
    }
  }

  return error == hipSuccess;
}
  
bool CUDATestDriver::run_tests(const ArgumentSet &args, const ArgumentMap &kwargs)
{
  bool verbose = kwargs.count("verbose");
  bool concise = kwargs.count("concise");

  if(verbose && concise)
  {
    std::cout << "--verbose and --concise cannot be used together" << std::endl;
    exit(EXIT_FAILURE);
  }

  // check error status before doing anything
  if(check_cuda_error(concise)) return false;
  
  bool result = true;

  if(kwargs.count("verbose"))
  {
    list_devices();
  }
  
  // figure out which devices to target
  std::vector<int> devices = target_devices(kwargs);
  
  // target each device
  for(std::vector<int>::iterator device = devices.begin();
      device != devices.end();
      ++device)
  {
    // set the device
    hipSetDevice(*device);

    // check if a binary exists for this device
    // if none exists, skip the device silently unless this is the only one we're targeting
    if(devices.size() > 1 && !binary_exists_for_current_device())
    {
      continue;     
    }

    if(!concise)
    {
      // note which device we're testing
      hipDeviceProp_t deviceProp;
      hipGetDeviceProperties(&deviceProp, *device);
      
      std::cout << "Testing Device " << *device << ": \"" << deviceProp.name << "\"" << std::endl;
    }

    // check error status before running any tests
    if(check_cuda_error(concise)) return false;
    
    // run tests
    result &= UnitTestDriver::run_tests(args, kwargs);
    
    if(!concise && my_next(device) != devices.end())
    {
      // provide some separation between the output of separate tests
      std::cout << std::endl;
    }
  }
  
  return result;
}

int CUDATestDriver::current_device_architecture() const
{
  int current = -1;
  hipGetDevice(&current);
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, current);

  return 100 * deviceProp.major + 10 * deviceProp.minor;
}

UnitTestDriver &driver_instance(thrust::system::cuda::tag)
{
  static CUDATestDriver s_instance;
  return s_instance;
}

