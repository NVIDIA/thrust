#include <unittest/unittest.h>

#include <thrust/detail/seq.h>
#include <thrust/system/cpp/detail/par.h>
#include <thrust/system/omp/detail/par.h>
#include <thrust/system/tbb/detail/par.h>

#if THRUST_DEVICE_SYSTEM == THRUST_DEVICE_SYSTEM_CUDA
#include <thrust/system/cuda/detail/par.h>
#endif

template<typename T>
struct test_allocator_t
{
};

test_allocator_t<int> test_allocator = test_allocator_t<int>();
const test_allocator_t<int> const_test_allocator = test_allocator_t<int>();

struct test_memory_resource_t THRUST_FINAL : thrust::mr::memory_resource<>
{
    void * do_allocate(std::size_t size, std::size_t) THRUST_OVERRIDE
    {
        return reinterpret_cast<void *>(size);
    }

    void do_deallocate(void * ptr, std::size_t size, std::size_t) THRUST_OVERRIDE
    {
        ASSERT_EQUAL(ptr, reinterpret_cast<void *>(size));
    }
} test_memory_resource;

template<typename Policy, template <typename> class CRTPBase>
struct policy_info
{
    typedef Policy policy;

    template<template <typename, template <typename> class> class Template, typename Argument>
    struct apply_base_second
    {
        typedef Template<Argument, CRTPBase> type;
    };
};

template<typename PolicyInfo>
struct TestAllocatorAttachment
{
    template<typename Expected, typename T>
    static void assert_correct(T)
    {
        ASSERT_EQUAL(
            (thrust::detail::is_same<
                T,
                typename PolicyInfo::template apply_base_second<
                    thrust::detail::execute_with_allocator,
                    Expected
                >::type
            >::value), true);
    }

    template<typename ExpectedResource, typename T>
    static void assert_npa_correct(T)
    {
        ASSERT_EQUAL(
            (thrust::detail::is_same<
                T,
                typename PolicyInfo::template apply_base_second<
                    thrust::detail::execute_with_allocator,
                    thrust::mr::allocator<
                        thrust::detail::max_align_t,
                        ExpectedResource
                    >
                >::type
            >::value), true);
    }

    template<typename Policy>
    void test_temporary_allocation_valid(Policy policy)
    {
        using thrust::detail::get_temporary_buffer;

        return_temporary_buffer(
            policy,
            get_temporary_buffer<int>(
                policy,
                123
            ).first,
            123
        );
    }

    void operator()()
    {
        typename PolicyInfo::policy policy;

        // test correctness of attachment
        assert_correct<test_allocator_t<int> >(policy(test_allocator_t<int>()));
        assert_correct<test_allocator_t<int>&>(policy(test_allocator));
        assert_correct<test_allocator_t<int> >(policy(const_test_allocator));

        assert_npa_correct<test_memory_resource_t>(policy(&test_memory_resource));

        // test whether the resulting policy is actually usable
        // a real allocator is necessary here, unlike above
        std::allocator<int> alloc;
        const std::allocator<int> const_alloc;

        test_temporary_allocation_valid(policy(std::allocator<int>()));
        test_temporary_allocation_valid(policy(alloc));
        test_temporary_allocation_valid(policy(const_alloc));
        test_temporary_allocation_valid(policy(&test_memory_resource));

        #if THRUST_CPP_DIALECT >= 2011
        test_temporary_allocation_valid(policy(std::allocator<int>()).after(1));
        test_temporary_allocation_valid(policy(alloc).after(1));
        test_temporary_allocation_valid(policy(const_alloc).after(1));
        #endif
    }
};

typedef policy_info<
    thrust::detail::seq_t,
    thrust::system::detail::sequential::execution_policy
> sequential_info;
typedef policy_info<
    thrust::system::cpp::detail::par_t,
    thrust::system::cpp::detail::execution_policy
> cpp_par_info;
typedef policy_info<
    thrust::system::omp::detail::par_t,
    thrust::system::omp::detail::execution_policy
> omp_par_info;
typedef policy_info<
    thrust::system::tbb::detail::par_t,
    thrust::system::tbb::detail::execution_policy
> tbb_par_info;

#if THRUST_DEVICE_SYSTEM == THRUST_DEVICE_SYSTEM_CUDA
typedef policy_info<
    thrust::system::cuda::detail::par_t,
    thrust::cuda_cub::execute_on_stream_base
> cuda_par_info;
#endif

SimpleUnitTest<
    TestAllocatorAttachment,
    unittest::type_list<
        sequential_info,
#if THRUST_DEVICE_SYSTEM == THRUST_DEVICE_SYSTEM_CUDA
        cuda_par_info,
#endif
        cpp_par_info,
        omp_par_info,
        tbb_par_info
    >
> TestAllocatorAttachmentInstance;
