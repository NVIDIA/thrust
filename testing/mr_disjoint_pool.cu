#include <unittest/unittest.h>

#include <thrust/detail/config.h>
#include <thrust/mr/disjoint_pool.h>
#include <thrust/mr/new.h>

#if THRUST_CPP_DIALECT >= 2011
#include <thrust/mr/disjoint_sync_pool.h>
#endif

struct alloc_id
{
    std::size_t id;
    std::size_t size;
    std::size_t alignment;
    std::size_t offset;

    __host__ __device__
    bool operator==(const alloc_id & other) const
    {
        return id == other.id && size == other.size && alignment == other.alignment;
    }

    alloc_id operator+(std::size_t size) const
    {
        alloc_id ret;
        ret.id = id;
        ret.size = size;
        ret.alignment = alignment;
        ret.offset = size;
        return ret;
    }
};

namespace thrust { namespace detail {
template<>
struct pointer_traits<alloc_id>
{
    template<typename>
    struct rebind
    {
        typedef alloc_id other;
    };

    // implemented for the purposes of alignment test in disjoint pool's do_deallocate
    static void * get(const alloc_id & id)
    {
        return reinterpret_cast<void *>(id.alignment);
    }
};
}}

class dummy_resource THRUST_FINAL : public thrust::mr::memory_resource<alloc_id>
{
public:
    dummy_resource() : id_to_allocate(0), id_to_deallocate(0)
    {
    }

    ~dummy_resource()
    {
        ASSERT_EQUAL(id_to_allocate, 0u);
        ASSERT_EQUAL(id_to_deallocate, 0u);
    }

    virtual alloc_id do_allocate(std::size_t bytes, std::size_t alignment) THRUST_OVERRIDE
    {
        ASSERT_EQUAL(static_cast<bool>(id_to_allocate), true);

        alloc_id ret;
        ret.id = id_to_allocate;
        ret.size = bytes;
        ret.alignment = alignment;

        id_to_allocate = 0;

        return ret;
    }

    virtual void do_deallocate(alloc_id p, std::size_t bytes, std::size_t alignment) THRUST_OVERRIDE
    {
        ASSERT_EQUAL(p.size, bytes);
        ASSERT_EQUAL(p.alignment, alignment);

        if (id_to_deallocate != 0)
        {
            ASSERT_EQUAL(p.id, id_to_deallocate);
            id_to_deallocate = 0;
        }
    }

    std::size_t id_to_allocate;
    std::size_t id_to_deallocate;
};

template<template<typename, typename> class PoolTemplate>
void TestDisjointPool()
{
    dummy_resource upstream;
    thrust::mr::new_delete_resource bookkeeper;

    typedef PoolTemplate<
        dummy_resource,
        thrust::mr::new_delete_resource
    > Pool;

    thrust::mr::pool_options opts = Pool::get_default_options();
    opts.cache_oversized = false;

    // avoid having the destructor run when an assertion failure is raised
    // (the destructor will try to release, which in turn calls do_deallocate,
    // which may fail with an assertion failure exception...)
    Pool * pool = new Pool(&upstream, &bookkeeper, opts);

    upstream.id_to_allocate = 1;

    // first allocation
    alloc_id a1 = pool->do_allocate(12, THRUST_MR_DEFAULT_ALIGNMENT);
    ASSERT_EQUAL(a1.id, 1u);

    // due to chunking, the above allocation should be enough for the next one too
    alloc_id a2 = pool->do_allocate(16, THRUST_MR_DEFAULT_ALIGNMENT);
    ASSERT_EQUAL(a2.id, 1u);

    // deallocating and allocating back should give the same resource back
    pool->do_deallocate(a1, 12, THRUST_MR_DEFAULT_ALIGNMENT);
    alloc_id a3 = pool->do_allocate(12, THRUST_MR_DEFAULT_ALIGNMENT);
    ASSERT_EQUAL(a1.id, a3.id);
    ASSERT_EQUAL(a1.size, a3.size);
    ASSERT_EQUAL(a1.alignment, a3.alignment);
    ASSERT_EQUAL(a1.offset, a3.offset);

    // allocating over-aligned memory should give non-cached results
    upstream.id_to_allocate = 2;
    alloc_id a4 = pool->do_allocate(32, THRUST_MR_DEFAULT_ALIGNMENT * 2);
    ASSERT_EQUAL(a4.id, 2u);
    ASSERT_EQUAL(a4.size, 32u);
    ASSERT_EQUAL(a4.alignment, (std::size_t)THRUST_MR_DEFAULT_ALIGNMENT * 2);

    // and deallocating it should return it back to upstream
    upstream.id_to_deallocate = 2;
    pool->do_deallocate(a4, 32u, THRUST_MR_DEFAULT_ALIGNMENT * 2);
    ASSERT_EQUAL(upstream.id_to_deallocate, 0u);

    // release actually returns properly sized memory to upstream
    upstream.id_to_deallocate = 1;
    pool->release();
    ASSERT_EQUAL(upstream.id_to_deallocate, 0u);

    // and does the same for oversized/overaligned memory
    upstream.id_to_allocate = 3;
    alloc_id a5 = pool->do_allocate(1024, THRUST_MR_DEFAULT_ALIGNMENT * 2);
    ASSERT_EQUAL(upstream.id_to_allocate, 0u);
    ASSERT_EQUAL(a5.id, 3u);

    upstream.id_to_deallocate = 3;
    pool->release();
    ASSERT_EQUAL(upstream.id_to_deallocate, 0u);

    // and after that, the formerly cached memory isn't used anymore,
    // so new memory from upstream is returned back
    upstream.id_to_allocate = 4;
    alloc_id a6 = pool->do_allocate(16, THRUST_MR_DEFAULT_ALIGNMENT);
    ASSERT_EQUAL(upstream.id_to_allocate, 0u);
    ASSERT_EQUAL(a6.id, 4u);

    // destruction also returns memory
    upstream.id_to_deallocate = 4;

    // actually destroy the pool; reasons why RAII is not used outlined at the beginning
    // of this function
    delete pool;
    ASSERT_EQUAL(upstream.id_to_deallocate, 0u);
}

void TestDisjointUnsynchronizedPool()
{
    TestDisjointPool<thrust::mr::disjoint_unsynchronized_pool_resource>();
}
DECLARE_UNITTEST(TestDisjointUnsynchronizedPool);

#if THRUST_CPP_DIALECT >= 2011
void TestDisjointSynchronizedPool()
{
    TestDisjointPool<thrust::mr::disjoint_synchronized_pool_resource>();
}
DECLARE_UNITTEST(TestDisjointSynchronizedPool);
#endif

template<template<typename, typename> class PoolTemplate>
void TestDisjointPoolCachingOversized()
{
    dummy_resource upstream;
    thrust::mr::new_delete_resource bookkeeper;

    typedef PoolTemplate<
        dummy_resource,
        thrust::mr::new_delete_resource
    > Pool;

    thrust::mr::pool_options opts = Pool::get_default_options();
    opts.cache_oversized = true;
    opts.largest_block_size = 1024;

    Pool pool(&upstream, &bookkeeper, opts);

    upstream.id_to_allocate = 1;
    alloc_id a1 = pool.do_allocate(2048, 32);
    ASSERT_EQUAL(a1.id, 1u);

    upstream.id_to_allocate = 2;
    alloc_id a2 = pool.do_allocate(64, 32);
    ASSERT_EQUAL(a2.id, 2u);

    pool.do_deallocate(a2, 64, 32);
    pool.do_deallocate(a1, 2048, 32);

    // make sure a good fit is used from the cache
    alloc_id a3 = pool.do_allocate(32, 32);
    ASSERT_EQUAL(a3.id, 2u);

    alloc_id a4 = pool.do_allocate(1024, 32);
    ASSERT_EQUAL(a4.id, 1u);

    pool.do_deallocate(a4, 1024, 32);

    // make sure that a new block is allocated when there's nothing cached with
    // the required alignment
    upstream.id_to_allocate = 3;
    alloc_id a5 = pool.do_allocate(32, 64);
    ASSERT_EQUAL(a5.id, 3u);

    pool.release();

    // make sure that release actually clears caches
    upstream.id_to_allocate = 4;
    alloc_id a6 = pool.do_allocate(32, 64);
    ASSERT_EQUAL(a6.id, 4u);

    upstream.id_to_allocate = 5;
    alloc_id a7 = pool.do_allocate(2048, 1024);
    ASSERT_EQUAL(a7.id, 5u);

    pool.do_deallocate(a7, 2048, 1024);

    // make sure that the 'ridiculousness' factor for size (options.cached_size_cutoff_factor)
    // is respected
    upstream.id_to_allocate = 6;
    alloc_id a8 = pool.do_allocate(24, 1024);
    ASSERT_EQUAL(a8.id, 6u);

    // make sure that the 'ridiculousness' factor for alignment (options.cached_alignment_cutoff_factor)
    // is respected
    upstream.id_to_allocate = 7;
    alloc_id a9 = pool.do_allocate(2048, 32);
    ASSERT_EQUAL(a9.id, 7u);
}

void TestDisjointUnsynchronizedPoolCachingOversized()
{
    TestDisjointPoolCachingOversized<thrust::mr::disjoint_unsynchronized_pool_resource>();
}
DECLARE_UNITTEST(TestDisjointUnsynchronizedPoolCachingOversized);

#if THRUST_CPP_DIALECT >= 2011
void TestDisjointSynchronizedPoolCachingOversized()
{
    TestDisjointPoolCachingOversized<thrust::mr::disjoint_synchronized_pool_resource>();
}
DECLARE_UNITTEST(TestDisjointSynchronizedPoolCachingOversized);
#endif

template<template<typename, typename> class PoolTemplate>
void TestDisjointGlobalPool()
{
    typedef PoolTemplate<
        thrust::mr::new_delete_resource,
        thrust::mr::new_delete_resource
    > Pool;

    ASSERT_EQUAL(thrust::mr::get_global_resource<Pool>() != NULL, true);
}

void TestUnsynchronizedDisjointGlobalPool()
{
    TestDisjointGlobalPool<thrust::mr::disjoint_unsynchronized_pool_resource>();
}
DECLARE_UNITTEST(TestUnsynchronizedDisjointGlobalPool);

#if THRUST_CPP_DIALECT >= 2011
void TestSynchronizedDisjointGlobalPool()
{
    TestDisjointGlobalPool<thrust::mr::disjoint_synchronized_pool_resource>();
}
DECLARE_UNITTEST(TestSynchronizedDisjointGlobalPool);
#endif

