#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/scan.h>
#include <thrust/functional.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/retag.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>


template<typename T>
  struct max_functor
{
  __host__ __device__
  T operator()(T rhs, T lhs) const
  {
    return thrust::max(rhs,lhs);
  }
};


template <class Vector>
void TestScanSimple(void)
{
    typedef typename Vector::value_type T;
    
    typename Vector::iterator iter;

    Vector input(5);
    Vector result(5);
    Vector output(5);

    input[0] = 1; input[1] = 3; input[2] = -2; input[3] = 4; input[4] = -5;

    Vector input_copy(input);

    // inclusive scan
    iter = thrust::inclusive_scan(input.begin(), input.end(), output.begin());
    result[0] = 1; result[1] = 4; result[2] = 2; result[3] = 6; result[4] = 1;
    ASSERT_EQUAL(std::size_t(iter - output.begin()), input.size());
    ASSERT_EQUAL(input,  input_copy);
    ASSERT_EQUAL(output, result);
    
    // exclusive scan
    iter = thrust::exclusive_scan(input.begin(), input.end(), output.begin(), T(0));
    result[0] = 0; result[1] = 1; result[2] = 4; result[3] = 2; result[4] = 6;
    ASSERT_EQUAL(std::size_t(iter - output.begin()), input.size());
    ASSERT_EQUAL(input,  input_copy);
    ASSERT_EQUAL(output, result);
    
    // exclusive scan with init
    iter = thrust::exclusive_scan(input.begin(), input.end(), output.begin(), T(3));
    result[0] = 3; result[1] = 4; result[2] = 7; result[3] = 5; result[4] = 9;
    ASSERT_EQUAL(std::size_t(iter - output.begin()), input.size());
    ASSERT_EQUAL(input,  input_copy);
    ASSERT_EQUAL(output, result);
    
    // inclusive scan with op
    iter = thrust::inclusive_scan(input.begin(), input.end(), output.begin(), thrust::plus<T>());
    result[0] = 1; result[1] = 4; result[2] = 2; result[3] = 6; result[4] = 1;
    ASSERT_EQUAL(std::size_t(iter - output.begin()), input.size());
    ASSERT_EQUAL(input,  input_copy);
    ASSERT_EQUAL(output, result);

    // exclusive scan with init and op
    iter = thrust::exclusive_scan(input.begin(), input.end(), output.begin(), T(3), thrust::plus<T>());
    result[0] = 3; result[1] = 4; result[2] = 7; result[3] = 5; result[4] = 9;
    ASSERT_EQUAL(std::size_t(iter - output.begin()), input.size());
    ASSERT_EQUAL(input,  input_copy);
    ASSERT_EQUAL(output, result);

    // inplace inclusive scan
    input = input_copy;
    iter = thrust::inclusive_scan(input.begin(), input.end(), input.begin());
    result[0] = 1; result[1] = 4; result[2] = 2; result[3] = 6; result[4] = 1;
    ASSERT_EQUAL(std::size_t(iter - input.begin()), input.size());
    ASSERT_EQUAL(input, result);

    // inplace exclusive scan with init
    input = input_copy;
    iter = thrust::exclusive_scan(input.begin(), input.end(), input.begin(), T(3));
    result[0] = 3; result[1] = 4; result[2] = 7; result[3] = 5; result[4] = 9;
    ASSERT_EQUAL(std::size_t(iter - input.begin()), input.size());
    ASSERT_EQUAL(input, result);

    // inplace exclusive scan with implicit init=0
    input = input_copy;
    iter = thrust::exclusive_scan(input.begin(), input.end(), input.begin());
    result[0] = 0; result[1] = 1; result[2] = 4; result[3] = 2; result[4] = 6;
    ASSERT_EQUAL(std::size_t(iter - input.begin()), input.size());
    ASSERT_EQUAL(input, result);
}
DECLARE_VECTOR_UNITTEST(TestScanSimple);


template<typename InputIterator,
         typename OutputIterator>
OutputIterator inclusive_scan(my_system &system,
                              InputIterator,
                              InputIterator,
                              OutputIterator result)
{
    system.validate_dispatch();
    return result;
}

void TestInclusiveScanDispatchExplicit()
{
    thrust::device_vector<int> vec(1);

    my_system sys(0);
    thrust::inclusive_scan(sys,
                           vec.begin(),
                           vec.begin(),
                           vec.begin());

    ASSERT_EQUAL(true, sys.is_valid());
}
DECLARE_UNITTEST(TestInclusiveScanDispatchExplicit);


template<typename InputIterator,
         typename OutputIterator>
OutputIterator inclusive_scan(my_tag,
                              InputIterator,
                              InputIterator,
                              OutputIterator result)
{
    *result = 13;
    return result;
}

void TestInclusiveScanDispatchImplicit()
{
    thrust::device_vector<int> vec(1);

    thrust::inclusive_scan(thrust::retag<my_tag>(vec.begin()),
                           thrust::retag<my_tag>(vec.begin()),
                           thrust::retag<my_tag>(vec.begin()));

    ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestInclusiveScanDispatchImplicit);


template<typename InputIterator,
         typename OutputIterator>
OutputIterator exclusive_scan(my_system &system,
                              InputIterator,
                              InputIterator,
                              OutputIterator result)
{
    system.validate_dispatch();
    return result;
}

void TestExclusiveScanDispatchExplicit()
{
    thrust::device_vector<int> vec(1);

    my_system sys(0);
    thrust::exclusive_scan(sys,
                           vec.begin(),
                           vec.begin(),
                           vec.begin());

    ASSERT_EQUAL(true, sys.is_valid());
}
DECLARE_UNITTEST(TestExclusiveScanDispatchExplicit);


template<typename InputIterator,
         typename OutputIterator>
OutputIterator exclusive_scan(my_tag,
                              InputIterator,
                              InputIterator,
                              OutputIterator result)
{
    *result = 13;
    return result;
}

void TestExclusiveScanDispatchImplicit()
{
    thrust::device_vector<int> vec(1);

    thrust::exclusive_scan(thrust::retag<my_tag>(vec.begin()),
                           thrust::retag<my_tag>(vec.begin()),
                           thrust::retag<my_tag>(vec.begin()));

    ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestExclusiveScanDispatchImplicit);


void TestInclusiveScan32(void)
{
    typedef int T;
    size_t n = 32;

    thrust::host_vector<T>   h_input = unittest::random_integers<T>(n);
    thrust::device_vector<T> d_input = h_input;
    
    thrust::host_vector<T>   h_output(n);
    thrust::device_vector<T> d_output(n);

    thrust::inclusive_scan(h_input.begin(), h_input.end(), h_output.begin());
    thrust::inclusive_scan(d_input.begin(), d_input.end(), d_output.begin());

    ASSERT_EQUAL(d_output, h_output);
}
DECLARE_UNITTEST(TestInclusiveScan32);


void TestExclusiveScan32(void)
{
    typedef int T;
    size_t n = 32;
    T init = 13;

    thrust::host_vector<T>   h_input = unittest::random_integers<T>(n);
    thrust::device_vector<T> d_input = h_input;
    
    thrust::host_vector<T>   h_output(n);
    thrust::device_vector<T> d_output(n);

    thrust::exclusive_scan(h_input.begin(), h_input.end(), h_output.begin(), init);
    thrust::exclusive_scan(d_input.begin(), d_input.end(), d_output.begin(), init);

    ASSERT_EQUAL(d_output, h_output);
}
DECLARE_UNITTEST(TestExclusiveScan32);


template <class IntVector, class FloatVector>
void TestScanMixedTypes(void)
{
    // make sure we get types for default args and operators correct
    IntVector int_input(4);
    int_input[0] = 1;
    int_input[1] = 2;
    int_input[2] = 3;
    int_input[3] = 4;

    FloatVector float_input(4);
    float_input[0] = 1.5;
    float_input[1] = 2.5;
    float_input[2] = 3.5;
    float_input[3] = 4.5;

    IntVector   int_output(4);
    FloatVector float_output(4);

    // float -> int should use plus<void> operator and float accumulator by default
    thrust::inclusive_scan(float_input.begin(), float_input.end(), int_output.begin());
    ASSERT_EQUAL(int_output[0],  1); // in: 1.5 accum: 1.5f out: 1
    ASSERT_EQUAL(int_output[1],  4); // in: 2.5 accum: 4.0f out: 4
    ASSERT_EQUAL(int_output[2],  7); // in: 3.5 accum: 7.5f out: 7
    ASSERT_EQUAL(int_output[3], 12); // in: 4.5 accum: 12.f out: 12

    // float -> float with plus<int> operator (float accumulator)
    thrust::inclusive_scan(float_input.begin(), float_input.end(), float_output.begin(), thrust::plus<int>());
    ASSERT_EQUAL(float_output[0],  1.5f); // in: 1.5 accum: 1.5f out: 1.5f
    ASSERT_EQUAL(float_output[1],  3.0f); // in: 2.5 accum: 3.0f out: 3.0f
    ASSERT_EQUAL(float_output[2],  6.0f); // in: 3.5 accum: 6.0f out: 6.0f
    ASSERT_EQUAL(float_output[3], 10.0f); // in: 4.5 accum: 10.f out: 10.f

    // float -> int should use plus<void> operator and float accumulator by default
    thrust::exclusive_scan(float_input.begin(), float_input.end(), int_output.begin());
    ASSERT_EQUAL(int_output[0], 0); // out: 0.0f  in: 1.5 accum: 1.5f
    ASSERT_EQUAL(int_output[1], 1); // out: 1.5f  in: 2.5 accum: 4.0f
    ASSERT_EQUAL(int_output[2], 4); // out: 4.0f  in: 3.5 accum: 7.5f
    ASSERT_EQUAL(int_output[3], 7); // out: 7.5f  in: 4.5 accum: 12.f

    // float -> int should use plus<> operator and float accumulator by default
    thrust::exclusive_scan(float_input.begin(), float_input.end(), int_output.begin(), (float) 5.5);
    ASSERT_EQUAL(int_output[0],  5); // out: 5.5f  in: 1.5 accum: 7.0f
    ASSERT_EQUAL(int_output[1],  7); // out: 7.0f  in: 2.5 accum: 9.5f
    ASSERT_EQUAL(int_output[2],  9); // out: 9.5f  in: 3.5 accum: 13.0f
    ASSERT_EQUAL(int_output[3], 13); // out: 13.f  in: 4.5 accum: 17.4f

    // int -> float should use using plus<> operator and int accumulator by default
    thrust::inclusive_scan(int_input.begin(), int_input.end(), float_output.begin());
    ASSERT_EQUAL(float_output[0],  1.f); // in: 1 accum: 1  out: 1
    ASSERT_EQUAL(float_output[1],  3.f); // in: 2 accum: 3  out: 3
    ASSERT_EQUAL(float_output[2],  6.f); // in: 3 accum: 6  out: 6
    ASSERT_EQUAL(float_output[3], 10.f); // in: 4 accum: 10 out: 10

    // int -> float + float init_value should use using plus<> operator and
    // float accumulator by default
    thrust::exclusive_scan(int_input.begin(), int_input.end(), float_output.begin(), (float) 5.5);
    ASSERT_EQUAL(float_output[0],  5.5f); // out: 5.5f  in: 1 accum: 6.5f
    ASSERT_EQUAL(float_output[1],  6.5f); // out: 6.0f  in: 2 accum: 8.5f
    ASSERT_EQUAL(float_output[2],  8.5f); // out: 8.0f  in: 3 accum: 11.5f
    ASSERT_EQUAL(float_output[3], 11.5f); // out: 11.f  in: 4 accum: 15.5f
}
void TestScanMixedTypesHost(void)
{
    TestScanMixedTypes< thrust::host_vector<int>, thrust::host_vector<float> >();
}
DECLARE_UNITTEST(TestScanMixedTypesHost);
void TestScanMixedTypesDevice(void)
{
    TestScanMixedTypes< thrust::device_vector<int>, thrust::device_vector<float> >();
}
DECLARE_UNITTEST(TestScanMixedTypesDevice);


template <typename T>
struct TestScanWithOperator
{
  void operator()(const size_t n)
  {
    thrust::host_vector<T>   h_input = unittest::random_integers<T>(n);
    thrust::device_vector<T> d_input = h_input;

    thrust::host_vector<T>   h_output(n);
    thrust::device_vector<T> d_output(n);
    
    thrust::inclusive_scan(h_input.begin(), h_input.end(), h_output.begin(), max_functor<T>());
    thrust::inclusive_scan(d_input.begin(), d_input.end(), d_output.begin(), max_functor<T>());
    ASSERT_EQUAL(d_output, h_output);
    
    thrust::exclusive_scan(h_input.begin(), h_input.end(), h_output.begin(), T(13), max_functor<T>());
    thrust::exclusive_scan(d_input.begin(), d_input.end(), d_output.begin(), T(13), max_functor<T>());
    ASSERT_EQUAL(d_output, h_output);
  }
};
VariableUnitTest<TestScanWithOperator, SignedIntegralTypes> TestScanWithOperatorInstance;


template <typename T>
struct TestScanWithOperatorToDiscardIterator
{
  void operator()(const size_t n)
  {
    thrust::host_vector<T>   h_input = unittest::random_integers<T>(n);
    thrust::device_vector<T> d_input = h_input;

    thrust::discard_iterator<> reference(n);
    
    thrust::discard_iterator<> h_result =
      thrust::inclusive_scan(h_input.begin(), h_input.end(), thrust::make_discard_iterator(), max_functor<T>());

    thrust::discard_iterator<> d_result =
      thrust::inclusive_scan(d_input.begin(), d_input.end(), thrust::make_discard_iterator(), max_functor<T>());
    
    ASSERT_EQUAL_QUIET(reference, h_result);
    ASSERT_EQUAL_QUIET(reference, d_result);
    
    h_result =
      thrust::exclusive_scan(h_input.begin(), h_input.end(), thrust::make_discard_iterator(), T(13), max_functor<T>());

    d_result =
      thrust::exclusive_scan(d_input.begin(), d_input.end(), thrust::make_discard_iterator(), T(13), max_functor<T>());

    ASSERT_EQUAL_QUIET(reference, h_result);
    ASSERT_EQUAL_QUIET(reference, d_result);
  }
};
VariableUnitTest<TestScanWithOperatorToDiscardIterator, unittest::type_list<unittest::int8_t,unittest::int16_t,unittest::int32_t> > TestScanWithOperatorToDiscardIteratorInstance;


template <typename T>
struct TestScan
{
  void operator()(const size_t n)
  {
    thrust::host_vector<T>   h_input = unittest::random_integers<T>(n);
    thrust::device_vector<T> d_input = h_input;

    thrust::host_vector<T>   h_output(n);
    thrust::device_vector<T> d_output(n);
    
    thrust::inclusive_scan(h_input.begin(), h_input.end(), h_output.begin());
    thrust::inclusive_scan(d_input.begin(), d_input.end(), d_output.begin());
    ASSERT_EQUAL(d_output, h_output);
    
    thrust::exclusive_scan(h_input.begin(), h_input.end(), h_output.begin());
    thrust::exclusive_scan(d_input.begin(), d_input.end(), d_output.begin());
    ASSERT_EQUAL(d_output, h_output);
    
    thrust::exclusive_scan(h_input.begin(), h_input.end(), h_output.begin(), (T) 11);
    thrust::exclusive_scan(d_input.begin(), d_input.end(), d_output.begin(), (T) 11);
    ASSERT_EQUAL(d_output, h_output);
    
    // in-place scans
    h_output = h_input;
    d_output = d_input;
    thrust::inclusive_scan(h_output.begin(), h_output.end(), h_output.begin());
    thrust::inclusive_scan(d_output.begin(), d_output.end(), d_output.begin());
    ASSERT_EQUAL(d_output, h_output);
    
    h_output = h_input;
    d_output = d_input;
    thrust::exclusive_scan(h_output.begin(), h_output.end(), h_output.begin());
    thrust::exclusive_scan(d_output.begin(), d_output.end(), d_output.begin());
    ASSERT_EQUAL(d_output, h_output);
  }
};
VariableUnitTest<TestScan, IntegralTypes> TestScanInstance;


template <typename T>
struct TestScanToDiscardIterator
{
  void operator()(const size_t n)
  {
    thrust::host_vector<T>   h_input = unittest::random_integers<T>(n);
    thrust::device_vector<T> d_input = h_input;
    
    thrust::discard_iterator<> h_result =
      thrust::inclusive_scan(h_input.begin(), h_input.end(), thrust::make_discard_iterator());

    thrust::discard_iterator<> d_result =
      thrust::inclusive_scan(d_input.begin(), d_input.end(), thrust::make_discard_iterator());

    thrust::discard_iterator<> reference(n);

    ASSERT_EQUAL_QUIET(reference, h_result);
    ASSERT_EQUAL_QUIET(reference, d_result);
    
    h_result =
      thrust::exclusive_scan(h_input.begin(), h_input.end(), thrust::make_discard_iterator(), (T) 11);

    d_result =
      thrust::exclusive_scan(d_input.begin(), d_input.end(), thrust::make_discard_iterator(), (T) 11);

    ASSERT_EQUAL_QUIET(reference, h_result);
    ASSERT_EQUAL_QUIET(reference, d_result);
  }
};
VariableUnitTest<TestScanToDiscardIterator, unittest::type_list<unittest::int8_t,unittest::int16_t,unittest::int32_t> > TestScanToDiscardIteratorInstance;


void TestScanMixedTypes(void)
{
    const unsigned int n = 113;

    thrust::host_vector<unsigned int> h_input = unittest::random_integers<unsigned int>(n);
    for(size_t i = 0; i < n; i++)
        h_input[i] %= 10;
    thrust::device_vector<unsigned int> d_input = h_input;

    thrust::host_vector<float>   h_float_output(n);
    thrust::device_vector<float> d_float_output(n);
    thrust::host_vector<int>   h_int_output(n);
    thrust::device_vector<int> d_int_output(n);

    //mixed input/output types
    thrust::inclusive_scan(h_input.begin(), h_input.end(), h_float_output.begin());
    thrust::inclusive_scan(d_input.begin(), d_input.end(), d_float_output.begin());
    ASSERT_EQUAL(d_float_output, h_float_output);
    
    thrust::exclusive_scan(h_input.begin(), h_input.end(), h_float_output.begin(), (float) 3.5);
    thrust::exclusive_scan(d_input.begin(), d_input.end(), d_float_output.begin(), (float) 3.5);
    ASSERT_EQUAL(d_float_output, h_float_output);
    
    thrust::exclusive_scan(h_input.begin(), h_input.end(), h_float_output.begin(), (int) 3);
    thrust::exclusive_scan(d_input.begin(), d_input.end(), d_float_output.begin(), (int) 3);
    ASSERT_EQUAL(d_float_output, h_float_output);
    
    thrust::exclusive_scan(h_input.begin(), h_input.end(), h_int_output.begin(), (int) 3);
    thrust::exclusive_scan(d_input.begin(), d_input.end(), d_int_output.begin(), (int) 3);
    ASSERT_EQUAL(d_int_output, h_int_output);
    
    thrust::exclusive_scan(h_input.begin(), h_input.end(), h_int_output.begin(), (float) 3.5);
    thrust::exclusive_scan(d_input.begin(), d_input.end(), d_int_output.begin(), (float) 3.5);
    ASSERT_EQUAL(d_int_output, h_int_output);
}
DECLARE_UNITTEST(TestScanMixedTypes);


template <typename T, unsigned int N>
void _TestScanWithLargeTypes(void)
{
    size_t n = (1024 * 1024) / sizeof(FixedVector<T,N>);

    thrust::host_vector< FixedVector<T,N> > h_input(n);
    thrust::host_vector< FixedVector<T,N> > h_output(n);

    for(size_t i = 0; i < h_input.size(); i++)
        h_input[i] = FixedVector<T,N>(i);

    thrust::device_vector< FixedVector<T,N> > d_input = h_input;
    thrust::device_vector< FixedVector<T,N> > d_output(n);
    
    thrust::inclusive_scan(h_input.begin(), h_input.end(), h_output.begin());
    thrust::inclusive_scan(d_input.begin(), d_input.end(), d_output.begin());

    ASSERT_EQUAL_QUIET(h_output, d_output);
    
    thrust::exclusive_scan(h_input.begin(), h_input.end(), h_output.begin(), FixedVector<T,N>(0));
    thrust::exclusive_scan(d_input.begin(), d_input.end(), d_output.begin(), FixedVector<T,N>(0));
    
    ASSERT_EQUAL_QUIET(h_output, d_output);
}

void TestScanWithLargeTypes(void)
{
  _TestScanWithLargeTypes<int,  1>();

#if !defined(__QNX__)
  _TestScanWithLargeTypes<int,  8>();
  _TestScanWithLargeTypes<int, 64>();
#else
  KNOWN_FAILURE;
#endif
}
DECLARE_UNITTEST(TestScanWithLargeTypes);


template <typename T>
struct plus_mod3
{
    T * table;

    plus_mod3(T * table) : table(table) {}

    __host__ __device__
    T operator()(T a, T b)
    {
        return table[(int) (a + b)];
    }
};

template <typename Vector>
void TestInclusiveScanWithIndirection(void)
{
    // add numbers modulo 3 with external lookup table
    typedef typename Vector::value_type T;

    Vector data(7);
    data[0] = 0;
    data[1] = 1;
    data[2] = 2;
    data[3] = 1;
    data[4] = 2;
    data[5] = 0;
    data[6] = 1;

    Vector table(6);
    table[0] = 0;
    table[1] = 1;
    table[2] = 2;
    table[3] = 0;
    table[4] = 1;
    table[5] = 2;

    thrust::inclusive_scan(data.begin(), data.end(), data.begin(), plus_mod3<T>(thrust::raw_pointer_cast(&table[0])));
    
    ASSERT_EQUAL(data[0], T(0));
    ASSERT_EQUAL(data[1], T(1));
    ASSERT_EQUAL(data[2], T(0));
    ASSERT_EQUAL(data[3], T(1));
    ASSERT_EQUAL(data[4], T(0));
    ASSERT_EQUAL(data[5], T(0));
    ASSERT_EQUAL(data[6], T(1));
}
DECLARE_INTEGRAL_VECTOR_UNITTEST(TestInclusiveScanWithIndirection);

struct only_set_when_expected_it
{
    long long expected;
    bool * flag;

    __host__ __device__ only_set_when_expected_it operator++() const { return *this; }
    __host__ __device__ only_set_when_expected_it operator*() const { return *this; }
    template<typename Difference>
    __host__ __device__ only_set_when_expected_it operator+(Difference) const { return *this; }
    template<typename Index>
    __host__ __device__ only_set_when_expected_it operator[](Index) const { return *this; }

    __device__
    void operator=(long long value) const
    {
        if (value == expected)
        {
            *flag = true;
        }
    }
};

namespace thrust
{
template<>
struct iterator_traits<only_set_when_expected_it>
{
    typedef long long value_type;
    typedef only_set_when_expected_it reference;
};
}

void TestInclusiveScanWithBigIndexesHelper(int magnitude)
{
    thrust::constant_iterator<long long> begin(1);
    thrust::constant_iterator<long long> end = begin + (1ll << magnitude);
    ASSERT_EQUAL(thrust::distance(begin, end), 1ll << magnitude);

    thrust::device_ptr<bool> has_executed = thrust::device_malloc<bool>(1);
    *has_executed = false;

    only_set_when_expected_it out = { (1ll << magnitude), thrust::raw_pointer_cast(has_executed) };

    thrust::inclusive_scan(thrust::device, begin, end, out);

    bool has_executed_h = *has_executed;
    thrust::device_free(has_executed);

    ASSERT_EQUAL(has_executed_h, true);
}

void TestInclusiveScanWithBigIndexes()
{
  TestInclusiveScanWithBigIndexesHelper(30);
  TestInclusiveScanWithBigIndexesHelper(31);
  TestInclusiveScanWithBigIndexesHelper(32);
  TestInclusiveScanWithBigIndexesHelper(33);
}

DECLARE_UNITTEST(TestInclusiveScanWithBigIndexes);

void TestExclusiveScanWithBigIndexesHelper(int magnitude)
{
    thrust::constant_iterator<long long> begin(1);
    thrust::constant_iterator<long long> end = begin + (1ll << magnitude);
    ASSERT_EQUAL(thrust::distance(begin, end), 1ll << magnitude);

    thrust::device_ptr<bool> has_executed = thrust::device_malloc<bool>(1);
    *has_executed = false;

    only_set_when_expected_it out = { (1ll << magnitude) - 1, thrust::raw_pointer_cast(has_executed) };

    thrust::exclusive_scan(thrust::device, begin, end, out,0ll);

    bool has_executed_h = *has_executed;
    thrust::device_free(has_executed);

    ASSERT_EQUAL(has_executed_h, true);
}

void TestExclusiveScanWithBigIndexes()
{
  TestExclusiveScanWithBigIndexesHelper(30);
  TestExclusiveScanWithBigIndexesHelper(31);
  TestExclusiveScanWithBigIndexesHelper(32);
  TestExclusiveScanWithBigIndexesHelper(33);
}

DECLARE_UNITTEST(TestExclusiveScanWithBigIndexes);

#if THRUST_CPP_DIALECT >= 2011

struct Int {
    int i{};
    __host__ __device__ explicit Int(int num) : i(num) {}
    __host__ __device__ Int() : i{} {}
    __host__ __device__ Int operator+(Int const& o) const { return Int{this->i + o.i}; }
};

void TestInclusiveScanWithUserDefinedType()
{
    thrust::device_vector<Int> vec(5, Int{1});

    thrust::inclusive_scan(
        thrust::device,
        vec.cbegin(),
        vec.cend(),
        vec.begin());

    ASSERT_EQUAL(static_cast<Int>(vec.back()).i, 5);
}
DECLARE_UNITTEST(TestInclusiveScanWithUserDefinedType);

#endif // c++11
