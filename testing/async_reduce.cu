#define THRUST_ENABLE_FUTURE_RAW_DATA_MEMBER

#include <thrust/detail/config.h>

#if THRUST_CPP_DIALECT >= 2014

#include <unittest/unittest.h>
#include <unittest/util_async.h>

#include <thrust/async/reduce.h>
#include <thrust/async/copy.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

template <typename T>
struct custom_plus
{
  __host__ __device__
  T operator()(T lhs, T rhs) const
  {
    return lhs + rhs;
  }
};

#define DEFINE_STATEFUL_ASYNC_REDUCE_INVOKER(                                 \
    NAME, MEMBERS, CTOR, DTOR, VALIDATE, ...                                  \
  )                                                                           \
  template <typename T>                                                       \
  struct NAME                                                                 \
  {                                                                           \
    MEMBERS                                                                   \
                                                                              \
    NAME() { CTOR }                                                           \
                                                                              \
    ~NAME() { DTOR }                                                          \
                                                                              \
    template <typename Event>                                                 \
    void validate_event(Event& e)                                             \
    {                                                                         \
      THRUST_UNUSED_VAR(e);                                                   \
      VALIDATE                                                                \
    }                                                                         \
                                                                              \
    template <                                                                \
      typename ForwardIt, typename Sentinel                                   \
    >                                                                         \
    __host__                                                                  \
    auto operator()(                                                          \
      ForwardIt&& first, Sentinel&& last                                      \
    )                                                                         \
    THRUST_DECLTYPE_RETURNS(                                                  \
      ::thrust::async::reduce(                                                \
        __VA_ARGS__                                                           \
      )                                                                       \
    )                                                                         \
  };                                                                          \
  /**/

#define DEFINE_ASYNC_REDUCE_INVOKER(NAME, ...)                                \
  DEFINE_STATEFUL_ASYNC_REDUCE_INVOKER(                                       \
    NAME                                                                      \
  , THRUST_PP_EMPTY(), THRUST_PP_EMPTY(), THRUST_PP_EMPTY(), THRUST_PP_EMPTY()\
  , __VA_ARGS__                                                               \
  )                                                                           \
  /**/

#define DEFINE_SYNC_REDUCE_INVOKER(NAME, ...)                                 \
  template <typename T>                                                       \
  struct NAME                                                                 \
  {                                                                           \
                                                                              \
    template <                                                                \
      typename ForwardIt, typename Sentinel                                   \
    >                                                                         \
    __host__                                                                  \
    auto operator()(                                                          \
      ForwardIt&& first, Sentinel&& last                                      \
    )                                                                         \
    THRUST_RETURNS(                                                           \
      ::thrust::reduce(                                                       \
        __VA_ARGS__                                                           \
      )                                                                       \
    )                                                                         \
  };                                                                          \
  /**/

DEFINE_ASYNC_REDUCE_INVOKER(
  reduce_async_invoker
, THRUST_FWD(first), THRUST_FWD(last)
);
DEFINE_ASYNC_REDUCE_INVOKER(
  reduce_async_invoker_device
, thrust::device
, THRUST_FWD(first), THRUST_FWD(last)
);
DEFINE_ASYNC_REDUCE_INVOKER(
  reduce_async_invoker_device_allocator
, thrust::device(thrust::device_allocator<void>{})
, THRUST_FWD(first), THRUST_FWD(last)
);
DEFINE_STATEFUL_ASYNC_REDUCE_INVOKER(
  reduce_async_invoker_device_on
  // Members.
, hipStream_t stream_;
  // Constructor.
, thrust::cuda_cub::throw_on_error(
    hipStreamCreateWithFlags(&stream_, hipStreamNonBlocking)
  );
  // Destructor.
, thrust::cuda_cub::throw_on_error(
    hipStreamDestroy(stream_)
  );
  // `validate_event` member.
, ASSERT_EQUAL_QUIET(stream_, e.stream().native_handle());
  // Arguments to `thrust::async::reduce`.
, thrust::device.on(stream_)
, THRUST_FWD(first), THRUST_FWD(last)
);
DEFINE_STATEFUL_ASYNC_REDUCE_INVOKER(
  reduce_async_invoker_device_allocator_on
  // Members.
, hipStream_t stream_;
  // Constructor.
, thrust::cuda_cub::throw_on_error(
    hipStreamCreateWithFlags(&stream_, hipStreamNonBlocking)
  );
  // Destructor.
, thrust::cuda_cub::throw_on_error(
    hipStreamDestroy(stream_)
  );
  // `validate_event` member.
, ASSERT_EQUAL_QUIET(stream_, e.stream().native_handle());
  // Arguments to `thrust::async::reduce`.
, thrust::device(thrust::device_allocator<void>{}).on(stream_)
, THRUST_FWD(first), THRUST_FWD(last)
);

DEFINE_SYNC_REDUCE_INVOKER(
  reduce_sync_invoker
, THRUST_FWD(first), THRUST_FWD(last)
);

DEFINE_ASYNC_REDUCE_INVOKER(
  reduce_async_invoker_init
, THRUST_FWD(first), THRUST_FWD(last)
, unittest::random_integer<T>()
);
DEFINE_ASYNC_REDUCE_INVOKER(
  reduce_async_invoker_device_init
, thrust::device
, THRUST_FWD(first), THRUST_FWD(last)
, unittest::random_integer<T>()
);
DEFINE_ASYNC_REDUCE_INVOKER(
  reduce_async_invoker_device_allocator_init
, thrust::device(thrust::device_allocator<void>{})
, THRUST_FWD(first), THRUST_FWD(last)
, unittest::random_integer<T>()
);
DEFINE_STATEFUL_ASYNC_REDUCE_INVOKER(
  reduce_async_invoker_device_on_init
  // Members.
, hipStream_t stream_;
  // Constructor.
, thrust::cuda_cub::throw_on_error(
    hipStreamCreateWithFlags(&stream_, hipStreamNonBlocking)
  );
  // Destructor.
, thrust::cuda_cub::throw_on_error(
    hipStreamDestroy(stream_)
  );
  // `validate_event` member.
, ASSERT_EQUAL_QUIET(stream_, e.stream().native_handle());
  // Arguments to `thrust::async::reduce`.
, thrust::device.on(stream_)
, THRUST_FWD(first), THRUST_FWD(last)
, unittest::random_integer<T>()
);
DEFINE_STATEFUL_ASYNC_REDUCE_INVOKER(
  reduce_async_invoker_device_allocator_on_init
  // Members.
, hipStream_t stream_;
  // Constructor.
, thrust::cuda_cub::throw_on_error(
    hipStreamCreateWithFlags(&stream_, hipStreamNonBlocking)
  );
  // Destructor.
, thrust::cuda_cub::throw_on_error(
    hipStreamDestroy(stream_)
  );
  // `validate_event` member.
, ASSERT_EQUAL_QUIET(stream_, e.stream().native_handle());
  // Arguments to `thrust::async::reduce`.
, thrust::device(thrust::device_allocator<void>{}).on(stream_)
, THRUST_FWD(first), THRUST_FWD(last)
, unittest::random_integer<T>()
);

DEFINE_SYNC_REDUCE_INVOKER(
  reduce_sync_invoker_init
, THRUST_FWD(first), THRUST_FWD(last)
, unittest::random_integer<T>()
);

DEFINE_ASYNC_REDUCE_INVOKER(
  reduce_async_invoker_init_plus
, THRUST_FWD(first), THRUST_FWD(last)
, unittest::random_integer<T>()
, thrust::plus<T>()
);
DEFINE_ASYNC_REDUCE_INVOKER(
  reduce_async_invoker_device_init_plus
, thrust::device
, THRUST_FWD(first), THRUST_FWD(last)
, unittest::random_integer<T>()
, thrust::plus<T>()
);
DEFINE_ASYNC_REDUCE_INVOKER(
  reduce_async_invoker_device_allocator_init_plus
, thrust::device(thrust::device_allocator<void>{})
, THRUST_FWD(first), THRUST_FWD(last)
, unittest::random_integer<T>()
, thrust::plus<T>()
);
DEFINE_STATEFUL_ASYNC_REDUCE_INVOKER(
  reduce_async_invoker_device_on_init_plus
  // Members.
, hipStream_t stream_;
  // Constructor.
, thrust::cuda_cub::throw_on_error(
    hipStreamCreateWithFlags(&stream_, hipStreamNonBlocking)
  );
  // Destructor.
, thrust::cuda_cub::throw_on_error(
    hipStreamDestroy(stream_)
  );
  // `validate_event` member.
, ASSERT_EQUAL_QUIET(stream_, e.stream().native_handle());
  // Arguments to `thrust::async::reduce`.
, thrust::device.on(stream_)
, THRUST_FWD(first), THRUST_FWD(last)
, unittest::random_integer<T>()
, thrust::plus<T>()
);
DEFINE_STATEFUL_ASYNC_REDUCE_INVOKER(
  reduce_async_invoker_device_allocator_on_init_plus
  // Members.
, hipStream_t stream_;
  // Constructor.
, thrust::cuda_cub::throw_on_error(
    hipStreamCreateWithFlags(&stream_, hipStreamNonBlocking)
  );
  // Destructor.
, thrust::cuda_cub::throw_on_error(
    hipStreamDestroy(stream_)
  );
  // `validate_event` member.
, ASSERT_EQUAL_QUIET(stream_, e.stream().native_handle());
  // Arguments to `thrust::async::reduce`.
, thrust::device(thrust::device_allocator<void>{}).on(stream_)
, THRUST_FWD(first), THRUST_FWD(last)
, unittest::random_integer<T>()
, thrust::plus<T>()
);

DEFINE_SYNC_REDUCE_INVOKER(
  reduce_sync_invoker_init_plus
, THRUST_FWD(first), THRUST_FWD(last)
, unittest::random_integer<T>()
, thrust::plus<T>()
);

DEFINE_ASYNC_REDUCE_INVOKER(
  reduce_async_invoker_init_custom_plus
, THRUST_FWD(first), THRUST_FWD(last)
, unittest::random_integer<T>()
, custom_plus<T>()
);
DEFINE_ASYNC_REDUCE_INVOKER(
  reduce_async_invoker_device_init_custom_plus
, thrust::device
, THRUST_FWD(first), THRUST_FWD(last)
, unittest::random_integer<T>()
, custom_plus<T>()
);
DEFINE_ASYNC_REDUCE_INVOKER(
  reduce_async_invoker_device_allocator_init_custom_plus
, thrust::device(thrust::device_allocator<void>{})
, THRUST_FWD(first), THRUST_FWD(last)
, unittest::random_integer<T>()
, custom_plus<T>()
);
DEFINE_STATEFUL_ASYNC_REDUCE_INVOKER(
  reduce_async_invoker_device_on_init_custom_plus
  // Members.
, hipStream_t stream_;
  // Constructor.
, thrust::cuda_cub::throw_on_error(
    hipStreamCreateWithFlags(&stream_, hipStreamNonBlocking)
  );
  // Destructor.
, thrust::cuda_cub::throw_on_error(
    hipStreamDestroy(stream_)
  );
  // `validate_event` member.
, ASSERT_EQUAL_QUIET(stream_, e.stream().native_handle());
  // Arguments to `thrust::async::reduce`.
, thrust::device.on(stream_)
, THRUST_FWD(first), THRUST_FWD(last)
, unittest::random_integer<T>()
, custom_plus<T>()
);
DEFINE_STATEFUL_ASYNC_REDUCE_INVOKER(
  reduce_async_invoker_device_allocator_on_init_custom_plus
  // Members.
, hipStream_t stream_;
  // Constructor.
, thrust::cuda_cub::throw_on_error(
    hipStreamCreateWithFlags(&stream_, hipStreamNonBlocking)
  );
  // Destructor.
, thrust::cuda_cub::throw_on_error(
    hipStreamDestroy(stream_)
  );
  // `validate_event` member.
, ASSERT_EQUAL_QUIET(stream_, e.stream().native_handle());
  // Arguments to `thrust::async::reduce`.
, thrust::device(thrust::device_allocator<void>{}).on(stream_)
, THRUST_FWD(first), THRUST_FWD(last)
, unittest::random_integer<T>()
, custom_plus<T>()
);

DEFINE_SYNC_REDUCE_INVOKER(
  reduce_sync_invoker_init_custom_plus
, THRUST_FWD(first), THRUST_FWD(last)
, unittest::random_integer<T>()
, custom_plus<T>()
);

///////////////////////////////////////////////////////////////////////////////

template <
  template <typename> class AsyncReduceInvoker
, template <typename> class SyncReduceInvoker
>
struct test_async_reduce
{
  template <typename T>
  struct tester
  {
    __host__
    void operator()(std::size_t n)
    {
      thrust::host_vector<T>   h0(unittest::random_integers<T>(n));
      thrust::device_vector<T> d0a(h0);
      thrust::device_vector<T> d0b(h0);
      thrust::device_vector<T> d0c(h0);
      thrust::device_vector<T> d0d(h0);

      AsyncReduceInvoker<T> invoke_async;
      SyncReduceInvoker<T>  invoke_sync;

      ASSERT_EQUAL(h0, d0a);
      ASSERT_EQUAL(h0, d0b);
      ASSERT_EQUAL(h0, d0c);
      ASSERT_EQUAL(h0, d0d);

      auto f0a = invoke_async(d0a.begin(), d0a.end());
      auto f0b = invoke_async(d0b.begin(), d0b.end());
      auto f0c = invoke_async(d0c.begin(), d0c.end());
      auto f0d = invoke_async(d0d.begin(), d0d.end());

      invoke_async.validate_event(f0a);
      invoke_async.validate_event(f0b);
      invoke_async.validate_event(f0c);
      invoke_async.validate_event(f0d);

      // This potentially runs concurrently with the copies.
      auto const r0 = invoke_sync(h0.begin(), h0.end());

      auto const r1a = TEST_FUTURE_VALUE_RETRIEVAL(f0a);
      auto const r1b = TEST_FUTURE_VALUE_RETRIEVAL(f0b);
      auto const r1c = TEST_FUTURE_VALUE_RETRIEVAL(f0c);
      auto const r1d = TEST_FUTURE_VALUE_RETRIEVAL(f0d);

      ASSERT_EQUAL(r0, r1a);
      ASSERT_EQUAL(r0, r1b);
      ASSERT_EQUAL(r0, r1c);
      ASSERT_EQUAL(r0, r1d);
    }
  };
};
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_reduce<
      reduce_async_invoker
    , reduce_sync_invoker
    >::tester
  )
, NumericTypes
, test_async_reduce
);
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_reduce<
      reduce_async_invoker_device
    , reduce_sync_invoker
    >::tester
  )
, NumericTypes
, test_async_reduce_policy
);
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_reduce<
      reduce_async_invoker_device_allocator
    , reduce_sync_invoker
    >::tester
  )
, NumericTypes
, test_async_reduce_policy_allocator
);
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_reduce<
      reduce_async_invoker_device_on
    , reduce_sync_invoker
    >::tester
  )
, NumericTypes
, test_async_reduce_policy_on
);
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_reduce<
      reduce_async_invoker_device_allocator_on
    , reduce_sync_invoker
    >::tester
  )
, NumericTypes
, test_async_reduce_policy_allocator_on
);
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_reduce<
      reduce_async_invoker_init
    , reduce_sync_invoker_init
    >::tester
  )
, NumericTypes
, test_async_reduce_init
);
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_reduce<
      reduce_async_invoker_device_init
    , reduce_sync_invoker_init
    >::tester
  )
, NumericTypes
, test_async_reduce_policy_init
);
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_reduce<
      reduce_async_invoker_device_allocator_init
    , reduce_sync_invoker_init
    >::tester
  )
, NumericTypes
, test_async_reduce_policy_allocator_init
);
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_reduce<
      reduce_async_invoker_device_on_init
    , reduce_sync_invoker_init
    >::tester
  )
, NumericTypes
, test_async_reduce_policy_on_init
);
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_reduce<
      reduce_async_invoker_device_allocator_on_init
    , reduce_sync_invoker_init
    >::tester
  )
, NumericTypes
, test_async_reduce_policy_allocator_on_init
);
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_reduce<
      reduce_async_invoker_init_plus
    , reduce_sync_invoker_init_plus
    >::tester
  )
, NumericTypes
, test_async_reduce_init_plus
);
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_reduce<
      reduce_async_invoker_device_init_plus
    , reduce_sync_invoker_init_plus
    >::tester
  )
, NumericTypes
, test_async_reduce_policy_init_plus
);
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_reduce<
      reduce_async_invoker_device_allocator_init_plus
    , reduce_sync_invoker_init_plus
    >::tester
  )
, NumericTypes
, test_async_reduce_policy_allocator_init_plus
);
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_reduce<
      reduce_async_invoker_device_on_init_plus
    , reduce_sync_invoker_init_plus
    >::tester
  )
, NumericTypes
, test_async_reduce_policy_on_init_plus
);
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_reduce<
      reduce_async_invoker_device_allocator_on_init_plus
    , reduce_sync_invoker_init_plus
    >::tester
  )
, NumericTypes
, test_async_reduce_policy_allocator_on_init_plus
);
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_reduce<
      reduce_async_invoker_init_custom_plus
    , reduce_sync_invoker_init_custom_plus
    >::tester
  )
, NumericTypes
, test_async_reduce_init_custom_plus
);
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_reduce<
      reduce_async_invoker_device_init_custom_plus
    , reduce_sync_invoker_init_custom_plus
    >::tester
  )
, NumericTypes
, test_async_reduce_policy_init_custom_plus
);
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_reduce<
      reduce_async_invoker_device_allocator_init_custom_plus
    , reduce_sync_invoker_init_custom_plus
    >::tester
  )
, NumericTypes
, test_async_reduce_policy_allocator_init_custom_plus
);
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_reduce<
      reduce_async_invoker_device_on_init_custom_plus
    , reduce_sync_invoker_init_custom_plus
    >::tester
  )
, NumericTypes
, test_async_reduce_policy_on_init_custom_plus
);
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_reduce<
      reduce_async_invoker_device_allocator_on_init_custom_plus
    , reduce_sync_invoker_init_custom_plus
    >::tester
  )
, NumericTypes
, test_async_reduce_policy_allocator_on_init_custom_plus
);

///////////////////////////////////////////////////////////////////////////////

template <
  template <typename> class AsyncReduceInvoker
, template <typename> class SyncReduceInvoker
>
struct test_async_reduce_counting_iterator
{
  template <typename T>
  struct tester
  {
    __host__
    void operator()()
    {
      constexpr std::size_t n = 15 * sizeof(T);

      ASSERT_LEQUAL(T(n), unittest::truncate_to_max_representable<T>(n));

      thrust::counting_iterator<T> first(0);
      thrust::counting_iterator<T> last(n);

      AsyncReduceInvoker<T> invoke_async;
      SyncReduceInvoker<T>  invoke_sync;

      auto f0a = invoke_async(first, last);
      auto f0b = invoke_async(first, last);
      auto f0c = invoke_async(first, last);
      auto f0d = invoke_async(first, last);

      invoke_async.validate_event(f0a);
      invoke_async.validate_event(f0b);
      invoke_async.validate_event(f0c);
      invoke_async.validate_event(f0d);

      // This potentially runs concurrently with the copies.
      auto const r0 = invoke_sync(first, last);

      auto const r1a = TEST_FUTURE_VALUE_RETRIEVAL(f0a);
      auto const r1b = TEST_FUTURE_VALUE_RETRIEVAL(f0b);
      auto const r1c = TEST_FUTURE_VALUE_RETRIEVAL(f0c);
      auto const r1d = TEST_FUTURE_VALUE_RETRIEVAL(f0d);

      ASSERT_EQUAL(r0, r1a);
      ASSERT_EQUAL(r0, r1b);
      ASSERT_EQUAL(r0, r1c);
      ASSERT_EQUAL(r0, r1d);
    }
  };
};
DECLARE_GENERIC_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_reduce_counting_iterator<
      reduce_async_invoker
    , reduce_sync_invoker
    >::tester
  )
, BuiltinNumericTypes
, test_async_reduce_counting_iterator
);
DECLARE_GENERIC_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_reduce_counting_iterator<
      reduce_async_invoker_device
    , reduce_sync_invoker
    >::tester
  )
, BuiltinNumericTypes
, test_async_reduce_policy_counting_iterator
);
DECLARE_GENERIC_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_reduce_counting_iterator<
      reduce_async_invoker_init
    , reduce_sync_invoker_init
    >::tester
  )
, BuiltinNumericTypes
, test_async_reduce_counting_iterator_init
);
DECLARE_GENERIC_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_reduce_counting_iterator<
      reduce_async_invoker_device_init
    , reduce_sync_invoker_init
    >::tester
  )
, BuiltinNumericTypes
, test_async_reduce_policy_counting_iterator_init
);
DECLARE_GENERIC_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_reduce_counting_iterator<
      reduce_async_invoker_init_plus
    , reduce_sync_invoker_init_plus
    >::tester
  )
, BuiltinNumericTypes
, test_async_reduce_counting_iterator_init_plus
);
DECLARE_GENERIC_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_reduce_counting_iterator<
      reduce_async_invoker_device_init_plus
    , reduce_sync_invoker_init_plus
    >::tester
  )
, BuiltinNumericTypes
, test_async_reduce_policy_counting_iterator_init_plus
);
DECLARE_GENERIC_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_reduce_counting_iterator<
      reduce_async_invoker_init_custom_plus
    , reduce_sync_invoker_init_custom_plus
    >::tester
  )
, BuiltinNumericTypes
, test_async_reduce_counting_iterator_init_custom_plus
);
DECLARE_GENERIC_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_reduce_counting_iterator<
      reduce_async_invoker_device_init_custom_plus
    , reduce_sync_invoker_init_custom_plus
    >::tester
  )
, BuiltinNumericTypes
, test_async_reduce_policy_counting_iterator_init_custom_plus
);

///////////////////////////////////////////////////////////////////////////////

template <typename T>
struct test_async_reduce_using
{
  __host__
  void operator()(std::size_t n)
  {
    thrust::host_vector<T>   h0(unittest::random_integers<T>(n));
    thrust::device_vector<T> d0a(h0);
    thrust::device_vector<T> d0b(h0);

    ASSERT_EQUAL(h0, d0a);
    ASSERT_EQUAL(h0, d0b);

    thrust::device_future<T> f0a;
    thrust::device_future<T> f0b;

    // When you import the customization points into the global namespace,
    // they should be selected instead of the synchronous algorithms.
    {
      using namespace thrust::async;
      f0a = reduce(d0a.begin(), d0a.end());
    }
    {
      using thrust::async::reduce;
      f0b = reduce(d0b.begin(), d0b.end());
    }

    // ADL should find the synchronous algorithms.
    // This potentially runs concurrently with the copies.
    T const r0 = reduce(h0.begin(), h0.end());

    T const r1a = TEST_FUTURE_VALUE_RETRIEVAL(f0a);
    T const r1b = TEST_FUTURE_VALUE_RETRIEVAL(f0b);

    ASSERT_EQUAL(r0, r1a);
    ASSERT_EQUAL(r0, r1b);
  }
};
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES(
  test_async_reduce_using
, NumericTypes
);

///////////////////////////////////////////////////////////////////////////////

template <typename T>
struct test_async_reduce_after
{
  __host__
  void operator()(std::size_t n)
  {
    thrust::host_vector<T>   h0(unittest::random_integers<T>(n));
    thrust::device_vector<T> d0(h0);

    ASSERT_EQUAL(h0, d0);

    auto f0 = thrust::async::reduce(
      d0.begin(), d0.end()
    );

    ASSERT_EQUAL(true, f0.valid_stream());
 
    auto const f0_stream = f0.stream().native_handle();

    auto f1 = thrust::async::reduce(
      thrust::device.after(f0), d0.begin(), d0.end()
    );

    // Verify that double consumption of a future produces an exception.
    ASSERT_THROWS_EQUAL(
      auto x = thrust::async::reduce(
        thrust::device.after(f0), d0.begin(), d0.end()
      );
      THRUST_UNUSED_VAR(x)
    , thrust::event_error
    , thrust::event_error(thrust::event_errc::no_state)
    );

    ASSERT_EQUAL_QUIET(f0_stream, f1.stream().native_handle());

    auto after_policy2 = thrust::device.after(f1);

    auto f2 = thrust::async::reduce(
      after_policy2, d0.begin(), d0.end()
    );

    // Verify that double consumption of a policy produces an exception.
    ASSERT_THROWS_EQUAL(
      auto x = thrust::async::reduce(
        after_policy2, d0.begin(), d0.end()
      );
      THRUST_UNUSED_VAR(x)
    , thrust::event_error
    , thrust::event_error(thrust::event_errc::no_state)
    );

    ASSERT_EQUAL_QUIET(f0_stream, f2.stream().native_handle());

    // This potentially runs concurrently with the copies.
    T const r0 = thrust::reduce(h0.begin(), h0.end());

    T const r1 = TEST_FUTURE_VALUE_RETRIEVAL(f2);

    ASSERT_EQUAL(r0, r1);
  }
};
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES(
  test_async_reduce_after
, NumericTypes
);

///////////////////////////////////////////////////////////////////////////////

template <typename T>
struct test_async_reduce_on_then_after
{
  __host__
  void operator()(std::size_t n)
  {
    thrust::host_vector<T>   h0(unittest::random_integers<T>(n));
    thrust::device_vector<T> d0(h0);

    ASSERT_EQUAL(h0, d0);

    hipStream_t stream;
    thrust::cuda_cub::throw_on_error(
      hipStreamCreateWithFlags(&stream, hipStreamNonBlocking)
    );

    auto f0 = thrust::async::reduce(
      thrust::device.on(stream), d0.begin(), d0.end()
    );

    ASSERT_EQUAL_QUIET(stream, f0.stream().native_handle());

    auto f1 = thrust::async::reduce(
      thrust::device.after(f0), d0.begin(), d0.end()
    );

    // Verify that double consumption of a future produces an exception.
    ASSERT_THROWS_EQUAL(
      auto x = thrust::async::reduce(
        thrust::device.after(f0), d0.begin(), d0.end()
      );
      THRUST_UNUSED_VAR(x)
    , thrust::event_error
    , thrust::event_error(thrust::event_errc::no_state)
    );

    ASSERT_EQUAL_QUIET(stream, f1.stream().native_handle());

    auto after_policy2 = thrust::device.after(f1);

    auto f2 = thrust::async::reduce(
      after_policy2, d0.begin(), d0.end()
    );

    // Verify that double consumption of a policy produces an exception.
    ASSERT_THROWS_EQUAL(
      auto x = thrust::async::reduce(
        after_policy2, d0.begin(), d0.end()
      );
      THRUST_UNUSED_VAR(x)
    , thrust::event_error
    , thrust::event_error(thrust::event_errc::no_state)
    );

    ASSERT_EQUAL_QUIET(stream, f2.stream().native_handle());

    // This potentially runs concurrently with the copies.
    T const r0 = thrust::reduce(h0.begin(), h0.end());

    T const r1 = TEST_FUTURE_VALUE_RETRIEVAL(f2);

    ASSERT_EQUAL(r0, r1);

    thrust::cuda_cub::throw_on_error(
      hipStreamDestroy(stream)
    );
  }
};
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES(
  test_async_reduce_on_then_after
, NumericTypes
);

///////////////////////////////////////////////////////////////////////////////

template <typename T>
struct test_async_reduce_allocator_on_then_after
{
  __host__
  void operator()(std::size_t n)
  {
    thrust::host_vector<T>   h0(unittest::random_integers<T>(n));
    thrust::device_vector<T> d0(h0);

    ASSERT_EQUAL(h0, d0);

    hipStream_t stream0;
    thrust::cuda_cub::throw_on_error(
      hipStreamCreateWithFlags(&stream0, hipStreamNonBlocking)
    );

    hipStream_t stream1;
    thrust::cuda_cub::throw_on_error(
      hipStreamCreateWithFlags(&stream1, hipStreamNonBlocking)
    );

    auto f0 = thrust::async::reduce(
      thrust::device(thrust::device_allocator<void>{}).on(stream0)
    , d0.begin(), d0.end()
    );

    ASSERT_EQUAL_QUIET(stream0, f0.stream().native_handle());

    auto f1 = thrust::async::reduce(
      thrust::device(thrust::device_allocator<void>{}).after(f0)
    , d0.begin(), d0.end()
    );

    ASSERT_THROWS_EQUAL(
      auto x = thrust::async::reduce(
        thrust::device(thrust::device_allocator<void>{}).after(f0)
      , d0.begin(), d0.end()
      );
      THRUST_UNUSED_VAR(x)
    , thrust::event_error
    , thrust::event_error(thrust::event_errc::no_state)
    );

    ASSERT_EQUAL_QUIET(stream0, f1.stream().native_handle());

    auto f2 = thrust::async::reduce(
      thrust::device(thrust::device_allocator<void>{}).on(stream1).after(f1)
    , d0.begin(), d0.end()
    );

    ASSERT_THROWS_EQUAL(
      auto x = thrust::async::reduce(
        thrust::device(thrust::device_allocator<void>{}).on(stream1).after(f1)
      , d0.begin(), d0.end()
      );
      THRUST_UNUSED_VAR(x)
    , thrust::event_error
    , thrust::event_error(thrust::event_errc::no_state)
    );

    KNOWN_FAILURE;
    // FIXME: The below fails because you can't combine allocator attachment,
    // `.on`, and `.after`.
    ASSERT_EQUAL_QUIET(stream1, f2.stream().native_handle());

    // This potentially runs concurrently with the copies.
    T const r0 = thrust::reduce(h0.begin(), h0.end());

    T const r1 = TEST_FUTURE_VALUE_RETRIEVAL(f2);

    ASSERT_EQUAL(r0, r1);

    thrust::cuda_cub::throw_on_error(hipStreamDestroy(stream0));
    thrust::cuda_cub::throw_on_error(hipStreamDestroy(stream1));
  }
};
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES(
  test_async_reduce_allocator_on_then_after
, NumericTypes
);

///////////////////////////////////////////////////////////////////////////////

template <typename T>
struct test_async_reduce_caching
{
  __host__
  void operator()(std::size_t n)
  {
    constexpr std::int64_t m = 32;

    thrust::host_vector<T>   h0(unittest::random_integers<T>(n));
    thrust::device_vector<T> d0(h0);

    ASSERT_EQUAL(h0, d0);

    T const* f0_raw_data;

    {
      // Perform one reduction to ensure there's an entry in the caching
      // allocator.
      auto f0 = thrust::async::reduce(d0.begin(), d0.end());

      TEST_EVENT_WAIT(f0);

      f0_raw_data = f0.raw_data();
    }

    for (std::int64_t i = 0; i < m; ++i)
    {
      auto f1 = thrust::async::reduce(d0.begin(), d0.end());

      ASSERT_EQUAL(true, f1.valid_stream());
      ASSERT_EQUAL(true, f1.valid_content());

      ASSERT_EQUAL_QUIET(f0_raw_data, f1.raw_data());

      // This potentially runs concurrently with the copies.
      T const r0 = thrust::reduce(h0.begin(), h0.end());

      T const r1 = TEST_FUTURE_VALUE_RETRIEVAL(f1);

      ASSERT_EQUAL(r0, r1);
    }
  }
};
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES(
  test_async_reduce_caching
, NumericTypes
);

///////////////////////////////////////////////////////////////////////////////

template <typename T>
struct test_async_copy_then_reduce
{
  __host__
  void operator()(std::size_t n)
  {
    thrust::host_vector<T>   h0a(unittest::random_integers<T>(n));
    thrust::host_vector<T>   h0b(unittest::random_integers<T>(n));
    thrust::host_vector<T>   h0c(unittest::random_integers<T>(n));
    thrust::host_vector<T>   h0d(unittest::random_integers<T>(n));

    thrust::device_vector<T> d0a(n);
    thrust::device_vector<T> d0b(n);
    thrust::device_vector<T> d0c(n);
    thrust::device_vector<T> d0d(n);

    auto f0a = thrust::async::copy(h0a.begin(), h0a.end(), d0a.begin());
    auto f0b = thrust::async::copy(h0b.begin(), h0b.end(), d0b.begin());
    auto f0c = thrust::async::copy(h0c.begin(), h0c.end(), d0c.begin());
    auto f0d = thrust::async::copy(h0d.begin(), h0d.end(), d0d.begin());

    ASSERT_EQUAL(true, f0a.valid_stream());
    ASSERT_EQUAL(true, f0b.valid_stream());
    ASSERT_EQUAL(true, f0c.valid_stream());
    ASSERT_EQUAL(true, f0d.valid_stream());

    auto const f0a_stream = f0a.stream().native_handle();
    auto const f0b_stream = f0b.stream().native_handle();
    auto const f0c_stream = f0c.stream().native_handle();
    auto const f0d_stream = f0d.stream().native_handle();

    auto f1a = thrust::async::reduce(
      thrust::device.after(f0a), d0a.begin(), d0a.end()
    );
    auto f1b = thrust::async::reduce(
      thrust::device.after(f0b), d0b.begin(), d0b.end()
    );
    auto f1c = thrust::async::reduce(
      thrust::device.after(f0c), d0c.begin(), d0c.end()
    );
    auto f1d = thrust::async::reduce(
      thrust::device.after(f0d), d0d.begin(), d0d.end()
    );

    ASSERT_EQUAL(false, f0a.valid_stream());
    ASSERT_EQUAL(false, f0b.valid_stream());
    ASSERT_EQUAL(false, f0c.valid_stream());
    ASSERT_EQUAL(false, f0d.valid_stream());

    ASSERT_EQUAL(true, f1a.valid_stream());
    ASSERT_EQUAL(true, f1a.valid_content());
    ASSERT_EQUAL(true, f1b.valid_stream());
    ASSERT_EQUAL(true, f1b.valid_content());
    ASSERT_EQUAL(true, f1c.valid_stream());
    ASSERT_EQUAL(true, f1c.valid_content());
    ASSERT_EQUAL(true, f1d.valid_stream());
    ASSERT_EQUAL(true, f1d.valid_content());

    // Verify that streams were stolen.
    ASSERT_EQUAL_QUIET(f0a_stream, f1a.stream().native_handle());
    ASSERT_EQUAL_QUIET(f0b_stream, f1b.stream().native_handle());
    ASSERT_EQUAL_QUIET(f0c_stream, f1c.stream().native_handle());
    ASSERT_EQUAL_QUIET(f0d_stream, f1d.stream().native_handle());

    // This potentially runs concurrently with the copies.
    T const r0 = thrust::reduce(h0a.begin(), h0a.end());

    T const r1a = TEST_FUTURE_VALUE_RETRIEVAL(f1a);
    T const r1b = TEST_FUTURE_VALUE_RETRIEVAL(f1b);
    T const r1c = TEST_FUTURE_VALUE_RETRIEVAL(f1c);
    T const r1d = TEST_FUTURE_VALUE_RETRIEVAL(f1d);

    ASSERT_EQUAL(r0, r1a);
    ASSERT_EQUAL(r0, r1b);
    ASSERT_EQUAL(r0, r1c);
    ASSERT_EQUAL(r0, r1d);
  }
};
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES(
  test_async_copy_then_reduce
, BuiltinNumericTypes
);

///////////////////////////////////////////////////////////////////////////////

// TODO: when_all from reductions.

#endif

