#include <thrust/detail/config.h>

#if THRUST_CPP_DIALECT >= 2014

#include <unittest/unittest.h>
#include <unittest/util_async.h>

#include <thrust/async/transform.h>
#include <thrust/async/copy.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

template <typename T>
struct divide_by_2
{
  __host__ __device__
  T operator()(T x) const
  {
    return x / 2;
  }
};

#define DEFINE_STATEFUL_ASYNC_TRANSFORM_UNARY_INVOKER(                        \
    NAME, MEMBERS, CTOR, DTOR, VALIDATE, ...                                  \
  )                                                                           \
  template <typename T>                                                       \
  struct NAME                                                                 \
  {                                                                           \
    MEMBERS                                                                   \
                                                                              \
    NAME() { CTOR }                                                           \
                                                                              \
    ~NAME() { DTOR }                                                          \
                                                                              \
    template <typename Event>                                                 \
    void validate_event(Event& e)                                             \
    {                                                                         \
      THRUST_UNUSED_VAR(e);                                                   \
      VALIDATE                                                                \
    }                                                                         \
                                                                              \
    template <                                                                \
      typename ForwardIt, typename Sentinel, typename OutputIt                \
    , typename UnaryOperation                                                 \
    >                                                                         \
    __host__                                                                  \
    auto operator()(                                                          \
      ForwardIt&& first, Sentinel&& last, OutputIt&& output                   \
    , UnaryOperation&& op                                                     \
    )                                                                         \
    THRUST_DECLTYPE_RETURNS(                                                  \
      ::thrust::async::transform(                                             \
        __VA_ARGS__                                                           \
      )                                                                       \
    )                                                                         \
  };                                                                          \
  /**/

#define DEFINE_ASYNC_TRANSFORM_UNARY_INVOKER(NAME, ...)                       \
  DEFINE_STATEFUL_ASYNC_TRANSFORM_UNARY_INVOKER(                              \
    NAME                                                                      \
  , THRUST_PP_EMPTY(), THRUST_PP_EMPTY(), THRUST_PP_EMPTY(), THRUST_PP_EMPTY()\
  , __VA_ARGS__                                                               \
  )                                                                           \
  /**/

#define DEFINE_SYNC_TRANSFORM_UNARY_INVOKER(NAME, ...)                        \
  template <typename T>                                                       \
  struct NAME                                                                 \
  {                                                                           \
                                                                              \
    template <                                                                \
      typename ForwardIt, typename Sentinel, typename OutputIt                \
    , typename UnaryOperation                                                 \
    >                                                                         \
    __host__                                                                  \
    auto operator()(                                                          \
      ForwardIt&& first, Sentinel&& last, OutputIt&& output                   \
    , UnaryOperation&& op                                                     \
    )                                                                         \
    THRUST_RETURNS(                                                           \
      ::thrust::transform(                                                    \
        __VA_ARGS__                                                           \
      )                                                                       \
    )                                                                         \
  };                                                                          \
  /**/

DEFINE_ASYNC_TRANSFORM_UNARY_INVOKER(
  transform_unary_async_invoker
, THRUST_FWD(first), THRUST_FWD(last)
, THRUST_FWD(output)
, THRUST_FWD(op)
);
DEFINE_ASYNC_TRANSFORM_UNARY_INVOKER(
  transform_unary_async_invoker_device
, thrust::device
, THRUST_FWD(first), THRUST_FWD(last)
, THRUST_FWD(output)
, THRUST_FWD(op)
);
DEFINE_ASYNC_TRANSFORM_UNARY_INVOKER(
  transform_unary_async_invoker_device_allocator
, thrust::device(thrust::device_allocator<void>{})
, THRUST_FWD(first), THRUST_FWD(last)
, THRUST_FWD(output)
, THRUST_FWD(op)
);
DEFINE_STATEFUL_ASYNC_TRANSFORM_UNARY_INVOKER(
  transform_unary_async_invoker_device_on
  // Members.
, hipStream_t stream_;
  // Constructor.
, thrust::cuda_cub::throw_on_error(
    hipStreamCreateWithFlags(&stream_, hipStreamNonBlocking)
  );
  // Destructor.
, thrust::cuda_cub::throw_on_error(
    hipStreamDestroy(stream_)
  );
  // `validate_event` member.
, ASSERT_EQUAL_QUIET(stream_, e.stream().native_handle());
  // Arguments to `thrust::async::transform`.
, thrust::device.on(stream_)
, THRUST_FWD(first), THRUST_FWD(last)
, THRUST_FWD(output)
, THRUST_FWD(op)
);
DEFINE_STATEFUL_ASYNC_TRANSFORM_UNARY_INVOKER(
  transform_unary_async_invoker_device_allocator_on
  // Members.
, hipStream_t stream_;
  // Constructor.
, thrust::cuda_cub::throw_on_error(
    hipStreamCreateWithFlags(&stream_, hipStreamNonBlocking)
  );
  // Destructor.
, thrust::cuda_cub::throw_on_error(
    hipStreamDestroy(stream_)
  );
  // `validate_event` member.
, ASSERT_EQUAL_QUIET(stream_, e.stream().native_handle());
  // Arguments to `thrust::async::transform`.
, thrust::device(thrust::device_allocator<void>{}).on(stream_)
, THRUST_FWD(first), THRUST_FWD(last)
, THRUST_FWD(output)
, THRUST_FWD(op)
);

DEFINE_SYNC_TRANSFORM_UNARY_INVOKER(
  transform_unary_sync_invoker
, THRUST_FWD(first), THRUST_FWD(last)
, THRUST_FWD(output)
, THRUST_FWD(op)
);

///////////////////////////////////////////////////////////////////////////////

template <
  template <typename> class AsyncTransformUnaryInvoker
, template <typename> class SyncTransformUnaryInvoker
, template <typename> class UnaryOperation
>
struct test_async_transform_unary
{
  template <typename T>
  struct tester
  {
    __host__
    void operator()(std::size_t n)
    {
      thrust::host_vector<T>   h0(unittest::random_integers<T>(n));

      thrust::device_vector<T> d0a(h0);
      thrust::device_vector<T> d0b(h0);
      thrust::device_vector<T> d0c(h0);
      thrust::device_vector<T> d0d(h0);

      thrust::host_vector<T>   h1(n);

      thrust::device_vector<T> d1a(n);
      thrust::device_vector<T> d1b(n);
      thrust::device_vector<T> d1c(n);
      thrust::device_vector<T> d1d(n);

      AsyncTransformUnaryInvoker<T> invoke_async;
      SyncTransformUnaryInvoker<T>  invoke_sync;

      UnaryOperation<T> op;

      ASSERT_EQUAL(h0, d0a);
      ASSERT_EQUAL(h0, d0b);
      ASSERT_EQUAL(h0, d0c);
      ASSERT_EQUAL(h0, d0d);

      auto f0a = invoke_async(d0a.begin(), d0a.end(), d1a.begin(), op);
      auto f0b = invoke_async(d0b.begin(), d0b.end(), d1b.begin(), op);
      auto f0c = invoke_async(d0c.begin(), d0c.end(), d1c.begin(), op);
      auto f0d = invoke_async(d0d.begin(), d0d.end(), d1d.begin(), op);

      invoke_async.validate_event(f0a);
      invoke_async.validate_event(f0b);
      invoke_async.validate_event(f0c);
      invoke_async.validate_event(f0d);

      // This potentially runs concurrently with the copies.
      invoke_sync(h0.begin(), h0.end(), h1.begin(), op);

      TEST_EVENT_WAIT(thrust::when_all(f0a, f0b, f0c, f0d));

      ASSERT_EQUAL(h0, d0a);
      ASSERT_EQUAL(h0, d0b);
      ASSERT_EQUAL(h0, d0c);
      ASSERT_EQUAL(h0, d0d);

      ASSERT_EQUAL(h1, d1a);
      ASSERT_EQUAL(h1, d1b);
      ASSERT_EQUAL(h1, d1c);
      ASSERT_EQUAL(h1, d1d);
    }
  };
};
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_transform_unary<
      transform_unary_async_invoker
    , transform_unary_sync_invoker
    , divide_by_2
    >::tester
  )
, NumericTypes
, test_async_transform_unary_divide_by_2
);
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_transform_unary<
      transform_unary_async_invoker_device
    , transform_unary_sync_invoker
    , divide_by_2
    >::tester
  )
, NumericTypes
, test_async_transform_unary_policy_divide_by_2
);
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_transform_unary<
      transform_unary_async_invoker_device_allocator
    , transform_unary_sync_invoker
    , divide_by_2
    >::tester
  )
, NumericTypes
, test_async_transform_unary_policy_allocator_divide_by_2
);
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_transform_unary<
      transform_unary_async_invoker_device_on
    , transform_unary_sync_invoker
    , divide_by_2
    >::tester
  )
, NumericTypes
, test_async_transform_unary_policy_on_divide_by_2
);
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_transform_unary<
      transform_unary_async_invoker_device_allocator_on
    , transform_unary_sync_invoker
    , divide_by_2
    >::tester
  )
, NumericTypes
, test_async_transform_unary_policy_allocator_on_divide_by_2
);

///////////////////////////////////////////////////////////////////////////////

template <
  template <typename> class AsyncTransformUnaryInvoker
, template <typename> class SyncTransformUnaryInvoker
, template <typename> class UnaryOperation
>
struct test_async_transform_unary_inplace
{
  template <typename T>
  struct tester
  {
    __host__
    void operator()(std::size_t n)
    {
      thrust::host_vector<T>   h0(unittest::random_integers<T>(n));

      thrust::device_vector<T> d0a(h0);
      thrust::device_vector<T> d0b(h0);
      thrust::device_vector<T> d0c(h0);
      thrust::device_vector<T> d0d(h0);

      AsyncTransformUnaryInvoker<T> invoke_async;
      SyncTransformUnaryInvoker<T>  invoke_sync;

      UnaryOperation<T> op;

      ASSERT_EQUAL(h0, d0a);
      ASSERT_EQUAL(h0, d0b);
      ASSERT_EQUAL(h0, d0c);
      ASSERT_EQUAL(h0, d0d);

      auto f0a = invoke_async(d0a.begin(), d0a.end(), d0a.begin(), op);
      auto f0b = invoke_async(d0b.begin(), d0b.end(), d0b.begin(), op);
      auto f0c = invoke_async(d0c.begin(), d0c.end(), d0c.begin(), op);
      auto f0d = invoke_async(d0d.begin(), d0d.end(), d0d.begin(), op);

      invoke_async.validate_event(f0a);
      invoke_async.validate_event(f0b);
      invoke_async.validate_event(f0c);
      invoke_async.validate_event(f0d);

      // This potentially runs concurrently with the copies.
      invoke_sync(h0.begin(), h0.end(), h0.begin(), op);

      TEST_EVENT_WAIT(thrust::when_all(f0a, f0b, f0c, f0d));

      ASSERT_EQUAL(h0, d0a);
      ASSERT_EQUAL(h0, d0b);
      ASSERT_EQUAL(h0, d0c);
      ASSERT_EQUAL(h0, d0d);
    }
  };
};
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_transform_unary_inplace<
      transform_unary_async_invoker
    , transform_unary_sync_invoker
    , divide_by_2
    >::tester
  )
, NumericTypes
, test_async_transform_unary_inplace_divide_by_2
);
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_transform_unary_inplace<
      transform_unary_async_invoker_device
    , transform_unary_sync_invoker
    , divide_by_2
    >::tester
  )
, NumericTypes
, test_async_transform_unary_inplace_policy_divide_by_2
);
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_transform_unary_inplace<
      transform_unary_async_invoker_device_allocator
    , transform_unary_sync_invoker
    , divide_by_2
    >::tester
  )
, NumericTypes
, test_async_transform_unary_inplace_policy_allocator_divide_by_2
);
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_transform_unary_inplace<
      transform_unary_async_invoker_device_on
    , transform_unary_sync_invoker
    , divide_by_2
    >::tester
  )
, NumericTypes
, test_async_transform_unary_inplace_policy_on_divide_by_2
);
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_transform_unary_inplace<
      transform_unary_async_invoker_device_allocator_on
    , transform_unary_sync_invoker
    , divide_by_2
    >::tester
  )
, NumericTypes
, test_async_transform_unary_inplace_policy_allocator_on_divide_by_2
);

///////////////////////////////////////////////////////////////////////////////

template <
  template <typename> class AsyncTransformUnaryInvoker
, template <typename> class SyncTransformUnaryInvoker
, template <typename> class UnaryOperation
>
struct test_async_transform_unary_counting_iterator
{
  template <typename T>
  struct tester
  {
    __host__
    void operator()()
    {
      constexpr std::size_t n = 15 * sizeof(T);

      ASSERT_LEQUAL(T(n), unittest::truncate_to_max_representable<T>(n));

      thrust::counting_iterator<T> first(0);
      thrust::counting_iterator<T> last(n);

      thrust::host_vector<T>   h0(n);

      thrust::device_vector<T> d0a(n);
      thrust::device_vector<T> d0b(n);
      thrust::device_vector<T> d0c(n);
      thrust::device_vector<T> d0d(n);

      AsyncTransformUnaryInvoker<T> invoke_async;
      SyncTransformUnaryInvoker<T>  invoke_sync;

      UnaryOperation<T> op;

      auto f0a = invoke_async(first, last, d0a.begin(), op);
      auto f0b = invoke_async(first, last, d0b.begin(), op);
      auto f0c = invoke_async(first, last, d0c.begin(), op);
      auto f0d = invoke_async(first, last, d0d.begin(), op);

      invoke_async.validate_event(f0a);
      invoke_async.validate_event(f0b);
      invoke_async.validate_event(f0c);
      invoke_async.validate_event(f0d);

      // This potentially runs concurrently with the copies.
      invoke_sync(first, last, h0.begin(), op);

      TEST_EVENT_WAIT(thrust::when_all(f0a, f0b, f0c, f0d));

      ASSERT_EQUAL(h0, d0a);
      ASSERT_EQUAL(h0, d0b);
      ASSERT_EQUAL(h0, d0c);
      ASSERT_EQUAL(h0, d0d);
    }
  };
};
DECLARE_GENERIC_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_transform_unary_counting_iterator<
      transform_unary_async_invoker
    , transform_unary_sync_invoker
    , divide_by_2
    >::tester
  )
, BuiltinNumericTypes
, test_async_transform_unary_counting_iterator_divide_by_2
);
DECLARE_GENERIC_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_transform_unary_counting_iterator<
      transform_unary_async_invoker_device
    , transform_unary_sync_invoker
    , divide_by_2
    >::tester
  )
, BuiltinNumericTypes
, test_async_transform_unary_counting_iterator_policy_divide_by_2
);

///////////////////////////////////////////////////////////////////////////////

template <
  template <typename> class UnaryOperation
>
struct test_async_transform_using
{
  template <typename T>
  struct tester
  {
    __host__
    void operator()(std::size_t n)
    {
      thrust::host_vector<T>   h0(unittest::random_integers<T>(n));

      thrust::device_vector<T> d0a(h0);
      thrust::device_vector<T> d0b(h0);

      thrust::host_vector<T>   h1(n);

      thrust::device_vector<T> d1a(n);
      thrust::device_vector<T> d1b(n);

      UnaryOperation<T> op;

      ASSERT_EQUAL(h0, d0a);
      ASSERT_EQUAL(h0, d0b);

      thrust::device_event f0a;
      thrust::device_event f0b;

      // When you import the customization points into the global namespace,
      // they should be selected instead of the synchronous algorithms.
      {
        using namespace thrust::async;
        f0a = transform(d0a.begin(), d0a.end(), d1a.begin(), op);
      }
      {
        using thrust::async::transform;
        f0b = transform(d0b.begin(), d0b.end(), d1b.begin(), op);
      }

      // ADL should find the synchronous algorithms.
      // This potentially runs concurrently with the copies.
      transform(h0.begin(), h0.end(), h1.begin(), op);

      TEST_EVENT_WAIT(thrust::when_all(f0a, f0b));

      ASSERT_EQUAL(h0, d0a);
      ASSERT_EQUAL(h0, d0b);

      ASSERT_EQUAL(h1, d1a);
      ASSERT_EQUAL(h1, d1b);
    }
  };
};
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND(test_async_transform_using<divide_by_2>::tester)
, NumericTypes
, test_async_transform_using_divide_by_2
);

///////////////////////////////////////////////////////////////////////////////

#endif

