#include <unittest/unittest.h>
#include <thrust/remove.h>
#include <thrust/count.h>
#include <thrust/functional.h>
#include <stdexcept>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/zip_iterator.h>

// for testing dispatch
struct my_system : thrust::device_system<my_system> {};

template<typename T>
struct is_even
  : thrust::unary_function<T,bool>
{
    __host__ __device__
    bool operator()(T x) { return (static_cast<unsigned int>(x) & 1) == 0; }
};

template<typename T>
struct is_true
  : thrust::unary_function<T,bool>
{
    __host__ __device__
    bool operator()(T x) { return x ? true : false; }
};

template<typename Vector>
void TestRemoveSimple(void)
{
    typedef typename Vector::value_type T;

    Vector data(5);
    data[0] =  1; 
    data[1] =  2; 
    data[2] =  1;
    data[3] =  3; 
    data[4] =  2; 

    typename Vector::iterator end = thrust::remove(data.begin(), 
                                                    data.end(), 
                                                    (T) 2);

    ASSERT_EQUAL(end - data.begin(), 3);

    ASSERT_EQUAL(data[0], 1);
    ASSERT_EQUAL(data[1], 1);
    ASSERT_EQUAL(data[2], 3);
}
DECLARE_VECTOR_UNITTEST(TestRemoveSimple);


template<typename ForwardIterator,
         typename T>
ForwardIterator remove(my_system,
                       ForwardIterator first,
                       ForwardIterator,
                       const T &)
{
    *first = 13;
    return first;
}

void TestRemoveDispatchExplicit()
{
    thrust::device_vector<int> vec(1);

    my_system sys;
    thrust::remove(sys, vec.begin(), vec.end(), 0);

    ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestRemoveDispatchExplicit);

void TestRemoveDispatchImplicit()
{
    thrust::device_vector<int> vec(1);

    thrust::remove(thrust::retag<my_system>(vec.begin()),
                   thrust::retag<my_system>(vec.begin()),
                   0);

    ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestRemoveDispatchImplicit);


template<typename Vector>
void TestRemoveCopySimple(void)
{
    typedef typename Vector::value_type T;

    Vector data(5);
    data[0] =  1; 
    data[1] =  2; 
    data[2] =  1;
    data[3] =  3; 
    data[4] =  2; 

    Vector result(5);

    typename Vector::iterator end = thrust::remove_copy(data.begin(), 
                                                         data.end(), 
                                                         result.begin(), 
                                                         (T) 2);

    ASSERT_EQUAL(end - result.begin(), 3);

    ASSERT_EQUAL(result[0], 1);
    ASSERT_EQUAL(result[1], 1);
    ASSERT_EQUAL(result[2], 3);
}
DECLARE_VECTOR_UNITTEST(TestRemoveCopySimple);


template<typename InputIterator,
         typename OutputIterator,
         typename T>
OutputIterator remove_copy(my_system,
                           InputIterator,
                           InputIterator,
                           OutputIterator result,
                           const T &)
{
    *result = 13;
    return result;
}

void TestRemoveCopyDispatchExplicit()
{
    thrust::device_vector<int> vec(1);

    my_system sys;
    thrust::remove_copy(sys,
                        vec.begin(),
                        vec.begin(),
                        vec.begin(),
                        0);

    ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestRemoveCopyDispatchExplicit);

void TestRemoveCopyDispatchImplicit()
{
    thrust::device_vector<int> vec(1);

    thrust::remove_copy(thrust::retag<my_system>(vec.begin()),
                        thrust::retag<my_system>(vec.begin()),
                        thrust::retag<my_system>(vec.begin()),
                        0);

    ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestRemoveCopyDispatchImplicit);


template<typename Vector>
void TestRemoveIfSimple(void)
{
    typedef typename Vector::value_type T;

    Vector data(5);
    data[0] =  1; 
    data[1] =  2; 
    data[2] =  1;
    data[3] =  3; 
    data[4] =  2; 

    typename Vector::iterator end = thrust::remove_if(data.begin(), 
                                                      data.end(), 
                                                      is_even<T>());

    ASSERT_EQUAL(end - data.begin(), 3);

    ASSERT_EQUAL(data[0], 1);
    ASSERT_EQUAL(data[1], 1);
    ASSERT_EQUAL(data[2], 3);
}
DECLARE_VECTOR_UNITTEST(TestRemoveIfSimple);


template<typename ForwardIterator,
         typename Predicate>
ForwardIterator remove_if(my_system,
                          ForwardIterator first,
                          ForwardIterator,
                          Predicate pred)
{
    *first = 13;
    return first;
}

void TestRemoveIfDispatchExplicit()
{
    thrust::device_vector<int> vec(1);

    my_system sys;
    thrust::remove_if(sys, vec.begin(), vec.end(), 0);

    ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestRemoveIfDispatchExplicit);

void TestRemoveIfDispatchImplicit()
{
    thrust::device_vector<int> vec(1);

    thrust::remove_if(thrust::retag<my_system>(vec.begin()),
                      thrust::retag<my_system>(vec.begin()),
                      0);

    ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestRemoveIfDispatchImplicit);


template<typename Vector>
void TestRemoveIfStencilSimple(void)
{
    typedef typename Vector::value_type T;

    Vector data(5);
    data[0] =  1; 
    data[1] =  2; 
    data[2] =  1;
    data[3] =  3; 
    data[4] =  2; 

    Vector stencil(5);
    stencil[0] = 0;
    stencil[1] = 1;
    stencil[2] = 0;
    stencil[3] = 0;
    stencil[4] = 1;

    typename Vector::iterator end = thrust::remove_if(data.begin(), 
                                                      data.end(),
                                                      stencil.begin(),
                                                      thrust::identity<T>());

    ASSERT_EQUAL(end - data.begin(), 3);

    ASSERT_EQUAL(data[0], 1);
    ASSERT_EQUAL(data[1], 1);
    ASSERT_EQUAL(data[2], 3);
}
DECLARE_VECTOR_UNITTEST(TestRemoveIfStencilSimple);


template<typename ForwardIterator,
         typename InputIterator,
         typename Predicate>
ForwardIterator remove_if(my_system,
                          ForwardIterator first,
                          ForwardIterator,
                          InputIterator,
                          Predicate pred)
{
    *first = 13;
    return first;
}

void TestRemoveIfStencilDispatchExplicit()
{
    thrust::device_vector<int> vec(1);

    my_system sys;
    thrust::remove_if(sys,
                      vec.begin(),
                      vec.begin(),
                      vec.begin(),
                      0);

    ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestRemoveIfStencilDispatchExplicit);

void TestRemoveIfStencilDispatchImplicit()
{
    thrust::device_vector<int> vec(1);

    thrust::remove_if(thrust::retag<my_system>(vec.begin()),
                      thrust::retag<my_system>(vec.begin()),
                      thrust::retag<my_system>(vec.begin()),
                      0);

    ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestRemoveIfStencilDispatchImplicit);


template<typename Vector>
void TestRemoveCopyIfSimple(void)
{
    typedef typename Vector::value_type T;

    Vector data(5);
    data[0] =  1; 
    data[1] =  2; 
    data[2] =  1;
    data[3] =  3; 
    data[4] =  2; 

    Vector result(5);

    typename Vector::iterator end = thrust::remove_copy_if(data.begin(), 
                                                           data.end(), 
                                                           result.begin(), 
                                                           is_even<T>());

    ASSERT_EQUAL(end - result.begin(), 3);

    ASSERT_EQUAL(result[0], 1);
    ASSERT_EQUAL(result[1], 1);
    ASSERT_EQUAL(result[2], 3);
}
DECLARE_VECTOR_UNITTEST(TestRemoveCopyIfSimple);


template<typename InputIterator,
         typename OutputIterator,
         typename Predicate>
InputIterator remove_copy_if(my_system,
                             InputIterator first,
                             InputIterator,
                             OutputIterator,
                             Predicate)
{
    *first = 13;
    return first;
}

void TestRemoveCopyIfDispatchExplicit()
{
    thrust::device_vector<int> vec(1);

    my_system sys;
    thrust::remove_copy_if(sys,
                           vec.begin(),
                           vec.begin(),
                           vec.begin(),
                           0);

    ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestRemoveCopyIfDispatchExplicit);

void TestRemoveCopyIfDispatchImplicit()
{
    thrust::device_vector<int> vec(1);

    thrust::remove_copy_if(thrust::retag<my_system>(vec.begin()),
                           thrust::retag<my_system>(vec.begin()),
                           thrust::retag<my_system>(vec.begin()),
                           0);

    ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestRemoveCopyIfDispatchImplicit);


template<typename Vector>
void TestRemoveCopyIfStencilSimple(void)
{
    typedef typename Vector::value_type T;

    Vector data(5);
    data[0] =  1; 
    data[1] =  2; 
    data[2] =  1;
    data[3] =  3; 
    data[4] =  2; 

    Vector stencil(5);
    stencil[0] = 0;
    stencil[1] = 1;
    stencil[2] = 0;
    stencil[3] = 0;
    stencil[4] = 1;

    Vector result(5);

    typename Vector::iterator end = thrust::remove_copy_if(data.begin(), 
                                                           data.end(), 
                                                           stencil.begin(),
                                                           result.begin(), 
                                                           thrust::identity<T>());

    ASSERT_EQUAL(end - result.begin(), 3);

    ASSERT_EQUAL(result[0], 1);
    ASSERT_EQUAL(result[1], 1);
    ASSERT_EQUAL(result[2], 3);
}
DECLARE_VECTOR_UNITTEST(TestRemoveCopyIfStencilSimple);


template<typename InputIterator1,
         typename InputIterator2,
         typename OutputIterator,
         typename Predicate>
OutputIterator remove_copy_if(my_system,
                              InputIterator1,
                              InputIterator1,
                              InputIterator2,
                              OutputIterator result,
                              Predicate)
{
    *result = 13;
    return result;
}

void TestRemoveCopyIfStencilDispatchExplicit()
{
    thrust::device_vector<int> vec(1);

    my_system sys;
    thrust::remove_copy_if(sys,
                           vec.begin(),
                           vec.begin(),
                           vec.begin(),
                           vec.begin(),
                           0);

    ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestRemoveCopyIfStencilDispatchExplicit);

void TestRemoveCopyIfStencilDispatchImplicit()
{
    thrust::device_vector<int> vec(1);

    thrust::remove_copy_if(thrust::retag<my_system>(vec.begin()),
                           thrust::retag<my_system>(vec.begin()),
                           thrust::retag<my_system>(vec.begin()),
                           thrust::retag<my_system>(vec.begin()),
                           0);

    ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestRemoveCopyIfStencilDispatchImplicit);


template<typename T>
void TestRemove(const size_t n)
{
    thrust::host_vector<T>   h_data = unittest::random_samples<T>(n);
    thrust::device_vector<T> d_data = h_data;

    size_t h_size = thrust::remove(h_data.begin(), h_data.end(), T(0)) - h_data.begin();
    size_t d_size = thrust::remove(d_data.begin(), d_data.end(), T(0)) - d_data.begin();
    
    ASSERT_EQUAL(h_size, d_size);

    h_data.resize(h_size);
    d_data.resize(d_size);

    ASSERT_EQUAL(h_data, d_data);
}
DECLARE_VARIABLE_UNITTEST(TestRemove);


template<typename T>
void TestRemoveIf(const size_t n)
{
    thrust::host_vector<T>   h_data = unittest::random_samples<T>(n);
    thrust::device_vector<T> d_data = h_data;

    size_t h_size = thrust::remove_if(h_data.begin(), h_data.end(), is_true<T>()) - h_data.begin();
    size_t d_size = thrust::remove_if(d_data.begin(), d_data.end(), is_true<T>()) - d_data.begin();
   
    ASSERT_EQUAL(h_size, d_size);

    h_data.resize(h_size);
    d_data.resize(d_size);

    ASSERT_EQUAL(h_data, d_data);
}
DECLARE_VARIABLE_UNITTEST(TestRemoveIf);


template<typename T>
void TestRemoveIfStencil(const size_t n)
{
    thrust::host_vector<T>   h_data = unittest::random_samples<T>(n);
    thrust::device_vector<T> d_data = h_data;

    thrust::host_vector<bool>   h_stencil = unittest::random_integers<bool>(n);
    thrust::device_vector<bool> d_stencil = h_stencil;
    
    size_t h_size = thrust::remove_if(h_data.begin(), h_data.end(), h_stencil.begin(), is_true<T>()) - h_data.begin();
    size_t d_size = thrust::remove_if(d_data.begin(), d_data.end(), d_stencil.begin(), is_true<T>()) - d_data.begin();
   
    ASSERT_EQUAL(h_size, d_size);

    h_data.resize(h_size);
    d_data.resize(d_size);

    ASSERT_EQUAL(h_data, d_data);
}
DECLARE_VARIABLE_UNITTEST(TestRemoveIfStencil);


template<typename T>
void TestRemoveCopy(const size_t n)
{
    thrust::host_vector<T>   h_data = unittest::random_samples<T>(n);
    thrust::device_vector<T> d_data = h_data;
    
    thrust::host_vector<T>   h_result(n);
    thrust::device_vector<T> d_result(n);

    size_t h_size = thrust::remove_copy(h_data.begin(), h_data.end(), h_result.begin(), T(0)) - h_result.begin();
    size_t d_size = thrust::remove_copy(d_data.begin(), d_data.end(), d_result.begin(), T(0)) - d_result.begin();
    
    ASSERT_EQUAL(h_size, d_size);

    h_result.resize(h_size);
    d_result.resize(d_size);

    ASSERT_EQUAL(h_result, d_result);
}
DECLARE_VARIABLE_UNITTEST(TestRemoveCopy);


template<typename T>
void TestRemoveCopyToDiscardIterator(const size_t n)
{
    thrust::host_vector<T>   h_data = unittest::random_samples<T>(n);
    thrust::device_vector<T> d_data = h_data;

    size_t num_zeros = thrust::count(h_data.begin(), h_data.end(), T(0));
    size_t num_nonzeros = h_data.size() - num_zeros;

    thrust::discard_iterator<> h_result =
      thrust::remove_copy(h_data.begin(), h_data.end(), thrust::make_discard_iterator(), T(0));

    thrust::discard_iterator<> d_result =
      thrust::remove_copy(d_data.begin(), d_data.end(), thrust::make_discard_iterator(), T(0));

    thrust::discard_iterator<> reference(num_nonzeros);
    
    ASSERT_EQUAL_QUIET(reference, h_result);
    ASSERT_EQUAL_QUIET(reference, d_result);
}
DECLARE_VARIABLE_UNITTEST(TestRemoveCopyToDiscardIterator);


template<typename T>
void TestRemoveCopyToDiscardIteratorZipped(const size_t n)
{
    thrust::host_vector<T>   h_data = unittest::random_samples<T>(n);
    thrust::device_vector<T> d_data = h_data;

    thrust::host_vector<T> h_output(n);
    thrust::device_vector<T> d_output(n);

    size_t num_zeros = thrust::count(h_data.begin(), h_data.end(), T(0));
    size_t num_nonzeros = h_data.size() - num_zeros;

    typedef thrust::tuple<typename thrust::host_vector<T>::iterator, thrust::discard_iterator<> >   Tuple1;
    typedef thrust::tuple<typename thrust::device_vector<T>::iterator, thrust::discard_iterator<> > Tuple2;

    typedef thrust::zip_iterator<Tuple1> ZipIterator1;
    typedef thrust::zip_iterator<Tuple2> ZipIterator2;

    ZipIterator1 h_result =
      thrust::remove_copy(thrust::make_zip_iterator(thrust::make_tuple(h_data.begin(), h_data.begin())),
                          thrust::make_zip_iterator(thrust::make_tuple(h_data.end(), h_data.end())),
                          thrust::make_zip_iterator(thrust::make_tuple(h_output.begin(),thrust::make_discard_iterator())),
                          thrust::make_tuple(T(0),T(0)));

    ZipIterator2 d_result =
      thrust::remove_copy(thrust::make_zip_iterator(thrust::make_tuple(d_data.begin(), d_data.begin())),
                          thrust::make_zip_iterator(thrust::make_tuple(d_data.end(), d_data.end())),
                          thrust::make_zip_iterator(thrust::make_tuple(d_output.begin(),thrust::make_discard_iterator())),
                          thrust::make_tuple(T(0),T(0)));

    thrust::discard_iterator<> reference(num_nonzeros);
    
    ASSERT_EQUAL(h_output, d_output);
    ASSERT_EQUAL_QUIET(reference, thrust::get<1>(h_result.get_iterator_tuple()));
    ASSERT_EQUAL_QUIET(reference, thrust::get<1>(d_result.get_iterator_tuple()));
}
DECLARE_VARIABLE_UNITTEST(TestRemoveCopyToDiscardIteratorZipped);


template<typename T>
void TestRemoveCopyIf(const size_t n)
{
    thrust::host_vector<T>   h_data = unittest::random_samples<T>(n);
    thrust::device_vector<T> d_data = h_data;

    thrust::host_vector<T>   h_result(n);
    thrust::device_vector<T> d_result(n);
    
    size_t h_size = thrust::remove_copy_if(h_data.begin(), h_data.end(), h_result.begin(), is_true<T>()) - h_result.begin();
    size_t d_size = thrust::remove_copy_if(d_data.begin(), d_data.end(), d_result.begin(), is_true<T>()) - d_result.begin();
    
    ASSERT_EQUAL(h_size, d_size);

    h_result.resize(h_size);
    d_result.resize(d_size);

    ASSERT_EQUAL(h_result, d_result);
}
DECLARE_VARIABLE_UNITTEST(TestRemoveCopyIf);


template<typename T>
void TestRemoveCopyIfToDiscardIterator(const size_t n)
{
    thrust::host_vector<T>   h_data = unittest::random_samples<T>(n);
    thrust::device_vector<T> d_data = h_data;

    size_t num_false = thrust::count_if(h_data.begin(), h_data.end(), thrust::not1(is_true<T>()));

    thrust::discard_iterator<> h_result =
      thrust::remove_copy_if(h_data.begin(), h_data.end(), thrust::make_discard_iterator(), is_true<T>());

    thrust::discard_iterator<> d_result =
      thrust::remove_copy_if(d_data.begin(), d_data.end(), thrust::make_discard_iterator(), is_true<T>());

    thrust::discard_iterator<> reference(num_false);

    ASSERT_EQUAL_QUIET(reference, h_result);
    ASSERT_EQUAL_QUIET(reference, d_result);
}
DECLARE_VARIABLE_UNITTEST(TestRemoveCopyIfToDiscardIterator);


template<typename T>
void TestRemoveCopyIfStencil(const size_t n)
{
    thrust::host_vector<T>   h_data = unittest::random_samples<T>(n);
    thrust::device_vector<T> d_data = h_data;
    
    thrust::host_vector<bool>   h_stencil = unittest::random_integers<bool>(n);
    thrust::device_vector<bool> d_stencil = h_stencil;
    
    thrust::host_vector<T>   h_result(n);
    thrust::device_vector<T> d_result(n);

    size_t h_size = thrust::remove_copy_if(h_data.begin(), h_data.end(), h_stencil.begin(), h_result.begin(), is_true<T>()) - h_result.begin();
    size_t d_size = thrust::remove_copy_if(d_data.begin(), d_data.end(), d_stencil.begin(), d_result.begin(), is_true<T>()) - d_result.begin();
    
    ASSERT_EQUAL(h_size, d_size);

    h_result.resize(h_size);
    d_result.resize(d_size);

    ASSERT_EQUAL(h_result, d_result);
}
DECLARE_VARIABLE_UNITTEST(TestRemoveCopyIfStencil);


template<typename T>
void TestRemoveCopyIfStencilToDiscardIterator(const size_t n)
{
    thrust::host_vector<T>   h_data = unittest::random_samples<T>(n);
    thrust::device_vector<T> d_data = h_data;
    
    thrust::host_vector<bool>   h_stencil = unittest::random_integers<bool>(n);
    thrust::device_vector<bool> d_stencil = h_stencil;

    size_t num_false = thrust::count_if(h_stencil.begin(), h_stencil.end(), thrust::not1(is_true<T>()));

    thrust::discard_iterator<> h_result =
      thrust::remove_copy_if(h_data.begin(), h_data.end(), h_stencil.begin(), thrust::make_discard_iterator(), is_true<T>());

    thrust::discard_iterator<> d_result =
      thrust::remove_copy_if(d_data.begin(), d_data.end(), d_stencil.begin(), thrust::make_discard_iterator(), is_true<T>());

    thrust::discard_iterator<> reference(num_false);

    ASSERT_EQUAL_QUIET(reference, h_result);
    ASSERT_EQUAL_QUIET(reference, d_result);
}
DECLARE_VARIABLE_UNITTEST(TestRemoveCopyIfStencilToDiscardIterator);

