#include <thrust/detail/config.h>

#if THRUST_CPP_DIALECT >= 2014

#include <async/test_policy_overloads.h>

#include <async/inclusive_scan/mixin.h>

#include <algorithm>
#include <limits>

template <typename input_value_type,
          typename output_value_type   = input_value_type,
          typename alternate_binary_op = thrust::maximum<>>
struct invoker
    : testing::async::mixin::input::counting_iterator_from_0<input_value_type>
    , testing::async::mixin::output::device_vector<output_value_type>
    , testing::async::inclusive_scan::mixin::postfix_args::
        all_overloads<alternate_binary_op>
    , testing::async::inclusive_scan::mixin::invoke_reference::
        host_synchronous<input_value_type, output_value_type>
    , testing::async::inclusive_scan::mixin::invoke_async::simple
    , testing::async::mixin::compare_outputs::assert_almost_equal_if_fp_quiet
{
  static std::string description()
  {
    return "fancy input iterator (counting_iterator)";
  }
};

template <typename T>
struct test_counting_iterator
{
  void operator()(std::size_t num_values) const
  {
    num_values = static_cast<std::size_t>(unittest::truncate_to_max_representable<T>(num_values));
    testing::async::test_policy_overloads<invoker<T>>::run(num_values);
  }
};
// Use built-in types only, counting_iterator doesn't seem to be compatible with
// the custom_numeric.
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES(test_counting_iterator,
                                          BuiltinNumericTypes);

#endif // C++14
