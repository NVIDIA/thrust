#include <unittest/unittest.h>
#include <thrust/mr/new.h>
#include <thrust/fill.h>

template<typename MemoryResource>
void TestAlignment(MemoryResource memres, std::size_t size, std::size_t alignment)
{
    void * ptr = memres.do_allocate(size, alignment);
    ASSERT_EQUAL(reinterpret_cast<std::size_t>(ptr) % alignment, 0u);

    char * char_ptr = reinterpret_cast<char *>(ptr);
    thrust::fill(char_ptr, char_ptr + size, 0);

    memres.do_deallocate(ptr, size, alignment);
}

static const std::size_t MinTestedSize = 32;
static const std::size_t MaxTestedSize = 8 * 1024;
static const std::size_t TestedSizeStep = 1;

static const std::size_t MinTestedAlignment = 16;
static const std::size_t MaxTestedAlignment = 4 * 1024;
static const std::size_t TestedAlignmentShift = 1;

void TestNewDeleteResourceAlignedAllocation()
{
    for (std::size_t size = MinTestedSize; size <= MaxTestedSize; size += TestedSizeStep)
    {
        for (std::size_t alignment = MinTestedAlignment; alignment <= MaxTestedAlignment;
            alignment <<= TestedAlignmentShift)
        {
            TestAlignment(thrust::mr::new_delete_resource(), size, alignment);
        }
    }
}

void TestNewDeleteResourceEmptyAllocation()
{
    thrust::mr::new_delete_resource memres;
    ASSERT_EQUAL(nullptr, memres.do_allocate(0));
}

DECLARE_UNITTEST(TestNewDeleteResourceAlignedAllocation);
DECLARE_UNITTEST(TestNewDeleteResourceEmptyAllocation);

