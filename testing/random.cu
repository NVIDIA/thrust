#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/random.h>
#include <thrust/generate.h>
#include <sstream>

template<typename Engine>
  struct ValidateEngine
{
  __host__ __device__
  ValidateEngine(const typename Engine::result_type value_10000)
    : m_value_10000(value_10000)
  {}

  __host__ __device__
  bool operator()(void) const
  {
    Engine e;
    e.discard(9999);

    // get the 10Kth result
    return e() == m_value_10000;
  }

  const typename Engine::result_type m_value_10000;
}; // end ValidateEngine


template<typename Engine,
         bool trivial_min = (Engine::min == 0)>
  struct ValidateEngineMin
{
  __host__ __device__
  bool operator()(void) const
  {
    Engine e;

    bool result = true;

    for(int i = 0; i < 10000; ++i)
    {
      result &= (e() >= Engine::min);
    }

    return result;
  }
}; // end ValidateEngineMin

template<typename Engine>
  struct ValidateEngineMin<Engine,true>
{
  __host__ __device__
  bool operator()(void) const
  {
    return true;
  }
};


template<typename Engine>
  struct ValidateEngineMax
{
  __host__ __device__
  bool operator()(void) const
  {
    Engine e;

    bool result = true;

    for(int i = 0; i < 10000; ++i)
    {
      result &= (e() <= Engine::max);
    }

    return result;
  }
}; // end ValidateEngineMax


template<typename Engine>
  struct ValidateEngineEqual
{
  __host__ __device__
  bool operator()(void) const
  {
    bool result = true;

    // test from default constructor
    Engine e0, e1;
    result &= (e0 == e1);

    // advance engines
    e0.discard(10000);
    e1.discard(10000);
    result &= (e0 == e1);

    // test from identical seeds
    Engine e2(13), e3(13);
    result &= (e2 == e3);

    // test different seeds aren't equal
    Engine e4(7), e5(13);
    result &= !(e4 == e5);

    // test reseeding engine to the same seed causes equality
    e4.seed(13);
    result &= (e4 == e5);

    return result;
  }
};


template<typename Engine>
  struct ValidateEngineUnequal
{
  __host__ __device__
  bool operator()(void) const
  {
    bool result = true;

    // test from default constructor
    Engine e0, e1;
    result &= !(e0 != e1);

    // advance engines
    e0.discard(1000);
    e1.discard(1000);
    result &= !(e0 != e1);

    // test from identical seeds
    Engine e2(13), e3(13);
    result &= !(e2 != e3);

    // test different seeds aren't equal
    Engine e4(7), e5(13);
    result &= (e4 != e5);

    // test reseeding engine to the same seed causes equality
    e4.seed(13);
    result &= !(e4 != e5);

    // test different discards causes inequality
    Engine e6(13), e7(13);
    e6.discard(500);
    e7.discard(1000);
    result &= (e6 != e7);

    return result;
  }
};


template<typename Distribution, typename Engine>
  struct ValidateDistributionMin
{
  typedef Engine random_engine;

  __host__ __device__
  ValidateDistributionMin(const Distribution &dd)
    : d(dd)
  {}

  __host__ __device__
  bool operator()(void)
  {
    Engine e;

    bool result = true;

    for(int i = 0; i < 10000; ++i)
    {
      result &= (d(e) >= d.min());
    }

    return result;
  }

  Distribution d;
};


template<typename Distribution, typename Engine>
  struct ValidateDistributionMax
{
  typedef Engine random_engine;

  __host__ __device__
  ValidateDistributionMax(const Distribution &dd)
    : d(dd)
  {}

  __host__ __device__
  bool operator()(void)
  {
    Engine e;

    bool result = true;

    for(int i = 0; i < 10000; ++i)
    {
      result &= (d(e) <= d.max());
    }

    return result;
  }

  Distribution d;
};


template<typename Distribution>
  struct ValidateDistributionEqual
{
  __host__ __device__
  bool operator()(void) const
  {
    return d0 == d1;
  }

  Distribution d0, d1;
};


template<typename Distribution>
  struct ValidateDistributionUnqual
{
  __host__ __device__
  bool operator()(void) const
  {
    return d0 != d1;
  }

  Distribution d0, d1;
};


template<typename Engine, thrust::detail::uint64_t value_10000>
void TestEngineValidation(void)
{
  // test host
  thrust::host_vector<bool> h(1);
  thrust::generate(h.begin(), h.end(), ValidateEngine<Engine>(value_10000));

  ASSERT_EQUAL(true, h[0]);

  // test device
  thrust::device_vector<bool> d(1);
  thrust::generate(d.begin(), d.end(), ValidateEngine<Engine>(value_10000));

  ASSERT_EQUAL(true, d[0]);
}


template<typename Engine>
void TestEngineMax(void)
{
  // test host
  thrust::host_vector<bool> h(1);
  thrust::generate(h.begin(), h.end(), ValidateEngineMax<Engine>());

  ASSERT_EQUAL(true, h[0]);

  // test device
  thrust::device_vector<bool> d(1);
  thrust::generate(d.begin(), d.end(), ValidateEngineMax<Engine>());

  ASSERT_EQUAL(true, d[0]);
}


template<typename Engine>
void TestEngineMin(void)
{
  // test host
  thrust::host_vector<bool> h(1);
  thrust::generate(h.begin(), h.end(), ValidateEngineMin<Engine>());

  ASSERT_EQUAL(true, h[0]);

  // test device
  thrust::device_vector<bool> d(1);
  thrust::generate(d.begin(), d.end(), ValidateEngineMin<Engine>());

  ASSERT_EQUAL(true, d[0]);
}


template<typename Engine>
void TestEngineSaveRestore(void)
{
  // create a default engine
  Engine e0;

  // run it for a while
  e0.discard(10000);

  // save it
  std::stringstream ss;
  ss << e0;

  // run it a while longer
  e0.discard(10000);

  // restore old state
  Engine e1;
  ss >> e1;

  // run e1 a while longer
  e1.discard(10000);

  // both should return the same result

  ASSERT_EQUAL(e0(), e1());
}


template<typename Engine>
void TestEngineEqual(void)
{
  ValidateEngineEqual<Engine> f;

  // test host
  thrust::host_vector<bool> h(1);
  thrust::generate(h.begin(), h.end(), f);

  ASSERT_EQUAL(true, h[0]);

  // test device
  thrust::device_vector<bool> d(1);
  thrust::generate(d.begin(), d.end(), f);

  ASSERT_EQUAL(true, d[0]);
}


template<typename Engine>
void TestEngineUnequal(void)
{
  ValidateEngineUnequal<Engine> f;

  // test host
  thrust::host_vector<bool> h(1);
  thrust::generate(h.begin(), h.end(), f);

  ASSERT_EQUAL(true, h[0]);

  // test device
  thrust::device_vector<bool> d(1);
  thrust::generate(d.begin(), d.end(), f);

  ASSERT_EQUAL(true, d[0]);
}

void TestRanlux24BaseValidation(void)
{
  typedef thrust::random::ranlux24_base Engine;

  TestEngineValidation<Engine,7937952u>();
}
DECLARE_UNITTEST(TestRanlux24BaseValidation);


void TestRanlux24BaseMin(void)
{
  typedef thrust::random::ranlux24_base Engine;

  TestEngineMin<Engine>();
}
DECLARE_UNITTEST(TestRanlux24BaseMin);


void TestRanlux24BaseMax(void)
{
  typedef thrust::random::ranlux24_base Engine;

  TestEngineMax<Engine>();
}
DECLARE_UNITTEST(TestRanlux24BaseMax);


void TestRanlux24BaseSaveRestore(void)
{
  typedef thrust::random::ranlux24_base Engine;

  TestEngineSaveRestore<Engine>();
}
DECLARE_UNITTEST(TestRanlux24BaseSaveRestore);


void TestRanlux24BaseEqual(void)
{
  typedef thrust::random::ranlux24_base Engine;

  TestEngineEqual<Engine>();
}
DECLARE_UNITTEST(TestRanlux24BaseEqual);


void TestRanlux24BaseUnequal(void)
{
  typedef thrust::random::ranlux24_base Engine;

  TestEngineUnequal<Engine>();
}
DECLARE_UNITTEST(TestRanlux24BaseUnequal);


void TestRanlux48BaseValidation(void)
{
  typedef thrust::random::ranlux48_base Engine;

  TestEngineValidation<Engine,192113843633948ull>();
}
DECLARE_UNITTEST(TestRanlux48BaseValidation);


void TestRanlux48BaseMin(void)
{
  typedef thrust::random::ranlux48_base Engine;

  TestEngineMin<Engine>();
}
DECLARE_UNITTEST(TestRanlux48BaseMin);


void TestRanlux48BaseMax(void)
{
  typedef thrust::random::ranlux48_base Engine;

  TestEngineMax<Engine>();
}
DECLARE_UNITTEST(TestRanlux48BaseMax);


void TestRanlux48BaseSaveRestore(void)
{
  typedef thrust::random::ranlux48_base Engine;

  TestEngineSaveRestore<Engine>();
}
DECLARE_UNITTEST(TestRanlux48BaseSaveRestore);


void TestRanlux48BaseEqual(void)
{
  typedef thrust::random::ranlux48_base Engine;

  TestEngineEqual<Engine>();
}
DECLARE_UNITTEST(TestRanlux48BaseEqual);


void TestRanlux48BaseUnequal(void)
{
  typedef thrust::random::ranlux48_base Engine;

  TestEngineUnequal<Engine>();
}
DECLARE_UNITTEST(TestRanlux48BaseUnequal);


void TestMinstdRandValidation(void)
{
  typedef thrust::random::minstd_rand Engine;

  TestEngineValidation<Engine,399268537u>();
}
DECLARE_UNITTEST(TestMinstdRandValidation);


void TestMinstdRandMin(void)
{
  typedef thrust::random::minstd_rand Engine;

  TestEngineMin<Engine>();
}
DECLARE_UNITTEST(TestMinstdRandMin);


void TestMinstdRandMax(void)
{
  typedef thrust::random::minstd_rand Engine;

  TestEngineMax<Engine>();
}
DECLARE_UNITTEST(TestMinstdRandMax);


void TestMinstdRandSaveRestore(void)
{
  typedef thrust::random::minstd_rand Engine;

  TestEngineSaveRestore<Engine>();
}
DECLARE_UNITTEST(TestMinstdRandSaveRestore);


void TestMinstdRandEqual(void)
{
  typedef thrust::random::minstd_rand Engine;

  TestEngineEqual<Engine>();
}
DECLARE_UNITTEST(TestMinstdRandEqual);


void TestMinstdRandUnequal(void)
{
  typedef thrust::random::minstd_rand Engine;

  TestEngineUnequal<Engine>();
}
DECLARE_UNITTEST(TestMinstdRandUnequal);


void TestMinstdRand0Validation(void)
{
  typedef thrust::random::minstd_rand0 Engine;

  TestEngineValidation<Engine,1043618065u>();
}
DECLARE_UNITTEST(TestMinstdRand0Validation);


void TestMinstdRand0Min(void)
{
  typedef thrust::random::minstd_rand0 Engine;

  TestEngineMin<Engine>();
}
DECLARE_UNITTEST(TestMinstdRand0Min);


void TestMinstdRand0Max(void)
{
  typedef thrust::random::minstd_rand0 Engine;

  TestEngineMax<Engine>();
}
DECLARE_UNITTEST(TestMinstdRand0Max);


void TestMinstdRand0SaveRestore(void)
{
  typedef thrust::random::minstd_rand0 Engine;

  TestEngineSaveRestore<Engine>();
}
DECLARE_UNITTEST(TestMinstdRand0SaveRestore);


void TestMinstdRand0Equal(void)
{
  typedef thrust::random::minstd_rand0 Engine;

  TestEngineEqual<Engine>();
}
DECLARE_UNITTEST(TestMinstdRand0Equal);


void TestMinstdRand0Unequal(void)
{
  typedef thrust::random::minstd_rand0 Engine;

  TestEngineUnequal<Engine>();
}
DECLARE_UNITTEST(TestMinstdRand0Unequal);


void TestTaus88Validation(void)
{
  typedef thrust::random::taus88 Engine;

  TestEngineValidation<Engine,3535848941ull>();
}
DECLARE_UNITTEST(TestTaus88Validation);


void TestTaus88Min(void)
{
  typedef thrust::random::taus88 Engine;

  TestEngineMin<Engine>();
}
DECLARE_UNITTEST(TestTaus88Min);


void TestTaus88Max(void)
{
  typedef thrust::random::taus88 Engine;

  TestEngineMax<Engine>();
}
DECLARE_UNITTEST(TestTaus88Max);


void TestTaus88SaveRestore(void)
{
  typedef thrust::random::taus88 Engine;

  TestEngineSaveRestore<Engine>();
}
DECLARE_UNITTEST(TestTaus88SaveRestore);


void TestTaus88Equal(void)
{
  typedef thrust::random::taus88 Engine;

  TestEngineEqual<Engine>();
}
DECLARE_UNITTEST(TestTaus88Equal);


void TestTaus88Unequal(void)
{
  typedef thrust::random::taus88 Engine;

  TestEngineUnequal<Engine>();
}
DECLARE_UNITTEST(TestTaus88Unequal);


void TestRanlux24Validation(void)
{
  typedef thrust::random::ranlux24 Engine;

  TestEngineValidation<Engine,9901578>();
}
DECLARE_UNITTEST(TestRanlux24Validation);


void TestRanlux24Min(void)
{
  typedef thrust::random::ranlux24 Engine;

  TestEngineMin<Engine>();
}
DECLARE_UNITTEST(TestRanlux24Min);


void TestRanlux24Max(void)
{
  typedef thrust::random::ranlux24 Engine;

  TestEngineMax<Engine>();
}
DECLARE_UNITTEST(TestRanlux24Max);


void TestRanlux24SaveRestore(void)
{
  typedef thrust::random::ranlux24 Engine;

  TestEngineSaveRestore<Engine>();
}
DECLARE_UNITTEST(TestRanlux24SaveRestore);


void TestRanlux24Equal(void)
{
  typedef thrust::random::ranlux24 Engine;

  TestEngineEqual<Engine>();
}
DECLARE_UNITTEST(TestRanlux24Equal);


void TestRanlux24Unequal(void)
{
  typedef thrust::random::ranlux24 Engine;

  TestEngineUnequal<Engine>();
}
DECLARE_UNITTEST(TestRanlux24Unequal);



void TestRanlux48Validation(void)
{
  typedef thrust::random::ranlux48 Engine;

  TestEngineValidation<Engine,88229545517833ull>();
}
DECLARE_UNITTEST(TestRanlux48Validation);


void TestRanlux48Min(void)
{
  typedef thrust::random::ranlux48 Engine;

  TestEngineMin<Engine>();
}
DECLARE_UNITTEST(TestRanlux48Min);


void TestRanlux48Max(void)
{
  typedef thrust::random::ranlux48 Engine;

  TestEngineMax<Engine>();
}
DECLARE_UNITTEST(TestRanlux48Max);


void TestRanlux48SaveRestore(void)
{
  typedef thrust::random::ranlux48 Engine;

  TestEngineSaveRestore<Engine>();
}
DECLARE_UNITTEST(TestRanlux48SaveRestore);


void TestRanlux48Equal(void)
{
  typedef thrust::random::ranlux48 Engine;

  TestEngineEqual<Engine>();
}
DECLARE_UNITTEST(TestRanlux48Equal);


void TestRanlux48Unequal(void)
{
  typedef thrust::random::ranlux48 Engine;

  TestEngineUnequal<Engine>();
}
DECLARE_UNITTEST(TestRanlux48Unequal);


template<typename Distribution, typename Validator>
  void ValidateDistributionCharacteristic(void)
{
  typedef typename Validator::random_engine Engine;

  // test default-constructed Distribution

  // test host
  thrust::host_vector<bool> h(1);
  thrust::generate(h.begin(), h.end(), Validator(Distribution()));

  ASSERT_EQUAL(true, h[0]);

  // test device
  thrust::device_vector<bool> d(1);
  thrust::generate(d.begin(), d.end(), Validator(Distribution()));

  ASSERT_EQUAL(true, d[0]);


  // test distribution & engine with comparable ranges
  // only do this if they have the same result_type
  if(thrust::detail::is_same<typename Distribution::result_type, typename Engine::result_type>::value)
  {
    // test Distribution with same range as engine

    // test host
    thrust::generate(h.begin(), h.end(), Validator(Distribution(Engine::min, Engine::max)));

    ASSERT_EQUAL(true, h[0]);

    // test device
    thrust::generate(d.begin(), d.end(), Validator(Distribution(Engine::min, Engine::max)));

    ASSERT_EQUAL(true, d[0]);

    // test Distribution with smaller range than engine

    // test host
    typename Distribution::result_type engine_range = Engine::max - Engine::min;
    thrust::generate(h.begin(), h.end(), Validator(Distribution(engine_range/3, (2 * engine_range)/3)));

    ASSERT_EQUAL(true, h[0]);

    // test device
    thrust::generate(d.begin(), d.end(), Validator(Distribution(engine_range/3, (2 * engine_range)/3)));

    ASSERT_EQUAL(true, d[0]);
  }


  // test Distribution with a very small range

  // test host
  thrust::generate(h.begin(), h.end(), Validator(Distribution(1,6)));

  ASSERT_EQUAL(true, h[0]);

  // test device
  thrust::generate(d.begin(), d.end(), Validator(Distribution(1,6)));

  ASSERT_EQUAL(true, d[0]);
}


template<typename Distribution>
  void TestDistributionSaveRestore(void)
{
  // create a default distribution
  Distribution d0(7, 13);

  // save it
  std::stringstream ss;
  ss << d0;

  // restore old state
  Distribution d1;
  ss >> d1;

  ASSERT_EQUAL(d0, d1);
}


void TestUniformIntDistributionMin(void)
{
  typedef thrust::random::uniform_int_distribution<int>          int_dist;
  typedef thrust::random::uniform_int_distribution<unsigned int> uint_dist;
  
  ValidateDistributionCharacteristic<int_dist,  ValidateDistributionMin<int_dist,  thrust::minstd_rand> >();
  ValidateDistributionCharacteristic<uint_dist, ValidateDistributionMin<uint_dist, thrust::minstd_rand> >();
}
DECLARE_UNITTEST(TestUniformIntDistributionMin);


void TestUniformIntDistributionMax(void)
{
  typedef thrust::random::uniform_int_distribution<int>          int_dist;
  typedef thrust::random::uniform_int_distribution<unsigned int> uint_dist;
  
  ValidateDistributionCharacteristic<int_dist,  ValidateDistributionMax<int_dist,  thrust::minstd_rand> >();
  ValidateDistributionCharacteristic<uint_dist, ValidateDistributionMax<uint_dist, thrust::minstd_rand> >();
}
DECLARE_UNITTEST(TestUniformIntDistributionMax);


void TestUniformIntDistributionSaveRestore(void)
{
  typedef thrust::random::uniform_int_distribution<int>          int_dist;
  typedef thrust::random::uniform_int_distribution<unsigned int> uint_dist;

  TestDistributionSaveRestore<int_dist>();
  TestDistributionSaveRestore<uint_dist>();
}
DECLARE_UNITTEST(TestUniformIntDistributionSaveRestore);


void TestUniformRealDistributionMin(void)
{
  typedef thrust::random::uniform_real_distribution<float>  float_dist;
  typedef thrust::random::uniform_real_distribution<double> double_dist;
  
  ValidateDistributionCharacteristic<float_dist,  ValidateDistributionMin<float_dist,  thrust::minstd_rand> >();
  ValidateDistributionCharacteristic<double_dist, ValidateDistributionMin<double_dist, thrust::minstd_rand> >();
}
DECLARE_UNITTEST(TestUniformRealDistributionMin);


void TestUniformRealDistributionMax(void)
{
  typedef thrust::random::uniform_real_distribution<float>  float_dist;
  typedef thrust::random::uniform_real_distribution<double> double_dist;
  
  ValidateDistributionCharacteristic<float_dist,  ValidateDistributionMax<float_dist,  thrust::minstd_rand> >();
  ValidateDistributionCharacteristic<double_dist, ValidateDistributionMax<double_dist, thrust::minstd_rand> >();
}
DECLARE_UNITTEST(TestUniformRealDistributionMax);


void TestUniformRealDistributionSaveRestore(void)
{
  typedef thrust::random::uniform_real_distribution<float>  float_dist;
  typedef thrust::random::uniform_real_distribution<double> double_dist;

  TestDistributionSaveRestore<float_dist>();
  TestDistributionSaveRestore<double_dist>();
}
DECLARE_UNITTEST(TestUniformRealDistributionSaveRestore);


void TestNormalDistributionMin(void)
{
  typedef thrust::random::normal_distribution<float>  float_dist;
  typedef thrust::random::normal_distribution<double> double_dist;
  
  ValidateDistributionCharacteristic<float_dist,  ValidateDistributionMin<float_dist,  thrust::minstd_rand> >();
  ValidateDistributionCharacteristic<double_dist, ValidateDistributionMin<double_dist, thrust::minstd_rand> >();
}
DECLARE_UNITTEST(TestNormalDistributionMin);


void TestNormalDistributionMax(void)
{
  typedef thrust::random::normal_distribution<float>  float_dist;
  typedef thrust::random::normal_distribution<double> double_dist;
  
  ValidateDistributionCharacteristic<float_dist,  ValidateDistributionMax<float_dist,  thrust::minstd_rand> >();
  ValidateDistributionCharacteristic<double_dist, ValidateDistributionMax<double_dist, thrust::minstd_rand> >();
}
DECLARE_UNITTEST(TestNormalDistributionMax);


void TestNormalDistributionSaveRestore(void)
{
  typedef thrust::random::normal_distribution<float>  float_dist;
  typedef thrust::random::normal_distribution<double> double_dist;

  TestDistributionSaveRestore<float_dist>();
  TestDistributionSaveRestore<double_dist>();
}
DECLARE_UNITTEST(TestNormalDistributionSaveRestore);

