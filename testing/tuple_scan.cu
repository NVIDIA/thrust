#include <unittest/unittest.h>
#include <thrust/tuple.h>
#include <thrust/scan.h>
#include <thrust/transform.h>

#if THRUST_DEVICE_SYSTEM == THRUST_DEVICE_SYSTEM_CUDA
#include <backend/cuda/testframework.h>
#endif

using namespace unittest;

struct SumTupleFunctor
{
  template <typename Tuple>
  __host__ __device__
  Tuple operator()(const Tuple &lhs, const Tuple &rhs)
  {
    using thrust::get;
  
    return thrust::make_tuple(get<0>(lhs) + get<0>(rhs),
                              get<1>(lhs) + get<1>(rhs));
  }
};

struct MakeTupleFunctor
{
  template<typename T1, typename T2>
  __host__ __device__
  thrust::tuple<T1,T2> operator()(T1 &lhs, T2 &rhs)
  {
    return thrust::make_tuple(lhs, rhs);
  }
};


template <typename T>
struct TestTupleScan
{
  void operator()(const size_t n)
  {
     using namespace thrust;

     host_vector<T> h_t1 = unittest::random_integers<T>(n);
     host_vector<T> h_t2 = unittest::random_integers<T>(n);

     // initialize input
     host_vector< tuple<T,T> > h_input(n);
     transform(h_t1.begin(), h_t1.end(), h_t2.begin(), h_input.begin(), MakeTupleFunctor());
     device_vector< tuple<T,T> > d_input = h_input;
     
     // allocate output
     tuple<T,T> zero(0,0);
     host_vector  < tuple<T,T> > h_output(n, zero);
     device_vector< tuple<T,T> > d_output(n, zero);

     // inclusive_scan
     inclusive_scan(h_input.begin(), h_input.end(), h_output.begin(), SumTupleFunctor());
     inclusive_scan(d_input.begin(), d_input.end(), d_output.begin(), SumTupleFunctor());
     ASSERT_EQUAL_QUIET(h_output, d_output);

    // The tests below get miscompiled on Tesla hw for 8b types

#if THRUST_DEVICE_SYSTEM == THRUST_DEVICE_SYSTEM_CUDA
    if(const CUDATestDriver *driver = dynamic_cast<const CUDATestDriver*>(&UnitTestDriver::s_driver()))
    {
      if(sizeof(T) == sizeof(unittest::uint8_t) && driver->current_device_architecture() < 200)
      {
        KNOWN_FAILURE;
      } // end if
    } // end if
#endif

     // exclusive_scan
     tuple<T,T> init(13,17);
     exclusive_scan(h_input.begin(), h_input.end(), h_output.begin(), init, SumTupleFunctor());
     exclusive_scan(d_input.begin(), d_input.end(), d_output.begin(), init, SumTupleFunctor());

     ASSERT_EQUAL_QUIET(h_output, d_output);
  }
};
VariableUnitTest<TestTupleScan, IntegralTypes> TestTupleScanInstance;

