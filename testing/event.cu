#include <thrust/detail/config.h>

#if THRUST_CPP_DIALECT >= 2011 && !defined(THRUST_LEGACY_GCC)

#include <unittest/unittest.h>
#include <unittest/util_async.h>

#include <thrust/event.h>

///////////////////////////////////////////////////////////////////////////////

__host__
void test_event_default_constructed()
{
  THRUST_STATIC_ASSERT(
    (std::is_same<
      thrust::event<decltype(thrust::device)>
    , thrust::unique_eager_event<decltype(thrust::device)>
    >::value)
  );

  THRUST_STATIC_ASSERT(
    (std::is_same<
      thrust::event<decltype(thrust::device)>
    , thrust::device_event
    >::value)
  );

  THRUST_STATIC_ASSERT(
    (std::is_same<
      thrust::device_event
    , thrust::device_unique_eager_event
    >::value)
  );

  thrust::device_event e0;

  ASSERT_EQUAL(false, e0.valid_stream());

  ASSERT_THROWS_EQUAL(
    e0.wait()
  , thrust::event_error
  , thrust::event_error(thrust::event_errc::no_state)
  );

  ASSERT_THROWS_EQUAL(
    e0.stream()
  , thrust::event_error
  , thrust::event_error(thrust::event_errc::no_state)
  );
}
DECLARE_UNITTEST(test_event_default_constructed);

///////////////////////////////////////////////////////////////////////////////

__host__
void test_event_new_stream()
{
  auto e0 = thrust::device_event(thrust::new_stream);

  ASSERT_EQUAL(true, e0.valid_stream());

  ASSERT_NOT_EQUAL_QUIET(nullptr, e0.stream().native_handle());    

  e0.wait();

  ASSERT_EQUAL(true, e0.ready());
}
DECLARE_UNITTEST(test_event_new_stream);

///////////////////////////////////////////////////////////////////////////////

__host__
void test_event_linear_chaining()
{
  constexpr std::int64_t n = 1024;

  // Create a new stream.
  auto e0 = thrust::when_all();

  auto const e0_stream = e0.stream().native_handle();

  ASSERT_EQUAL(true, e0.valid_stream());

  ASSERT_NOT_EQUAL_QUIET(nullptr, e0_stream);

  thrust::device_event e1;

  for (std::int64_t i = 0; i < n; ++i)
  {
    ASSERT_EQUAL(true,  e0.valid_stream());

    ASSERT_EQUAL(false, e1.valid_stream());
    ASSERT_EQUAL(false, e1.ready());

    ASSERT_EQUAL_QUIET(e0_stream, e0.stream().native_handle());

    e1 = thrust::when_all(e0);

    ASSERT_EQUAL(false, e0.valid_stream());
    ASSERT_EQUAL(false, e0.ready());

    ASSERT_EQUAL(true,  e1.valid_stream());

    ASSERT_EQUAL(e0_stream, e1.stream().native_handle());

    std::swap(e0, e1);
  }
}
DECLARE_UNITTEST(test_event_linear_chaining);

///////////////////////////////////////////////////////////////////////////////

__host__
void test_event_when_all()
{
  // Create events with new streams.
  auto e0 = thrust::when_all();
  auto e1 = thrust::when_all();
  auto e2 = thrust::when_all();
  auto e3 = thrust::when_all();
  auto e4 = thrust::when_all();
  auto e5 = thrust::when_all();
  auto e6 = thrust::when_all();
  auto e7 = thrust::when_all();

  auto const e0_stream = e0.stream().native_handle();

  ASSERT_EQUAL(true, e0.valid_stream());
  ASSERT_EQUAL(true, e1.valid_stream());
  ASSERT_EQUAL(true, e2.valid_stream());
  ASSERT_EQUAL(true, e3.valid_stream());
  ASSERT_EQUAL(true, e4.valid_stream());
  ASSERT_EQUAL(true, e5.valid_stream());
  ASSERT_EQUAL(true, e6.valid_stream());
  ASSERT_EQUAL(true, e7.valid_stream());

  ASSERT_NOT_EQUAL_QUIET(nullptr, e0_stream);
  ASSERT_NOT_EQUAL_QUIET(nullptr, e1.stream().native_handle());
  ASSERT_NOT_EQUAL_QUIET(nullptr, e2.stream().native_handle());
  ASSERT_NOT_EQUAL_QUIET(nullptr, e3.stream().native_handle());
  ASSERT_NOT_EQUAL_QUIET(nullptr, e4.stream().native_handle());
  ASSERT_NOT_EQUAL_QUIET(nullptr, e5.stream().native_handle());
  ASSERT_NOT_EQUAL_QUIET(nullptr, e6.stream().native_handle());
  ASSERT_NOT_EQUAL_QUIET(nullptr, e7.stream().native_handle());

  auto e8 = thrust::when_all(e0, e1, e2, e3, e4, e5, e6, e7);

  ASSERT_EQUAL(false, e0.valid_stream());
  ASSERT_EQUAL(false, e1.valid_stream());
  ASSERT_EQUAL(false, e2.valid_stream());
  ASSERT_EQUAL(false, e3.valid_stream());
  ASSERT_EQUAL(false, e4.valid_stream());
  ASSERT_EQUAL(false, e5.valid_stream());
  ASSERT_EQUAL(false, e6.valid_stream());
  ASSERT_EQUAL(false, e7.valid_stream());

  ASSERT_EQUAL(true, e8.valid_stream());

  ASSERT_EQUAL(e0_stream, e8.stream().native_handle());

  e8.wait();

  ASSERT_EQUAL(false, e0.ready());
  ASSERT_EQUAL(false, e1.ready());
  ASSERT_EQUAL(false, e2.ready());
  ASSERT_EQUAL(false, e3.ready());
  ASSERT_EQUAL(false, e4.ready());
  ASSERT_EQUAL(false, e5.ready());
  ASSERT_EQUAL(false, e6.ready());
  ASSERT_EQUAL(false, e7.ready());

  ASSERT_EQUAL(true,  e8.ready());
}
DECLARE_UNITTEST(test_event_when_all);

///////////////////////////////////////////////////////////////////////////////
 
#endif

