#include <unittest/unittest.h>
#include <thrust/mismatch.h>

template <class Vector>
void TestMismatchSimple(void)
{
    typedef typename Vector::value_type T;

    Vector a(4); Vector b(4);
    a[0] = 1; b[0] = 1;
    a[1] = 2; b[1] = 2;
    a[2] = 3; b[2] = 4;
    a[3] = 4; b[3] = 3;

    ASSERT_EQUAL(thrust::mismatch(a.begin(), a.end(), b.begin()).first  - a.begin(), 2);
    ASSERT_EQUAL(thrust::mismatch(a.begin(), a.end(), b.begin()).second - b.begin(), 2);

    b[2] = 3;
    
    ASSERT_EQUAL(thrust::mismatch(a.begin(), a.end(), b.begin()).first  - a.begin(), 3);
    ASSERT_EQUAL(thrust::mismatch(a.begin(), a.end(), b.begin()).second - b.begin(), 3);
    
    b[3] = 4;
    
    ASSERT_EQUAL(thrust::mismatch(a.begin(), a.end(), b.begin()).first  - a.begin(), 4);
    ASSERT_EQUAL(thrust::mismatch(a.begin(), a.end(), b.begin()).second - b.begin(), 4);
}
DECLARE_VECTOR_UNITTEST(TestMismatchSimple);


template <typename InputIterator1, typename InputIterator2>
thrust::pair<InputIterator1, InputIterator2> mismatch(my_system &system,
                                                      InputIterator1 first,
                                                      InputIterator1,
                                                      InputIterator2)
{
    system.validate_dispatch();
    return thrust::make_pair(first,first);
}

void TestMismatchDispatchExplicit()
{
    thrust::device_vector<int> vec(1);

    my_system sys(0);
    thrust::mismatch(sys,
                     vec.begin(),
                     vec.begin(),
                     vec.begin());

    ASSERT_EQUAL(true, sys.is_valid());
}
DECLARE_UNITTEST(TestMismatchDispatchExplicit);


template <typename InputIterator1, typename InputIterator2>
thrust::pair<InputIterator1, InputIterator2> mismatch(my_tag,
                                                      InputIterator1 first,
                                                      InputIterator1,
                                                      InputIterator2)
{
    *first = 13;
    return thrust::make_pair(first,first);
}

void TestMismatchDispatchImplicit()
{
    thrust::device_vector<int> vec(1);

    thrust::mismatch(thrust::retag<my_tag>(vec.begin()),
                     thrust::retag<my_tag>(vec.begin()),
                     thrust::retag<my_tag>(vec.begin()));

    ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestMismatchDispatchImplicit);

