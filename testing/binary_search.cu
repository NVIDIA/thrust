#include <unittest/unittest.h>
#include <thrust/binary_search.h>

#include <thrust/sequence.h>
#include <thrust/sort.h>

__THRUST_DISABLE_MSVC_POSSIBLE_LOSS_OF_DATA_WARNING_BEGIN

//////////////////////
// Scalar Functions //
//////////////////////

template <class Vector>
void TestScalarLowerBoundSimple(void)
{
    typedef typename Vector::value_type T;

    Vector vec(5);

    vec[0] = 0;
    vec[1] = 2;
    vec[2] = 5;
    vec[3] = 7;
    vec[4] = 8;

    ASSERT_EQUAL(thrust::lower_bound(vec.begin(), vec.end(), 0) - vec.begin(), 0);
    ASSERT_EQUAL(thrust::lower_bound(vec.begin(), vec.end(), 1) - vec.begin(), 1);
    ASSERT_EQUAL(thrust::lower_bound(vec.begin(), vec.end(), 2) - vec.begin(), 1);
    ASSERT_EQUAL(thrust::lower_bound(vec.begin(), vec.end(), 3) - vec.begin(), 2);
    ASSERT_EQUAL(thrust::lower_bound(vec.begin(), vec.end(), 4) - vec.begin(), 2);
    ASSERT_EQUAL(thrust::lower_bound(vec.begin(), vec.end(), 5) - vec.begin(), 2);
    ASSERT_EQUAL(thrust::lower_bound(vec.begin(), vec.end(), 6) - vec.begin(), 3);
    ASSERT_EQUAL(thrust::lower_bound(vec.begin(), vec.end(), 7) - vec.begin(), 3);
    ASSERT_EQUAL(thrust::lower_bound(vec.begin(), vec.end(), 8) - vec.begin(), 4);
    ASSERT_EQUAL(thrust::lower_bound(vec.begin(), vec.end(), 9) - vec.begin(), 5);
}
DECLARE_VECTOR_UNITTEST(TestScalarLowerBoundSimple);


template<typename ForwardIterator, typename LessThanComparable>
ForwardIterator lower_bound(my_system &system, ForwardIterator first, ForwardIterator last, const LessThanComparable &value)
{
    system.validate_dispatch();
    return first;
}

void TestScalarLowerBoundDispatchExplicit()
{
    thrust::device_vector<int> vec(1);

    my_system sys(0);
    thrust::lower_bound(sys,
                        vec.begin(),
                        vec.end(),
                        0);

    ASSERT_EQUAL(true, sys.is_valid());
}
DECLARE_UNITTEST(TestScalarLowerBoundDispatchExplicit);


template<typename ForwardIterator, typename LessThanComparable>
ForwardIterator lower_bound(my_tag, ForwardIterator first, ForwardIterator last, const LessThanComparable &value)
{
    *first = 13;
    return first;
}


void TestScalarLowerBoundDispatchImplicit()
{
    thrust::device_vector<int> vec(1);

    thrust::lower_bound(thrust::retag<my_tag>(vec.begin()),
                        thrust::retag<my_tag>(vec.end()),
                        0);

    ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestScalarLowerBoundDispatchImplicit);


template <class Vector>
void TestScalarUpperBoundSimple(void)
{
    typedef typename Vector::value_type T;

    Vector vec(5);

    vec[0] = 0;
    vec[1] = 2;
    vec[2] = 5;
    vec[3] = 7;
    vec[4] = 8;

    ASSERT_EQUAL(thrust::upper_bound(vec.begin(), vec.end(), 0) - vec.begin(), 1);
    ASSERT_EQUAL(thrust::upper_bound(vec.begin(), vec.end(), 1) - vec.begin(), 1);
    ASSERT_EQUAL(thrust::upper_bound(vec.begin(), vec.end(), 2) - vec.begin(), 2);
    ASSERT_EQUAL(thrust::upper_bound(vec.begin(), vec.end(), 3) - vec.begin(), 2);
    ASSERT_EQUAL(thrust::upper_bound(vec.begin(), vec.end(), 4) - vec.begin(), 2);
    ASSERT_EQUAL(thrust::upper_bound(vec.begin(), vec.end(), 5) - vec.begin(), 3);
    ASSERT_EQUAL(thrust::upper_bound(vec.begin(), vec.end(), 6) - vec.begin(), 3);
    ASSERT_EQUAL(thrust::upper_bound(vec.begin(), vec.end(), 7) - vec.begin(), 4);
    ASSERT_EQUAL(thrust::upper_bound(vec.begin(), vec.end(), 8) - vec.begin(), 5);
    ASSERT_EQUAL(thrust::upper_bound(vec.begin(), vec.end(), 9) - vec.begin(), 5);
}
DECLARE_VECTOR_UNITTEST(TestScalarUpperBoundSimple);


template<typename ForwardIterator, typename LessThanComparable>
ForwardIterator upper_bound(my_system &system, ForwardIterator first, ForwardIterator last, const LessThanComparable &value)
{
    system.validate_dispatch();
    return first;
}

void TestScalarUpperBoundDispatchExplicit()
{
    thrust::device_vector<int> vec(1);

    my_system sys(0);
    thrust::upper_bound(sys,
                        vec.begin(),
                        vec.end(),
                        0);

    ASSERT_EQUAL(true, sys.is_valid());
}
DECLARE_UNITTEST(TestScalarUpperBoundDispatchExplicit);


template<typename ForwardIterator, typename LessThanComparable>
ForwardIterator upper_bound(my_tag, ForwardIterator first, ForwardIterator last, const LessThanComparable &value)
{
    *first = 13;
    return first;
}

void TestScalarUpperBoundDispatchImplicit()
{
    thrust::device_vector<int> vec(1);

    thrust::upper_bound(thrust::retag<my_tag>(vec.begin()),
                        thrust::retag<my_tag>(vec.end()),
                        0);

    ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestScalarUpperBoundDispatchImplicit);


template <class Vector>
void TestScalarBinarySearchSimple(void)
{
    typedef typename Vector::value_type T;

    Vector vec(5);

    vec[0] = 0;
    vec[1] = 2;
    vec[2] = 5;
    vec[3] = 7;
    vec[4] = 8;

    ASSERT_EQUAL(thrust::binary_search(vec.begin(), vec.end(), 0),  true);
    ASSERT_EQUAL(thrust::binary_search(vec.begin(), vec.end(), 1), false);
    ASSERT_EQUAL(thrust::binary_search(vec.begin(), vec.end(), 2),  true);
    ASSERT_EQUAL(thrust::binary_search(vec.begin(), vec.end(), 3), false);
    ASSERT_EQUAL(thrust::binary_search(vec.begin(), vec.end(), 4), false);
    ASSERT_EQUAL(thrust::binary_search(vec.begin(), vec.end(), 5),  true);
    ASSERT_EQUAL(thrust::binary_search(vec.begin(), vec.end(), 6), false);
    ASSERT_EQUAL(thrust::binary_search(vec.begin(), vec.end(), 7),  true);
    ASSERT_EQUAL(thrust::binary_search(vec.begin(), vec.end(), 8),  true);
    ASSERT_EQUAL(thrust::binary_search(vec.begin(), vec.end(), 9), false);
}
DECLARE_VECTOR_UNITTEST(TestScalarBinarySearchSimple);


template<typename ForwardIterator, typename LessThanComparable>
bool binary_search(my_system &system, ForwardIterator first, ForwardIterator last, const LessThanComparable &value)
{
    system.validate_dispatch();
    return false;
}

void TestScalarBinarySearchDispatchExplicit()
{
    thrust::device_vector<int> vec(1);

    my_system sys(0);
    thrust::binary_search(sys,
                          vec.begin(),
                          vec.end(),
                          0);

    ASSERT_EQUAL(true, sys.is_valid());
}
DECLARE_UNITTEST(TestScalarBinarySearchDispatchExplicit);


template<typename ForwardIterator, typename LessThanComparable>
bool binary_search(my_tag, ForwardIterator first, ForwardIterator last, const LessThanComparable &value)
{
    *first = 13;
    return false;
}

void TestScalarBinarySearchDispatchImplicit()
{
    thrust::device_vector<int> vec(1);

    thrust::binary_search(thrust::retag<my_tag>(vec.begin()),
                          thrust::retag<my_tag>(vec.end()),
                          0);

    ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestScalarBinarySearchDispatchImplicit);


template <class Vector>
void TestScalarEqualRangeSimple(void)
{
    typedef typename Vector::value_type T;

    Vector vec(5);

    vec[0] = 0;
    vec[1] = 2;
    vec[2] = 5;
    vec[3] = 7;
    vec[4] = 8;

    ASSERT_EQUAL(thrust::equal_range(vec.begin(), vec.end(), 0).first - vec.begin(), 0);
    ASSERT_EQUAL(thrust::equal_range(vec.begin(), vec.end(), 1).first - vec.begin(), 1);
    ASSERT_EQUAL(thrust::equal_range(vec.begin(), vec.end(), 2).first - vec.begin(), 1);
    ASSERT_EQUAL(thrust::equal_range(vec.begin(), vec.end(), 3).first - vec.begin(), 2);
    ASSERT_EQUAL(thrust::equal_range(vec.begin(), vec.end(), 4).first - vec.begin(), 2);
    ASSERT_EQUAL(thrust::equal_range(vec.begin(), vec.end(), 5).first - vec.begin(), 2);
    ASSERT_EQUAL(thrust::equal_range(vec.begin(), vec.end(), 6).first - vec.begin(), 3);
    ASSERT_EQUAL(thrust::equal_range(vec.begin(), vec.end(), 7).first - vec.begin(), 3);
    ASSERT_EQUAL(thrust::equal_range(vec.begin(), vec.end(), 8).first - vec.begin(), 4);
    ASSERT_EQUAL(thrust::equal_range(vec.begin(), vec.end(), 9).first - vec.begin(), 5);
    
    ASSERT_EQUAL(thrust::equal_range(vec.begin(), vec.end(), 0).second - vec.begin(), 1);
    ASSERT_EQUAL(thrust::equal_range(vec.begin(), vec.end(), 1).second - vec.begin(), 1);
    ASSERT_EQUAL(thrust::equal_range(vec.begin(), vec.end(), 2).second - vec.begin(), 2);
    ASSERT_EQUAL(thrust::equal_range(vec.begin(), vec.end(), 3).second - vec.begin(), 2);
    ASSERT_EQUAL(thrust::equal_range(vec.begin(), vec.end(), 4).second - vec.begin(), 2);
    ASSERT_EQUAL(thrust::equal_range(vec.begin(), vec.end(), 5).second - vec.begin(), 3);
    ASSERT_EQUAL(thrust::equal_range(vec.begin(), vec.end(), 6).second - vec.begin(), 3);
    ASSERT_EQUAL(thrust::equal_range(vec.begin(), vec.end(), 7).second - vec.begin(), 4);
    ASSERT_EQUAL(thrust::equal_range(vec.begin(), vec.end(), 8).second - vec.begin(), 5);
    ASSERT_EQUAL(thrust::equal_range(vec.begin(), vec.end(), 9).second - vec.begin(), 5);
}
DECLARE_VECTOR_UNITTEST(TestScalarEqualRangeSimple);


template<typename ForwardIterator, typename LessThanComparable>
thrust::pair<ForwardIterator,ForwardIterator> equal_range(my_system &system, ForwardIterator first, ForwardIterator last, const LessThanComparable &value)
{
    system.validate_dispatch();
    return thrust::make_pair(first,first);
}

void TestScalarEqualRangeDispatchExplicit()
{
    thrust::device_vector<int> vec(1);

    my_system sys(0);
    thrust::equal_range(sys,
                        vec.begin(),
                        vec.end(),
                        0);

    ASSERT_EQUAL(true, sys.is_valid());
}
DECLARE_UNITTEST(TestScalarEqualRangeDispatchExplicit);


template<typename ForwardIterator, typename LessThanComparable>
thrust::pair<ForwardIterator,ForwardIterator> equal_range(my_tag, ForwardIterator first, ForwardIterator last, const LessThanComparable &value)
{
    *first = 13;
    return thrust::make_pair(first,first);
}

void TestScalarEqualRangeDispatchImplicit()
{
    thrust::device_vector<int> vec(1);

    thrust::equal_range(thrust::retag<my_tag>(vec.begin()),
                        thrust::retag<my_tag>(vec.end()),
                        0);

    ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestScalarEqualRangeDispatchImplicit);

__THRUST_DISABLE_MSVC_POSSIBLE_LOSS_OF_DATA_WARNING_END
