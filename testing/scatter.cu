#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/scatter.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/sequence.h>
#include <thrust/fill.h>

struct my_system : thrust::device_system<my_system> {};

template <class Vector>
void TestScatterSimple(void)
{
    typedef typename Vector::value_type T;

    Vector map(5);  // scatter indices
    Vector src(5);  // source vector
    Vector dst(8);  // destination vector

    map[0] = 6; map[1] = 3; map[2] = 1; map[3] = 7; map[4] = 2;
    src[0] = 0; src[1] = 1; src[2] = 2; src[3] = 3; src[4] = 4;
    dst[0] = 0; dst[1] = 0; dst[2] = 0; dst[3] = 0; dst[4] = 0; dst[5] = 0; dst[6] = 0; dst[7] = 0;

    thrust::scatter(src.begin(), src.end(), map.begin(), dst.begin());

    ASSERT_EQUAL(dst[0], 0);
    ASSERT_EQUAL(dst[1], 2);
    ASSERT_EQUAL(dst[2], 4);
    ASSERT_EQUAL(dst[3], 1);
    ASSERT_EQUAL(dst[4], 0);
    ASSERT_EQUAL(dst[5], 0);
    ASSERT_EQUAL(dst[6], 0);
    ASSERT_EQUAL(dst[7], 3);
}
DECLARE_VECTOR_UNITTEST(TestScatterSimple);


template<typename InputIterator1,
         typename InputIterator2,
         typename RandomAccessIterator>
void scatter(my_system,
             InputIterator1,
             InputIterator1,
             InputIterator2,
             RandomAccessIterator output)
{
    *output = 13;
}


void TestScatterDispatchExplicit()
{
    thrust::device_vector<int> vec(1);

    my_system sys;
    thrust::scatter(sys,
                    vec.begin(),
                    vec.begin(),
                    vec.begin(),
                    vec.begin());

    ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestScatterDispatchExplicit);

void TestScatterDispatchImplicit()
{
    thrust::device_vector<int> vec(1);

    thrust::scatter(thrust::retag<my_system>(vec.begin()),
                    thrust::retag<my_system>(vec.begin()),
                    thrust::retag<my_system>(vec.begin()),
                    thrust::retag<my_system>(vec.begin()));

    ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestScatterDispatchImplicit);


template <typename T>
void TestScatter(const size_t n)
{
    const size_t output_size = std::min((size_t) 10, 2 * n);
    
    thrust::host_vector<T> h_input(n, (T) 1);
    thrust::device_vector<T> d_input(n, (T) 1);
   
    thrust::host_vector<unsigned int> h_map = unittest::random_integers<unsigned int>(n);

    for(size_t i = 0; i < n; i++)
        h_map[i] =  h_map[i] % output_size;
    
    thrust::device_vector<unsigned int> d_map = h_map;

    thrust::host_vector<T>   h_output(output_size, (T) 0);
    thrust::device_vector<T> d_output(output_size, (T) 0);

    thrust::scatter(h_input.begin(), h_input.end(), h_map.begin(), h_output.begin());
    thrust::scatter(d_input.begin(), d_input.end(), d_map.begin(), d_output.begin());

    ASSERT_EQUAL(h_output, d_output);
}
DECLARE_VARIABLE_UNITTEST(TestScatter);


template <typename T>
void TestScatterToDiscardIterator(const size_t n)
{
    const size_t output_size = std::min((size_t) 10, 2 * n);
    
    thrust::host_vector<T> h_input(n, (T) 1);
    thrust::device_vector<T> d_input(n, (T) 1);
   
    thrust::host_vector<unsigned int> h_map = unittest::random_integers<unsigned int>(n);

    for(size_t i = 0; i < n; i++)
        h_map[i] =  h_map[i] % output_size;
    
    thrust::device_vector<unsigned int> d_map = h_map;

    thrust::scatter(h_input.begin(), h_input.end(), h_map.begin(), thrust::make_discard_iterator());
    thrust::scatter(d_input.begin(), d_input.end(), d_map.begin(), thrust::make_discard_iterator());

    // there's nothing to check -- just make sure it compiles
}
DECLARE_VARIABLE_UNITTEST(TestScatterToDiscardIterator);


template <class Vector>
void TestScatterIfSimple(void)
{
    typedef typename Vector::value_type T;

    Vector flg(5);  // predicate array
    Vector map(5);  // scatter indices
    Vector src(5);  // source vector
    Vector dst(8);  // destination vector

    flg[0] = 0; flg[1] = 1; flg[2] = 0; flg[3] = 1; flg[4] = 0;
    map[0] = 6; map[1] = 3; map[2] = 1; map[3] = 7; map[4] = 2;
    src[0] = 0; src[1] = 1; src[2] = 2; src[3] = 3; src[4] = 4;
    dst[0] = 0; dst[1] = 0; dst[2] = 0; dst[3] = 0; dst[4] = 0; dst[5] = 0; dst[6] = 0; dst[7] = 0;

    thrust::scatter_if(src.begin(), src.end(), map.begin(), flg.begin(), dst.begin());

    ASSERT_EQUAL(dst[0], 0);
    ASSERT_EQUAL(dst[1], 0);
    ASSERT_EQUAL(dst[2], 0);
    ASSERT_EQUAL(dst[3], 1);
    ASSERT_EQUAL(dst[4], 0);
    ASSERT_EQUAL(dst[5], 0);
    ASSERT_EQUAL(dst[6], 0);
    ASSERT_EQUAL(dst[7], 3);
}
DECLARE_VECTOR_UNITTEST(TestScatterIfSimple);


template<typename InputIterator1,
         typename InputIterator2,
         typename InputIterator3,
         typename RandomAccessIterator>
void scatter_if(my_system,
                InputIterator1,
                InputIterator1,
                InputIterator2,
                InputIterator3,
                RandomAccessIterator output)
{
    *output = 13;
}

void TestScatterIfDispatchExplicit()
{
    thrust::device_vector<int> vec(1);

    my_system sys;
    thrust::scatter_if(sys,
                       vec.begin(),
                       vec.begin(),
                       vec.begin(),
                       vec.begin(),
                       vec.begin());

    ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestScatterIfDispatchExplicit);


void TestScatterIfDispatchImplicit()
{
    thrust::device_vector<int> vec(1);

    thrust::scatter_if(thrust::retag<my_system>(vec.begin()),
                       thrust::retag<my_system>(vec.begin()),
                       thrust::retag<my_system>(vec.begin()),
                       thrust::retag<my_system>(vec.begin()),
                       thrust::retag<my_system>(vec.begin()));

    ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestScatterIfDispatchImplicit);


template <typename T>
class is_even_scatter_if
{
    public:
    __host__ __device__ bool operator()(const T i) const { return (i % 2) == 0; }
};

template <typename T>
void TestScatterIf(const size_t n)
{
    const size_t output_size = std::min((size_t) 10, 2 * n);
    
    thrust::host_vector<T> h_input(n, (T) 1);
    thrust::device_vector<T> d_input(n, (T) 1);
   
    thrust::host_vector<unsigned int> h_map = unittest::random_integers<unsigned int>(n);

    for(size_t i = 0; i < n; i++)
        h_map[i] =  h_map[i] % output_size;
    
    thrust::device_vector<unsigned int> d_map = h_map;

    thrust::host_vector<T>   h_output(output_size, (T) 0);
    thrust::device_vector<T> d_output(output_size, (T) 0);

    thrust::scatter_if(h_input.begin(), h_input.end(), h_map.begin(), h_map.begin(), h_output.begin(), is_even_scatter_if<unsigned int>());
    thrust::scatter_if(d_input.begin(), d_input.end(), d_map.begin(), d_map.begin(), d_output.begin(), is_even_scatter_if<unsigned int>());

    ASSERT_EQUAL(h_output, d_output);
}
DECLARE_VARIABLE_UNITTEST(TestScatterIf);


template <typename T>
void TestScatterIfToDiscardIterator(const size_t n)
{
    const size_t output_size = std::min((size_t) 10, 2 * n);
    
    thrust::host_vector<T> h_input(n, (T) 1);
    thrust::device_vector<T> d_input(n, (T) 1);
   
    thrust::host_vector<unsigned int> h_map = unittest::random_integers<unsigned int>(n);

    for(size_t i = 0; i < n; i++)
        h_map[i] =  h_map[i] % output_size;
    
    thrust::device_vector<unsigned int> d_map = h_map;

    thrust::scatter_if(h_input.begin(), h_input.end(), h_map.begin(), h_map.begin(), thrust::make_discard_iterator(), is_even_scatter_if<unsigned int>());
    thrust::scatter_if(d_input.begin(), d_input.end(), d_map.begin(), d_map.begin(), thrust::make_discard_iterator(), is_even_scatter_if<unsigned int>());
}
DECLARE_VARIABLE_UNITTEST(TestScatterIfToDiscardIterator);


template <typename Vector>
void TestScatterCountingIterator(void)
{
    typedef typename Vector::value_type T;

    Vector source(10);
    thrust::sequence(source.begin(), source.end(), 0);

    Vector map(10);
    thrust::sequence(map.begin(), map.end(), 0);

    Vector output(10);

    // source has any_system_tag
    thrust::fill(output.begin(), output.end(), 0);
    thrust::scatter(thrust::make_counting_iterator(0), thrust::make_counting_iterator(10),
                    map.begin(),
                    output.begin());

    ASSERT_EQUAL(output, map);
    
    // map has any_system_tag
    thrust::fill(output.begin(), output.end(), 0);
    thrust::scatter(source.begin(), source.end(),
                    thrust::make_counting_iterator(0),
                    output.begin());

    ASSERT_EQUAL(output, map);
    
    // source and map have any_system_tag
    thrust::fill(output.begin(), output.end(), 0);
    thrust::scatter(thrust::make_counting_iterator(0), thrust::make_counting_iterator(10),
                    thrust::make_counting_iterator(0),
                    output.begin());

    ASSERT_EQUAL(output, map);
}
DECLARE_VECTOR_UNITTEST(TestScatterCountingIterator);


template <typename Vector>
void TestScatterIfCountingIterator(void)
{
    typedef typename Vector::value_type T;

    Vector source(10);
    thrust::sequence(source.begin(), source.end(), 0);

    Vector map(10);
    thrust::sequence(map.begin(), map.end(), 0);
    
    Vector stencil(10, 1);

    Vector output(10);

    // source has any_system_tag
    thrust::fill(output.begin(), output.end(), 0);
    thrust::scatter_if(thrust::make_counting_iterator(0), thrust::make_counting_iterator(10),
                       map.begin(),
                       stencil.begin(),
                       output.begin());

    ASSERT_EQUAL(output, map);
    
    // map has any_system_tag
    thrust::fill(output.begin(), output.end(), 0);
    thrust::scatter_if(source.begin(), source.end(),
                       thrust::make_counting_iterator(0),
                       stencil.begin(),
                       output.begin());

    ASSERT_EQUAL(output, map);
    
    // source and map have any_system_tag
    thrust::fill(output.begin(), output.end(), 0);
    thrust::scatter_if(thrust::make_counting_iterator(0), thrust::make_counting_iterator(10),
                       thrust::make_counting_iterator(0),
                       stencil.begin(),
                       output.begin());

    ASSERT_EQUAL(output, map);
}
DECLARE_VECTOR_UNITTEST(TestScatterIfCountingIterator);

