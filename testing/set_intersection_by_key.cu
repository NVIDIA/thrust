#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/set_operations.h>
#include <thrust/functional.h>
#include <thrust/sort.h>


template<typename InputIterator1,
         typename InputIterator2,
         typename InputIterator3,
         typename OutputIterator1,
         typename OutputIterator2>
thrust::pair<OutputIterator1,OutputIterator2>
  set_intersection_by_key(my_system &system,
                          InputIterator1,
                          InputIterator1,
                          InputIterator2,
                          InputIterator2,
                          InputIterator3,
                          OutputIterator1 keys_result,
                          OutputIterator2 values_result)
{
  system.validate_dispatch();
  return thrust::make_pair(keys_result, values_result);
}

void TestSetIntersectionByKeyDispatchExplicit()
{
  thrust::device_vector<int> vec(1);

  my_system sys(0);
  thrust::set_intersection_by_key(sys,
                                  vec.begin(),
                                  vec.begin(),
                                  vec.begin(),
                                  vec.begin(),
                                  vec.begin(),
                                  vec.begin(),
                                  vec.begin());

  ASSERT_EQUAL(true, sys.is_valid());
}
DECLARE_UNITTEST(TestSetIntersectionByKeyDispatchExplicit);


template<typename InputIterator1,
         typename InputIterator2,
         typename InputIterator3,
         typename OutputIterator1,
         typename OutputIterator2>
thrust::pair<OutputIterator1,OutputIterator2>
  set_intersection_by_key(my_tag,
                          InputIterator1,
                          InputIterator1,
                          InputIterator2,
                          InputIterator2,
                          InputIterator3,
                          OutputIterator1 keys_result,
                          OutputIterator2 values_result)
{
  *keys_result = 13;
  return thrust::make_pair(keys_result,values_result);
}

void TestSetIntersectionByKeyDispatchImplicit()
{
  thrust::device_vector<int> vec(1);

  thrust::set_intersection_by_key(thrust::retag<my_tag>(vec.begin()),
                                  thrust::retag<my_tag>(vec.begin()),
                                  thrust::retag<my_tag>(vec.begin()),
                                  thrust::retag<my_tag>(vec.begin()),
                                  thrust::retag<my_tag>(vec.begin()),
                                  thrust::retag<my_tag>(vec.begin()),
                                  thrust::retag<my_tag>(vec.begin()));

  ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestSetIntersectionByKeyDispatchImplicit);


template<typename Vector>
void TestSetIntersectionByKeySimple(void)
{
  typedef typename Vector::iterator Iterator;

  Vector a_key(3), b_key(4);
  Vector a_val(3);

  a_key[0] = 0; a_key[1] = 2; a_key[2] = 4;
  a_val[0] = 0; a_val[1] = 0; a_val[2] = 0;

  b_key[0] = 0; b_key[1] = 3; b_key[2] = 3; b_key[3] = 4;

  Vector ref_key(2), ref_val(2);
  ref_key[0] = 0; ref_key[1] = 4;
  ref_val[0] = 0; ref_val[1] = 0;

  Vector result_key(2), result_val(2);

  thrust::pair<Iterator,Iterator> end =
    thrust::set_intersection_by_key(a_key.begin(), a_key.end(),
                                    b_key.begin(), b_key.end(),
                                    a_val.begin(),
                                    result_key.begin(),
                                    result_val.begin());

  ASSERT_EQUAL_QUIET(result_key.end(), end.first);
  ASSERT_EQUAL_QUIET(result_val.end(), end.second);
  ASSERT_EQUAL(ref_key, result_key);
  ASSERT_EQUAL(ref_val, result_val);
}
DECLARE_VECTOR_UNITTEST(TestSetIntersectionByKeySimple);


template<typename T>
void TestSetIntersectionByKey(const size_t n)
{
  thrust::host_vector<T> random_keys = unittest::random_integers<unittest::int8_t>(n);
  thrust::host_vector<T> random_vals = unittest::random_integers<unittest::int8_t>(n);

  size_t denominators[] = {1, 2, 3, 4, 5, 6, 7, 8, 9};
  size_t num_denominators = sizeof(denominators) / sizeof(size_t);

  for(size_t i = 0; i < num_denominators; ++i)
  {
    size_t size_a = n / denominators[i];

    thrust::host_vector<T> h_a_keys(random_keys.begin(), random_keys.begin() + size_a);
    thrust::host_vector<T> h_b_keys(random_keys.begin() + size_a, random_keys.end());

    thrust::host_vector<T> h_a_vals(random_vals.begin(), random_vals.begin() + size_a);

    thrust::stable_sort(h_a_keys.begin(), h_a_keys.end());
    thrust::stable_sort(h_b_keys.begin(), h_b_keys.end());

    thrust::device_vector<T> d_a_keys = h_a_keys;
    thrust::device_vector<T> d_b_keys = h_b_keys;

    thrust::device_vector<T> d_a_vals = h_a_vals;

    thrust::host_vector<T> h_result_keys(n);
    thrust::host_vector<T> h_result_vals(n);

    thrust::device_vector<T> d_result_keys(n);
    thrust::device_vector<T> d_result_vals(n);


    thrust::pair<
      typename thrust::host_vector<T>::iterator,
      typename thrust::host_vector<T>::iterator
    > h_end;

    thrust::pair<
      typename thrust::device_vector<T>::iterator,
      typename thrust::device_vector<T>::iterator
    > d_end;


    h_end = thrust::set_intersection_by_key(h_a_keys.begin(), h_a_keys.end(),
                                            h_b_keys.begin(), h_b_keys.end(),
                                            h_a_vals.begin(),
                                            h_result_keys.begin(),
                                            h_result_vals.begin());
    h_result_keys.erase(h_end.first, h_result_keys.end());
    h_result_vals.erase(h_end.second, h_result_vals.end());

    d_end = thrust::set_intersection_by_key(d_a_keys.begin(), d_a_keys.end(),
                                            d_b_keys.begin(), d_b_keys.end(),
                                            d_a_vals.begin(),
                                            d_result_keys.begin(),
                                            d_result_vals.begin());
    d_result_keys.erase(d_end.first, d_result_keys.end());
    d_result_vals.erase(d_end.second, d_result_vals.end());

    ASSERT_EQUAL(h_result_keys, d_result_keys);
    ASSERT_EQUAL(h_result_vals, d_result_vals);
  }
}
DECLARE_VARIABLE_UNITTEST(TestSetIntersectionByKey);


template<typename T>
void TestSetIntersectionByKeyEquivalentRanges(const size_t n)
{
  thrust::host_vector<T> temp = unittest::random_integers<T>(n);

  thrust::host_vector<T> h_a_key = temp;
  thrust::sort(h_a_key.begin(), h_a_key.end());
  thrust::host_vector<T> h_b_key = h_a_key;

  thrust::host_vector<T> h_a_val = unittest::random_integers<T>(n);

  thrust::device_vector<T> d_a_key = h_a_key;
  thrust::device_vector<T> d_b_key = h_b_key;

  thrust::device_vector<T> d_a_val = h_a_val;

  thrust::host_vector<T>   h_result_key(n), h_result_val(n);
  thrust::device_vector<T> d_result_key(n), d_result_val(n);

  thrust::pair<
    typename thrust::host_vector<T>::iterator,
    typename thrust::host_vector<T>::iterator
  > h_end;
  
  thrust::pair<
    typename thrust::device_vector<T>::iterator,
    typename thrust::device_vector<T>::iterator
  > d_end;
  
  h_end = thrust::set_intersection_by_key(h_a_key.begin(), h_a_key.end(),
                                          h_b_key.begin(), h_b_key.end(),
                                          h_a_val.begin(),
                                          h_result_key.begin(),
                                          h_result_val.begin());
  h_result_key.erase(h_end.first,  h_result_key.end());
  h_result_val.erase(h_end.second, h_result_val.end());

  d_end = thrust::set_intersection_by_key(d_a_key.begin(), d_a_key.end(),
                                          d_b_key.begin(), d_b_key.end(),
                                          d_a_val.begin(),
                                          d_result_key.begin(),
                                          d_result_val.begin());
  d_result_key.erase(d_end.first,  d_result_key.end());
  d_result_val.erase(d_end.second, d_result_val.end());

  ASSERT_EQUAL(h_result_key, d_result_key);
  ASSERT_EQUAL(h_result_val, d_result_val);
}
DECLARE_VARIABLE_UNITTEST(TestSetIntersectionByKeyEquivalentRanges);


template<typename T>
void TestSetIntersectionByKeyMultiset(const size_t n)
{
  thrust::host_vector<T> temp = unittest::random_integers<T>(2 * n);

  // restrict elements to [min,13)
  for(typename thrust::host_vector<T>::iterator i = temp.begin();
      i != temp.end();
      ++i)
  {
    int temp = static_cast<int>(*i);
    temp %= 13;
    *i = temp;
  }

  thrust::host_vector<T> h_a_key(temp.begin(), temp.begin() + n);
  thrust::host_vector<T> h_b_key(temp.begin() + n, temp.end());

  thrust::sort(h_a_key.begin(), h_a_key.end());
  thrust::sort(h_b_key.begin(), h_b_key.end());

  thrust::host_vector<T> h_a_val = unittest::random_integers<T>(n);

  thrust::device_vector<T> d_a_key = h_a_key;
  thrust::device_vector<T> d_b_key = h_b_key;

  thrust::device_vector<T> d_a_val = h_a_val;

  thrust::host_vector<T>   h_result_key(n), h_result_val(n);
  thrust::device_vector<T> d_result_key(n), d_result_val(n);

  thrust::pair<
    typename thrust::host_vector<T>::iterator,
    typename thrust::host_vector<T>::iterator
  > h_end;

  thrust::pair<
    typename thrust::device_vector<T>::iterator,
    typename thrust::device_vector<T>::iterator
  > d_end;
  
  h_end = thrust::set_intersection_by_key(h_a_key.begin(), h_a_key.end(),
                                          h_b_key.begin(), h_b_key.end(),
                                          h_a_val.begin(),
                                          h_result_key.begin(),
                                          h_result_val.begin());
  h_result_key.erase(h_end.first,  h_result_key.end());
  h_result_val.erase(h_end.second, h_result_val.end());

  d_end = thrust::set_intersection_by_key(d_a_key.begin(), d_a_key.end(),
                                          d_b_key.begin(), d_b_key.end(),
                                          d_a_val.begin(),
                                          d_result_key.begin(),
                                          d_result_val.begin());
  d_result_key.erase(d_end.first,  d_result_key.end());
  d_result_val.erase(d_end.second, d_result_val.end());

  ASSERT_EQUAL(h_result_key, d_result_key);
  ASSERT_EQUAL(h_result_val, d_result_val);
}
DECLARE_VARIABLE_UNITTEST(TestSetIntersectionByKeyMultiset);

