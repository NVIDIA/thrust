#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/for_each.h>
#include <thrust/device_ptr.h>
#include <thrust/iterator/counting_iterator.h>


__THRUST_DISABLE_MSVC_POSSIBLE_LOSS_OF_DATA_WARNING_BEGIN

struct my_tag : thrust::device_system_tag {};

template <typename T>
class mark_present_for_each
{
    public:
        T * ptr;
        __host__ __device__ void operator()(T x){ ptr[(int) x] = 1; }
};

template <class Vector>
void TestForEachSimple(void)
{
    typedef typename Vector::value_type T;

    Vector input(5);
    Vector output(7, (T) 0);
    
    input[0] = 3; input[1] = 2; input[2] = 3; input[3] = 4; input[4] = 6;

    mark_present_for_each<T> f;
    f.ptr = thrust::raw_pointer_cast(output.data());

    typename Vector::iterator result = thrust::for_each(input.begin(), input.end(), f);

    ASSERT_EQUAL(output[0], 0);
    ASSERT_EQUAL(output[1], 0);
    ASSERT_EQUAL(output[2], 1);
    ASSERT_EQUAL(output[3], 1);
    ASSERT_EQUAL(output[4], 1);
    ASSERT_EQUAL(output[5], 0);
    ASSERT_EQUAL(output[6], 1);
    ASSERT_EQUAL_QUIET(result, input.end());
}
DECLARE_VECTOR_UNITTEST(TestForEachSimple);


template<typename InputIterator, typename Function>
InputIterator for_each(my_tag, InputIterator first, InputIterator, Function)
{
    *first = 13;
    return first;
}

void TestForEachDispatch()
{
    thrust::device_vector<int> vec(1);

    thrust::for_each(thrust::retag<my_tag>(vec.begin()),
                     thrust::retag<my_tag>(vec.end()),
                     0);

    ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestForEachDispatch);


template <class Vector>
void TestForEachNSimple(void)
{
    typedef typename Vector::value_type T;

    Vector input(5);
    Vector output(7, (T) 0);
    
    input[0] = 3; input[1] = 2; input[2] = 3; input[3] = 4; input[4] = 6;

    mark_present_for_each<T> f;
    f.ptr = thrust::raw_pointer_cast(output.data());

    typename Vector::iterator result = thrust::for_each_n(input.begin(), input.size(), f);

    ASSERT_EQUAL(output[0], 0);
    ASSERT_EQUAL(output[1], 0);
    ASSERT_EQUAL(output[2], 1);
    ASSERT_EQUAL(output[3], 1);
    ASSERT_EQUAL(output[4], 1);
    ASSERT_EQUAL(output[5], 0);
    ASSERT_EQUAL(output[6], 1);
    ASSERT_EQUAL_QUIET(result, input.end());
}
DECLARE_VECTOR_UNITTEST(TestForEachNSimple);


template<typename InputIterator, typename Size, typename Function>
InputIterator for_each_n(my_tag, InputIterator first, Size, Function)
{
    *first = 13;
    return first;
}

void TestForEachNDispatch()
{
    thrust::device_vector<int> vec(1);

    thrust::for_each_n(thrust::retag<my_tag>(vec.begin()),
                       vec.size(),
                       0);

    ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestForEachNDispatch);


void TestForEachSimpleAnySystem(void)
{
    thrust::device_vector<int> output(7, 0);

    mark_present_for_each<int> f;
    f.ptr = thrust::raw_pointer_cast(output.data());

    thrust::counting_iterator<int> result = thrust::for_each(thrust::make_counting_iterator(0), thrust::make_counting_iterator(5), f);

    ASSERT_EQUAL(output[0], 1);
    ASSERT_EQUAL(output[1], 1);
    ASSERT_EQUAL(output[2], 1);
    ASSERT_EQUAL(output[3], 1);
    ASSERT_EQUAL(output[4], 1);
    ASSERT_EQUAL(output[5], 0);
    ASSERT_EQUAL(output[6], 0);
    ASSERT_EQUAL_QUIET(result, thrust::make_counting_iterator(5));
}
DECLARE_UNITTEST(TestForEachSimpleAnySystem);


void TestForEachNSimpleAnySystem(void)
{
    thrust::device_vector<int> output(7, 0);

    mark_present_for_each<int> f;
    f.ptr = thrust::raw_pointer_cast(output.data());

    thrust::counting_iterator<int> result = thrust::for_each_n(thrust::make_counting_iterator(0), 5, f);

    ASSERT_EQUAL(output[0], 1);
    ASSERT_EQUAL(output[1], 1);
    ASSERT_EQUAL(output[2], 1);
    ASSERT_EQUAL(output[3], 1);
    ASSERT_EQUAL(output[4], 1);
    ASSERT_EQUAL(output[5], 0);
    ASSERT_EQUAL(output[6], 0);
    ASSERT_EQUAL_QUIET(result, thrust::make_counting_iterator(5));
}
DECLARE_UNITTEST(TestForEachNSimpleAnySystem);


template <typename T>
void TestForEach(const size_t n)
{
    const size_t output_size = std::min((size_t) 10, 2 * n);
    
    thrust::host_vector<T> h_input = unittest::random_integers<T>(n);

    for(size_t i = 0; i < n; i++)
        h_input[i] =  ((size_t) h_input[i]) % output_size;
    
    thrust::device_vector<T> d_input = h_input;

    thrust::host_vector<T>   h_output(output_size, (T) 0);
    thrust::device_vector<T> d_output(output_size, (T) 0);

    mark_present_for_each<T> h_f;
    mark_present_for_each<T> d_f;
    h_f.ptr = &h_output[0];
    d_f.ptr = (&d_output[0]).get();
    
    typename thrust::host_vector<T>::iterator h_result =
      thrust::for_each(h_input.begin(), h_input.end(), h_f);

    typename thrust::device_vector<T>::iterator d_result =
      thrust::for_each(d_input.begin(), d_input.end(), d_f);

    ASSERT_EQUAL(h_output, d_output);
    ASSERT_EQUAL_QUIET(h_result, h_input.end());
    ASSERT_EQUAL_QUIET(d_result, d_input.end());
}
DECLARE_VARIABLE_UNITTEST(TestForEach);


template <typename T>
void TestForEachN(const size_t n)
{
    const size_t output_size = std::min((size_t) 10, 2 * n);
    
    thrust::host_vector<T> h_input = unittest::random_integers<T>(n);

    for(size_t i = 0; i < n; i++)
        h_input[i] =  ((size_t) h_input[i]) % output_size;
    
    thrust::device_vector<T> d_input = h_input;

    thrust::host_vector<T>   h_output(output_size, (T) 0);
    thrust::device_vector<T> d_output(output_size, (T) 0);

    mark_present_for_each<T> h_f;
    mark_present_for_each<T> d_f;
    h_f.ptr = &h_output[0];
    d_f.ptr = (&d_output[0]).get();
    
    typename thrust::host_vector<T>::iterator h_result =
      thrust::for_each_n(h_input.begin(), h_input.size(), h_f);

    typename thrust::device_vector<T>::iterator d_result =
      thrust::for_each_n(d_input.begin(), d_input.size(), d_f);

    ASSERT_EQUAL(h_output, d_output);
    ASSERT_EQUAL_QUIET(h_result, h_input.end());
    ASSERT_EQUAL_QUIET(d_result, d_input.end());
}
DECLARE_VARIABLE_UNITTEST(TestForEachN);


template <typename T, unsigned int N>
struct SetFixedVectorToConstant
{
    FixedVector<T,N> exemplar;

    SetFixedVectorToConstant(T scalar) : exemplar(scalar) {} 

    __host__ __device__
    void operator()(FixedVector<T,N>& t)
    {
        t = exemplar;
    }
};


template <typename T, unsigned int N>
void _TestForEachWithLargeTypes(void)
{
    size_t n = (64 * 1024) / sizeof(FixedVector<T,N>);

    thrust::host_vector< FixedVector<T,N> > h_data(n);

    for(size_t i = 0; i < h_data.size(); i++)
        h_data[i] = FixedVector<T,N>(i);

    thrust::device_vector< FixedVector<T,N> > d_data = h_data;
   
    SetFixedVectorToConstant<T,N> func(123);

    thrust::for_each(h_data.begin(), h_data.end(), func);
    thrust::for_each(d_data.begin(), d_data.end(), func);

    ASSERT_EQUAL_QUIET(h_data, d_data);
}


void TestForEachWithLargeTypes(void)
{
    _TestForEachWithLargeTypes<int,    1>();
    _TestForEachWithLargeTypes<int,    2>();
    _TestForEachWithLargeTypes<int,    4>();
    _TestForEachWithLargeTypes<int,    8>();
    _TestForEachWithLargeTypes<int,   16>();

    KNOWN_FAILURE;

    //_TestForEachWithLargeTypes<int,   32>();  // fails on Linux 32 w/ gcc 4.1
    //_TestForEachWithLargeTypes<int,   64>();
    //_TestForEachWithLargeTypes<int,  128>();
    //_TestForEachWithLargeTypes<int,  256>();
    //_TestForEachWithLargeTypes<int,  512>();
    //_TestForEachWithLargeTypes<int, 1024>();  // fails on Vista 64 w/ VS2008
}
DECLARE_UNITTEST(TestForEachWithLargeTypes);


template <typename T, unsigned int N>
void _TestForEachNWithLargeTypes(void)
{
    size_t n = (64 * 1024) / sizeof(FixedVector<T,N>);

    thrust::host_vector< FixedVector<T,N> > h_data(n);

    for(size_t i = 0; i < h_data.size(); i++)
        h_data[i] = FixedVector<T,N>(i);

    thrust::device_vector< FixedVector<T,N> > d_data = h_data;
   
    SetFixedVectorToConstant<T,N> func(123);

    thrust::for_each_n(h_data.begin(), h_data.size(), func);
    thrust::for_each_n(d_data.begin(), d_data.size(), func);

    ASSERT_EQUAL_QUIET(h_data, d_data);
}


void TestForEachNWithLargeTypes(void)
{
    _TestForEachNWithLargeTypes<int,    1>();
    _TestForEachNWithLargeTypes<int,    2>();
    _TestForEachNWithLargeTypes<int,    4>();
    _TestForEachNWithLargeTypes<int,    8>();
    _TestForEachNWithLargeTypes<int,   16>();

    KNOWN_FAILURE;

    //_TestForEachNWithLargeTypes<int,   32>();  // fails on Linux 32 w/ gcc 4.1
    //_TestForEachNWithLargeTypes<int,   64>();
    //_TestForEachNWithLargeTypes<int,  128>();
    //_TestForEachNWithLargeTypes<int,  256>();
    //_TestForEachNWithLargeTypes<int,  512>();
    //_TestForEachNWithLargeTypes<int, 1024>();  // fails on Vista 64 w/ VS2008
}
DECLARE_UNITTEST(TestForEachNWithLargeTypes);

__THRUST_DISABLE_MSVC_POSSIBLE_LOSS_OF_DATA_WARNING_END
