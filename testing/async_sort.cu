#include <thrust/detail/config.h>

// Disabled on MSVC && NVCC < 11.1 for GH issue #1098.
#if (THRUST_HOST_COMPILER == THRUST_HOST_COMPILER_MSVC) && defined(__HIPCC__)
#if (__CUDACC_VER_MAJOR__ < 11) || (__CUDACC_VER_MAJOR__ == 11 && __CUDACC_VER_MINOR__ < 1)
#define THRUST_BUG_1098_ACTIVE
#endif // NVCC version check
#endif // MSVC + NVCC check

#if THRUST_CPP_DIALECT >= 2014 && !defined(THRUST_BUG_1098_ACTIVE)

#include <unittest/unittest.h>

#include <thrust/async/sort.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

enum wait_policy
{
  wait_for_futures
, do_not_wait_for_futures
};

template <typename T>
struct custom_greater
{
  __host__ __device__
  bool operator()(T rhs, T lhs) const
  {
    return lhs > rhs;
  }
};

#define DEFINE_SORT_INVOKER(name, ...)                                        \
  template <typename T>                                                       \
  struct name                                                                 \
  {                                                                           \
    template <                                                                \
      typename ForwardIt, typename Sentinel                                   \
    >                                                                         \
    __host__                                                                  \
    static void sync(                                                         \
      ForwardIt&& first, Sentinel&& last                                      \
    )                                                                         \
    {                                                                         \
      ::thrust::sort(                                                         \
        THRUST_FWD(first), THRUST_FWD(last)                                   \
      );                                                                      \
    }                                                                         \
                                                                              \
    template <                                                                \
      typename ForwardIt, typename Sentinel                                   \
    >                                                                         \
    __host__                                                                  \
    static auto async(                                                        \
      ForwardIt&& first, Sentinel&& last                                      \
    )                                                                         \
    THRUST_RETURNS(                                                           \
      ::thrust::async::sort(                                                  \
        __VA_ARGS__                                                           \
        THRUST_PP_COMMA_IF(THRUST_PP_ARITY(__VA_ARGS__))                      \
        THRUST_FWD(first), THRUST_FWD(last)                                   \
      )                                                                       \
    )                                                                         \
  };                                                                          \
  /**/

DEFINE_SORT_INVOKER(
  sort_invoker
);
DEFINE_SORT_INVOKER(
  sort_invoker_device, thrust::device
);

#define DEFINE_SORT_OP_INVOKER(name, op, ...)                                 \
  template <typename T>                                                       \
  struct name                                                                 \
  {                                                                           \
    template <                                                                \
      typename ForwardIt, typename Sentinel                                   \
    >                                                                         \
    __host__                                                                  \
    static void sync(                                                         \
      ForwardIt&& first, Sentinel&& last                                      \
    )                                                                         \
    {                                                                         \
      ::thrust::sort(                                                         \
        THRUST_FWD(first), THRUST_FWD(last), op<T>{}                          \
      );                                                                      \
    }                                                                         \
                                                                              \
    template <                                                                \
      typename ForwardIt, typename Sentinel                                   \
    >                                                                         \
    __host__                                                                  \
    static auto async(                                                        \
      ForwardIt&& first, Sentinel&& last                                      \
    )                                                                         \
    THRUST_RETURNS(                                                           \
      ::thrust::async::sort(                                                  \
        __VA_ARGS__                                                           \
        THRUST_PP_COMMA_IF(THRUST_PP_ARITY(__VA_ARGS__))                      \
        THRUST_FWD(first), THRUST_FWD(last), op<T>{}                          \
      )                                                                       \
    )                                                                         \
  };                                                                          \
  /**/

DEFINE_SORT_OP_INVOKER(
  sort_invoker_less,        thrust::less
);
DEFINE_SORT_OP_INVOKER(
  sort_invoker_less_device, thrust::less, thrust::device 
);

DEFINE_SORT_OP_INVOKER(
  sort_invoker_greater,        thrust::greater
);
DEFINE_SORT_OP_INVOKER(
  sort_invoker_greater_device, thrust::greater, thrust::device 
);

DEFINE_SORT_OP_INVOKER(
  sort_invoker_custom_greater,        custom_greater
);
DEFINE_SORT_OP_INVOKER(
  sort_invoker_custom_greater_device, custom_greater, thrust::device 
);

#undef DEFINE_SORT_INVOKER
#undef DEFINE_SORT_OP_INVOKER

///////////////////////////////////////////////////////////////////////////////

template <template <typename> class SortInvoker, wait_policy WaitPolicy>
struct test_async_sort
{
  template <typename T>
  struct tester
  {
    __host__
    void operator()(std::size_t n)
    {
      thrust::host_vector<T>   h0_data(unittest::random_integers<T>(n));
      thrust::device_vector<T> d0_data(h0_data);

      ASSERT_EQUAL(h0_data, d0_data);

      SortInvoker<T>::sync(
        h0_data.begin(), h0_data.end()
      );

      auto f0 = SortInvoker<T>::async(
        d0_data.begin(), d0_data.end()
      );

      if (wait_for_futures == WaitPolicy)
      {
        f0.wait();

        ASSERT_EQUAL(h0_data, d0_data);
      }
    }
  };
};
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_sort<
      sort_invoker
    , wait_for_futures
    >::tester
  )
, NumericTypes
, test_async_sort
);
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_sort<
      sort_invoker
    , do_not_wait_for_futures
    >::tester
  )
, NumericTypes
, test_async_sort_no_wait
);
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_sort<
      sort_invoker_device
    , wait_for_futures
    >::tester
  )
, NumericTypes
, test_async_sort_policy
);
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_sort<
      sort_invoker_device
    , do_not_wait_for_futures
    >::tester
  )
, NumericTypes
, test_async_sort_policy_no_wait
);
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_sort<
      sort_invoker_less
    , wait_for_futures
    >::tester
  )
, NumericTypes
, test_async_sort_less
);
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_sort<
      sort_invoker_less
    , do_not_wait_for_futures
    >::tester
  )
, NumericTypes
, test_async_sort_less_no_wait
);
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_sort<
      sort_invoker_less_device
    , wait_for_futures
    >::tester
  )
, NumericTypes
, test_async_sort_policy_less
);
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_sort<
      sort_invoker_less_device
    , do_not_wait_for_futures
    >::tester
  )
, NumericTypes
, test_async_sort_policy_less_no_wait
);
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_sort<
      sort_invoker_greater
    , wait_for_futures
    >::tester
  )
, NumericTypes
, test_async_sort_greater
);
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_sort<
      sort_invoker_greater
    , do_not_wait_for_futures
    >::tester
  )
, NumericTypes
, test_async_sort_greater_no_wait
);
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_sort<
      sort_invoker_greater_device
    , wait_for_futures
    >::tester
  )
, NumericTypes
, test_async_sort_policy_greater
);
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_sort<
      sort_invoker_greater_device
    , do_not_wait_for_futures
    >::tester
  )
, NumericTypes
, test_async_sort_policy_greater_no_wait
);
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_sort<
      sort_invoker_custom_greater
    , wait_for_futures
    >::tester
  )
, NumericTypes
, test_async_sort_custom_greater
);
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_sort<
      sort_invoker_custom_greater
    , do_not_wait_for_futures
    >::tester
  )
, NumericTypes
, test_async_sort_custom_greater_no_wait
);
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_sort<
      sort_invoker_custom_greater_device
    , wait_for_futures
    >::tester
  )
, NumericTypes
, test_async_sort_policy_custom_greater
);
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  THRUST_PP_EXPAND_ARGS(
    test_async_sort<
      sort_invoker_custom_greater_device
    , do_not_wait_for_futures
    >::tester
  )
, NumericTypes
, test_async_sort_policy_custom_greater_no_wait
);

///////////////////////////////////////////////////////////////////////////////

// TODO: Async copy then sort.

// TODO: Test future return type.

#endif

