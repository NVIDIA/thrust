#include <unittest/unittest.h>
#include <thrust/tuple.h>
#include <thrust/generate.h>
#include <thrust/swap.h>

using namespace unittest;

template <typename T>
struct TestTupleConstructor
{
  void operator()(void)
  {
    using namespace thrust;

    host_vector<T> data = random_integers<T>(10);

    tuple<T> t1(data[0]);
    ASSERT_EQUAL(data[0], get<0>(t1));

    tuple<T,T> t2(data[0], data[1]);
    ASSERT_EQUAL(data[0], get<0>(t2));
    ASSERT_EQUAL(data[1], get<1>(t2));

    tuple<T,T,T> t3(data[0], data[1], data[2]);
    ASSERT_EQUAL(data[0], get<0>(t3));
    ASSERT_EQUAL(data[1], get<1>(t3));
    ASSERT_EQUAL(data[2], get<2>(t3));

    tuple<T,T,T,T> t4(data[0], data[1], data[2], data[3]);
    ASSERT_EQUAL(data[0], get<0>(t4));
    ASSERT_EQUAL(data[1], get<1>(t4));
    ASSERT_EQUAL(data[2], get<2>(t4));
    ASSERT_EQUAL(data[3], get<3>(t4));

    tuple<T,T,T,T,T> t5(data[0], data[1], data[2], data[3], data[4]);
    ASSERT_EQUAL(data[0], get<0>(t5));
    ASSERT_EQUAL(data[1], get<1>(t5));
    ASSERT_EQUAL(data[2], get<2>(t5));
    ASSERT_EQUAL(data[3], get<3>(t5));
    ASSERT_EQUAL(data[4], get<4>(t5));

    tuple<T,T,T,T,T,T> t6(data[0], data[1], data[2], data[3], data[4], data[5]);
    ASSERT_EQUAL(data[0], get<0>(t6));
    ASSERT_EQUAL(data[1], get<1>(t6));
    ASSERT_EQUAL(data[2], get<2>(t6));
    ASSERT_EQUAL(data[3], get<3>(t6));
    ASSERT_EQUAL(data[4], get<4>(t6));
    ASSERT_EQUAL(data[5], get<5>(t6));

    tuple<T,T,T,T,T,T,T> t7(data[0], data[1], data[2], data[3], data[4], data[5], data[6]);
    ASSERT_EQUAL(data[0], get<0>(t7));
    ASSERT_EQUAL(data[1], get<1>(t7));
    ASSERT_EQUAL(data[2], get<2>(t7));
    ASSERT_EQUAL(data[3], get<3>(t7));
    ASSERT_EQUAL(data[4], get<4>(t7));
    ASSERT_EQUAL(data[5], get<5>(t7));
    ASSERT_EQUAL(data[6], get<6>(t7));

    tuple<T,T,T,T,T,T,T,T> t8(data[0], data[1], data[2], data[3], data[4], data[5], data[6], data[7]);
    ASSERT_EQUAL(data[0], get<0>(t8));
    ASSERT_EQUAL(data[1], get<1>(t8));
    ASSERT_EQUAL(data[2], get<2>(t8));
    ASSERT_EQUAL(data[3], get<3>(t8));
    ASSERT_EQUAL(data[4], get<4>(t8));
    ASSERT_EQUAL(data[5], get<5>(t8));
    ASSERT_EQUAL(data[6], get<6>(t8));
    ASSERT_EQUAL(data[7], get<7>(t8));

    tuple<T,T,T,T,T,T,T,T,T> t9(data[0], data[1], data[2], data[3], data[4], data[5], data[6], data[7], data[8]);
    ASSERT_EQUAL(data[0], get<0>(t9));
    ASSERT_EQUAL(data[1], get<1>(t9));
    ASSERT_EQUAL(data[2], get<2>(t9));
    ASSERT_EQUAL(data[3], get<3>(t9));
    ASSERT_EQUAL(data[4], get<4>(t9));
    ASSERT_EQUAL(data[5], get<5>(t9));
    ASSERT_EQUAL(data[6], get<6>(t9));
    ASSERT_EQUAL(data[7], get<7>(t9));
    ASSERT_EQUAL(data[8], get<8>(t9));

    tuple<T,T,T,T,T,T,T,T,T,T> t10(data[0], data[1], data[2], data[3], data[4], data[5], data[6], data[7], data[8], data[9]);
    ASSERT_EQUAL(data[0], get<0>(t10));
    ASSERT_EQUAL(data[1], get<1>(t10));
    ASSERT_EQUAL(data[2], get<2>(t10));
    ASSERT_EQUAL(data[3], get<3>(t10));
    ASSERT_EQUAL(data[4], get<4>(t10));
    ASSERT_EQUAL(data[5], get<5>(t10));
    ASSERT_EQUAL(data[6], get<6>(t10));
    ASSERT_EQUAL(data[7], get<7>(t10));
    ASSERT_EQUAL(data[8], get<8>(t10));
    ASSERT_EQUAL(data[9], get<9>(t10));
  }
};
SimpleUnitTest<TestTupleConstructor, NumericTypes> TestTupleConstructorInstance;

template <typename T>
struct TestMakeTuple
{
  void operator()(void)
  {
    using namespace thrust;

    host_vector<T> data = random_integers<T>(10);

    tuple<T> t1 = make_tuple(data[0]);
    ASSERT_EQUAL(data[0], get<0>(t1));

    tuple<T,T> t2 = make_tuple(data[0], data[1]);
    ASSERT_EQUAL(data[0], get<0>(t2));
    ASSERT_EQUAL(data[1], get<1>(t2));

    tuple<T,T,T> t3 = make_tuple(data[0], data[1], data[2]);
    ASSERT_EQUAL(data[0], get<0>(t3));
    ASSERT_EQUAL(data[1], get<1>(t3));
    ASSERT_EQUAL(data[2], get<2>(t3));

    tuple<T,T,T,T> t4 = make_tuple(data[0], data[1], data[2], data[3]);
    ASSERT_EQUAL(data[0], get<0>(t4));
    ASSERT_EQUAL(data[1], get<1>(t4));
    ASSERT_EQUAL(data[2], get<2>(t4));
    ASSERT_EQUAL(data[3], get<3>(t4));

    tuple<T,T,T,T,T> t5 = make_tuple(data[0], data[1], data[2], data[3], data[4]);
    ASSERT_EQUAL(data[0], get<0>(t5));
    ASSERT_EQUAL(data[1], get<1>(t5));
    ASSERT_EQUAL(data[2], get<2>(t5));
    ASSERT_EQUAL(data[3], get<3>(t5));
    ASSERT_EQUAL(data[4], get<4>(t5));

    tuple<T,T,T,T,T,T> t6 = make_tuple(data[0], data[1], data[2], data[3], data[4], data[5]);
    ASSERT_EQUAL(data[0], get<0>(t6));
    ASSERT_EQUAL(data[1], get<1>(t6));
    ASSERT_EQUAL(data[2], get<2>(t6));
    ASSERT_EQUAL(data[3], get<3>(t6));
    ASSERT_EQUAL(data[4], get<4>(t6));
    ASSERT_EQUAL(data[5], get<5>(t6));

    tuple<T,T,T,T,T,T,T> t7 = make_tuple(data[0], data[1], data[2], data[3], data[4], data[5], data[6]);
    ASSERT_EQUAL(data[0], get<0>(t7));
    ASSERT_EQUAL(data[1], get<1>(t7));
    ASSERT_EQUAL(data[2], get<2>(t7));
    ASSERT_EQUAL(data[3], get<3>(t7));
    ASSERT_EQUAL(data[4], get<4>(t7));
    ASSERT_EQUAL(data[5], get<5>(t7));
    ASSERT_EQUAL(data[6], get<6>(t7));

    tuple<T,T,T,T,T,T,T,T> t8 = make_tuple(data[0], data[1], data[2], data[3], data[4], data[5], data[6], data[7]);
    ASSERT_EQUAL(data[0], get<0>(t8));
    ASSERT_EQUAL(data[1], get<1>(t8));
    ASSERT_EQUAL(data[2], get<2>(t8));
    ASSERT_EQUAL(data[3], get<3>(t8));
    ASSERT_EQUAL(data[4], get<4>(t8));
    ASSERT_EQUAL(data[5], get<5>(t8));
    ASSERT_EQUAL(data[6], get<6>(t8));
    ASSERT_EQUAL(data[7], get<7>(t8));

    tuple<T,T,T,T,T,T,T,T,T> t9 = make_tuple(data[0], data[1], data[2], data[3], data[4], data[5], data[6], data[7], data[8]);
    ASSERT_EQUAL(data[0], get<0>(t9));
    ASSERT_EQUAL(data[1], get<1>(t9));
    ASSERT_EQUAL(data[2], get<2>(t9));
    ASSERT_EQUAL(data[3], get<3>(t9));
    ASSERT_EQUAL(data[4], get<4>(t9));
    ASSERT_EQUAL(data[5], get<5>(t9));
    ASSERT_EQUAL(data[6], get<6>(t9));
    ASSERT_EQUAL(data[7], get<7>(t9));
    ASSERT_EQUAL(data[8], get<8>(t9));

    tuple<T,T,T,T,T,T,T,T,T,T> t10 = make_tuple(data[0], data[1], data[2], data[3], data[4], data[5], data[6], data[7], data[8], data[9]);
    ASSERT_EQUAL(data[0], get<0>(t10));
    ASSERT_EQUAL(data[1], get<1>(t10));
    ASSERT_EQUAL(data[2], get<2>(t10));
    ASSERT_EQUAL(data[3], get<3>(t10));
    ASSERT_EQUAL(data[4], get<4>(t10));
    ASSERT_EQUAL(data[5], get<5>(t10));
    ASSERT_EQUAL(data[6], get<6>(t10));
    ASSERT_EQUAL(data[7], get<7>(t10));
    ASSERT_EQUAL(data[8], get<8>(t10));
    ASSERT_EQUAL(data[9], get<9>(t10));
  }
};
SimpleUnitTest<TestMakeTuple, NumericTypes> TestMakeTupleInstance;

template <typename T>
struct TestTupleGet
{
  void operator()(void)
  {
    using namespace thrust;
    host_vector<T> data = random_integers<T>(10);

    tuple<T> t1(data[0]);
    ASSERT_EQUAL(data[0], thrust::get<0>(t1));

    tuple<T,T> t2(data[0], data[1]);
    ASSERT_EQUAL(data[0], thrust::get<0>(t2));
    ASSERT_EQUAL(data[1], thrust::get<1>(t2));

    tuple<T,T,T> t3 = make_tuple(data[0], data[1], data[2]);
    ASSERT_EQUAL(data[0], thrust::get<0>(t3));
    ASSERT_EQUAL(data[1], thrust::get<1>(t3));
    ASSERT_EQUAL(data[2], thrust::get<2>(t3));

    tuple<T,T,T,T> t4 = make_tuple(data[0], data[1], data[2], data[3]);
    ASSERT_EQUAL(data[0], thrust::get<0>(t4));
    ASSERT_EQUAL(data[1], thrust::get<1>(t4));
    ASSERT_EQUAL(data[2], thrust::get<2>(t4));
    ASSERT_EQUAL(data[3], thrust::get<3>(t4));

    tuple<T,T,T,T,T> t5 = make_tuple(data[0], data[1], data[2], data[3], data[4]);
    ASSERT_EQUAL(data[0], thrust::get<0>(t5));
    ASSERT_EQUAL(data[1], thrust::get<1>(t5));
    ASSERT_EQUAL(data[2], thrust::get<2>(t5));
    ASSERT_EQUAL(data[3], thrust::get<3>(t5));
    ASSERT_EQUAL(data[4], thrust::get<4>(t5));

    tuple<T,T,T,T,T,T> t6 = make_tuple(data[0], data[1], data[2], data[3], data[4], data[5]);
    ASSERT_EQUAL(data[0], thrust::get<0>(t6));
    ASSERT_EQUAL(data[1], thrust::get<1>(t6));
    ASSERT_EQUAL(data[2], thrust::get<2>(t6));
    ASSERT_EQUAL(data[3], thrust::get<3>(t6));
    ASSERT_EQUAL(data[4], thrust::get<4>(t6));
    ASSERT_EQUAL(data[5], thrust::get<5>(t6));

    tuple<T,T,T,T,T,T,T> t7 = make_tuple(data[0], data[1], data[2], data[3], data[4], data[5], data[6]);
    ASSERT_EQUAL(data[0], thrust::get<0>(t7));
    ASSERT_EQUAL(data[1], thrust::get<1>(t7));
    ASSERT_EQUAL(data[2], thrust::get<2>(t7));
    ASSERT_EQUAL(data[3], thrust::get<3>(t7));
    ASSERT_EQUAL(data[4], thrust::get<4>(t7));
    ASSERT_EQUAL(data[5], thrust::get<5>(t7));
    ASSERT_EQUAL(data[6], thrust::get<6>(t7));

    tuple<T,T,T,T,T,T,T,T> t8 = make_tuple(data[0], data[1], data[2], data[3], data[4], data[5], data[6], data[7]);
    ASSERT_EQUAL(data[0], thrust::get<0>(t8));
    ASSERT_EQUAL(data[1], thrust::get<1>(t8));
    ASSERT_EQUAL(data[2], thrust::get<2>(t8));
    ASSERT_EQUAL(data[3], thrust::get<3>(t8));
    ASSERT_EQUAL(data[4], thrust::get<4>(t8));
    ASSERT_EQUAL(data[5], thrust::get<5>(t8));
    ASSERT_EQUAL(data[6], thrust::get<6>(t8));
    ASSERT_EQUAL(data[7], thrust::get<7>(t8));

    tuple<T,T,T,T,T,T,T,T,T> t9 = make_tuple(data[0], data[1], data[2], data[3], data[4], data[5], data[6], data[7], data[8]);
    ASSERT_EQUAL(data[0], thrust::get<0>(t9));
    ASSERT_EQUAL(data[1], thrust::get<1>(t9));
    ASSERT_EQUAL(data[2], thrust::get<2>(t9));
    ASSERT_EQUAL(data[3], thrust::get<3>(t9));
    ASSERT_EQUAL(data[4], thrust::get<4>(t9));
    ASSERT_EQUAL(data[5], thrust::get<5>(t9));
    ASSERT_EQUAL(data[6], thrust::get<6>(t9));
    ASSERT_EQUAL(data[7], thrust::get<7>(t9));
    ASSERT_EQUAL(data[8], thrust::get<8>(t9));

    tuple<T,T,T,T,T,T,T,T,T,T> t10 = make_tuple(data[0], data[1], data[2], data[3], data[4], data[5], data[6], data[7], data[8], data[9]);
    ASSERT_EQUAL(data[0], thrust::get<0>(t10));
    ASSERT_EQUAL(data[1], thrust::get<1>(t10));
    ASSERT_EQUAL(data[2], thrust::get<2>(t10));
    ASSERT_EQUAL(data[3], thrust::get<3>(t10));
    ASSERT_EQUAL(data[4], thrust::get<4>(t10));
    ASSERT_EQUAL(data[5], thrust::get<5>(t10));
    ASSERT_EQUAL(data[6], thrust::get<6>(t10));
    ASSERT_EQUAL(data[7], thrust::get<7>(t10));
    ASSERT_EQUAL(data[8], thrust::get<8>(t10));
    ASSERT_EQUAL(data[9], thrust::get<9>(t10));
  }
};
SimpleUnitTest<TestTupleGet, NumericTypes> TestTupleGetInstance;



template <typename T>
struct TestTupleComparison
{
  void operator()(void)
  {
    using namespace thrust;

    tuple<T,T,T,T,T> lhs(0, 0, 0, 0, 0), rhs(0, 0, 0, 0, 0);

    // equality
    ASSERT_EQUAL(true,  lhs == rhs);
    get<0>(rhs) = 1;
    ASSERT_EQUAL(false,  lhs == rhs);

    // inequality
    ASSERT_EQUAL(true,  lhs != rhs);
    lhs = rhs;
    ASSERT_EQUAL(false, lhs != rhs);

    // less than
    lhs = make_tuple(0,0,0,0,0);
    rhs = make_tuple(0,0,1,0,0);
    ASSERT_EQUAL(true,  lhs < rhs);
    get<0>(lhs) = 2;
    ASSERT_EQUAL(false, lhs < rhs);

    // less than equal
    lhs = make_tuple(0,0,0,0,0);
    rhs = lhs;
    ASSERT_EQUAL(true,  lhs <= rhs); // equal
    get<2>(rhs) = 1;
    ASSERT_EQUAL(true,  lhs <= rhs); // less than
    get<2>(lhs) = 2;
    ASSERT_EQUAL(false, lhs <= rhs);

    // greater than
    lhs = make_tuple(1,0,0,0,0);
    rhs = make_tuple(0,1,1,1,1);
    ASSERT_EQUAL(true,  lhs > rhs);
    get<0>(rhs) = 2;
    ASSERT_EQUAL(false, lhs > rhs);

    // greater than equal
    lhs = make_tuple(0,0,0,0,0);
    rhs = lhs;
    ASSERT_EQUAL(true,  lhs >= rhs); // equal
    get<4>(lhs) = 1;
    ASSERT_EQUAL(true,  lhs >= rhs); // greater than
    get<3>(rhs) = 1;
    ASSERT_EQUAL(false, lhs >= rhs);
  }
};
SimpleUnitTest<TestTupleComparison, NumericTypes> TestTupleComparisonInstance;


template <typename T>
struct TestTupleTieFunctor
{
  __host__ __device__
  void clear(T *data) const
  {
    for(int i = 0; i < 10; ++i)
      data[i] = 13;
  }

  __host__ __device__
  bool operator()() const
  {
    using namespace thrust;

    bool result = true;

    T data[10];
    clear(data);

    tie(data[0]) = make_tuple(0);;
    result &= data[0] == 0;
    clear(data);

    tie(data[0], data[1]) = make_tuple(0,1);
    result &= data[0] == 0;
    result &= data[1] == 1;
    clear(data);

    tie(data[0], data[1], data[2]) = make_tuple(0,1,2);
    result &= data[0] == 0;
    result &= data[1] == 1;
    result &= data[2] == 2;
    clear(data);

    tie(data[0], data[1], data[2], data[3]) = make_tuple(0,1,2,3);
    result &= data[0] == 0;
    result &= data[1] == 1;
    result &= data[2] == 2;
    result &= data[3] == 3;
    clear(data);

    tie(data[0], data[1], data[2], data[3], data[4]) = make_tuple(0,1,2,3,4);
    result &= data[0] == 0;
    result &= data[1] == 1;
    result &= data[2] == 2;
    result &= data[3] == 3;
    result &= data[4] == 4;
    clear(data);

    tie(data[0], data[1], data[2], data[3], data[4], data[5]) = make_tuple(0,1,2,3,4,5);
    result &= data[0] == 0;
    result &= data[1] == 1;
    result &= data[2] == 2;
    result &= data[3] == 3;
    result &= data[4] == 4;
    result &= data[5] == 5;
    clear(data);

    tie(data[0], data[1], data[2], data[3], data[4], data[5], data[6]) = make_tuple(0,1,2,3,4,5,6);
    result &= data[0] == 0;
    result &= data[1] == 1;
    result &= data[2] == 2;
    result &= data[3] == 3;
    result &= data[4] == 4;
    result &= data[5] == 5;
    result &= data[6] == 6;
    clear(data);

    tie(data[0], data[1], data[2], data[3], data[4], data[5], data[6], data[7]) = make_tuple(0,1,2,3,4,5,6,7);
    result &= data[0] == 0;
    result &= data[1] == 1;
    result &= data[2] == 2;
    result &= data[3] == 3;
    result &= data[4] == 4;
    result &= data[5] == 5;
    result &= data[6] == 6;
    result &= data[7] == 7;
    clear(data);

    tie(data[0], data[1], data[2], data[3], data[4], data[5], data[6], data[7], data[8]) = make_tuple(0,1,2,3,4,5,6,7,8);
    result &= data[0] == 0;
    result &= data[1] == 1;
    result &= data[2] == 2;
    result &= data[3] == 3;
    result &= data[4] == 4;
    result &= data[5] == 5;
    result &= data[6] == 6;
    result &= data[7] == 7;
    result &= data[8] == 8;
    clear(data);

    tie(data[0], data[1], data[2], data[3], data[4], data[5], data[6], data[7], data[8], data[9]) = make_tuple(0,1,2,3,4,5,6,7,8,9);
    result &= data[0] == 0;
    result &= data[1] == 1;
    result &= data[2] == 2;
    result &= data[3] == 3;
    result &= data[4] == 4;
    result &= data[5] == 5;
    result &= data[6] == 6;
    result &= data[7] == 7;
    result &= data[8] == 8;
    result &= data[9] == 9;
    clear(data);

    return result;
  }
};

template <typename T>
struct TestTupleTie
{
  void operator()(void)
  {
    thrust::host_vector<bool> h_result(1);
    thrust::generate(h_result.begin(), h_result.end(), TestTupleTieFunctor<T>());

    thrust::device_vector<bool> d_result(1);
    thrust::generate(d_result.begin(), d_result.end(), TestTupleTieFunctor<T>());

    ASSERT_EQUAL(true, h_result[0]);
    ASSERT_EQUAL(true, d_result[0]);
  }
};
SimpleUnitTest<TestTupleTie, NumericTypes> TestTupleTieInstance;

void TestTupleSwap(void)
{
  int a = 7;
  int b = 13;
  int c = 42;

  int x = 77;
  int y = 1313;
  int z = 4242;

  thrust::tuple<int,int,int> t1(a,b,c);
  thrust::tuple<int,int,int> t2(x,y,z);

  thrust::swap(t1,t2);

  ASSERT_EQUAL(x, thrust::get<0>(t1));
  ASSERT_EQUAL(y, thrust::get<1>(t1));
  ASSERT_EQUAL(z, thrust::get<2>(t1));
  ASSERT_EQUAL(a, thrust::get<0>(t2));
  ASSERT_EQUAL(b, thrust::get<1>(t2));
  ASSERT_EQUAL(c, thrust::get<2>(t2));


  typedef thrust::tuple<user_swappable,user_swappable,user_swappable,user_swappable> swappable_tuple;

  thrust::host_vector<swappable_tuple>   h_v1(1), h_v2(1);
  thrust::device_vector<swappable_tuple> d_v1(1), d_v2(1);

  thrust::swap_ranges(h_v1.begin(), h_v1.end(), h_v2.begin());
  thrust::swap_ranges(d_v1.begin(), d_v1.end(), d_v2.begin());

  swappable_tuple ref(user_swappable(true),user_swappable(true),user_swappable(true),user_swappable(true));

  ASSERT_EQUAL_QUIET(ref, h_v1[0]);
  ASSERT_EQUAL_QUIET(ref, h_v1[0]);
  ASSERT_EQUAL_QUIET(ref, (swappable_tuple)d_v1[0]);
  ASSERT_EQUAL_QUIET(ref, (swappable_tuple)d_v1[0]);
}
DECLARE_UNITTEST(TestTupleSwap);


