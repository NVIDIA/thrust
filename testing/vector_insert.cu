#include <unittest/unittest.h>
#include <thrust/sequence.h>
#include <thrust/device_malloc_allocator.h>

template <class Vector>
struct TestVectorRangeInsertSimple
{
    void operator()(size_t)
    {
        Vector v1(5);
        thrust::sequence(v1.begin(), v1.end());

        // test when insertion range fits inside capacity
        // and the size of the insertion is greater than the number
        // of displaced elements
        Vector v2(3);
        v2.reserve(10);
        thrust::sequence(v2.begin(), v2.end());

        size_t new_size = v2.size() + v1.size();
        size_t insertion_size = v1.end() - v1.begin();
        size_t num_displaced = v2.end() - (v2.begin() + 1);

        ASSERT_EQUAL(true, v2.capacity()   >= new_size);
        ASSERT_EQUAL(true, insertion_size  >  num_displaced);

        v2.insert(v2.begin() + 1,
                  v1.begin(), v1.end());

        ASSERT_EQUAL(0, v2[0]);

        ASSERT_EQUAL(0, v2[1]);
        ASSERT_EQUAL(1, v2[2]);
        ASSERT_EQUAL(2, v2[3]);
        ASSERT_EQUAL(3, v2[4]);
        ASSERT_EQUAL(4, v2[5]);

        ASSERT_EQUAL(1, v2[6]);
        ASSERT_EQUAL(2, v2[7]);
        
        ASSERT_EQUAL(8,  v2.size());
        ASSERT_EQUAL(10, v2.capacity());

        // test when insertion range fits inside capacity
        // and the size of the insertion is equal to the number
        // of displaced elements
        Vector v3(5);
        v3.reserve(10);
        thrust::sequence(v3.begin(), v3.end());

        new_size = v3.size() + v1.size();
        insertion_size = v1.end() - v1.begin();
        num_displaced = v3.end() - v3.begin();

        ASSERT_EQUAL(true, v3.capacity()   >=  new_size);
        ASSERT_EQUAL(true, insertion_size  ==  num_displaced);

        v3.insert(v3.begin(),
                  v1.begin(), v1.end());

        ASSERT_EQUAL(0, v3[0]);
        ASSERT_EQUAL(1, v3[1]);
        ASSERT_EQUAL(2, v3[2]);
        ASSERT_EQUAL(3, v3[3]);
        ASSERT_EQUAL(4, v3[4]);

        ASSERT_EQUAL(0, v3[5]);
        ASSERT_EQUAL(1, v3[6]);
        ASSERT_EQUAL(2, v3[7]);
        ASSERT_EQUAL(3, v3[8]);
        ASSERT_EQUAL(4, v3[9]);

        ASSERT_EQUAL(10, v3.size());
        ASSERT_EQUAL(10, v3.capacity());

        // test when insertion range fits inside capacity
        // and the size of the insertion is less than the
        // number of displaced elements
        Vector v4(5);
        v4.reserve(10);
        thrust::sequence(v4.begin(), v4.end());

        new_size = v4.size() + v1.size();
        insertion_size = (v1.begin() + 3) - v1.begin();
        num_displaced = v4.end() - (v4.begin() + 1);

        ASSERT_EQUAL(true, v4.capacity()   >=  new_size);
        ASSERT_EQUAL(true, insertion_size  <   num_displaced);

        v4.insert(v4.begin() + 1,
                  v1.begin(), v1.begin() + 3);

        ASSERT_EQUAL(0, v4[0]);

        ASSERT_EQUAL(0, v4[1]);
        ASSERT_EQUAL(1, v4[2]);
        ASSERT_EQUAL(2, v4[3]);

        ASSERT_EQUAL(1, v4[4]);
        ASSERT_EQUAL(2, v4[5]);
        ASSERT_EQUAL(3, v4[6]);
        ASSERT_EQUAL(4, v4[7]);

        ASSERT_EQUAL(8, v4.size());
        ASSERT_EQUAL(10, v4.capacity());

        // test when insertion range does not fit inside capacity
        Vector v5(5);
        thrust::sequence(v5.begin(), v5.end());

        new_size = v5.size() + v1.size();

        ASSERT_EQUAL(true, v5.capacity() < new_size);

        v5.insert(v5.begin() + 1,
                  v1.begin(), v1.end());

        ASSERT_EQUAL(0, v5[0]);

        ASSERT_EQUAL(0, v5[1]);
        ASSERT_EQUAL(1, v5[2]);
        ASSERT_EQUAL(2, v5[3]);
        ASSERT_EQUAL(3, v5[4]);
        ASSERT_EQUAL(4, v5[5]);

        ASSERT_EQUAL(1, v5[6]);
        ASSERT_EQUAL(2, v5[7]);
        ASSERT_EQUAL(3, v5[8]);
        ASSERT_EQUAL(4, v5[9]);

        ASSERT_EQUAL(10, v5.size());
    }
}; // end TestVectorRangeInsertSimple
VectorUnitTest<TestVectorRangeInsertSimple, NumericTypes, thrust::device_vector, thrust::device_malloc_allocator> TestVectorRangeInsertSimpleDeviceInstance;
VectorUnitTest<TestVectorRangeInsertSimple, NumericTypes, thrust::host_vector,   std::allocator>                  TestVectorRangeInsertSimpleHostInstance;

template <class T>
struct TestVectorRangeInsert
{
    void operator()(size_t n)
    {
        thrust::host_vector<T>   h_src = unittest::random_samples<T>(n + 3);
        thrust::host_vector<T>   h_dst = unittest::random_samples<T>(n);

        thrust::device_vector<T> d_src = h_src;
        thrust::device_vector<T> d_dst = h_dst;

        // choose insertion range at random
        size_t begin = n > 0 ? (size_t)h_src[n]   % n : 0;
        size_t end   = n > 0 ? (size_t)h_src[n+1] % n : 0;
        if(end < begin) thrust::swap(begin,end);

        // choose insertion position at random
        size_t position = n > 0 ? (size_t)h_src[n+2] % n : 0;

        // insert on host
        h_dst.insert(h_dst.begin() + position,
                     h_src.begin() + begin,
                     h_src.begin() + end);

        // insert on device
        d_dst.insert(d_dst.begin() + position,
                     d_src.begin() + begin,
                     d_src.begin() + end);

        ASSERT_EQUAL(h_dst, d_dst);
    }
}; // end TestVectorRangeInsert
VariableUnitTest<TestVectorRangeInsert, IntegralTypes> TestVectorRangeInsertInstance;

template <class Vector>
struct TestVectorFillInsertSimple
{
    void operator()(size_t)
    {
        // test when insertion range fits inside capacity
        // and the size of the insertion is greater than the number
        // of displaced elements
        Vector v1(3);
        v1.reserve(10);
        thrust::sequence(v1.begin(), v1.end());

        size_t insertion_size = 5;
        size_t new_size = v1.size() + insertion_size;
        size_t num_displaced = v1.end() - (v1.begin() + 1);

        ASSERT_EQUAL(true, v1.capacity()   >= new_size);
        ASSERT_EQUAL(true, insertion_size  >  num_displaced);

        v1.insert(v1.begin() + 1, insertion_size, 13);

        ASSERT_EQUAL(0, v1[0]);

        ASSERT_EQUAL(13, v1[1]);
        ASSERT_EQUAL(13, v1[2]);
        ASSERT_EQUAL(13, v1[3]);
        ASSERT_EQUAL(13, v1[4]);
        ASSERT_EQUAL(13, v1[5]);

        ASSERT_EQUAL(1, v1[6]);
        ASSERT_EQUAL(2, v1[7]);
        
        ASSERT_EQUAL(8,  v1.size());
        ASSERT_EQUAL(10, v1.capacity());

        // test when insertion range fits inside capacity
        // and the size of the insertion is equal to the number
        // of displaced elements
        Vector v2(5);
        v2.reserve(10);
        thrust::sequence(v2.begin(), v2.end());

        insertion_size = 5;
        new_size = v2.size() + insertion_size;
        num_displaced = v2.end() - v2.begin();

        ASSERT_EQUAL(true, v2.capacity()   >=  new_size);
        ASSERT_EQUAL(true, insertion_size  ==  num_displaced);

        v2.insert(v2.begin(), insertion_size, 13);

        ASSERT_EQUAL(13, v2[0]);
        ASSERT_EQUAL(13, v2[1]);
        ASSERT_EQUAL(13, v2[2]);
        ASSERT_EQUAL(13, v2[3]);
        ASSERT_EQUAL(13, v2[4]);

        ASSERT_EQUAL(0, v2[5]);
        ASSERT_EQUAL(1, v2[6]);
        ASSERT_EQUAL(2, v2[7]);
        ASSERT_EQUAL(3, v2[8]);
        ASSERT_EQUAL(4, v2[9]);

        ASSERT_EQUAL(10, v2.size());
        ASSERT_EQUAL(10, v2.capacity());

        // test when insertion range fits inside capacity
        // and the size of the insertion is less than the
        // number of displaced elements
        Vector v3(5);
        v3.reserve(10);
        thrust::sequence(v3.begin(), v3.end());

        insertion_size = 3;
        new_size = v3.size() + insertion_size;
        num_displaced = v3.end() - (v3.begin() + 1);

        ASSERT_EQUAL(true, v3.capacity()   >=  new_size);
        ASSERT_EQUAL(true, insertion_size  <   num_displaced);

        v3.insert(v3.begin() + 1, insertion_size, 13);

        ASSERT_EQUAL(0, v3[0]);

        ASSERT_EQUAL(13, v3[1]);
        ASSERT_EQUAL(13, v3[2]);
        ASSERT_EQUAL(13, v3[3]);

        ASSERT_EQUAL(1, v3[4]);
        ASSERT_EQUAL(2, v3[5]);
        ASSERT_EQUAL(3, v3[6]);
        ASSERT_EQUAL(4, v3[7]);

        ASSERT_EQUAL(8, v3.size());
        ASSERT_EQUAL(10, v3.capacity());

        // test when insertion range does not fit inside capacity
        Vector v4(5);
        thrust::sequence(v4.begin(), v4.end());

        insertion_size = 5;
        new_size = v4.size() + insertion_size;

        ASSERT_EQUAL(true, v4.capacity() < new_size);

        v4.insert(v4.begin() + 1, insertion_size, 13);

        ASSERT_EQUAL(0, v4[0]);

        ASSERT_EQUAL(13, v4[1]);
        ASSERT_EQUAL(13, v4[2]);
        ASSERT_EQUAL(13, v4[3]);
        ASSERT_EQUAL(13, v4[4]);
        ASSERT_EQUAL(13, v4[5]);

        ASSERT_EQUAL(1, v4[6]);
        ASSERT_EQUAL(2, v4[7]);
        ASSERT_EQUAL(3, v4[8]);
        ASSERT_EQUAL(4, v4[9]);

        ASSERT_EQUAL(10, v4.size());
    }
}; // end TestVectorFillInsertSimple
VectorUnitTest<TestVectorFillInsertSimple, NumericTypes, thrust::device_vector, thrust::device_malloc_allocator> TestVectorFillInsertSimpleDeviceInstance;
VectorUnitTest<TestVectorFillInsertSimple, NumericTypes, thrust::host_vector,   std::allocator>                  TestVectorFillInsertSimpleHostInstance;

template <class T>
struct TestVectorFillInsert
{
    void operator()(size_t n)
    {
        thrust::host_vector<T>   h_dst = unittest::random_samples<T>(n + 2);

        thrust::device_vector<T> d_dst = h_dst;

        // choose insertion position at random
        size_t position = n > 0 ? (size_t)h_dst[n] % n : 0;

        // choose insertion size at random
        size_t insertion_size = n > 0 ? (size_t)h_dst[n] % n : 13;

        // insert on host
        h_dst.insert(h_dst.begin() + position,
                     insertion_size,
                     13);

        // insert on device
        d_dst.insert(d_dst.begin() + position,
                     insertion_size,
                     13);

        ASSERT_EQUAL(h_dst, d_dst);
    }
}; // end TestVectorFillInsert
VariableUnitTest<TestVectorFillInsert, IntegralTypes> TestVectorFillInsertInstance;

