#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/set_operations.h>
#include <thrust/functional.h>
#include <thrust/sort.h>


template<typename InputIterator1,
         typename InputIterator2,
         typename OutputIterator>
OutputIterator set_symmetric_difference(my_system &system,
                                        InputIterator1,
                                        InputIterator1,
                                        InputIterator2,
                                        InputIterator2,
                                        OutputIterator result)
{
  system.validate_dispatch();
  return result;
}

void TestSetSymmetricDifferenceDispatchExplicit()
{
  thrust::device_vector<int> vec(1);

  my_system sys(0);
  thrust::set_symmetric_difference(sys,
                                   vec.begin(),
                                   vec.begin(),
                                   vec.begin(),
                                   vec.begin(),
                                   vec.begin());

  ASSERT_EQUAL(true, sys.is_valid());
}
DECLARE_UNITTEST(TestSetSymmetricDifferenceDispatchExplicit);


template<typename InputIterator1,
         typename InputIterator2,
         typename OutputIterator>
OutputIterator set_symmetric_difference(my_tag,
                                        InputIterator1,
                                        InputIterator1,
                                        InputIterator2,
                                        InputIterator2,
                                        OutputIterator result)
{
  *result = 13;
  return result;
}

void TestSetSymmetricDifferenceDispatchImplicit()
{
  thrust::device_vector<int> vec(1);

  thrust::set_symmetric_difference(thrust::retag<my_tag>(vec.begin()),
                                   thrust::retag<my_tag>(vec.begin()),
                                   thrust::retag<my_tag>(vec.begin()),
                                   thrust::retag<my_tag>(vec.begin()),
                                   thrust::retag<my_tag>(vec.begin()));

  ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestSetSymmetricDifferenceDispatchImplicit);


template<typename Vector>
void TestSetSymmetricDifferenceSimple(void)
{
  typedef typename Vector::iterator Iterator;

  Vector a(4), b(5);

  a[0] = 0; a[1] = 2; a[2] = 4; a[3] = 6;
  b[0] = 0; b[1] = 3; b[2] = 3; b[3] = 4; b[4] = 7;

  Vector ref(5);
  ref[0] = 2; ref[1] = 3; ref[2] = 3; ref[3] = 6; ref[4] = 7;

  Vector result(5);

  Iterator end = thrust::set_symmetric_difference(a.begin(), a.end(),
                                                  b.begin(), b.end(),
                                                  result.begin());

  ASSERT_EQUAL_QUIET(result.end(), end);
  ASSERT_EQUAL(ref, result);
}
DECLARE_VECTOR_UNITTEST(TestSetSymmetricDifferenceSimple);


template<typename T>
void TestSetSymmetricDifference(const size_t n)
{
  size_t sizes[]   = {0, 1, n / 2, n, n + 1, 2 * n};
  size_t num_sizes = sizeof(sizes) / sizeof(size_t);

  thrust::host_vector<T> random = unittest::random_integers<unittest::int8_t>(n + *thrust::max_element(sizes, sizes + num_sizes));

  thrust::host_vector<T> h_a(random.begin(), random.begin() + n);
  thrust::host_vector<T> h_b(random.begin() + n, random.end());

  thrust::stable_sort(h_a.begin(), h_a.end());
  thrust::stable_sort(h_b.begin(), h_b.end());
  
  thrust::device_vector<T> d_a = h_a;
  thrust::device_vector<T> d_b = h_b;

  for (size_t i = 0; i < num_sizes; i++)
  {
      size_t size = sizes[i];
      
      thrust::host_vector<T>   h_result(n + size);
      thrust::device_vector<T> d_result(n + size);

      typename thrust::host_vector<T>::iterator   h_end;
      typename thrust::device_vector<T>::iterator d_end;
      
      h_end = thrust::set_symmetric_difference(h_a.begin(), h_a.end(),
                                               h_b.begin(), h_b.begin() + size,
                                               h_result.begin());
      h_result.resize(h_end - h_result.begin());

      d_end = thrust::set_symmetric_difference(d_a.begin(), d_a.end(),
                                               d_b.begin(), d_b.begin() + size,
                                               d_result.begin());
      d_result.resize(d_end - d_result.begin());

      ASSERT_EQUAL(h_result, d_result);
  }
}
DECLARE_VARIABLE_UNITTEST(TestSetSymmetricDifference);


template<typename T>
void TestSetSymmetricDifferenceEquivalentRanges(const size_t n)
{
  thrust::host_vector<T> temp = unittest::random_integers<T>(n);
  thrust::host_vector<T> h_a = temp; thrust::sort(h_a.begin(), h_a.end());
  thrust::host_vector<T> h_b = h_a;

  thrust::device_vector<T> d_a = h_a;
  thrust::device_vector<T> d_b = h_b;

  thrust::host_vector<T>   h_result(h_a.size() + h_b.size());
  thrust::device_vector<T> d_result(h_result.size());

  typename thrust::host_vector<T>::iterator   h_end;
  typename thrust::device_vector<T>::iterator d_end;
  
  h_end = thrust::set_symmetric_difference(h_a.begin(), h_a.end(),
                                           h_b.begin(), h_b.end(),
                                           h_result.begin());
  h_result.erase(h_end, h_result.end());

  d_end = thrust::set_symmetric_difference(d_a.begin(), d_a.end(),
                                           d_b.begin(), d_b.end(),
                                           d_result.begin());
  d_result.erase(d_end, d_result.end());

  ASSERT_EQUAL(h_result, d_result);
}
DECLARE_VARIABLE_UNITTEST(TestSetSymmetricDifferenceEquivalentRanges);


template<typename T>
void TestSetSymmetricDifferenceMultiset(const size_t n)
{
  thrust::host_vector<T> temp = unittest::random_integers<T>(2 * n);

  // restrict elements to [min,13)
  for(typename thrust::host_vector<T>::iterator i = temp.begin();
      i != temp.end();
      ++i)
  {
    int temp = static_cast<int>(*i);
    temp %= 13;
    *i = temp;
  }

  thrust::host_vector<T> h_a(temp.begin(), temp.begin() + n);
  thrust::host_vector<T> h_b(temp.begin() + n, temp.end());

  thrust::sort(h_a.begin(), h_a.end());
  thrust::sort(h_b.begin(), h_b.end());

  thrust::device_vector<T> d_a = h_a;
  thrust::device_vector<T> d_b = h_b;

  thrust::host_vector<T> h_result(h_a.size() + h_b.size());
  thrust::device_vector<T> d_result(h_result.size());

  typename thrust::host_vector<T>::iterator h_end;
  typename thrust::device_vector<T>::iterator d_end;
  
  h_end = thrust::set_difference(h_a.begin(), h_a.end(),
                                 h_b.begin(), h_b.end(),
                                 h_result.begin());
  h_result.erase(h_end, h_result.end());

  d_end = thrust::set_difference(d_a.begin(), d_a.end(),
                                 d_b.begin(), d_b.end(),
                                 d_result.begin());
  d_result.erase(d_end, d_result.end());

  ASSERT_EQUAL(h_result, d_result);
}
DECLARE_VARIABLE_UNITTEST(TestSetSymmetricDifferenceMultiset);


template<typename U>
  void TestSetSymmetricDifferenceKeyValue(size_t n)
{
  typedef key_value<U,U> T;

  thrust::host_vector<U> h_keys_a   = unittest::random_integers<U>(n);
  thrust::host_vector<U> h_values_a = unittest::random_integers<U>(n);

  thrust::host_vector<U> h_keys_b   = unittest::random_integers<U>(n);
  thrust::host_vector<U> h_values_b = unittest::random_integers<U>(n);

  thrust::host_vector<T> h_a(n), h_b(n);
  for(size_t i = 0; i < n; ++i)
  {
    h_a[i] = T(h_keys_a[i], h_values_a[i]);
    h_b[i] = T(h_keys_b[i], h_values_b[i]);
  }

  thrust::stable_sort(h_a.begin(), h_a.end());
  thrust::stable_sort(h_b.begin(), h_b.end());

  thrust::device_vector<T> d_a = h_a;
  thrust::device_vector<T> d_b = h_b;

  thrust::host_vector<T> h_result(h_a.size() + h_b.size());
  thrust::device_vector<T> d_result(h_result.size());

  typename thrust::host_vector<T>::iterator h_end;
  typename thrust::device_vector<T>::iterator d_end;
  
  h_end = thrust::set_symmetric_difference(h_a.begin(), h_a.end(),
                                           h_b.begin(), h_b.end(),
                                           h_result.begin());
  h_result.erase(h_end, h_result.begin());

  d_end = thrust::set_symmetric_difference(d_a.begin(), d_a.end(),
                                           d_b.begin(), d_b.end(),
                                           d_result.begin());

  d_result.erase(d_end, d_result.begin());

  ASSERT_EQUAL_QUIET(h_result, d_result);
}
DECLARE_VARIABLE_UNITTEST(TestSetSymmetricDifferenceKeyValue);

