#include "hip/hip_runtime.h"
#include <thrust/detail/config.h>

#if THRUST_CPP_DIALECT >= 2014

#include <unittest/unittest.h>
#include <unittest/util_async.h>

#include <thrust/limits.h>
#include <thrust/async/copy.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#define DEFINE_ASYNC_COPY_CALLABLE(name, ...)                                 \
  struct THRUST_PP_CAT2(name, _fn)                                            \
  {                                                                           \
    template <typename ForwardIt, typename Sentinel, typename OutputIt>       \
    __host__                                                                  \
    auto operator()(                                                          \
      ForwardIt&& first, Sentinel&& last, OutputIt&& output                   \
    ) const                                                                   \
    THRUST_RETURNS(                                                           \
      ::thrust::async::copy(                                                  \
        __VA_ARGS__                                                           \
        THRUST_PP_COMMA_IF(THRUST_PP_ARITY(__VA_ARGS__))                      \
        THRUST_FWD(first), THRUST_FWD(last), THRUST_FWD(output)               \
      )                                                                       \
    )                                                                         \
  };                                                                          \
  /**/

DEFINE_ASYNC_COPY_CALLABLE(
  invoke_async_copy
);

DEFINE_ASYNC_COPY_CALLABLE(
  invoke_async_copy_host,   thrust::host
);
DEFINE_ASYNC_COPY_CALLABLE(
  invoke_async_copy_device, thrust::device
);

DEFINE_ASYNC_COPY_CALLABLE(
  invoke_async_copy_host_to_device,    thrust::host,   thrust::device
);
DEFINE_ASYNC_COPY_CALLABLE(
  invoke_async_copy_device_to_host,    thrust::device, thrust::host
);
DEFINE_ASYNC_COPY_CALLABLE(
  invoke_async_copy_host_to_host,      thrust::host,   thrust::host
);
DEFINE_ASYNC_COPY_CALLABLE(
  invoke_async_copy_device_to_device,  thrust::device, thrust::device
);

#undef DEFINE_ASYNC_COPY_CALLABLE

///////////////////////////////////////////////////////////////////////////////

template <typename AsyncCopyCallable>
struct test_async_copy_host_to_device
{
  template <typename T>
  struct tester
  {
    __host__
    void operator()(std::size_t n)
    {
      thrust::host_vector<T>   h0(unittest::random_integers<T>(n));
      thrust::device_vector<T> d0(n);

      auto f0 = AsyncCopyCallable{}(
        h0.begin(), h0.end(), d0.begin()
      );

      f0.wait();

      ASSERT_EQUAL(h0, d0);
    }
  };
};
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  test_async_copy_host_to_device<invoke_async_copy_fn>::tester
, BuiltinNumericTypes
, test_async_copy_trivially_relocatable_elements_host_to_device
);
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  test_async_copy_host_to_device<invoke_async_copy_host_to_device_fn>::tester
, BuiltinNumericTypes
, test_async_copy_trivially_relocatable_elements_host_to_device_policies
);

///////////////////////////////////////////////////////////////////////////////

template <typename AsyncCopyCallable>
struct test_async_copy_device_to_host
{
  template <typename T>
  struct tester
  {
    __host__
    void operator()(std::size_t n)
    {
      thrust::host_vector<T>   h0(unittest::random_integers<T>(n));
      thrust::host_vector<T>   h1(n);
      thrust::device_vector<T> d0(n);

      thrust::copy(h0.begin(), h0.end(), d0.begin());

      ASSERT_EQUAL(h0, d0);

      auto f0 = AsyncCopyCallable{}(
        d0.begin(), d0.end(), h1.begin()
      );

      f0.wait();

      ASSERT_EQUAL(h0, d0);
      ASSERT_EQUAL(d0, h1);
    }
  };
};
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  test_async_copy_device_to_host<invoke_async_copy_fn>::tester
, BuiltinNumericTypes
, test_async_copy_trivially_relocatable_elements_device_to_host
);
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  test_async_copy_device_to_host<invoke_async_copy_device_to_host_fn>::tester
, BuiltinNumericTypes
, test_async_copy_trivially_relocatable_elements_device_to_host_policies
);

///////////////////////////////////////////////////////////////////////////////

template <typename AsyncCopyCallable>
struct test_async_copy_device_to_device
{
  template <typename T>
  struct tester
  {
    __host__
    void operator()(std::size_t n)
    {
      thrust::host_vector<T>   h0(unittest::random_integers<T>(n));
      thrust::device_vector<T> d0(n);
      thrust::device_vector<T> d1(n);

      thrust::copy(h0.begin(), h0.end(), d0.begin());

      ASSERT_EQUAL(h0, d0);

      auto f0 = AsyncCopyCallable{}(
        d0.begin(), d0.end(), d1.begin()
      );

      f0.wait();

      ASSERT_EQUAL(h0, d0);
      ASSERT_EQUAL(d0, d1);
    }
  };
};
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  test_async_copy_device_to_device<invoke_async_copy_fn>::tester
, NumericTypes
, test_async_copy_device_to_device
);
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  test_async_copy_device_to_device<invoke_async_copy_device_fn>::tester
, NumericTypes
, test_async_copy_device_to_device_policy
);
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  test_async_copy_device_to_device<invoke_async_copy_device_to_device_fn>::tester
, NumericTypes
, test_async_copy_device_to_device_policies
);

///////////////////////////////////////////////////////////////////////////////

// Non ContiguousIterator input.
template <typename AsyncCopyCallable>
struct test_async_copy_counting_iterator_input_to_device_vector
{
  template <typename T>
  struct tester
  {
    __host__
    void operator()(std::size_t n)
    {
      thrust::counting_iterator<T> first(0);
      thrust::counting_iterator<T> last(
        unittest::truncate_to_max_representable<T>(n)
      );

      thrust::device_vector<T> d0(n);
      thrust::device_vector<T> d1(n);

      thrust::copy(first, last, d0.begin());

      auto f0 = AsyncCopyCallable{}(
        first, last, d1.begin()
      );

      f0.wait();

      ASSERT_EQUAL(d0, d1);
    }
  };
};
// TODO: Re-add custom_numeric when it supports counting iterators.
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  test_async_copy_counting_iterator_input_to_device_vector<
    invoke_async_copy_fn
  >::tester
, BuiltinNumericTypes
, test_async_copy_counting_iterator_input_trivially_relocatable_elements_device_to_device
);
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  test_async_copy_counting_iterator_input_to_device_vector<
    invoke_async_copy_device_fn
  >::tester
, BuiltinNumericTypes
, test_async_copy_counting_iterator_input_trivially_relocatable_elements_device_to_device_policy
);
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  test_async_copy_counting_iterator_input_to_device_vector<
    invoke_async_copy_device_to_device_fn
  >::tester
, BuiltinNumericTypes
, test_async_copy_counting_iterator_input_trivially_relocatable_elements_device_to_device_policies
);
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  test_async_copy_counting_iterator_input_to_device_vector<
    invoke_async_copy_host_to_device_fn
  >::tester
, BuiltinNumericTypes
, test_async_copy_counting_iterator_input_host_to_device_policies
);

///////////////////////////////////////////////////////////////////////////////

// Non ContiguousIterator input.
template <typename AsyncCopyCallable>
struct test_async_copy_counting_iterator_input_to_host_vector
{
  template <typename T>
  struct tester
  {
    __host__
    void operator()(std::size_t n)
    {
      thrust::counting_iterator<T> first(0);
      thrust::counting_iterator<T> last(
        unittest::truncate_to_max_representable<T>(n)
      );

      thrust::host_vector<T> d0(n);
      thrust::host_vector<T> d1(n);

      thrust::copy(first, last, d0.begin());

      auto f0 = AsyncCopyCallable{}(
        first, last, d1.begin()
      );

      f0.wait();

      ASSERT_EQUAL(d0, d1);
    }
  };
};
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  test_async_copy_counting_iterator_input_to_host_vector<
    invoke_async_copy_fn
  >::tester
, BuiltinNumericTypes
, test_async_copy_counting_iterator_input_trivially_relocatable_elements_device_to_host
);
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  test_async_copy_counting_iterator_input_to_host_vector<
    invoke_async_copy_device_to_host_fn
  >::tester
, BuiltinNumericTypes
, test_async_copy_counting_iterator_input_trivially_relocatable_elements_device_to_host_policies
);

///////////////////////////////////////////////////////////////////////////////

template <typename T>
struct test_async_copy_roundtrip
{
  __host__
  void operator()(std::size_t n)
  {
    thrust::host_vector<T>   h0(unittest::random_integers<T>(n));
    thrust::device_vector<T> d0(n);

    auto e0 = thrust::async::copy(
      thrust::host, thrust::device
    , h0.begin(), h0.end(), d0.begin()
    );

    auto e1 = thrust::async::copy(
      thrust::device.after(e0), thrust::host
    , d0.begin(), d0.end(), h0.begin()
    );

    TEST_EVENT_WAIT(e1);

    ASSERT_EQUAL(h0, d0);
  }
};
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES_AND_NAME(
  test_async_copy_roundtrip
, BuiltinNumericTypes
, test_async_copy_trivially_relocatable_elements_roundtrip
);

///////////////////////////////////////////////////////////////////////////////

template <typename T>
struct test_async_copy_after
{
  __host__
  void operator()(std::size_t n)
  {
    thrust::host_vector<T>   h0(unittest::random_integers<T>(n));
    thrust::host_vector<T>   h1(n);
    thrust::device_vector<T> d0(n);
    thrust::device_vector<T> d1(n);
    thrust::device_vector<T> d2(n);

    auto e0 = thrust::async::copy(
      h0.begin(), h0.end(), d0.begin()
    );

    ASSERT_EQUAL(true, e0.valid_stream());

    auto const e0_stream = e0.stream().native_handle();

    auto e1 = thrust::async::copy(
      thrust::device.after(e0), d0.begin(), d0.end(), d1.begin()
    );

    // Verify that double consumption of a future produces an exception.
    ASSERT_THROWS_EQUAL(
      auto x = thrust::async::copy(
        thrust::device.after(e0), d0.begin(), d0.end(), d1.begin()
      );
      THRUST_UNUSED_VAR(x)
    , thrust::event_error
    , thrust::event_error(thrust::event_errc::no_state)
    );

    ASSERT_EQUAL_QUIET(e0_stream, e1.stream().native_handle());

    auto after_policy2 = thrust::device.after(e1);

    auto e2 = thrust::async::copy(
      thrust::host, after_policy2
    , h0.begin(), h0.end(), d2.begin()
    );

    // Verify that double consumption of a policy produces an exception.
    ASSERT_THROWS_EQUAL(
      auto x = thrust::async::copy(
        thrust::host, after_policy2
      , h0.begin(), h0.end(), d2.begin()
      );
      THRUST_UNUSED_VAR(x)
    , thrust::event_error
    , thrust::event_error(thrust::event_errc::no_state)
    );

    ASSERT_EQUAL_QUIET(e0_stream, e2.stream().native_handle());

    auto e3 = thrust::async::copy(
      thrust::device.after(e2), thrust::host
    , d1.begin(), d1.end(), h1.begin()
    );

    ASSERT_EQUAL_QUIET(e0_stream, e3.stream().native_handle());

    TEST_EVENT_WAIT(e3);

    ASSERT_EQUAL(h0, h1);
    ASSERT_EQUAL(h0, d0);
    ASSERT_EQUAL(h0, d1);
    ASSERT_EQUAL(h0, d2);
  }
};
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES(
  test_async_copy_after
, BuiltinNumericTypes
);

///////////////////////////////////////////////////////////////////////////////

// TODO: device_to_device NonContiguousIterator output (discard_iterator).

// TODO: host_to_device non trivially relocatable.

// TODO: device_to_host non trivially relocatable.

// TODO: host_to_device NonContiguousIterator input (counting_iterator).

// TODO: host_to_device NonContiguousIterator output (discard_iterator).

// TODO: device_to_host NonContiguousIterator input (counting_iterator).

// TODO: device_to_host NonContiguousIterator output (discard_iterator).

// TODO: Mixed types, needs loosening of `is_trivially_relocatable_to` logic.

// TODO: H->D copy, then dependent D->H copy (round trip).
// Can't do this today because we can't do cross-system with explicit policies.

#endif

