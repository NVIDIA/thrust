#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/transform_reduce.h>
#include <thrust/execution_policy.h>


template<typename Iterator1, typename Function1, typename T, typename Function2, typename Iterator2>
__global__
void transform_reduce_kernel(Iterator1 first, Iterator1 last, Function1 f1, T init, Function2 f2, Iterator2 result)
{
  *result = thrust::transform_reduce(thrust::seq, first, last, f1, init, f2);
}


void TestTransformReduceDeviceSeq()
{
  typedef thrust::device_vector<int> Vector;
  typedef typename Vector::value_type T;
  
  Vector data(3);
  data[0] = 1; data[1] = -2; data[2] = 3;
  
  T init = 10;

  thrust::device_vector<T> result(1);

  transform_reduce_kernel<<<1,1>>>(data.begin(), data.end(), thrust::negate<T>(), init, thrust::plus<T>(), result.begin());
  
  ASSERT_EQUAL(8, (T)result[0]);
}
DECLARE_UNITTEST(TestTransformReduceDeviceSeq);

