#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/partition.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>


template<typename Iterator1, typename Predicate, typename Iterator2>
__global__
void partition_point_kernel(Iterator1 first, Iterator1 last, Predicate pred, Iterator2 result)
{
  *result = thrust::partition_point(thrust::seq, first, last, pred);
}


template<typename T>
struct is_even
{
  __host__ __device__
  bool operator()(T x) const { return ((int) x % 2) == 0; }
};


template<typename T>
void TestPartitionPointDeviceSeq(size_t n)
{
  thrust::device_vector<T> v = unittest::random_integers<T>(n);
  typedef typename thrust::device_vector<T>::iterator iterator;

  iterator ref = thrust::stable_partition(v.begin(), v.end(), is_even<T>());

  thrust::device_vector<iterator> result(1);
  partition_point_kernel<<<1,1>>>(v.begin(), v.end(), is_even<T>(), result.begin());

  ASSERT_EQUAL(ref - v.begin(), (iterator)result[0] - v.begin());
}
DECLARE_VARIABLE_UNITTEST(TestPartitionPointDeviceSeq);

