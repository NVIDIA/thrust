#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/set_operations.h>
#include <thrust/execution_policy.h>


template<typename Iterator1, typename Iterator2, typename Iterator3, typename Iterator4>
__global__
void set_symmetric_difference_kernel(Iterator1 first1, Iterator1 last1,
                                     Iterator2 first2, Iterator2 last2,
                                     Iterator3 result1,
                                     Iterator4 result2)
{
  *result2 = thrust::set_symmetric_difference(thrust::seq, first1, last1, first2, last2, result1);
}


void TestSetSymmetricDifferenceDeviceSeq()
{
  typedef thrust::device_vector<int> Vector;
  typedef typename Vector::iterator Iterator;

  Vector a(4), b(5);

  a[0] = 0; a[1] = 2; a[2] = 4; a[3] = 6;
  b[0] = 0; b[1] = 3; b[2] = 3; b[3] = 4; b[4] = 7;

  Vector ref(5);
  ref[0] = 2; ref[1] = 3; ref[2] = 3; ref[3] = 6; ref[4] = 7;

  Vector result(5);
  thrust::device_vector<Iterator> end_vec(1);

  set_symmetric_difference_kernel<<<1,1>>>(a.begin(), a.end(),
                                           b.begin(), b.end(),
                                           result.begin(),
                                           end_vec.begin());
  Iterator end = end_vec[0];

  ASSERT_EQUAL_QUIET(result.end(), end);
  ASSERT_EQUAL(ref, result);
}
DECLARE_UNITTEST(TestSetSymmetricDifferenceDeviceSeq);

