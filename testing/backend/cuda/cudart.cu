#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <hip/hip_runtime_api.h>
#include <thrust/detail/util/align.h>

void TestCudaMemcpyD2DNullPointerError(void)
{
  hipError_t result1 = hipMemcpy((void*)0, (void*)0, 1, hipMemcpyDeviceToDevice);
  hipError_t result2 = hipGetLastError();

  ASSERT_EQUAL(hipErrorInvalidValue, result1);
  ASSERT_EQUAL(hipErrorInvalidValue, result2);
}
DECLARE_UNITTEST(TestCudaMemcpyD2DNullPointerError);

template<typename T>
void TestCudaMallocResultAligned(const std::size_t n)
{
  T *ptr = 0;
  hipMalloc(&ptr, n * sizeof(T));
  hipFree(ptr);

  ASSERT_EQUAL(true, thrust::detail::util::is_aligned(ptr));
}
DECLARE_VARIABLE_UNITTEST(TestCudaMallocResultAligned);

__global__ void segfault(int *dst, int *src)
{
  *dst = *src;
}

void TestNullPtrDereferenceYieldsError(void)
{
  segfault<<<1,1>>>((int*)0, (int*)0);
  hipError_t result = hipDeviceSynchronize();

  // kill the context so it can revive later
  hipDeviceReset();

  ASSERT_EQUAL(true, (result != hipSuccess));
}
DECLARE_UNITTEST(TestNullPtrDereferenceYieldsError);

