#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>


template<typename Iterator, typename T, typename Iterator2>
__global__
void reduce_kernel(Iterator first, Iterator last, T init, Iterator2 result)
{
  *result = thrust::reduce(thrust::seq, first, last, init);
}


template<typename T>
struct TestReduceDeviceSeq
{
  void operator()(const size_t n)
  {
    thrust::host_vector<T>   h_data = unittest::random_integers<T>(n);
    thrust::device_vector<T> d_data = h_data;

    thrust::device_vector<T> d_result(1);
    
    T init = 13;
    
    T h_result = thrust::reduce(h_data.begin(), h_data.end(), init);

    reduce_kernel<<<1,1>>>(d_data.begin(), d_data.end(), init, d_result.begin());
    
    ASSERT_EQUAL(h_result, d_result[0]);
  }
};
VariableUnitTest<TestReduceDeviceSeq, IntegralTypes> TestReduceDeviceSeqInstance;

