#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/for_each.h>
#include <thrust/execution_policy.h>
#include <algorithm>

static const size_t NUM_REGISTERS = 64;

template <size_t N> __host__ __device__ void f   (int * x) { int temp = *x; f<N - 1>(x + 1); *x = temp;};
template <>         __host__ __device__ void f<0>(int *)   { }
template <size_t N>
struct CopyFunctorWithManyRegisters
{
  __host__ __device__
  void operator()(int * ptr)
  {
      f<N>(ptr);
  }
};


void TestForEachLargeRegisterFootprint()
{
  int current_device = -1;
  hipGetDevice(&current_device);
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, current_device);

  thrust::device_vector<int> data(NUM_REGISTERS, 12345);

  thrust::device_vector<int *> input(1, thrust::raw_pointer_cast(&data[0])); // length is irrelevant
  
  thrust::for_each(input.begin(), input.end(), CopyFunctorWithManyRegisters<NUM_REGISTERS>());
}
DECLARE_UNITTEST(TestForEachLargeRegisterFootprint);


void TestForEachNLargeRegisterFootprint()
{
  int current_device = -1;
  hipGetDevice(&current_device);
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, current_device);

  thrust::device_vector<int> data(NUM_REGISTERS, 12345);

  thrust::device_vector<int *> input(1, thrust::raw_pointer_cast(&data[0])); // length is irrelevant
  
  thrust::for_each_n(input.begin(), input.size(), CopyFunctorWithManyRegisters<NUM_REGISTERS>());
}
DECLARE_UNITTEST(TestForEachNLargeRegisterFootprint);


template <typename T>
struct mark_present_for_each
{
  T * ptr;
  __host__ __device__ void
  operator()(T x){ ptr[(int) x] = 1; }
};


template<typename ExecutionPolicy, typename Iterator, typename Function>
__global__ void for_each_kernel(ExecutionPolicy exec, Iterator first, Iterator last, Function f)
{
  thrust::for_each(exec, first, last, f);
}


template<typename T>
void TestForEachDeviceSeq(const size_t n)
{
  const size_t output_size = std::min((size_t) 10, 2 * n);
  
  thrust::host_vector<T> h_input = unittest::random_integers<T>(n);
  
  for(size_t i = 0; i < n; i++)
    h_input[i] =  ((size_t) h_input[i]) % output_size;
  
  thrust::device_vector<T> d_input = h_input;
  
  thrust::host_vector<T>   h_output(output_size, (T) 0);
  thrust::device_vector<T> d_output(output_size, (T) 0);
  
  mark_present_for_each<T> h_f;
  mark_present_for_each<T> d_f;
  h_f.ptr = &h_output[0];
  d_f.ptr = (&d_output[0]).get();
  
  thrust::for_each(h_input.begin(), h_input.end(), h_f);
  
  for_each_kernel<<<1,1>>>(thrust::seq, d_input.begin(), d_input.end(), d_f);
  
  ASSERT_EQUAL(h_output, d_output);
}
DECLARE_VARIABLE_UNITTEST(TestForEachDeviceSeq);


template<typename T>
void TestForEachDeviceDevice(const size_t n)
{
  const size_t output_size = std::min((size_t) 10, 2 * n);
  
  thrust::host_vector<T> h_input = unittest::random_integers<T>(n);
  
  for(size_t i = 0; i < n; i++)
    h_input[i] =  ((size_t) h_input[i]) % output_size;
  
  thrust::device_vector<T> d_input = h_input;
  
  thrust::host_vector<T>   h_output(output_size, (T) 0);
  thrust::device_vector<T> d_output(output_size, (T) 0);
  
  mark_present_for_each<T> h_f;
  mark_present_for_each<T> d_f;
  h_f.ptr = &h_output[0];
  d_f.ptr = (&d_output[0]).get();
  
  thrust::for_each(h_input.begin(), h_input.end(), h_f);
  
  for_each_kernel<<<1,1>>>(thrust::device, d_input.begin(), d_input.end(), d_f);
  
  ASSERT_EQUAL(h_output, d_output);
}
DECLARE_VARIABLE_UNITTEST(TestForEachDeviceDevice);


template<typename ExecutionPolicy, typename Iterator, typename Size, typename Function>
__global__
void for_each_n_kernel(ExecutionPolicy exec, Iterator first, Size n, Function f)
{
  thrust::for_each_n(exec, first, n, f);
}


template<typename T>
void TestForEachNDeviceSeq(const size_t n)
{
  const size_t output_size = std::min((size_t) 10, 2 * n);
  
  thrust::host_vector<T> h_input = unittest::random_integers<T>(n);
  
  for(size_t i = 0; i < n; i++)
    h_input[i] =  ((size_t) h_input[i]) % output_size;
  
  thrust::device_vector<T> d_input = h_input;
  
  thrust::host_vector<T>   h_output(output_size, (T) 0);
  thrust::device_vector<T> d_output(output_size, (T) 0);
  
  mark_present_for_each<T> h_f;
  mark_present_for_each<T> d_f;
  h_f.ptr = &h_output[0];
  d_f.ptr = (&d_output[0]).get();
  
  thrust::for_each_n(h_input.begin(), h_input.size(), h_f);
  
  for_each_n_kernel<<<1,1>>>(thrust::seq, d_input.begin(), d_input.size(), d_f);
  
  ASSERT_EQUAL(h_output, d_output);
}
DECLARE_VARIABLE_UNITTEST(TestForEachNDeviceSeq);


template<typename T>
void TestForEachNDeviceDevice(const size_t n)
{
  const size_t output_size = std::min((size_t) 10, 2 * n);
  
  thrust::host_vector<T> h_input = unittest::random_integers<T>(n);
  
  for(size_t i = 0; i < n; i++)
    h_input[i] =  ((size_t) h_input[i]) % output_size;
  
  thrust::device_vector<T> d_input = h_input;
  
  thrust::host_vector<T>   h_output(output_size, (T) 0);
  thrust::device_vector<T> d_output(output_size, (T) 0);
  
  mark_present_for_each<T> h_f;
  mark_present_for_each<T> d_f;
  h_f.ptr = &h_output[0];
  d_f.ptr = (&d_output[0]).get();
  
  thrust::for_each_n(h_input.begin(), h_input.size(), h_f);
  
  for_each_n_kernel<<<1,1>>>(thrust::device, d_input.begin(), d_input.size(), d_f);
  
  ASSERT_EQUAL(h_output, d_output);
}
DECLARE_VARIABLE_UNITTEST(TestForEachNDeviceDevice);


void TestForEachCudaStreams()
{
  hipStream_t s;
  hipStreamCreate(&s);
  
  thrust::device_vector<int> input(5);
  thrust::device_vector<int> output(7, 0);
  
  input[0] = 3; input[1] = 2; input[2] = 3; input[3] = 4; input[4] = 6;
  
  mark_present_for_each<int> f;
  f.ptr = thrust::raw_pointer_cast(output.data());
  
  thrust::for_each(thrust::cuda::par.on(s), input.begin(), input.end(), f);

  hipStreamSynchronize(s);
  
  ASSERT_EQUAL(output[0], 0);
  ASSERT_EQUAL(output[1], 0);
  ASSERT_EQUAL(output[2], 1);
  ASSERT_EQUAL(output[3], 1);
  ASSERT_EQUAL(output[4], 1);
  ASSERT_EQUAL(output[5], 0);
  ASSERT_EQUAL(output[6], 1);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestForEachCudaStreams);

