#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/set_operations.h>
#include <thrust/execution_policy.h>


template<typename Iterator1, typename Iterator2, typename Iterator3, typename Iterator4>
__global__
void set_difference_kernel(Iterator1 first1, Iterator1 last1, Iterator2 first2, Iterator2 last2, Iterator3 result1, Iterator4 result2)
{
  *result2 = thrust::set_difference(thrust::seq, first1, last1, first2, last2, result1);
}


void TestSetDifferenceDeviceSeq()
{
  typedef thrust::device_vector<int> Vector;
  typedef typename Vector::iterator Iterator;

  Vector a(4), b(5);

  a[0] = 0; a[1] = 2; a[2] = 4; a[3] = 5;
  b[0] = 0; b[1] = 3; b[2] = 3; b[3] = 4; b[4] = 6;

  Vector ref(2);
  ref[0] = 2; ref[1] = 5;

  Vector result(2);

  thrust::device_vector<Iterator> end_vec(1);

  set_difference_kernel<<<1,1>>>(a.begin(), a.end(), b.begin(), b.end(), result.begin(), end_vec.begin());

  Iterator end = end_vec.front();

  ASSERT_EQUAL_QUIET(result.end(), end);
  ASSERT_EQUAL(ref, result);
}
DECLARE_UNITTEST(TestSetDifferenceDeviceSeq);

