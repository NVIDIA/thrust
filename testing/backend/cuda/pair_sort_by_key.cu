#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/pair.h>
#include <thrust/sort.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>
#include <thrust/execution_policy.h>


template<typename Iterator1, typename Iterator2, typename Iterator3>
__global__
void stable_sort_by_key_kernel(Iterator1 keys_first, Iterator1 keys_last, Iterator2 values_first, Iterator3 is_supported)
{
#if (__CUDA_ARCH__ >= 200)
  *is_supported = true;
  thrust::stable_sort_by_key(thrust::seq, keys_first, keys_last, values_first);
#else
  *is_supported = false;
#endif
}


struct make_pair_functor
{
  template<typename T1, typename T2>
  __host__ __device__
    thrust::pair<T1,T2> operator()(const T1 &x, const T2 &y)
  {
    return thrust::make_pair(x,y);
  } // end operator()()
}; // end make_pair_functor


template <typename T>
  struct TestPairStableSortByKeyDeviceSeq
{
  void operator()(const size_t n)
  {
    typedef thrust::pair<T,T> P;

    // host arrays
    thrust::host_vector<T>   h_p1 = unittest::random_integers<T>(n);
    thrust::host_vector<T>   h_p2 = unittest::random_integers<T>(n);
    thrust::host_vector<P>   h_pairs(n);

    thrust::host_vector<int> h_values(n);
    thrust::sequence(h_values.begin(), h_values.end());

    // zip up pairs on the host
    thrust::transform(h_p1.begin(), h_p1.end(), h_p2.begin(), h_pairs.begin(), make_pair_functor());

    // device arrays
    thrust::device_vector<P>   d_pairs = h_pairs;
    thrust::device_vector<int> d_values = h_values;

    thrust::device_vector<bool> is_supported(1);

    // sort on the device
    stable_sort_by_key_kernel<<<1,1>>>(d_pairs.begin(), d_pairs.end(), d_values.begin(), is_supported.begin());

    if(is_supported[0])
    {
      // sort on the host
      thrust::stable_sort_by_key(h_pairs.begin(), h_pairs.end(), h_values.begin());

      ASSERT_EQUAL_QUIET(h_pairs,  d_pairs);
      ASSERT_EQUAL(h_values, d_values);
    }
  }
};
VariableUnitTest<TestPairStableSortByKeyDeviceSeq, unittest::type_list<unittest::int8_t,unittest::int16_t,unittest::int32_t> > TestPairStableSortByKeyDeviceSeqInstance;

