#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/merge.h>
#include <thrust/functional.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>


template<typename Iterator1,
         typename Iterator2,
         typename Iterator3,
         typename Iterator4,
         typename Iterator5,
         typename Iterator6,
         typename Iterator7>
__global__
void merge_by_key_kernel(Iterator1 keys_first1, Iterator1 keys_last1,
                         Iterator2 keys_first2, Iterator2 keys_last2,
                         Iterator3 values_first1,
                         Iterator4 values_first2,
                         Iterator5 keys_result,
                         Iterator6 values_result,
                         Iterator7 result)
{
  *result = thrust::merge_by_key(thrust::seq, keys_first1, keys_last1, keys_first2, keys_last2, values_first1, values_first2, keys_result, values_result);
}


template<typename T>
void TestMergeByKeyDeviceSeq(size_t n)
{
  thrust::host_vector<T> random_keys = unittest::random_integers<unittest::int8_t>(n);
  thrust::host_vector<T> random_vals = unittest::random_integers<unittest::int8_t>(n);

  size_t denominators[] = {1, 2, 3, 4, 5, 6, 7, 8, 9};
  size_t num_denominators = sizeof(denominators) / sizeof(size_t);

  for(size_t i = 0; i < num_denominators; ++i)
  {
    size_t size_a = n / denominators[i];

    thrust::host_vector<T> h_a_keys(random_keys.begin(), random_keys.begin() + size_a);
    thrust::host_vector<T> h_b_keys(random_keys.begin() + size_a, random_keys.end());

    thrust::host_vector<T> h_a_vals(random_vals.begin(), random_vals.begin() + size_a);
    thrust::host_vector<T> h_b_vals(random_vals.begin() + size_a, random_vals.end());

    thrust::stable_sort(h_a_keys.begin(), h_a_keys.end());
    thrust::stable_sort(h_b_keys.begin(), h_b_keys.end());

    thrust::device_vector<T> d_a_keys = h_a_keys;
    thrust::device_vector<T> d_b_keys = h_b_keys;

    thrust::device_vector<T> d_a_vals = h_a_vals;
    thrust::device_vector<T> d_b_vals = h_b_vals;

    thrust::host_vector<T> h_result_keys(n);
    thrust::host_vector<T> h_result_vals(n);

    thrust::device_vector<T> d_result_keys(n);
    thrust::device_vector<T> d_result_vals(n);

    thrust::pair<
      typename thrust::host_vector<T>::iterator,
      typename thrust::host_vector<T>::iterator
    > h_end;

    typedef thrust::pair<
      typename thrust::device_vector<T>::iterator,
      typename thrust::device_vector<T>::iterator
    > iter_pair_type;

    thrust::device_vector<iter_pair_type> d_end_vec(1);

    h_end = thrust::merge_by_key(h_a_keys.begin(), h_a_keys.end(),
                                 h_b_keys.begin(), h_b_keys.end(),
                                 h_a_vals.begin(),
                                 h_b_vals.begin(),
                                 h_result_keys.begin(),
                                 h_result_vals.begin());
    h_result_keys.erase(h_end.first, h_result_keys.end());
    h_result_vals.erase(h_end.second, h_result_vals.end());

    merge_by_key_kernel<<<1,1>>>(d_a_keys.begin(), d_a_keys.end(),
                                 d_b_keys.begin(), d_b_keys.end(),
                                 d_a_vals.begin(),
                                 d_b_vals.begin(),
                                 d_result_keys.begin(),
                                 d_result_vals.begin(),
                                 d_end_vec.begin());

    iter_pair_type d_end = d_end_vec[0];

    d_result_keys.erase(d_end.first, d_result_keys.end());
    d_result_vals.erase(d_end.second, d_result_vals.end());

    ASSERT_EQUAL(h_result_keys, d_result_keys);
    ASSERT_EQUAL(h_result_vals, d_result_vals);
  }
}
DECLARE_VARIABLE_UNITTEST(TestMergeByKeyDeviceSeq);

