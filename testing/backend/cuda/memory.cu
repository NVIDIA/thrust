#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/system/cuda/memory.h>
#include <thrust/system/cpp/memory.h>
#include <thrust/memory.h>
#include <thrust/execution_policy.h>
#include <thrust/logical.h>


template<typename T1, typename T2>
bool are_same_type(const T1 &, const T2 &)
{
  return false;
}


template<typename T>
bool are_same_type(const T &, const T &)
{
  return true;
}


void TestSelectSystemCudaToCpp()
{
  using thrust::system::detail::generic::select_system;

  thrust::cuda::tag cuda_tag;
  thrust::cpp::tag cpp_tag;
  thrust::system::cuda::detail::cross_system<thrust::cuda::tag,thrust::cpp::tag> cuda_to_cpp(cuda_tag, cpp_tag);

  // select_system(cuda::tag, thrust::host_system_tag) should return cuda_to_cpp
  bool is_cuda_to_cpp = are_same_type(cuda_to_cpp, select_system(cuda_tag, cpp_tag));
  ASSERT_EQUAL(true, is_cuda_to_cpp);
}
DECLARE_UNITTEST(TestSelectSystemCudaToCpp);


template<typename Iterator>
__global__ void get_temporary_buffer_kernel(size_t n, Iterator result)
{
  *result = thrust::get_temporary_buffer<int>(thrust::seq, n);
}


template<typename Pointer>
__global__ void return_temporary_buffer_kernel(Pointer ptr)
{
  thrust::return_temporary_buffer(thrust::seq, ptr);
}


void TestGetTemporaryBufferDeviceSeq()
{
  const size_t n = 9001;

  typedef thrust::pointer<int, thrust::detail::seq_t> pointer;
  typedef thrust::pair<pointer, std::ptrdiff_t> ptr_and_sz_type;
  thrust::device_vector<ptr_and_sz_type> d_result(1);
  
  get_temporary_buffer_kernel<<<1,1>>>(n, d_result.begin());

  ptr_and_sz_type ptr_and_sz = d_result[0];

  if(ptr_and_sz.second > 0)
  {
    ASSERT_EQUAL(ptr_and_sz.second, n);

    const int ref_val = 13;
    thrust::device_vector<int> ref(n, ref_val);

    thrust::fill_n(thrust::device, ptr_and_sz.first, n, ref_val);

    ASSERT_EQUAL(true, thrust::all_of(thrust::device, ptr_and_sz.first, ptr_and_sz.first + n, thrust::placeholders::_1 == ref_val));

    return_temporary_buffer_kernel<<<1,1>>>(ptr_and_sz.first);
  }
}
DECLARE_UNITTEST(TestGetTemporaryBufferDeviceSeq);


template<typename Iterator>
__global__ void malloc_kernel(size_t n, Iterator result)
{
  *result = static_cast<int*>(thrust::malloc(thrust::seq, sizeof(int) * n).get());
}


template<typename Pointer>
__global__ void free_kernel(Pointer ptr)
{
  thrust::free(thrust::seq, ptr);
}


void TestMallocDeviceSeq()
{
  const size_t n = 9001;

  typedef thrust::pointer<int, thrust::detail::seq_t> pointer;
  thrust::device_vector<pointer> d_result(1);
  
  malloc_kernel<<<1,1>>>(n, d_result.begin());

  pointer ptr = d_result[0];

  if(ptr.get() != 0)
  {
    const int ref_val = 13;
    thrust::device_vector<int> ref(n, ref_val);

    thrust::fill_n(thrust::device, ptr, n, ref_val);

    ASSERT_EQUAL(true, thrust::all_of(thrust::device, ptr, ptr + n, thrust::placeholders::_1 == ref_val));

    free_kernel<<<1,1>>>(ptr);
  }
}
DECLARE_UNITTEST(TestMallocDeviceSeq);

