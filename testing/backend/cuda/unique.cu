#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/unique.h>
#include <thrust/execution_policy.h>


template<typename Iterator1, typename Iterator2>
__global__
void unique_kernel(Iterator1 first, Iterator1 last, Iterator2 result)
{
  *result = thrust::unique(thrust::seq, first, last);
}


template<typename Iterator1, typename BinaryPredicate, typename Iterator2>
__global__
void unique_kernel(Iterator1 first, Iterator1 last, BinaryPredicate pred, Iterator2 result)
{
  *result = thrust::unique(thrust::seq, first, last, pred);
}


template<typename T>
struct is_equal_div_10_unique
{
  __host__ __device__
  bool operator()(const T x, const T& y) const { return ((int) x / 10) == ((int) y / 10); }
};


void TestUniqueDeviceSeq()
{
  typedef thrust::device_vector<int> Vector;
  typedef Vector::value_type T;

  Vector data(10);
  data[0] = 11; 
  data[1] = 11; 
  data[2] = 12;
  data[3] = 20; 
  data[4] = 29; 
  data[5] = 21; 
  data[6] = 21; 
  data[7] = 31; 
  data[8] = 31; 
  data[9] = 37; 

  thrust::device_vector<Vector::iterator> new_last_vec(1);
  Vector::iterator new_last;
  
  unique_kernel<<<1,1>>>(data.begin(), data.end(), new_last_vec.begin());
  new_last = new_last_vec[0];

  ASSERT_EQUAL(new_last - data.begin(), 7);
  ASSERT_EQUAL(data[0], 11);
  ASSERT_EQUAL(data[1], 12);
  ASSERT_EQUAL(data[2], 20);
  ASSERT_EQUAL(data[3], 29);
  ASSERT_EQUAL(data[4], 21);
  ASSERT_EQUAL(data[5], 31);
  ASSERT_EQUAL(data[6], 37);

  unique_kernel<<<1,1>>>(data.begin(), new_last, is_equal_div_10_unique<T>(), new_last_vec.begin());
  new_last = new_last_vec[0];

  ASSERT_EQUAL(new_last - data.begin(), 3);
  ASSERT_EQUAL(data[0], 11);
  ASSERT_EQUAL(data[1], 20);
  ASSERT_EQUAL(data[2], 31);
}
DECLARE_UNITTEST(TestUniqueDeviceSeq);


template<typename Iterator1, typename Iterator2, typename Iterator3>
__global__
void unique_copy_kernel(Iterator1 first, Iterator1 last, Iterator2 result1, Iterator3 result2)
{
  *result2 = thrust::unique_copy(thrust::seq, first, last, result1);
}


template<typename Iterator1, typename Iterator2, typename BinaryPredicate, typename Iterator3>
__global__
void unique_copy_kernel(Iterator1 first, Iterator1 last, Iterator2 result1, BinaryPredicate pred, Iterator3 result2)
{
  *result2 = thrust::unique_copy(thrust::seq, first, last, result1, pred);
}


void TestUniqueCopyDeviceSeq()
{
  typedef thrust::device_vector<int> Vector;
  typedef Vector::value_type T;

  Vector data(10);
  data[0] = 11; 
  data[1] = 11; 
  data[2] = 12;
  data[3] = 20; 
  data[4] = 29; 
  data[5] = 21; 
  data[6] = 21; 
  data[7] = 31; 
  data[8] = 31; 
  data[9] = 37; 
  
  Vector output(10, -1);

  thrust::device_vector<Vector::iterator> new_last_vec(1);
  Vector::iterator new_last;
  
  unique_copy_kernel<<<1,1>>>(data.begin(), data.end(), output.begin(), new_last_vec.begin());
  new_last = new_last_vec[0];

  ASSERT_EQUAL(new_last - output.begin(), 7);
  ASSERT_EQUAL(output[0], 11);
  ASSERT_EQUAL(output[1], 12);
  ASSERT_EQUAL(output[2], 20);
  ASSERT_EQUAL(output[3], 29);
  ASSERT_EQUAL(output[4], 21);
  ASSERT_EQUAL(output[5], 31);
  ASSERT_EQUAL(output[6], 37);

  unique_copy_kernel<<<1,1>>>(output.begin(), new_last, data.begin(), is_equal_div_10_unique<T>(), new_last_vec.begin());
  new_last = new_last_vec[0];

  ASSERT_EQUAL(new_last - data.begin(), 3);
  ASSERT_EQUAL(data[0], 11);
  ASSERT_EQUAL(data[1], 20);
  ASSERT_EQUAL(data[2], 31);
}
DECLARE_UNITTEST(TestUniqueCopyDeviceSeq);

