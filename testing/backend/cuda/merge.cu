#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/merge.h>
#include <thrust/functional.h>
#include <thrust/sort.h>
#include <thrust/extrema.h>
#include <thrust/execution_policy.h>


template<typename Iterator1, typename Iterator2, typename Iterator3, typename Iterator4>
__global__
void merge_kernel(Iterator1 first1, Iterator1 last1,
                  Iterator2 first2, Iterator2 last2,
                  Iterator3 result1,
                  Iterator4 result2)
{
  *result2 = thrust::merge(thrust::seq, first1, last1, first2, last2, result1);
}


template<typename T>
  void TestMergeDeviceSeq(size_t n)
{
  size_t sizes[]   = {0, 1, n / 2, n, n + 1, 2 * n};
  size_t num_sizes = sizeof(sizes) / sizeof(size_t);

  thrust::host_vector<T> random = unittest::random_integers<unittest::int8_t>(n + *thrust::max_element(sizes, sizes + num_sizes));

  thrust::host_vector<T> h_a(random.begin(), random.begin() + n);
  thrust::host_vector<T> h_b(random.begin() + n, random.end());

  thrust::stable_sort(h_a.begin(), h_a.end());
  thrust::stable_sort(h_b.begin(), h_b.end());
  
  thrust::device_vector<T> d_a = h_a;
  thrust::device_vector<T> d_b = h_b;

  for(size_t i = 0; i < num_sizes; i++)
  {
    size_t size = sizes[i];
    
    thrust::host_vector<T>   h_result(n + size);
    thrust::device_vector<T> d_result(n + size);

    typename thrust::host_vector<T>::iterator   h_end;

    typedef typename thrust::device_vector<T>::iterator iter_type;
    thrust::device_vector<iter_type> d_end(1);
    
    h_end = thrust::merge(h_a.begin(), h_a.end(),
                          h_b.begin(), h_b.begin() + size,
                          h_result.begin());
    h_result.resize(h_end - h_result.begin());

    merge_kernel<<<1,1>>>(d_a.begin(), d_a.end(),
                          d_b.begin(), d_b.begin() + size,
                          d_result.begin(),
                          d_end.begin());
    d_result.resize((iter_type)d_end[0] - d_result.begin());

    ASSERT_EQUAL(h_result, d_result);
  }
}
DECLARE_VARIABLE_UNITTEST(TestMergeDeviceSeq);

