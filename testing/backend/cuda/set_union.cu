#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/set_operations.h>
#include <thrust/execution_policy.h>


template<typename Iterator1, typename Iterator2, typename Iterator3, typename Iterator4>
__global__
void set_union_kernel(Iterator1 first1, Iterator1 last1,
                      Iterator2 first2, Iterator2 last2,
                      Iterator3 result1,
                      Iterator4 result2)
{
  *result2 = thrust::set_union(thrust::seq, first1, last1, first2, last2, result1);
}


void TestSetUnionDeviceSeq()
{
  typedef thrust::device_vector<int> Vector;
  typedef typename Vector::iterator Iterator;

  Vector a(3), b(4);

  a[0] = 0; a[1] = 2; a[2] = 4;
  b[0] = 0; b[1] = 3; b[2] = 3; b[3] = 4;

  Vector ref(5);
  ref[0] = 0; ref[1] = 2; ref[2] = 3; ref[3] = 3; ref[4] = 4;

  Vector result(5);
  thrust::device_vector<Iterator> end_vec(1);

  set_union_kernel<<<1,1>>>(a.begin(), a.end(),
                            b.begin(), b.end(),
                            result.begin(),
                            end_vec.begin());
  Iterator end = end_vec[0];

  ASSERT_EQUAL_QUIET(result.end(), end);
  ASSERT_EQUAL(ref, result);
}
DECLARE_UNITTEST(TestSetUnionDeviceSeq);

