#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/remove.h>
#include <thrust/execution_policy.h>


template<typename Iterator, typename T, typename Iterator2>
__global__
void remove_kernel(Iterator first, Iterator last, T val, Iterator2 result)
{
  *result = thrust::remove(thrust::seq, first, last, val);
}


template<typename Iterator, typename Predicate, typename Iterator2>
__global__
void remove_if_kernel(Iterator first, Iterator last, Predicate pred, Iterator2 result)
{
  *result = thrust::remove_if(thrust::seq, first, last, pred);
}


template<typename Iterator1, typename Iterator2, typename Predicate, typename Iterator3>
__global__
void remove_if_kernel(Iterator1 first, Iterator1 last, Iterator2 stencil_first, Predicate pred, Iterator3 result)
{
  *result = thrust::remove_if(thrust::seq, first, last, stencil_first, pred);
}


template<typename Iterator1, typename Iterator2, typename T, typename Iterator3>
__global__
void remove_copy_kernel(Iterator1 first, Iterator1 last, Iterator2 result1, T val, Iterator3 result2)
{
  *result2 = thrust::remove_copy(thrust::seq, first, last, result1, val);
}


template<typename Iterator1, typename Iterator2, typename Predicate, typename Iterator3>
__global__
void remove_copy_if_kernel(Iterator1 first, Iterator1 last, Iterator2 result, Predicate pred, Iterator3 result_end)
{
  *result_end = thrust::remove_copy_if(thrust::seq, first, last, result, pred);
}


template<typename Iterator1, typename Iterator2, typename Iterator3, typename Predicate, typename Iterator4>
__global__
void remove_copy_if_kernel(Iterator1 first, Iterator1 last, Iterator2 stencil_first, Iterator3 result, Predicate pred, Iterator4 result_end)
{
  *result_end = thrust::remove_copy_if(thrust::seq, first, last, stencil_first, result, pred);
}


template<typename T>
struct is_even
  : thrust::unary_function<T,bool>
{
  __host__ __device__
  bool operator()(T x) { return (static_cast<unsigned int>(x) & 1) == 0; }
};


template<typename T>
struct is_true
  : thrust::unary_function<T,bool>
{
  __host__ __device__
  bool operator()(T x) { return x ? true : false; }
};


template<typename T>
void TestRemoveDeviceSeq(const size_t n)
{
  thrust::host_vector<T>   h_data = unittest::random_samples<T>(n);
  thrust::device_vector<T> d_data = h_data;

  typedef typename thrust::device_vector<T>::iterator iterator;
  thrust::device_vector<iterator> d_result(1);
  
  size_t h_size = thrust::remove(h_data.begin(), h_data.end(), T(0)) - h_data.begin();
  remove_kernel<<<1,1>>>(d_data.begin(), d_data.end(), T(0), d_result.begin());
  size_t d_size = (iterator)d_result[0] - d_data.begin();
  
  ASSERT_EQUAL(h_size, d_size);
  
  h_data.resize(h_size);
  d_data.resize(d_size);
  
  ASSERT_EQUAL(h_data, d_data);
}
DECLARE_VARIABLE_UNITTEST(TestRemoveDeviceSeq);


template<typename T>
void TestRemoveIfDeviceSeq(const size_t n)
{
  thrust::host_vector<T>   h_data = unittest::random_samples<T>(n);
  thrust::device_vector<T> d_data = h_data;

  typedef typename thrust::device_vector<T>::iterator iterator;
  thrust::device_vector<iterator> d_result(1);
  
  size_t h_size = thrust::remove_if(h_data.begin(), h_data.end(), is_true<T>()) - h_data.begin();
  remove_if_kernel<<<1,1>>>(d_data.begin(), d_data.end(), is_true<T>(), d_result.begin());
  size_t d_size = (iterator)d_result[0] - d_data.begin();
  
  ASSERT_EQUAL(h_size, d_size);
  
  h_data.resize(h_size);
  d_data.resize(d_size);
  
  ASSERT_EQUAL(h_data, d_data);
}
DECLARE_VARIABLE_UNITTEST(TestRemoveIfDeviceSeq);


template<typename T>
void TestRemoveIfStencilDeviceSeq(const size_t n)
{
  thrust::host_vector<T>   h_data = unittest::random_samples<T>(n);
  thrust::device_vector<T> d_data = h_data;

  typedef typename thrust::device_vector<T>::iterator iterator;
  thrust::device_vector<iterator> d_result(1);
  
  thrust::host_vector<bool>   h_stencil = unittest::random_integers<bool>(n);
  thrust::device_vector<bool> d_stencil = h_stencil;
  
  size_t h_size = thrust::remove_if(h_data.begin(), h_data.end(), h_stencil.begin(), is_true<T>()) - h_data.begin();

  remove_if_kernel<<<1,1>>>(d_data.begin(), d_data.end(), d_stencil.begin(), is_true<T>(), d_result.begin());
  size_t d_size = (iterator)d_result[0] - d_data.begin();
  
  ASSERT_EQUAL(h_size, d_size);
  
  h_data.resize(h_size);
  d_data.resize(d_size);
  
  ASSERT_EQUAL(h_data, d_data);
}
DECLARE_VARIABLE_UNITTEST(TestRemoveIfStencilDeviceSeq);


template<typename T>
void TestRemoveCopyDeviceSeq(const size_t n)
{
  thrust::host_vector<T>   h_data = unittest::random_samples<T>(n);
  thrust::device_vector<T> d_data = h_data;
  
  thrust::host_vector<T>   h_result(n);
  thrust::device_vector<T> d_result(n);

  typedef typename thrust::device_vector<T>::iterator iterator;
  thrust::device_vector<iterator> d_new_end(1);
  
  size_t h_size = thrust::remove_copy(h_data.begin(), h_data.end(), h_result.begin(), T(0)) - h_result.begin();

  remove_copy_kernel<<<1,1>>>(d_data.begin(), d_data.end(), d_result.begin(), T(0), d_new_end.begin());
  size_t d_size = (iterator)d_new_end[0] - d_result.begin();
  
  ASSERT_EQUAL(h_size, d_size);
  
  h_result.resize(h_size);
  d_result.resize(d_size);
  
  ASSERT_EQUAL(h_result, d_result);
}
DECLARE_VARIABLE_UNITTEST(TestRemoveCopyDeviceSeq);


template<typename T>
void TestRemoveCopyIfDeviceSeq(const size_t n)
{
  thrust::host_vector<T>   h_data = unittest::random_samples<T>(n);
  thrust::device_vector<T> d_data = h_data;
  
  thrust::host_vector<T>   h_result(n);
  thrust::device_vector<T> d_result(n);

  typedef typename thrust::device_vector<T>::iterator iterator;
  thrust::device_vector<iterator> d_new_end(1);
  
  size_t h_size = thrust::remove_copy_if(h_data.begin(), h_data.end(), h_result.begin(), is_true<T>()) - h_result.begin();

  remove_copy_if_kernel<<<1,1>>>(d_data.begin(), d_data.end(), d_result.begin(), is_true<T>(), d_new_end.begin());
  size_t d_size = (iterator)d_new_end[0] - d_result.begin();
  
  ASSERT_EQUAL(h_size, d_size);
  
  h_result.resize(h_size);
  d_result.resize(d_size);
  
  ASSERT_EQUAL(h_result, d_result);
}
DECLARE_VARIABLE_UNITTEST(TestRemoveCopyIfDeviceSeq);


template<typename T>
void TestRemoveCopyIfStencilDeviceSeq(const size_t n)
{
  thrust::host_vector<T>   h_data = unittest::random_samples<T>(n);
  thrust::device_vector<T> d_data = h_data;
  
  thrust::host_vector<T>   h_result(n);
  thrust::device_vector<T> d_result(n);

  typedef typename thrust::device_vector<T>::iterator iterator;
  thrust::device_vector<iterator> d_new_end(1);

  thrust::host_vector<bool>   h_stencil = unittest::random_integers<bool>(n);
  thrust::device_vector<bool> d_stencil = h_stencil;
  
  size_t h_size = thrust::remove_copy_if(h_data.begin(), h_data.end(), h_stencil.begin(), h_result.begin(), is_true<T>()) - h_result.begin();

  remove_copy_if_kernel<<<1,1>>>(d_data.begin(), d_data.end(), d_stencil.begin(), d_result.begin(), is_true<T>(), d_new_end.begin());
  size_t d_size = (iterator)d_new_end[0] - d_result.begin();
  
  ASSERT_EQUAL(h_size, d_size);
  
  h_result.resize(h_size);
  d_result.resize(d_size);
  
  ASSERT_EQUAL(h_result, d_result);
}
DECLARE_VARIABLE_UNITTEST(TestRemoveCopyIfStencilDeviceSeq);

