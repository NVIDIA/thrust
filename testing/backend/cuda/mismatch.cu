#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/mismatch.h>
#include <thrust/execution_policy.h>


template<typename Iterator1, typename Iterator2, typename Iterator3>
__global__ void mismatch_kernel(Iterator1 first1, Iterator1 last1, Iterator2 first2, Iterator3 result)
{
  *result = thrust::mismatch(thrust::seq, first1, last1, first2);
}


void TestMismatchDeviceSeq()
{
  thrust::device_vector<int> a(4);
  thrust::device_vector<int> b(4);
  a[0] = 1; b[0] = 1;
  a[1] = 2; b[1] = 2;
  a[2] = 3; b[2] = 4;
  a[3] = 4; b[3] = 3;

  typedef thrust::pair<
    typename thrust::device_vector<int>::iterator,
    typename thrust::device_vector<int>::iterator
  > pair_type;

  thrust::device_vector<pair_type> d_result(1);
  
  mismatch_kernel<<<1,1>>>(a.begin(), a.end(), b.begin(), d_result.begin());

  ASSERT_EQUAL(2, ((pair_type)d_result[0]).first  - a.begin());
  ASSERT_EQUAL(2, ((pair_type)d_result[0]).second - b.begin());
  
  b[2] = 3;
  
  mismatch_kernel<<<1,1>>>(a.begin(), a.end(), b.begin(), d_result.begin());
  ASSERT_EQUAL(3, ((pair_type)d_result[0]).first  - a.begin());
  ASSERT_EQUAL(3, ((pair_type)d_result[0]).second - b.begin());
  
  b[3] = 4;
  
  mismatch_kernel<<<1,1>>>(a.begin(), a.end(), b.begin(), d_result.begin());
  ASSERT_EQUAL(4, ((pair_type)d_result[0]).first  - a.begin());
  ASSERT_EQUAL(4, ((pair_type)d_result[0]).second - b.begin());
}
DECLARE_UNITTEST(TestMismatchDeviceSeq);

