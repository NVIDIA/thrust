#include <unittest/unittest.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/transform.h>

using namespace unittest;

template<typename T>
  struct TestZipIteratorManipulation
{
  template<typename Vector>
  void test(void)
  {
    using namespace thrust;

    Vector v0(4);
    Vector v1(4);
    Vector v2(4);

    // initialize input
    sequence(v0.begin(), v0.end());
    sequence(v1.begin(), v1.end());
    sequence(v2.begin(), v2.end());

    typedef tuple<typename Vector::iterator, typename Vector::iterator> IteratorTuple;

    IteratorTuple t = make_tuple(v0.begin(), v1.begin());

    typedef zip_iterator<IteratorTuple> ZipIterator;

    // test construction
    ZipIterator iter0 = make_zip_iterator(t);

    ASSERT_EQUAL_QUIET(v0.begin(), get<0>(iter0.get_iterator_tuple()));
    ASSERT_EQUAL_QUIET(v1.begin(), get<1>(iter0.get_iterator_tuple()));

    // test dereference
    ASSERT_EQUAL(*v0.begin(), get<0>(*iter0));
    ASSERT_EQUAL(*v1.begin(), get<1>(*iter0));

    // test equality
    ZipIterator iter1 = iter0;
    ZipIterator iter2 = make_zip_iterator(make_tuple(v0.begin(), v2.begin()));
    ZipIterator iter3 = make_zip_iterator(make_tuple(v1.begin(), v2.begin()));
    ASSERT_EQUAL(true,  iter0 == iter1);
    ASSERT_EQUAL(true,  iter0 == iter2);
    ASSERT_EQUAL(false, iter0 == iter3);

    // test inequality
    ASSERT_EQUAL(false, iter0 != iter1);
    ASSERT_EQUAL(false, iter0 != iter2);
    ASSERT_EQUAL(true,  iter0 != iter3);

    // test advance
    ZipIterator iter4 = iter0 + 1;
    ASSERT_EQUAL_QUIET(v0.begin() + 1, get<0>(iter4.get_iterator_tuple()));
    ASSERT_EQUAL_QUIET(v1.begin() + 1, get<1>(iter4.get_iterator_tuple()));

    // test pre-increment
    ++iter4;
    ASSERT_EQUAL_QUIET(v0.begin() + 2, get<0>(iter4.get_iterator_tuple()));
    ASSERT_EQUAL_QUIET(v1.begin() + 2, get<1>(iter4.get_iterator_tuple()));

    // test post-increment
    iter4++;
    ASSERT_EQUAL_QUIET(v0.begin() + 3, get<0>(iter4.get_iterator_tuple()));
    ASSERT_EQUAL_QUIET(v1.begin() + 3, get<1>(iter4.get_iterator_tuple()));

    // test pre-decrement
    --iter4;
    ASSERT_EQUAL_QUIET(v0.begin() + 2, get<0>(iter4.get_iterator_tuple()));
    ASSERT_EQUAL_QUIET(v1.begin() + 2, get<1>(iter4.get_iterator_tuple()));

    // test post-decrement
    iter4--;
    ASSERT_EQUAL_QUIET(v0.begin() + 1, get<0>(iter4.get_iterator_tuple()));
    ASSERT_EQUAL_QUIET(v1.begin() + 1, get<1>(iter4.get_iterator_tuple()));

    // test difference
    ASSERT_EQUAL( 1, iter4 - iter0);
    ASSERT_EQUAL(-1, iter0 - iter4);
  }

  void operator()(void)
  {
    test<   thrust::host_vector<T> >();
    test< thrust::device_vector<T> >();
  }
};
SimpleUnitTest<TestZipIteratorManipulation, type_list<int> > TestZipIteratorManipulationInstance;

template <typename T>
  struct TestZipIteratorReference
{
  void operator()(void)
  {
    using namespace thrust;

    // test host types
    typedef typename host_vector<T>::iterator          Iterator1;
    typedef typename host_vector<T>::const_iterator    Iterator2;
    typedef tuple<Iterator1,Iterator2>                 IteratorTuple1;
    typedef zip_iterator<IteratorTuple1> ZipIterator1;

    typedef typename iterator_reference<ZipIterator1>::type zip_iterator_reference_type1;

    host_vector<T> h_variable(1);

    typedef tuple<T&,const T&> reference_type1;

    reference_type1               ref1(*h_variable.begin(),*h_variable.cbegin());
    zip_iterator_reference_type1 test1(*h_variable.begin(),*h_variable.cbegin());

    ASSERT_EQUAL_QUIET(ref1, test1);
    ASSERT_EQUAL( get<0>(ref1),  get<0>(test1));
    ASSERT_EQUAL( get<1>(ref1),  get<1>(test1));


    // test device types
    typedef typename device_vector<T>::iterator        Iterator3;
    typedef typename device_vector<T>::const_iterator  Iterator4;
    typedef tuple<Iterator3,Iterator4>                 IteratorTuple2;
    typedef zip_iterator<IteratorTuple2> ZipIterator2;

    typedef typename iterator_reference<ZipIterator2>::type zip_iterator_reference_type2;

    device_vector<T> d_variable(1);

    typedef tuple< device_reference<T>, device_reference<const T> > reference_type2;

    reference_type2               ref2(*d_variable.begin(),*d_variable.cbegin());
    zip_iterator_reference_type2 test2(*d_variable.begin(),*d_variable.cbegin());

    ASSERT_EQUAL_QUIET(ref2, test2);
    ASSERT_EQUAL( get<0>(ref2),  get<0>(test2));
    ASSERT_EQUAL( get<1>(ref2),  get<1>(test2));
  } // end operator()()
};
SimpleUnitTest<TestZipIteratorReference, NumericTypes> TestZipIteratorReferenceInstance;


template <typename T>
  struct TestZipIteratorTraversal
{
  void operator()(void)
  {
    using namespace thrust;

    // test host types
    typedef typename host_vector<T>::iterator          Iterator1;
    typedef typename host_vector<T>::const_iterator    Iterator2;
    typedef tuple<Iterator1,Iterator2>                 IteratorTuple1;
    typedef zip_iterator<IteratorTuple1> ZipIterator1;

    typedef typename iterator_traversal<ZipIterator1>::type zip_iterator_traversal_type1;

    //ASSERT_EQUAL(true, (detail::is_convertible<zip_iterator_traversal_type1, random_access_traversal_tag>::value) );


    // test device types
    typedef typename device_vector<T>::iterator        Iterator3;
    typedef typename device_vector<T>::const_iterator  Iterator4;
    typedef tuple<Iterator3,Iterator4>                 IteratorTuple2;
    typedef zip_iterator<IteratorTuple2> ZipIterator2;

    typedef typename iterator_traversal<ZipIterator2>::type zip_iterator_traversal_type2;

    //ASSERT_EQUAL(true, (detail::is_convertible<zip_iterator_traversal_type2, thrust::random_access_traversal_tag>::value) );
  } // end operator()()
};
SimpleUnitTest<TestZipIteratorTraversal, NumericTypes> TestZipIteratorTraversalInstance;


template <typename T>
  struct TestZipIteratorSystem
{
  void operator()(void)
  {
    using namespace thrust;

    // XXX these assertions complain about undefined references to integral_constant<...>::value

    // test host types
    typedef typename host_vector<T>::iterator          Iterator1;
    typedef typename host_vector<T>::const_iterator    Iterator2;
    typedef tuple<Iterator1,Iterator2>                 IteratorTuple1;
    typedef zip_iterator<IteratorTuple1> ZipIterator1;

    typedef typename iterator_system<ZipIterator1>::type zip_iterator_system_type1;

    //ASSERT_EQUAL(true, (detail::is_same<zip_iterator_system_type1, experimental::space::host>::value) );


    // test device types
    typedef typename device_vector<T>::iterator        Iterator3;
    typedef typename device_vector<T>::const_iterator  Iterator4;
    typedef tuple<Iterator3,Iterator4>                 IteratorTuple2;
    typedef zip_iterator<IteratorTuple1> ZipIterator2;

    typedef typename iterator_system<ZipIterator2>::type zip_iterator_system_type2;

    //ASSERT_EQUAL(true, (detail::is_convertible<zip_iterator_system_type2, experimental::space::device>::value) );


    // test any
    typedef counting_iterator<T>         Iterator5;
    typedef counting_iterator<const T>   Iterator6;
    typedef tuple<Iterator5, Iterator6>                IteratorTuple3;
    typedef zip_iterator<IteratorTuple3> ZipIterator3;

    typedef typename iterator_system<ZipIterator3>::type zip_iterator_system_type3;

    //ASSERT_EQUAL(true, (detail::is_convertible<zip_iterator_system_type3, thrust::experimental::space::any>::value) );

    
    // test host/any
    typedef tuple<Iterator1, Iterator5>                IteratorTuple4;
    typedef zip_iterator<IteratorTuple4> ZipIterator4;

    typedef typename iterator_system<ZipIterator4>::type zip_iterator_system_type4;

    //ASSERT_EQUAL(true, (detail::is_convertible<zip_iterator_system_type4, thrust::host_system_tag>::value) );


    // test any/host
    typedef tuple<Iterator5, Iterator1>                IteratorTuple5;
    typedef zip_iterator<IteratorTuple5> ZipIterator5;

    typedef typename iterator_system<ZipIterator5>::type zip_iterator_system_type5;

    //ASSERT_EQUAL(true, (detail::is_convertible<zip_iterator_system_type5, thrust::host_system_tag>::value) );


    // test device/any
    typedef tuple<Iterator3, Iterator5>                IteratorTuple6;
    typedef zip_iterator<IteratorTuple6> ZipIterator6;

    typedef typename iterator_system<ZipIterator6>::type zip_iterator_system_type6;

    //ASSERT_EQUAL(true, (detail::is_convertible<zip_iterator_system_type6, thrust::device_system_tag>::value) );


    // test any/device
    typedef tuple<Iterator5, Iterator3>                IteratorTuple7;
    typedef zip_iterator<IteratorTuple7> ZipIterator7;

    typedef typename iterator_system<ZipIterator7>::type zip_iterator_system_type7;

    //ASSERT_EQUAL(true, (detail::is_convertible<zip_iterator_system_type7, thrust::device_system_tag>::value) );
  } // end operator()()
};
SimpleUnitTest<TestZipIteratorSystem, NumericTypes> TestZipIteratorSystemInstance;


template <typename Vector>
void TestZipIteratorCopy(void)
{
  using namespace thrust;

  Vector input0(4),  input1(4);
  Vector output0(4), output1(4);

  // initialize input
  sequence(input0.begin(), input0.end(),  0);
  sequence(input1.begin(), input1.end(), 13);

  copy( make_zip_iterator(make_tuple(input0.begin(),  input1.begin())),
        make_zip_iterator(make_tuple(input0.end(),    input1.end())),
        make_zip_iterator(make_tuple(output0.begin(), output1.begin())));

  ASSERT_EQUAL(input0, output0);
  ASSERT_EQUAL(input1, output1);
}
DECLARE_VECTOR_UNITTEST(TestZipIteratorCopy);


struct SumTwoTuple
{
  template<typename Tuple>
  __host__ __device__
  typename thrust::detail::remove_reference<typename thrust::tuple_element<0,Tuple>::type>::type
    operator()(Tuple x) const
  {
    return thrust::get<0>(x) + thrust::get<1>(x);
  }
}; // end SumTwoTuple

struct SumThreeTuple
{
  template<typename Tuple>
  __host__ __device__
  typename thrust::detail::remove_reference<typename thrust::tuple_element<0,Tuple>::type>::type
    operator()(Tuple x) const
  {
    return thrust::get<0>(x) + thrust::get<1>(x) + thrust::get<2>(x);
  }
}; // end SumThreeTuple


template <typename T>
struct TestZipIteratorTransform
{
  void operator()(const size_t n)
  {
    using namespace thrust;

    host_vector<T> h_data0 = unittest::random_samples<T>(n);
    host_vector<T> h_data1 = unittest::random_samples<T>(n);
    host_vector<T> h_data2 = unittest::random_samples<T>(n);

    device_vector<T> d_data0 = h_data0;
    device_vector<T> d_data1 = h_data1;
    device_vector<T> d_data2 = h_data2;

    host_vector<T>   h_result(n);
    device_vector<T> d_result(n);

    // Tuples with 2 elements
    transform( make_zip_iterator(make_tuple(h_data0.begin(), h_data1.begin())),
               make_zip_iterator(make_tuple(h_data0.end(),   h_data1.end())),
               h_result.begin(),
               SumTwoTuple());
    transform( make_zip_iterator(make_tuple(d_data0.begin(), d_data1.begin())),
               make_zip_iterator(make_tuple(d_data0.end(),   d_data1.end())),
               d_result.begin(),
               SumTwoTuple());
    ASSERT_EQUAL(h_result, d_result);
    
    
    // Tuples with 3 elements
    transform( make_zip_iterator(make_tuple(h_data0.begin(), h_data1.begin(), h_data2.begin())),
               make_zip_iterator(make_tuple(h_data0.end(),   h_data1.end(),   h_data2.end())),
               h_result.begin(),
               SumThreeTuple());
    transform( make_zip_iterator(make_tuple(d_data0.begin(), d_data1.begin(), d_data2.begin())),
               make_zip_iterator(make_tuple(d_data0.end(),   d_data1.end(),   d_data2.end())),
               d_result.begin(),
               SumThreeTuple());
    ASSERT_EQUAL(h_result, d_result);
  }
};
VariableUnitTest<TestZipIteratorTransform, ThirtyTwoBitTypes> TestZipIteratorTransformInstance;


void TestZipIteratorCopyAoSToSoA(void)
{
  using namespace thrust;

  const size_t n = 1;

  typedef tuple<int,int> structure;
  typedef host_vector<structure>   host_array_of_structures;
  typedef device_vector<structure> device_array_of_structures;

  typedef zip_iterator<
    tuple<host_vector<int>::iterator, host_vector<int>::iterator>
  > host_structure_of_arrays;

  typedef zip_iterator<
    tuple<device_vector<int>::iterator, device_vector<int>::iterator>
  > device_structure_of_arrays;

  host_array_of_structures   h_aos(n, make_tuple(7, 13) );
  device_array_of_structures d_aos(n, make_tuple(7, 13) );



  // host to host
  host_vector<int> h_field0(n), h_field1(n);
  host_structure_of_arrays h_soa = make_zip_iterator( make_tuple(h_field0.begin(), h_field1.begin()) );

  thrust::copy(h_aos.begin(), h_aos.end(), h_soa);
  ASSERT_EQUAL_QUIET(make_tuple(7, 13), h_soa[0]);



  // host to device
  device_vector<int> d_field0(n), d_field1(n);
  device_structure_of_arrays d_soa = make_zip_iterator( make_tuple(d_field0.begin(), d_field1.begin()) );

  thrust::copy(h_aos.begin(), h_aos.end(), d_soa);
  ASSERT_EQUAL_QUIET(make_tuple(7, 13), d_soa[0]);



  // device to device
  thrust::fill(d_field0.begin(), d_field0.end(), 0);
  thrust::fill(d_field1.begin(), d_field1.end(), 0);

  thrust::copy(d_aos.begin(), d_aos.end(), d_soa);
  ASSERT_EQUAL_QUIET(make_tuple(7, 13), d_soa[0]);


  // device to host
  thrust::fill(h_field0.begin(), h_field0.end(), 0);
  thrust::fill(h_field1.begin(), h_field1.end(), 0);

  thrust::copy(d_aos.begin(), d_aos.end(), h_soa);
  ASSERT_EQUAL_QUIET(make_tuple(7, 13), h_soa[0]);
};
DECLARE_UNITTEST(TestZipIteratorCopyAoSToSoA);



void TestZipIteratorCopySoAToAoS(void)
{
  using namespace thrust;

  const size_t n = 1;

  typedef tuple<int,int> structure;
  typedef host_vector<structure>   host_array_of_structures;
  typedef device_vector<structure> device_array_of_structures;

  typedef zip_iterator<
    tuple<host_vector<int>::iterator, host_vector<int>::iterator>
  > host_structure_of_arrays;

  typedef zip_iterator<
    tuple<device_vector<int>::iterator, device_vector<int>::iterator>
  > device_structure_of_arrays;

  host_vector<int>   h_field0(n, 7), h_field1(n, 13);
  device_vector<int> d_field0(n, 7), d_field1(n, 13);

  host_structure_of_arrays   h_soa = make_zip_iterator(make_tuple(h_field0.begin(), h_field1.begin()));
  device_structure_of_arrays d_soa = make_zip_iterator(make_tuple(d_field0.begin(), d_field1.begin()));

  host_array_of_structures   h_aos(n);
  device_array_of_structures d_aos(n);



  // host to host
  thrust::fill(h_aos.begin(), h_aos.end(), make_tuple(0,0));

  thrust::copy(h_soa, h_soa + n, h_aos.begin());
  ASSERT_EQUAL_QUIET(7,  get<0>(h_soa[0]));
  ASSERT_EQUAL_QUIET(13, get<1>(h_soa[0]));



  // host to device
  thrust::fill(d_aos.begin(), d_aos.end(), make_tuple(0,0));

  thrust::copy(h_soa, h_soa + n, d_aos.begin());
  ASSERT_EQUAL_QUIET(7,  get<0>(d_soa[0]));
  ASSERT_EQUAL_QUIET(13, get<1>(d_soa[0]));



  // device to device
  thrust::fill(d_aos.begin(), d_aos.end(), make_tuple(0,0));

  thrust::copy(d_soa, d_soa + n, d_aos.begin());
  ASSERT_EQUAL_QUIET(7,  get<0>(d_soa[0]));
  ASSERT_EQUAL_QUIET(13, get<1>(d_soa[0]));



  // device to host
  thrust::fill(h_aos.begin(), h_aos.end(), make_tuple(0,0));

  thrust::copy(d_soa, d_soa + n, h_aos.begin());
  ASSERT_EQUAL_QUIET(7,  get<0>(h_soa[0]));
  ASSERT_EQUAL_QUIET(13, get<1>(h_soa[0]));
};
DECLARE_UNITTEST(TestZipIteratorCopySoAToAoS);

