#include <unittest/unittest.h>
#include <thrust/functional.h>
#include <thrust/transform.h>

#include <functional>
#include <algorithm>

__THRUST_DISABLE_MSVC_POSSIBLE_LOSS_OF_DATA_WARNING_BEGIN
    
const size_t NUM_SAMPLES = 10000;

template <class InputVector, class OutputVector, class Operator, class ReferenceOperator>
void TestUnaryFunctional(void)
{
    typedef typename InputVector::value_type  InputType;
    typedef typename OutputVector::value_type OutputType;
    
    thrust::host_vector<InputType>  std_input = unittest::random_samples<InputType>(NUM_SAMPLES);
    thrust::host_vector<OutputType> std_output(NUM_SAMPLES);

    InputVector  input = std_input;
    OutputVector output(NUM_SAMPLES);

    thrust::transform(    input.begin(),     input.end(),     output.begin(),          Operator());
    thrust::transform(std_input.begin(), std_input.end(), std_output.begin(), ReferenceOperator());

    ASSERT_EQUAL(output, std_output);
}

template <class InputVector, class OutputVector, class Operator, class ReferenceOperator>
void TestBinaryFunctional(void)
{
    typedef typename InputVector::value_type  InputType;
    typedef typename OutputVector::value_type OutputType;
    
    thrust::host_vector<InputType>  std_input1 = unittest::random_samples<InputType>(NUM_SAMPLES);
    thrust::host_vector<InputType>  std_input2 = unittest::random_samples<InputType>(NUM_SAMPLES);
    thrust::host_vector<OutputType> std_output(NUM_SAMPLES);

    // Replace zeros to avoid divide by zero exceptions
    std::replace(std_input2.begin(), std_input2.end(), (InputType) 0, (InputType) 1);

    InputVector input1 = std_input1; 
    InputVector input2 = std_input2; 
    OutputVector output(NUM_SAMPLES);

    thrust::transform(    input1.begin(),     input1.end(),      input2.begin(),     output.begin(),          Operator());
    thrust::transform(std_input1.begin(), std_input1.end(),  std_input2.begin(), std_output.begin(), ReferenceOperator());

    // Note: FP division is not bit-equal, even when nvcc is invoked with --prec-div
    ASSERT_ALMOST_EQUAL(output, std_output);
}



// XXX add bool to list
// Instantiate a macro for all integer-like data types
#define INSTANTIATE_INTEGER_TYPES(Macro, vector_type, operator_name)   \
Macro(vector_type, operator_name, unittest::int8_t  )                  \
Macro(vector_type, operator_name, unittest::uint8_t )                  \
Macro(vector_type, operator_name, unittest::int16_t )                  \
Macro(vector_type, operator_name, unittest::uint16_t)                  \
Macro(vector_type, operator_name, unittest::int32_t )                  \
Macro(vector_type, operator_name, unittest::uint32_t)                  \
Macro(vector_type, operator_name, unittest::int64_t )                  \
Macro(vector_type, operator_name, unittest::uint64_t)

// Instantiate a macro for all integer and floating point data types
#define INSTANTIATE_ALL_TYPES(Macro, vector_type, operator_name)       \
INSTANTIATE_INTEGER_TYPES(Macro, vector_type, operator_name)           \
Macro(vector_type, operator_name, float)


// op(T) -> T
#define INSTANTIATE_UNARY_ARITHMETIC_FUNCTIONAL_TEST(vector_type, operator_name, data_type) \
    TestUnaryFunctional< thrust::vector_type<data_type>,                                   \
                         thrust::vector_type<data_type>,                                   \
                         thrust::operator_name<data_type>,                                 \
                         std::operator_name<data_type> >();
// XXX revert OutputVector<T> back to bool
// op(T) -> bool
#define INSTANTIATE_UNARY_LOGICAL_FUNCTIONAL_TEST(vector_type, operator_name, data_type) \
    TestUnaryFunctional< thrust::vector_type<data_type>,                                \
                         thrust::vector_type<data_type>,                                \
                         thrust::operator_name<data_type>,                              \
                         std::operator_name<data_type> >();
// op(T,T) -> T
#define INSTANTIATE_BINARY_ARITHMETIC_FUNCTIONAL_TEST(vector_type, operator_name, data_type) \
    TestBinaryFunctional< thrust::vector_type<data_type>,                                   \
                          thrust::vector_type<data_type>,                                   \
                          thrust::operator_name<data_type>,                                 \
                          std::operator_name<data_type> >();
// XXX revert OutputVector<T> back to bool
// op(T,T) -> bool
#define INSTANTIATE_BINARY_LOGICAL_FUNCTIONAL_TEST(vector_type, operator_name, data_type) \
    TestBinaryFunctional< thrust::vector_type<data_type>,                                \
                          thrust::vector_type<data_type>,                                \
                          thrust::operator_name<data_type>,                              \
                          std::operator_name<data_type> >();




// op(T) -> T
#define DECLARE_UNARY_ARITHMETIC_FUNCTIONAL_UNITTEST(operator_name, OperatorName)                          \
void Test##OperatorName##FunctionalHost(void)                                                              \
{                                                                                                          \
    INSTANTIATE_ALL_TYPES( INSTANTIATE_UNARY_ARITHMETIC_FUNCTIONAL_TEST, host_vector,   operator_name);    \
}                                                                                                          \
DECLARE_UNITTEST(Test##OperatorName##FunctionalHost);                                                      \
void Test##OperatorName##FunctionalDevice(void)                                                            \
{                                                                                                          \
    INSTANTIATE_ALL_TYPES( INSTANTIATE_UNARY_ARITHMETIC_FUNCTIONAL_TEST, device_vector, operator_name);    \
}                                                                                                          \
DECLARE_UNITTEST(Test##OperatorName##FunctionalDevice);

// op(T) -> bool
#define DECLARE_UNARY_LOGICAL_FUNCTIONAL_UNITTEST(operator_name, OperatorName)                             \
void Test##OperatorName##FunctionalHost(void)                                                              \
{                                                                                                          \
    INSTANTIATE_ALL_TYPES( INSTANTIATE_UNARY_LOGICAL_FUNCTIONAL_TEST, host_vector,   operator_name);       \
}                                                                                                          \
DECLARE_UNITTEST(Test##OperatorName##FunctionalHost);                                                      \
void Test##OperatorName##FunctionalDevice(void)                                                            \
{                                                                                                          \
    INSTANTIATE_ALL_TYPES( INSTANTIATE_UNARY_LOGICAL_FUNCTIONAL_TEST, device_vector, operator_name);       \
}                                                                                                          \
DECLARE_UNITTEST(Test##OperatorName##FunctionalDevice);

// op(T,T) -> T
#define DECLARE_BINARY_ARITHMETIC_FUNCTIONAL_UNITTEST(operator_name, OperatorName)                         \
void Test##OperatorName##FunctionalHost(void)                                                              \
{                                                                                                          \
    INSTANTIATE_ALL_TYPES( INSTANTIATE_BINARY_ARITHMETIC_FUNCTIONAL_TEST, host_vector,   operator_name);   \
}                                                                                                          \
DECLARE_UNITTEST(Test##OperatorName##FunctionalHost);                                                      \
void Test##OperatorName##FunctionalDevice(void)                                                            \
{                                                                                                          \
    INSTANTIATE_ALL_TYPES( INSTANTIATE_BINARY_ARITHMETIC_FUNCTIONAL_TEST, device_vector, operator_name);   \
}                                                                                                          \
DECLARE_UNITTEST(Test##OperatorName##FunctionalDevice);

// op(T,T) -> T (for integer T only)
#define DECLARE_BINARY_INTEGER_ARITHMETIC_FUNCTIONAL_UNITTEST(operator_name, OperatorName)                     \
void Test##OperatorName##FunctionalHost(void)                                                                  \
{                                                                                                              \
    INSTANTIATE_INTEGER_TYPES( INSTANTIATE_BINARY_ARITHMETIC_FUNCTIONAL_TEST, host_vector,   operator_name);   \
}                                                                                                              \
DECLARE_UNITTEST(Test##OperatorName##FunctionalHost);                                                          \
void Test##OperatorName##FunctionalDevice(void)                                                                \
{                                                                                                              \
    INSTANTIATE_INTEGER_TYPES( INSTANTIATE_BINARY_ARITHMETIC_FUNCTIONAL_TEST, device_vector, operator_name);   \
}                                                                                                              \
DECLARE_UNITTEST(Test##OperatorName##FunctionalDevice);

// op(T,T) -> bool
#define DECLARE_BINARY_LOGICAL_FUNCTIONAL_UNITTEST(operator_name, OperatorName)                         \
void Test##OperatorName##FunctionalHost(void)                                                           \
{                                                                                                       \
    INSTANTIATE_ALL_TYPES( INSTANTIATE_BINARY_LOGICAL_FUNCTIONAL_TEST, host_vector,   operator_name);   \
}                                                                                                       \
DECLARE_UNITTEST(Test##OperatorName##FunctionalHost);                                                   \
void Test##OperatorName##FunctionalDevice(void)                                                         \
{                                                                                                       \
    INSTANTIATE_ALL_TYPES( INSTANTIATE_BINARY_LOGICAL_FUNCTIONAL_TEST, device_vector, operator_name);   \
}                                                                                                       \
DECLARE_UNITTEST(Test##OperatorName##FunctionalDevice);




// Create the unit tests
DECLARE_UNARY_ARITHMETIC_FUNCTIONAL_UNITTEST(negate, Negate);
DECLARE_UNARY_LOGICAL_FUNCTIONAL_UNITTEST(logical_not, LogicalNot);

// Ad-hoc testing for other functionals
template <class Vector>
void TestIdentityFunctional(void)
{
    typedef typename Vector::value_type T;

    Vector input(3);
    input[0] = 0; input[1] = 1; input[2] = 2;

    Vector output(3);

    thrust::transform(input.begin(), input.end(), output.begin(), thrust::identity<T>());

    ASSERT_EQUAL(input, output);
}
DECLARE_VECTOR_UNITTEST(TestIdentityFunctional);

template <class Vector>
void TestProject1stFunctional(void)
{
    typedef typename Vector::value_type T;

    Vector lhs(3);
    Vector rhs(3);
    lhs[0] = 0;  rhs[0] = 3; 
    lhs[1] = 1;  rhs[1] = 4;
    lhs[2] = 2;  rhs[2] = 5;

    Vector output(3);

    thrust::transform(lhs.begin(), lhs.end(), rhs.begin(), output.begin(), thrust::project1st<T,T>());

    ASSERT_EQUAL(output, lhs);
}
DECLARE_VECTOR_UNITTEST(TestProject1stFunctional);

template <class Vector>
void TestProject2ndFunctional(void)
{
    typedef typename Vector::value_type T;

    Vector lhs(3);
    Vector rhs(3);
    lhs[0] = 0;  rhs[0] = 3; 
    lhs[1] = 1;  rhs[1] = 4;
    lhs[2] = 2;  rhs[2] = 5;

    Vector output(3);

    thrust::transform(lhs.begin(), lhs.end(), rhs.begin(), output.begin(), thrust::project2nd<T,T>());

    ASSERT_EQUAL(output, rhs);
}
DECLARE_VECTOR_UNITTEST(TestProject2ndFunctional);

template <class Vector>
void TestMaximumFunctional(void)
{
    typedef typename Vector::value_type T;

    Vector input1(3);
    Vector input2(3);
    input1[0] = 8; input1[1] = 3; input1[2] = 7;
    input2[0] = 5; input2[1] = 6; input2[2] = 9;

    Vector output(3);

    thrust::transform(input1.begin(), input1.end(), 
                      input2.begin(), 
                      output.begin(), 
                      thrust::maximum<T>());

    ASSERT_EQUAL(output[0], 8);
    ASSERT_EQUAL(output[1], 6);
    ASSERT_EQUAL(output[2], 9);
}
DECLARE_VECTOR_UNITTEST(TestMaximumFunctional);

template <class Vector>
void TestMinimumFunctional(void)
{
    typedef typename Vector::value_type T;

    Vector input1(3);
    Vector input2(3);
    input1[0] = 8; input1[1] = 3; input1[2] = 7;
    input2[0] = 5; input2[1] = 6; input2[2] = 9;

    Vector output(3);

    thrust::transform(input1.begin(), input1.end(), 
                      input2.begin(), 
                      output.begin(), 
                      thrust::minimum<T>());

    ASSERT_EQUAL(output[0], 5);
    ASSERT_EQUAL(output[1], 3);
    ASSERT_EQUAL(output[2], 7);
}
DECLARE_VECTOR_UNITTEST(TestMinimumFunctional);

template <class Vector>
void TestNot1(void)
{
    typedef typename Vector::value_type T;

    Vector input(5);
    input[0] = 1; input[1] = 0; input[2] = 1; input[3] = 1; input[4] = 0;

    Vector output(5);

    thrust::transform(input.begin(), input.end(), 
                      output.begin(), 
                      thrust::not1(thrust::identity<T>()));

    ASSERT_EQUAL(output[0], 0);
    ASSERT_EQUAL(output[1], 1);
    ASSERT_EQUAL(output[2], 0);
    ASSERT_EQUAL(output[3], 0);
    ASSERT_EQUAL(output[4], 1);
}
DECLARE_VECTOR_UNITTEST(TestNot1);

template <class Vector>
void TestNot2(void)
{
    typedef typename Vector::value_type T;

    Vector input1(5);
    Vector input2(5);
    input1[0] = 1; input1[1] = 0; input1[2] = 1; input1[3] = 1; input1[4] = 0;
    input2[0] = 1; input2[1] = 1; input2[2] = 0; input2[3] = 1; input2[4] = 1;

    Vector output(5);

    thrust::transform(input1.begin(), input1.end(), 
                      input2.begin(),
                      output.begin(), 
                      thrust::not2(thrust::equal_to<T>()));

    ASSERT_EQUAL(output[0], 0);
    ASSERT_EQUAL(output[1], 1);
    ASSERT_EQUAL(output[2], 1);
    ASSERT_EQUAL(output[3], 0);
    ASSERT_EQUAL(output[4], 1);
}
DECLARE_VECTOR_UNITTEST(TestNot2);

__THRUST_DISABLE_MSVC_POSSIBLE_LOSS_OF_DATA_WARNING_END
