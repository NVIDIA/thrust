#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/set_operations.h>
#include <thrust/functional.h>
#include <thrust/sort.h>


template<typename InputIterator1,
         typename InputIterator2,
         typename OutputIterator>
OutputIterator set_difference(my_system &system,
                              InputIterator1,
                              InputIterator1,
                              InputIterator2,
                              InputIterator2,
                              OutputIterator result)
{
  system.validate_dispatch();
  return result;
}

void TestSetDifferenceDispatchExplicit()
{
  thrust::device_vector<int> vec(1);

  my_system sys(0);
  thrust::set_difference(sys,
                         vec.begin(),
                         vec.begin(),
                         vec.begin(),
                         vec.begin(),
                         vec.begin());

  ASSERT_EQUAL(true, sys.is_valid());
}
DECLARE_UNITTEST(TestSetDifferenceDispatchExplicit);


template<typename InputIterator1,
         typename InputIterator2,
         typename OutputIterator>
OutputIterator set_difference(my_tag,
                              InputIterator1,
                              InputIterator1,
                              InputIterator2,
                              InputIterator2,
                              OutputIterator result)
{
  *result = 13;
  return result;
}

void TestSetDifferenceDispatchImplicit()
{
  thrust::device_vector<int> vec(1);

  thrust::set_difference(thrust::retag<my_tag>(vec.begin()),
                         thrust::retag<my_tag>(vec.begin()),
                         thrust::retag<my_tag>(vec.begin()),
                         thrust::retag<my_tag>(vec.begin()),
                         thrust::retag<my_tag>(vec.begin()));

  ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestSetDifferenceDispatchImplicit);


template<typename Vector>
void TestSetDifferenceSimple(void)
{
  typedef typename Vector::iterator Iterator;

  Vector a(4), b(5);

  a[0] = 0; a[1] = 2; a[2] = 4; a[3] = 5;
  b[0] = 0; b[1] = 3; b[2] = 3; b[3] = 4; b[4] = 6;

  Vector ref(2);
  ref[0] = 2; ref[1] = 5;

  Vector result(2);

  Iterator end = thrust::set_difference(a.begin(), a.end(),
                                        b.begin(), b.end(),
                                        result.begin());

  ASSERT_EQUAL_QUIET(result.end(), end);
  ASSERT_EQUAL(ref, result);
}
DECLARE_VECTOR_UNITTEST(TestSetDifferenceSimple);


template<typename T>
void TestSetDifference(const size_t n)
{
  size_t sizes[]   = {0, 1, n / 2, n, n + 1, 2 * n};
  size_t num_sizes = sizeof(sizes) / sizeof(size_t);

  thrust::host_vector<T> random = unittest::random_integers<unittest::int8_t>(n + *thrust::max_element(sizes, sizes + num_sizes));

  thrust::host_vector<T> h_a(random.begin(), random.begin() + n);
  thrust::host_vector<T> h_b(random.begin() + n, random.end());
  
  thrust::stable_sort(h_a.begin(), h_a.end());
  thrust::stable_sort(h_b.begin(), h_b.end());
  
  thrust::device_vector<T> d_a = h_a;
  thrust::device_vector<T> d_b = h_b;

  for (size_t i = 0; i < num_sizes; i++)
  {
    size_t size = sizes[i];
    
    thrust::host_vector<T>   h_result(n + size);
    thrust::device_vector<T> d_result(n + size);

    typename thrust::host_vector<T>::iterator   h_end;
    typename thrust::device_vector<T>::iterator d_end;
    
    h_end = thrust::set_difference(h_a.begin(), h_a.end(),
                                   h_b.begin(), h_b.begin() + size,
                                   h_result.begin());
    h_result.resize(h_end - h_result.begin());

    d_end = thrust::set_difference(d_a.begin(), d_a.end(),
                                   d_b.begin(), d_b.begin() + size,
                                   d_result.begin());
    d_result.resize(d_end - d_result.begin());

    ASSERT_EQUAL(h_result, d_result);
  }
}
DECLARE_VARIABLE_UNITTEST(TestSetDifference);


template<typename T>
void TestSetDifferenceEquivalentRanges(const size_t n)
{
  thrust::host_vector<T> temp = unittest::random_integers<T>(n);
  thrust::host_vector<T> h_a = temp; thrust::sort(h_a.begin(), h_a.end());
  thrust::host_vector<T> h_b = h_a;

  thrust::device_vector<T> d_a = h_a;
  thrust::device_vector<T> d_b = h_b;

  thrust::host_vector<T>   h_result(n);
  thrust::device_vector<T> d_result(n);

  typename thrust::host_vector<T>::iterator   h_end;
  typename thrust::device_vector<T>::iterator d_end;
  
  h_end = thrust::set_difference(h_a.begin(), h_a.end(),
                                 h_b.begin(), h_b.end(),
                                 h_result.begin());
  h_result.resize(h_end - h_result.begin());

  d_end = thrust::set_difference(d_a.begin(), d_a.end(),
                                 d_b.begin(), d_b.end(),
                                 d_result.begin());

  d_result.resize(d_end - d_result.begin());

  ASSERT_EQUAL(h_result, d_result);
}
DECLARE_VARIABLE_UNITTEST(TestSetDifferenceEquivalentRanges);


template<typename T>
void TestSetDifferenceMultiset(const size_t n)
{
  thrust::host_vector<T> temp = unittest::random_integers<T>(2 * n);

  // restrict elements to [min,13)
  for(typename thrust::host_vector<T>::iterator i = temp.begin();
      i != temp.end();
      ++i)
  {
    int temp = static_cast<int>(*i);
    temp %= 13;
    *i = temp;
  }

  thrust::host_vector<T> h_a(temp.begin(), temp.begin() + n);
  thrust::host_vector<T> h_b(temp.begin() + n, temp.end());

  thrust::sort(h_a.begin(), h_a.end());
  thrust::sort(h_b.begin(), h_b.end());

  thrust::device_vector<T> d_a = h_a;
  thrust::device_vector<T> d_b = h_b;

  thrust::host_vector<T> h_result(n);
  thrust::device_vector<T> d_result(n);

  typename thrust::host_vector<T>::iterator h_end;
  typename thrust::device_vector<T>::iterator d_end;
  
  h_end = thrust::set_difference(h_a.begin(), h_a.end(),
                                 h_b.begin(), h_b.end(),
                                 h_result.begin());
  h_result.resize(h_end - h_result.begin());

  d_end = thrust::set_difference(d_a.begin(), d_a.end(),
                                 d_b.begin(), d_b.end(),
                                 d_result.begin());

  d_result.resize(d_end - d_result.begin());

  ASSERT_EQUAL(h_result, d_result);
}
DECLARE_VARIABLE_UNITTEST(TestSetDifferenceMultiset);

