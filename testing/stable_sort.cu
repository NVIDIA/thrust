#include <unittest/unittest.h>
#include <thrust/sort.h>
#include <thrust/functional.h>


template<typename RandomAccessIterator>
void stable_sort(my_system &system, RandomAccessIterator, RandomAccessIterator)
{
    system.validate_dispatch();
}

void TestStableSortDispatchExplicit()
{
    thrust::device_vector<int> vec(1);

    my_system sys(0);
    thrust::stable_sort(sys, vec.begin(), vec.begin());

    ASSERT_EQUAL(true, sys.is_valid());
}
DECLARE_UNITTEST(TestStableSortDispatchExplicit);


template<typename RandomAccessIterator>
void stable_sort(my_tag, RandomAccessIterator first, RandomAccessIterator)
{
    *first = 13;
}

void TestStableSortDispatchImplicit()
{
    thrust::device_vector<int> vec(1);

    thrust::stable_sort(thrust::retag<my_tag>(vec.begin()),
                        thrust::retag<my_tag>(vec.begin()));

    ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestStableSortDispatchImplicit);


template <typename T>
struct less_div_10
{
  __host__ __device__ bool operator()(const T &lhs, const T &rhs) const {return ((int) lhs) / 10 < ((int) rhs) / 10;}
};

template <class Vector>
void InitializeSimpleStableKeySortTest(Vector& unsorted_keys, Vector& sorted_keys)
{
    unsorted_keys.resize(9);   
    unsorted_keys[0] = 25; 
    unsorted_keys[1] = 14; 
    unsorted_keys[2] = 35; 
    unsorted_keys[3] = 16; 
    unsorted_keys[4] = 26; 
    unsorted_keys[5] = 34; 
    unsorted_keys[6] = 36; 
    unsorted_keys[7] = 24; 
    unsorted_keys[8] = 15; 
    
    sorted_keys.resize(9);
    sorted_keys[0] = 14; 
    sorted_keys[1] = 16; 
    sorted_keys[2] = 15; 
    sorted_keys[3] = 25; 
    sorted_keys[4] = 26; 
    sorted_keys[5] = 24; 
    sorted_keys[6] = 35; 
    sorted_keys[7] = 34; 
    sorted_keys[8] = 36; 
}


template <class Vector>
void TestStableSortSimple(void)
{
    typedef typename Vector::value_type T;

    Vector unsorted_keys;
    Vector   sorted_keys;

    InitializeSimpleStableKeySortTest(unsorted_keys, sorted_keys);

    thrust::stable_sort(unsorted_keys.begin(), unsorted_keys.end(), less_div_10<T>());

    ASSERT_EQUAL(unsorted_keys,   sorted_keys);
}
DECLARE_VECTOR_UNITTEST(TestStableSortSimple);


template <typename T>
struct TestStableSort
{
    void operator()(const size_t n)
    {
        thrust::host_vector<T>   h_data = unittest::random_integers<T>(n);
        thrust::device_vector<T> d_data = h_data;

        thrust::stable_sort(h_data.begin(), h_data.end(), less_div_10<T>());
        thrust::stable_sort(d_data.begin(), d_data.end(), less_div_10<T>());

        ASSERT_EQUAL(h_data, d_data);
    }
};
VariableUnitTest<TestStableSort, SignedIntegralTypes> TestStableSortInstance;


template <typename T>
struct TestStableSortSemantics
{
    void operator()(const size_t n)
    {
        thrust::host_vector<T>   h_data = unittest::random_integers<T>(n);
        thrust::device_vector<T> d_data = h_data;

        thrust::stable_sort(h_data.begin(), h_data.end(), less_div_10<T>());
        thrust::stable_sort(d_data.begin(), d_data.end(), less_div_10<T>());

        ASSERT_EQUAL(h_data, d_data);
    }
};
VariableUnitTest<TestStableSortSemantics, unittest::type_list<unittest::int8_t,unittest::int16_t,unittest::int32_t> > TestStableSortSemanticsInstance;


template <typename T>
struct comp_mod3
{
    T * table;

    comp_mod3(T * table) : table(table) {}

    __host__ __device__
    bool operator()(T a, T b)
    {
        return table[(int) a] < table[(int) b];
    }
};

template <typename Vector>
void TestStableSortWithIndirection(void)
{
    // add numbers modulo 3 with external lookup table
    typedef typename Vector::value_type T;

    Vector data(7);
    data[0] = 1;
    data[1] = 3;
    data[2] = 5;
    data[3] = 3;
    data[4] = 0;
    data[5] = 2;
    data[6] = 1;

    Vector table(6);
    table[0] = 0;
    table[1] = 1;
    table[2] = 2;
    table[3] = 0;
    table[4] = 1;
    table[5] = 2;

    thrust::stable_sort(data.begin(), data.end(), comp_mod3<T>(thrust::raw_pointer_cast(&table[0])));
    
    ASSERT_EQUAL(data[0], T(3));
    ASSERT_EQUAL(data[1], T(3));
    ASSERT_EQUAL(data[2], T(0));
    ASSERT_EQUAL(data[3], T(1));
    ASSERT_EQUAL(data[4], T(1));
    ASSERT_EQUAL(data[5], T(5));
    ASSERT_EQUAL(data[6], T(2));
}
DECLARE_VECTOR_UNITTEST(TestStableSortWithIndirection);

