#include <unittest/unittest.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/iterator/constant_iterator.h>

#define BINARY_FUNCTIONAL_PLACEHOLDERS_TEST(name, op, reference_functor, type_list) \
template<typename Vector> \
  struct TestFunctionalPlaceholders##name \
{ \
  void operator()(const size_t) \
  { \
    static const size_t num_samples = 10000; \
    const size_t zero = 0; \
    typedef typename Vector::value_type T; \
    Vector lhs = unittest::random_samples<T>(num_samples); \
    Vector rhs = unittest::random_samples<T>(num_samples); \
    thrust::replace(rhs.begin(), rhs.end(), T(0), T(1)); \
\
    Vector reference(lhs.size()); \
    Vector result(lhs.size()); \
    using namespace thrust::placeholders; \
\
    thrust::transform(lhs.begin(), lhs.end(), rhs.begin(), reference.begin(), reference_functor<T>()); \
    thrust::transform(lhs.begin(), lhs.end(), rhs.begin(), result.begin(), _1 op _2); \
    ASSERT_ALMOST_EQUAL(reference, result); \
\
    thrust::transform(lhs.begin(), lhs.end(), thrust::make_constant_iterator<T>(1), reference.begin(), reference_functor<T>()); \
    thrust::transform(lhs.begin(), lhs.end(), result.begin(), _1 op T(1)); \
    ASSERT_ALMOST_EQUAL(reference, result); \
\
    thrust::transform(thrust::make_constant_iterator<T>(1,zero), thrust::make_constant_iterator<T>(1,num_samples), rhs.begin(), reference.begin(), reference_functor<T>()); \
    thrust::transform(rhs.begin(), rhs.end(), result.begin(), T(1) op _1); \
    ASSERT_ALMOST_EQUAL(reference, result); \
  } \
}; \
VectorUnitTest<TestFunctionalPlaceholders##name, type_list, thrust::device_vector, thrust::device_malloc_allocator> TestFunctionalPlaceholders##name##DeviceInstance; \
VectorUnitTest<TestFunctionalPlaceholders##name, type_list, thrust::host_vector, std::allocator> TestFunctionalPlaceholders##name##HostInstance;

BINARY_FUNCTIONAL_PLACEHOLDERS_TEST(Plus,       +, thrust::plus,       ThirtyTwoBitTypes);
BINARY_FUNCTIONAL_PLACEHOLDERS_TEST(Minus,      -, thrust::minus,      ThirtyTwoBitTypes);
BINARY_FUNCTIONAL_PLACEHOLDERS_TEST(Multiplies, *, thrust::multiplies, ThirtyTwoBitTypes);
BINARY_FUNCTIONAL_PLACEHOLDERS_TEST(Divides,    /, thrust::divides,    ThirtyTwoBitTypes);
BINARY_FUNCTIONAL_PLACEHOLDERS_TEST(Modulus,    %, thrust::modulus,    SmallIntegralTypes);

#define UNARY_FUNCTIONAL_PLACEHOLDERS_TEST(name, reference_operator, functor) \
template<typename Vector> \
  void TestFunctionalPlaceholders##name(void) \
{ \
  static const size_t num_samples = 10000; \
  typedef typename Vector::value_type T; \
  Vector input = unittest::random_samples<T>(num_samples); \
\
  Vector reference(input.size()); \
  thrust::transform(input.begin(), input.end(), reference.begin(), functor<T>()); \
\
  using namespace thrust::placeholders; \
  Vector result(input.size()); \
  thrust::transform(input.begin(), input.end(), result.begin(), reference_operator _1); \
\
  ASSERT_EQUAL(reference, result); \
} \
DECLARE_VECTOR_UNITTEST(TestFunctionalPlaceholders##name);

template<typename T>
  struct unary_plus_reference
{
  __host__ __device__ T operator()(const T &x) const
  {
    return +x;
  }
};

UNARY_FUNCTIONAL_PLACEHOLDERS_TEST(UnaryPlus, +, unary_plus_reference);
UNARY_FUNCTIONAL_PLACEHOLDERS_TEST(Negate,    -, thrust::negate);

