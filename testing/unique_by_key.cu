#include <unittest/unittest.h>
#include <thrust/unique.h>
#include <thrust/functional.h>
#include <thrust/iterator/discard_iterator.h>

struct my_tag : thrust::device_system_tag {};

template <typename ForwardIterator1,
          typename ForwardIterator2>
thrust::pair<ForwardIterator1,ForwardIterator2>
unique_by_key(my_tag,
              ForwardIterator1 keys_first, 
              ForwardIterator1,
              ForwardIterator2 values_first)
{
    *keys_first = 13;
    return thrust::make_pair(keys_first,values_first);
}

void TestUniqueByKeyDispatch()
{
    thrust::device_vector<int> vec(1);

    thrust::unique_by_key(thrust::retag<my_tag>(vec.begin()), 
                          thrust::retag<my_tag>(vec.begin()),
                          thrust::retag<my_tag>(vec.begin()));

    ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestUniqueByKeyDispatch);


template <typename InputIterator1,
          typename InputIterator2,
          typename OutputIterator1,
          typename OutputIterator2>
thrust::pair<OutputIterator1,OutputIterator2>
unique_by_key_copy(my_tag,
                   InputIterator1, 
                   InputIterator1,
                   InputIterator2,
                   OutputIterator1 keys_output,
                   OutputIterator2 values_output)
{
    *keys_output = 13;
    return thrust::make_pair(keys_output, values_output);
}

void TestUniqueByKeyCopyDispatch()
{
    thrust::device_vector<int> vec(1);

    thrust::unique_by_key_copy(thrust::retag<my_tag>(vec.begin()),
                               thrust::retag<my_tag>(vec.begin()),
                               thrust::retag<my_tag>(vec.begin()),
                               thrust::retag<my_tag>(vec.begin()),
                               thrust::retag<my_tag>(vec.begin()));

    ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestUniqueByKeyCopyDispatch);


template<typename T>
struct is_equal_div_10_unique
{
    __host__ __device__
    bool operator()(const T x, const T& y) const { return ((int) x / 10) == ((int) y / 10); }
};

template <typename Vector>
void initialize_keys(Vector& keys)
{
    keys.resize(9);
    keys[0] = 11;
    keys[1] = 11;
    keys[2] = 21;
    keys[3] = 20;
    keys[4] = 21;
    keys[5] = 21;
    keys[6] = 21;
    keys[7] = 37;
    keys[8] = 37;
}

template <typename Vector>
void initialize_values(Vector& values)
{
    values.resize(9);
    values[0] = 0; 
    values[1] = 1;
    values[2] = 2;
    values[3] = 3;
    values[4] = 4;
    values[5] = 5;
    values[6] = 6;
    values[7] = 7;
    values[8] = 8;
}


template<typename Vector>
void TestUniqueByKeySimple(void)
{
    typedef typename Vector::value_type T;

    Vector keys;
    Vector values;

    typename thrust::pair<typename Vector::iterator, typename Vector::iterator> new_last;

    // basic test
    initialize_keys(keys);  initialize_values(values);

    new_last = thrust::unique_by_key(keys.begin(), keys.end(), values.begin());

    ASSERT_EQUAL(new_last.first  - keys.begin(),   5);
    ASSERT_EQUAL(new_last.second - values.begin(), 5);
    ASSERT_EQUAL(keys[0], 11);
    ASSERT_EQUAL(keys[1], 21);
    ASSERT_EQUAL(keys[2], 20);
    ASSERT_EQUAL(keys[3], 21);
    ASSERT_EQUAL(keys[4], 37);
    
    ASSERT_EQUAL(values[0], 0);
    ASSERT_EQUAL(values[1], 2);
    ASSERT_EQUAL(values[2], 3);
    ASSERT_EQUAL(values[3], 4);
    ASSERT_EQUAL(values[4], 7);

    // test BinaryPredicate
    initialize_keys(keys);  initialize_values(values);
    
    new_last = thrust::unique_by_key(keys.begin(), keys.end(), values.begin(), is_equal_div_10_unique<T>());

    ASSERT_EQUAL(new_last.first  - keys.begin(),   3);
    ASSERT_EQUAL(new_last.second - values.begin(), 3);
    ASSERT_EQUAL(keys[0], 11);
    ASSERT_EQUAL(keys[1], 21);
    ASSERT_EQUAL(keys[2], 37);
    
    ASSERT_EQUAL(values[0], 0);
    ASSERT_EQUAL(values[1], 2);
    ASSERT_EQUAL(values[2], 7);
}
DECLARE_VECTOR_UNITTEST(TestUniqueByKeySimple);


template<typename Vector>
void TestUniqueCopyByKeySimple(void)
{
    typedef typename Vector::value_type T;

    Vector keys;
    Vector values;

    typename thrust::pair<typename Vector::iterator, typename Vector::iterator> new_last;

    // basic test
    initialize_keys(keys);  initialize_values(values);

    Vector output_keys(keys.size());
    Vector output_values(values.size());

    new_last = thrust::unique_by_key_copy(keys.begin(), keys.end(), values.begin(), output_keys.begin(), output_values.begin());

    ASSERT_EQUAL(new_last.first  - output_keys.begin(),   5);
    ASSERT_EQUAL(new_last.second - output_values.begin(), 5);
    ASSERT_EQUAL(output_keys[0], 11);
    ASSERT_EQUAL(output_keys[1], 21);
    ASSERT_EQUAL(output_keys[2], 20);
    ASSERT_EQUAL(output_keys[3], 21);
    ASSERT_EQUAL(output_keys[4], 37);
    
    ASSERT_EQUAL(output_values[0], 0);
    ASSERT_EQUAL(output_values[1], 2);
    ASSERT_EQUAL(output_values[2], 3);
    ASSERT_EQUAL(output_values[3], 4);
    ASSERT_EQUAL(output_values[4], 7);

    // test BinaryPredicate
    initialize_keys(keys);  initialize_values(values);
    
    new_last = thrust::unique_by_key_copy(keys.begin(), keys.end(), values.begin(), output_keys.begin(), output_values.begin(), is_equal_div_10_unique<T>());

    ASSERT_EQUAL(new_last.first  - output_keys.begin(),   3);
    ASSERT_EQUAL(new_last.second - output_values.begin(), 3);
    ASSERT_EQUAL(output_keys[0], 11);
    ASSERT_EQUAL(output_keys[1], 21);
    ASSERT_EQUAL(output_keys[2], 37);
    
    ASSERT_EQUAL(output_values[0], 0);
    ASSERT_EQUAL(output_values[1], 2);
    ASSERT_EQUAL(output_values[2], 7);
}
DECLARE_VECTOR_UNITTEST(TestUniqueCopyByKeySimple);


template<typename K>
struct TestUniqueByKey
{
    void operator()(const size_t n)
    {
        typedef unsigned int V; // ValueType

        thrust::host_vector<K>   h_keys = unittest::random_integers<bool>(n);
        thrust::host_vector<V>   h_vals = unittest::random_integers<V>(n);
        thrust::device_vector<K> d_keys = h_keys;
        thrust::device_vector<V> d_vals = h_vals;

        typedef typename thrust::host_vector<K>::iterator   HostKeyIterator;
        typedef typename thrust::host_vector<V>::iterator   HostValIterator;
        typedef typename thrust::device_vector<K>::iterator DeviceKeyIterator;
        typedef typename thrust::device_vector<V>::iterator DeviceValIterator;

        typedef typename thrust::pair<HostKeyIterator,  HostValIterator>   HostIteratorPair;
        typedef typename thrust::pair<DeviceKeyIterator,DeviceValIterator> DeviceIteratorPair;

        HostIteratorPair   h_last = thrust::unique_by_key(h_keys.begin(), h_keys.end(), h_vals.begin());
        DeviceIteratorPair d_last = thrust::unique_by_key(d_keys.begin(), d_keys.end(), d_vals.begin());

        ASSERT_EQUAL(h_last.first  - h_keys.begin(), d_last.first  - d_keys.begin());
        ASSERT_EQUAL(h_last.second - h_vals.begin(), d_last.second - d_vals.begin());
       
        size_t N = h_last.first - h_keys.begin();

        h_keys.resize(N);
        h_vals.resize(N);
        d_keys.resize(N);
        d_vals.resize(N);

        ASSERT_EQUAL(h_keys, d_keys);
        ASSERT_EQUAL(h_vals, d_vals);
    }
};
VariableUnitTest<TestUniqueByKey, IntegralTypes> TestUniqueByKeyInstance;


template<typename K>
struct TestUniqueCopyByKey
{
    void operator()(const size_t n)
    {
        typedef unsigned int V; // ValueType

        thrust::host_vector<K>   h_keys = unittest::random_integers<bool>(n);
        thrust::host_vector<V>   h_vals = unittest::random_integers<V>(n);
        thrust::device_vector<K> d_keys = h_keys;
        thrust::device_vector<V> d_vals = h_vals;

        thrust::host_vector<K>   h_keys_output(n);
        thrust::host_vector<V>   h_vals_output(n);
        thrust::device_vector<K> d_keys_output(n);
        thrust::device_vector<V> d_vals_output(n);

        typedef typename thrust::host_vector<K>::iterator   HostKeyIterator;
        typedef typename thrust::host_vector<V>::iterator   HostValIterator;
        typedef typename thrust::device_vector<K>::iterator DeviceKeyIterator;
        typedef typename thrust::device_vector<V>::iterator DeviceValIterator;

        typedef typename thrust::pair<HostKeyIterator,  HostValIterator>   HostIteratorPair;
        typedef typename thrust::pair<DeviceKeyIterator,DeviceValIterator> DeviceIteratorPair;

        HostIteratorPair   h_last = thrust::unique_by_key_copy(h_keys.begin(), h_keys.end(), h_vals.begin(), h_keys_output.begin(), h_vals_output.begin());
        DeviceIteratorPair d_last = thrust::unique_by_key_copy(d_keys.begin(), d_keys.end(), d_vals.begin(), d_keys_output.begin(), d_vals_output.begin());

        ASSERT_EQUAL(h_last.first  - h_keys_output.begin(), d_last.first  - d_keys_output.begin());
        ASSERT_EQUAL(h_last.second - h_vals_output.begin(), d_last.second - d_vals_output.begin());
       
        size_t N = h_last.first - h_keys_output.begin();

        h_keys_output.resize(N);
        h_vals_output.resize(N);
        d_keys_output.resize(N);
        d_vals_output.resize(N);

        ASSERT_EQUAL(h_keys_output, d_keys_output);
        ASSERT_EQUAL(h_vals_output, d_vals_output);
    }
};
VariableUnitTest<TestUniqueCopyByKey, IntegralTypes> TestUniqueCopyByKeyInstance;

template<typename K>
struct TestUniqueCopyByKeyToDiscardIterator
{
    void operator()(const size_t n)
    {
        typedef unsigned int V; // ValueType

        thrust::host_vector<K>   h_keys = unittest::random_integers<bool>(n);
        thrust::host_vector<V>   h_vals = unittest::random_integers<V>(n);
        thrust::device_vector<K> d_keys = h_keys;
        thrust::device_vector<V> d_vals = h_vals;

        thrust::host_vector<V>   h_vals_output(n);
        thrust::device_vector<V> d_vals_output(n);

        thrust::host_vector<K>   h_keys_output(n);
        thrust::device_vector<K> d_keys_output(n);

        thrust::host_vector<K> h_unique_keys = h_keys;
        h_unique_keys.erase(thrust::unique(h_unique_keys.begin(), h_unique_keys.end()), h_unique_keys.end());

        size_t num_unique_keys = h_unique_keys.size();


        // mask both outputs
        thrust::pair<thrust::discard_iterator<>, thrust::discard_iterator<> > h_result1 =
          thrust::unique_by_key_copy(h_keys.begin(), h_keys.end(),
                                     h_vals.begin(),
                                     thrust::make_discard_iterator(),
                                     thrust::make_discard_iterator());

        thrust::pair<thrust::discard_iterator<>, thrust::discard_iterator<> > d_result1 =
          thrust::unique_by_key_copy(d_keys.begin(), d_keys.end(),
                                     d_vals.begin(),
                                     thrust::make_discard_iterator(),
                                     thrust::make_discard_iterator());

        thrust::pair<thrust::discard_iterator<>, thrust::discard_iterator<> > reference1 =
          thrust::make_pair(thrust::make_discard_iterator(num_unique_keys),
                            thrust::make_discard_iterator(num_unique_keys));

        ASSERT_EQUAL_QUIET(reference1, h_result1);
        ASSERT_EQUAL_QUIET(reference1, d_result1);


        // mask values output
        thrust::pair<typename thrust::host_vector<K>::iterator, thrust::discard_iterator<> > h_result2 =
          thrust::unique_by_key_copy(h_keys.begin(), h_keys.end(),
                                     h_vals.begin(),
                                     h_keys_output.begin(),
                                     thrust::make_discard_iterator());

        thrust::pair<typename thrust::device_vector<K>::iterator, thrust::discard_iterator<> > d_result2 =
          thrust::unique_by_key_copy(d_keys.begin(), d_keys.end(),
                                     d_vals.begin(),
                                     d_keys_output.begin(),
                                     thrust::make_discard_iterator());

        thrust::pair<typename thrust::host_vector<K>::iterator, thrust::discard_iterator<> > h_reference2 =
          thrust::make_pair(h_keys_output.begin() + num_unique_keys,
                            thrust::make_discard_iterator(num_unique_keys));

        thrust::pair<typename thrust::device_vector<K>::iterator, thrust::discard_iterator<> > d_reference2 =
          thrust::make_pair(d_keys_output.begin() + num_unique_keys,
                            thrust::make_discard_iterator(num_unique_keys));

        ASSERT_EQUAL(h_keys_output, d_keys_output);
        ASSERT_EQUAL_QUIET(h_reference2, h_result2);
        ASSERT_EQUAL_QUIET(d_reference2, d_result2);


        // mask keys output
        thrust::pair<thrust::discard_iterator<>, typename thrust::host_vector<V>::iterator> h_result3 =
          thrust::unique_by_key_copy(h_keys.begin(), h_keys.end(),
                                     h_vals.begin(),
                                     thrust::make_discard_iterator(),
                                     h_vals_output.begin());

        thrust::pair<thrust::discard_iterator<>, typename thrust::device_vector<V>::iterator> d_result3 =
          thrust::unique_by_key_copy(d_keys.begin(), d_keys.end(),
                                     d_vals.begin(),
                                     thrust::make_discard_iterator(),
                                     d_vals_output.begin());

        thrust::pair<thrust::discard_iterator<>, typename thrust::host_vector<V>::iterator> h_reference3 =
          thrust::make_pair(thrust::make_discard_iterator(num_unique_keys),
                            h_vals_output.begin() + num_unique_keys);

        thrust::pair<thrust::discard_iterator<>, typename thrust::device_vector<V>::iterator> d_reference3 =
          thrust::make_pair(thrust::make_discard_iterator(num_unique_keys),
                            d_vals_output.begin() + num_unique_keys);

        ASSERT_EQUAL(h_vals_output, d_vals_output);
        ASSERT_EQUAL_QUIET(h_reference3, h_result3);
        ASSERT_EQUAL_QUIET(d_reference3, d_result3);
    }
};
VariableUnitTest<TestUniqueCopyByKeyToDiscardIterator, IntegralTypes> TestUniqueCopyByKeyToDiscardIteratorInstance;

