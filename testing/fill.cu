#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/fill.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/discard_iterator.h>

__THRUST_DISABLE_MSVC_POSSIBLE_LOSS_OF_DATA_WARNING_BEGIN

template <class Vector>
void TestFillSimple(void)
{
    typedef typename Vector::value_type T;

    Vector v(5);
    v[0] = 0; v[1] = 1; v[2] = 2; v[3] = 3; v[4] = 4;

    thrust::fill(v.begin() + 1, v.begin() + 4, (T) 7);

    ASSERT_EQUAL(v[0], 0);
    ASSERT_EQUAL(v[1], 7);
    ASSERT_EQUAL(v[2], 7);
    ASSERT_EQUAL(v[3], 7);
    ASSERT_EQUAL(v[4], 4);
    
    thrust::fill(v.begin() + 0, v.begin() + 3, (T) 8);
    
    ASSERT_EQUAL(v[0], 8);
    ASSERT_EQUAL(v[1], 8);
    ASSERT_EQUAL(v[2], 8);
    ASSERT_EQUAL(v[3], 7);
    ASSERT_EQUAL(v[4], 4);
    
    thrust::fill(v.begin() + 2, v.end(), (T) 9);
    
    ASSERT_EQUAL(v[0], 8);
    ASSERT_EQUAL(v[1], 8);
    ASSERT_EQUAL(v[2], 9);
    ASSERT_EQUAL(v[3], 9);
    ASSERT_EQUAL(v[4], 9);

    thrust::fill(v.begin(), v.end(), (T) 1);
    
    ASSERT_EQUAL(v[0], 1);
    ASSERT_EQUAL(v[1], 1);
    ASSERT_EQUAL(v[2], 1);
    ASSERT_EQUAL(v[3], 1);
    ASSERT_EQUAL(v[4], 1);
}
DECLARE_VECTOR_UNITTEST(TestFillSimple);


void TestFillDiscardIterator(void)
{
    // there's no result to check because fill returns void
    thrust::fill(thrust::discard_iterator<thrust::host_system_tag>(),
                 thrust::discard_iterator<thrust::host_system_tag>(10),
                 13);

    thrust::fill(thrust::discard_iterator<thrust::device_system_tag>(),
                 thrust::discard_iterator<thrust::device_system_tag>(10),
                 13);
}
DECLARE_UNITTEST(TestFillDiscardIterator);


template <class Vector>
void TestFillMixedTypes(void)
{
    typedef typename Vector::value_type T;

    Vector v(4);

    thrust::fill(v.begin(), v.end(), (long) 10);
    
    ASSERT_EQUAL(v[0], 10);
    ASSERT_EQUAL(v[1], 10);
    ASSERT_EQUAL(v[2], 10);
    ASSERT_EQUAL(v[3], 10);
    
    thrust::fill(v.begin(), v.end(), (float) 20);
    
    ASSERT_EQUAL(v[0], 20);
    ASSERT_EQUAL(v[1], 20);
    ASSERT_EQUAL(v[2], 20);
    ASSERT_EQUAL(v[3], 20);
}
DECLARE_VECTOR_UNITTEST(TestFillMixedTypes);


template <typename T>
void TestFill(size_t n)
{
    thrust::host_vector<T>   h_data = unittest::random_integers<T>(n);
    thrust::device_vector<T> d_data = h_data;

    thrust::fill(h_data.begin() + std::min((size_t)1, n), h_data.begin() + std::min((size_t)3, n), (T) 0);
    thrust::fill(d_data.begin() + std::min((size_t)1, n), d_data.begin() + std::min((size_t)3, n), (T) 0);

    ASSERT_EQUAL(h_data, d_data);

    thrust::fill(h_data.begin() + std::min((size_t)117, n), h_data.begin() + std::min((size_t)367, n), (T) 1);
    thrust::fill(d_data.begin() + std::min((size_t)117, n), d_data.begin() + std::min((size_t)367, n), (T) 1);

    ASSERT_EQUAL(h_data, d_data);
    
    thrust::fill(h_data.begin() + std::min((size_t)8, n), h_data.begin() + std::min((size_t)259, n), (T) 2);
    thrust::fill(d_data.begin() + std::min((size_t)8, n), d_data.begin() + std::min((size_t)259, n), (T) 2);

    ASSERT_EQUAL(h_data, d_data);
    
    thrust::fill(h_data.begin() + std::min((size_t)3, n), h_data.end(), (T) 3);
    thrust::fill(d_data.begin() + std::min((size_t)3, n), d_data.end(), (T) 3);

    ASSERT_EQUAL(h_data, d_data);
    
    thrust::fill(h_data.begin(), h_data.end(), (T) 4);
    thrust::fill(d_data.begin(), d_data.end(), (T) 4);

    ASSERT_EQUAL(h_data, d_data);
}
DECLARE_VARIABLE_UNITTEST(TestFill);

template <class Vector>
void TestFillNSimple(void)
{
    typedef typename Vector::value_type T;

    Vector v(5);
    v[0] = 0; v[1] = 1; v[2] = 2; v[3] = 3; v[4] = 4;

    typename Vector::iterator iter = thrust::fill_n(v.begin() + 1, 3, (T) 7);

    ASSERT_EQUAL(v[0], 0);
    ASSERT_EQUAL(v[1], 7);
    ASSERT_EQUAL(v[2], 7);
    ASSERT_EQUAL(v[3], 7);
    ASSERT_EQUAL(v[4], 4);
    ASSERT_EQUAL_QUIET(v.begin() + 4, iter);
    
    iter = thrust::fill_n(v.begin() + 0, 3, (T) 8);
    
    ASSERT_EQUAL(v[0], 8);
    ASSERT_EQUAL(v[1], 8);
    ASSERT_EQUAL(v[2], 8);
    ASSERT_EQUAL(v[3], 7);
    ASSERT_EQUAL(v[4], 4);
    ASSERT_EQUAL_QUIET(v.begin() + 3, iter);
    
    iter = thrust::fill_n(v.begin() + 2, 3, (T) 9);
    
    ASSERT_EQUAL(v[0], 8);
    ASSERT_EQUAL(v[1], 8);
    ASSERT_EQUAL(v[2], 9);
    ASSERT_EQUAL(v[3], 9);
    ASSERT_EQUAL(v[4], 9);
    ASSERT_EQUAL_QUIET(v.end(), iter);

    iter = thrust::fill_n(v.begin(), v.size(), (T) 1);
    
    ASSERT_EQUAL(v[0], 1);
    ASSERT_EQUAL(v[1], 1);
    ASSERT_EQUAL(v[2], 1);
    ASSERT_EQUAL(v[3], 1);
    ASSERT_EQUAL(v[4], 1);
    ASSERT_EQUAL_QUIET(v.end(), iter);
}
DECLARE_VECTOR_UNITTEST(TestFillNSimple);


void TestFillNDiscardIterator(void)
{
  thrust::discard_iterator<thrust::host_system_tag> h_result =
    thrust::fill_n(thrust::discard_iterator<thrust::host_system_tag>(),
                   10,
                   13);

  thrust::discard_iterator<thrust::device_system_tag> d_result =
    thrust::fill_n(thrust::discard_iterator<thrust::device_system_tag>(),
                   10,
                   13);

  thrust::discard_iterator<> reference(10);

  ASSERT_EQUAL_QUIET(reference, h_result);
  ASSERT_EQUAL_QUIET(reference, d_result);
}
DECLARE_UNITTEST(TestFillNDiscardIterator);


template <class Vector>
void TestFillNMixedTypes(void)
{
    typedef typename Vector::value_type T;

    Vector v(4);

    typename Vector::iterator iter = thrust::fill_n(v.begin(), v.size(), (long) 10);
    
    ASSERT_EQUAL(v[0], 10);
    ASSERT_EQUAL(v[1], 10);
    ASSERT_EQUAL(v[2], 10);
    ASSERT_EQUAL(v[3], 10);
    ASSERT_EQUAL_QUIET(v.end(), iter);
    
    iter = thrust::fill_n(v.begin(), v.size(), (float) 20);
    
    ASSERT_EQUAL(v[0], 20);
    ASSERT_EQUAL(v[1], 20);
    ASSERT_EQUAL(v[2], 20);
    ASSERT_EQUAL(v[3], 20);
    ASSERT_EQUAL_QUIET(v.end(), iter);
}
DECLARE_VECTOR_UNITTEST(TestFillNMixedTypes);


template <typename T>
void TestFillN(size_t n)
{
    thrust::host_vector<T>   h_data = unittest::random_integers<T>(n);
    thrust::device_vector<T> d_data = h_data;

    size_t begin_offset = std::min<size_t>(1,n);
    thrust::fill_n(h_data.begin() + begin_offset, std::min((size_t)3, n) - begin_offset, (T) 0);
    thrust::fill_n(d_data.begin() + begin_offset, std::min((size_t)3, n) - begin_offset, (T) 0);

    ASSERT_EQUAL(h_data, d_data);

    begin_offset = std::min<size_t>(117, n);
    thrust::fill_n(h_data.begin() + begin_offset, std::min((size_t)367, n) - begin_offset, (T) 1);
    thrust::fill_n(d_data.begin() + begin_offset, std::min((size_t)367, n) - begin_offset, (T) 1);

    ASSERT_EQUAL(h_data, d_data);
    
    begin_offset = std::min<size_t>(8, n);
    thrust::fill_n(h_data.begin() + begin_offset, std::min((size_t)259, n) - begin_offset, (T) 2);
    thrust::fill_n(d_data.begin() + begin_offset, std::min((size_t)259, n) - begin_offset, (T) 2);

    ASSERT_EQUAL(h_data, d_data);
    
    begin_offset = std::min<size_t>(3, n);
    thrust::fill_n(h_data.begin() + begin_offset, h_data.size() - begin_offset, (T) 3);
    thrust::fill_n(d_data.begin() + begin_offset, d_data.size() - begin_offset, (T) 3);

    ASSERT_EQUAL(h_data, d_data);
    
    thrust::fill_n(h_data.begin(), h_data.size(), (T) 4);
    thrust::fill_n(d_data.begin(), d_data.size(), (T) 4);

    ASSERT_EQUAL(h_data, d_data);
}
DECLARE_VARIABLE_UNITTEST(TestFillN);


template <typename Vector>
void TestFillZipIterator(void)
{
    typedef typename Vector::value_type T;

    Vector v1(3,T(0));
    Vector v2(3,T(0));
    Vector v3(3,T(0));

    thrust::fill(thrust::make_zip_iterator(thrust::make_tuple(v1.begin(),v2.begin(),v3.begin())),
                 thrust::make_zip_iterator(thrust::make_tuple(v1.end(),v2.end(),v3.end())),
                 thrust::tuple<T,T,T>(4,7,13));

    ASSERT_EQUAL(4,  v1[0]);
    ASSERT_EQUAL(4,  v1[1]);
    ASSERT_EQUAL(4,  v1[2]);
    ASSERT_EQUAL(7,  v2[0]);
    ASSERT_EQUAL(7,  v2[1]);
    ASSERT_EQUAL(7,  v2[2]);
    ASSERT_EQUAL(13, v3[0]);
    ASSERT_EQUAL(13, v3[1]);
    ASSERT_EQUAL(13, v3[2]);
};
DECLARE_VECTOR_UNITTEST(TestFillZipIterator);


void TestFillTuple(void)
{
    typedef int T;
    typedef thrust::tuple<T,T> Tuple;

    thrust::host_vector<Tuple>   h(3, Tuple(0,0));
    thrust::device_vector<Tuple> d(3, Tuple(0,0));

    thrust::fill(h.begin(), h.end(), Tuple(4,7));
    thrust::fill(d.begin(), d.end(), Tuple(4,7));

    ASSERT_EQUAL_QUIET(h, d);
};
DECLARE_UNITTEST(TestFillTuple);


struct TypeWithTrivialAssigment
{
  int x, y, z;
};

void TestFillWithTrivialAssignment(void)
{
    typedef TypeWithTrivialAssigment T;

    thrust::host_vector<T>   h(1);
    thrust::device_vector<T> d(1);
    
    ASSERT_EQUAL(h[0].x, 0);
    ASSERT_EQUAL(h[0].y, 0);
    ASSERT_EQUAL(h[0].z, 0);
    ASSERT_EQUAL(static_cast<T>(d[0]).x, 0);
    ASSERT_EQUAL(static_cast<T>(d[0]).y, 0);
    ASSERT_EQUAL(static_cast<T>(d[0]).z, 0);

    T val;
    val.x = 10;
    val.y = 20;
    val.z = -1;

    thrust::fill(h.begin(), h.end(), val);
    thrust::fill(d.begin(), d.end(), val);

    ASSERT_EQUAL(h[0].x, 10);
    ASSERT_EQUAL(h[0].y, 20);
    ASSERT_EQUAL(h[0].z, -1);
    ASSERT_EQUAL(static_cast<T>(d[0]).x, 10);
    ASSERT_EQUAL(static_cast<T>(d[0]).y, 20);
    ASSERT_EQUAL(static_cast<T>(d[0]).z, -1);
};
DECLARE_UNITTEST(TestFillWithTrivialAssignment);


struct TypeWithNonTrivialAssigment
{
  int x, y, z;

  __host__ __device__
  TypeWithNonTrivialAssigment() : x(0), y(0), z(0) {}

  __host__ __device__
  TypeWithNonTrivialAssigment& operator=(const TypeWithNonTrivialAssigment& t)
  {
    x = t.x;
    y = t.y;
    z = t.x + t.y;
    return *this;
  }
  
  __host__ __device__
  bool operator==(const TypeWithNonTrivialAssigment& t) const
  {
    return x == t.x && y == t.y && z == t.z;
  }
};

void TestFillWithNonTrivialAssignment(void)
{
    typedef TypeWithNonTrivialAssigment T;

    thrust::host_vector<T>   h(1);
    thrust::device_vector<T> d(1);
    
    ASSERT_EQUAL(h[0].x, 0);
    ASSERT_EQUAL(h[0].y, 0);
    ASSERT_EQUAL(h[0].z, 0);
    ASSERT_EQUAL(static_cast<T>(d[0]).x, 0);
    ASSERT_EQUAL(static_cast<T>(d[0]).y, 0);
    ASSERT_EQUAL(static_cast<T>(d[0]).z, 0);

    T val;
    val.x = 10;
    val.y = 20;
    val.z = -1;

    thrust::fill(h.begin(), h.end(), val);
    thrust::fill(d.begin(), d.end(), val);

    ASSERT_EQUAL(h[0].x, 10);
    ASSERT_EQUAL(h[0].y, 20);
    ASSERT_EQUAL(h[0].z, 30);
    ASSERT_EQUAL(static_cast<T>(d[0]).x, 10);
    ASSERT_EQUAL(static_cast<T>(d[0]).y, 20);
    ASSERT_EQUAL(static_cast<T>(d[0]).z, 30);
};
DECLARE_UNITTEST(TestFillWithNonTrivialAssignment);


struct my_tag : thrust::device_system_tag {};

template<typename ForwardIterator, typename T>
void fill(my_tag, ForwardIterator first, ForwardIterator, const T&)
{
    *first = 13;
}

void TestFillDispatch()
{
    thrust::device_vector<int> vec(1);

    thrust::fill(thrust::retag<my_tag>(vec.begin()),
                 thrust::retag<my_tag>(vec.end()),
                 0);

    ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestFillDispatch);


template<typename OutputIterator, typename Size, typename T>
OutputIterator fill_n(my_tag, OutputIterator first, Size, const T&)
{
    *first = 13;
    return first;
}

void TestFillNDispatch()
{
    thrust::device_vector<int> vec(1);

    thrust::fill_n(thrust::retag<my_tag>(vec.begin()),
                   vec.size(),
                   0);

    ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestFillNDispatch);


__THRUST_DISABLE_MSVC_POSSIBLE_LOSS_OF_DATA_WARNING_END
