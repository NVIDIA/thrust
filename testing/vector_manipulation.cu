#include <unittest/unittest.h>
#include <thrust/device_malloc_allocator.h>
#include <vector>

template <class Vector>
void TestVectorManipulation(size_t n)
{
    typedef typename Vector::iterator   Iterator;
    typedef typename Vector::value_type T;

    thrust::host_vector<T> src = unittest::random_samples<T>(n);
    ASSERT_EQUAL(src.size(), n);

    // basic initialization
    Vector test0(n);
    Vector test1(n, (T) 3);
    ASSERT_EQUAL(test0.size(), n);
    ASSERT_EQUAL(test1.size(), n);
    ASSERT_EQUAL((test1 == std::vector<T>(n, (T) 3)), true);

#if (THRUST_HOST_COMPILER == THRUST_HOST_COMPILER_MSVC) && (_MSC_VER <= 1400)
    // XXX MSVC 2005's STL unintentionally uses adl to dispatch advance which
    //     produces an ambiguity between std::advance & thrust::advance
    //     don't produce a KNOWN_FAILURE, just ignore the issue
#else
    // initializing from other vector
    std::vector<T> stl_vector(src.begin(), src.end());
    Vector cpy0 = src;
    Vector cpy1(stl_vector);
    Vector cpy2(stl_vector.begin(), stl_vector.end());
    ASSERT_EQUAL(cpy0, src);
    ASSERT_EQUAL(cpy1, src);
    ASSERT_EQUAL(cpy2, src);
#endif

    // resizing
    Vector vec1(src);
    vec1.resize(n + 3);
    ASSERT_EQUAL(vec1.size(), n + 3);
    vec1.resize(n);
    ASSERT_EQUAL(vec1.size(), n);
    ASSERT_EQUAL(vec1, src); 
    
    vec1.resize(n + 20, (T) 11);
    Vector tail(vec1.begin() + n, vec1.end());
    ASSERT_EQUAL( (tail == std::vector<T>(20, (T) 11)), true);

    // shrinking a vector should not invalidate iterators
    Iterator first = vec1.begin();
    vec1.resize(10);
    ASSERT_EQUAL_QUIET(first, vec1.begin());

    vec1.resize(0);
    ASSERT_EQUAL(vec1.size(), 0);
    ASSERT_EQUAL(vec1.empty(), true);
    vec1.resize(10);
    ASSERT_EQUAL(vec1.size(), 10);
    vec1.clear();
    ASSERT_EQUAL(vec1.size(), 0);
    vec1.resize(5);
    ASSERT_EQUAL(vec1.size(), 5);

    // push_back
    Vector vec2;
    for(size_t i = 0; i < 10; ++i)
    {
        ASSERT_EQUAL(vec2.size(), i);
        vec2.push_back( (T) i );
        ASSERT_EQUAL(vec2.size(), i + 1);
        for(size_t j = 0; j <= i; j++)
            ASSERT_EQUAL(vec2[j],     j);
        ASSERT_EQUAL(vec2.back(), i);
    }

    // pop_back
    for(size_t i = 10; i > 0; --i)
    {
        ASSERT_EQUAL(vec2.size(), i);
        ASSERT_EQUAL(vec2.back(), i-1);
        vec2.pop_back();
        ASSERT_EQUAL(vec2.size(), i-1);
        for(size_t j = 0; j < i; j++)
            ASSERT_EQUAL(vec2[j], j);
    }

    //TODO test swap, erase(pos), erase(begin, end)
}

template <typename T>
void TestVectorManipulationHost(size_t n)
{
    TestVectorManipulation< thrust::host_vector<T> >(n);
}
DECLARE_VARIABLE_UNITTEST(TestVectorManipulationHost);

template <typename T>
void TestVectorManipulationDevice(size_t n)
{
    TestVectorManipulation< thrust::device_vector<T> >(n);
}
DECLARE_VARIABLE_UNITTEST(TestVectorManipulationDevice);

