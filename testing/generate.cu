#include <unittest/unittest.h>
#include <thrust/generate.h>
#include <thrust/iterator/discard_iterator.h>

__THRUST_DISABLE_MSVC_POSSIBLE_LOSS_OF_DATA_WARNING_BEGIN

// for testing dispatch
struct my_tag : thrust::device_system_tag {};

template<typename T>
struct return_value
{
    T val;

    return_value(void){}
    return_value(T v):val(v){}

    __host__ __device__
    T operator()(void){ return val; }
};

template<class Vector>
void TestGenerateSimple(void)
{
    typedef typename Vector::value_type T;

    Vector result(5);

    T value = 13;

    return_value<T> f(value);

    thrust::generate(result.begin(), result.end(), f);

    ASSERT_EQUAL(result[0], value);
    ASSERT_EQUAL(result[1], value);
    ASSERT_EQUAL(result[2], value);
    ASSERT_EQUAL(result[3], value);
    ASSERT_EQUAL(result[4], value);
}
DECLARE_VECTOR_UNITTEST(TestGenerateSimple);

template<typename ForwardIterator, typename Generator>
void generate(my_tag, ForwardIterator first, ForwardIterator, Generator)
{
    *first = 13;
}

void TestGenerateDispatch()
{
    thrust::device_vector<int> vec(1);

    thrust::generate(thrust::retag<my_tag>(vec.begin()),
                     thrust::retag<my_tag>(vec.end()),
                     0);

    ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestGenerateDispatch);

template <typename T>
void TestGenerate(const size_t n)
{
    thrust::host_vector<T> h_result(n);
    thrust::device_vector<T> d_result(n);

    T value = 13;
    return_value<T> f(value);

    thrust::generate(h_result.begin(), h_result.end(), f);
    thrust::generate(d_result.begin(), d_result.end(), f);

    ASSERT_EQUAL(h_result, d_result);
}
DECLARE_VARIABLE_UNITTEST(TestGenerate);

template <typename T>
void TestGenerateToDiscardIterator(const size_t n)
{
    T value = 13;
    return_value<T> f(value);

    thrust::discard_iterator<thrust::host_system_tag> h_first;
    thrust::generate(h_first, h_first + 10, f);

    thrust::discard_iterator<thrust::device_system_tag> d_first;
    thrust::generate(d_first, d_first + 10, f);

    // there's nothing to actually check except that it compiles
}
DECLARE_VARIABLE_UNITTEST(TestGenerateToDiscardIterator);

template<class Vector>
void TestGenerateNSimple(void)
{
    typedef typename Vector::value_type T;

    Vector result(5);

    T value = 13;

    return_value<T> f(value);

    thrust::generate_n(result.begin(), result.size(), f);

    ASSERT_EQUAL(result[0], value);
    ASSERT_EQUAL(result[1], value);
    ASSERT_EQUAL(result[2], value);
    ASSERT_EQUAL(result[3], value);
    ASSERT_EQUAL(result[4], value);
}
DECLARE_VECTOR_UNITTEST(TestGenerateNSimple);

template<typename ForwardIterator, typename Size, typename Generator>
ForwardIterator generate_n(my_tag, ForwardIterator first, Size, Generator)
{
    *first = 13;
    return first;
}

void TestGenerateNDispatch()
{
    thrust::device_vector<int> vec(1);

    thrust::generate_n(thrust::retag<my_tag>(vec.begin()),
                       vec.size(),
                       0);

    ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestGenerateNDispatch);

template <typename T>
void TestGenerateNToDiscardIterator(const size_t n)
{
    T value = 13;
    return_value<T> f(value);

    thrust::discard_iterator<thrust::host_system_tag> h_result = 
      thrust::generate_n(thrust::discard_iterator<thrust::host_system_tag>(), n, f);

    thrust::discard_iterator<thrust::device_system_tag> d_result = 
      thrust::generate_n(thrust::discard_iterator<thrust::device_system_tag>(), n, f);

    thrust::discard_iterator<> reference(n);

    ASSERT_EQUAL_QUIET(reference, h_result);
    ASSERT_EQUAL_QUIET(reference, d_result);
}
DECLARE_VARIABLE_UNITTEST(TestGenerateNToDiscardIterator);


template <typename Vector>
void TestGenerateZipIterator(void)
{
    typedef typename Vector::value_type T;

    Vector v1(3,T(0));
    Vector v2(3,T(0));

    thrust::generate(thrust::make_zip_iterator(thrust::make_tuple(v1.begin(),v2.begin())),
                     thrust::make_zip_iterator(thrust::make_tuple(v1.end(),v2.end())),
                     return_value< thrust::tuple<T,T> > (thrust::tuple<T,T>(4,7)));

    ASSERT_EQUAL(v1[0], 4);
    ASSERT_EQUAL(v1[1], 4);
    ASSERT_EQUAL(v1[2], 4);
    ASSERT_EQUAL(v2[0], 7);
    ASSERT_EQUAL(v2[1], 7);
    ASSERT_EQUAL(v2[2], 7);
};
DECLARE_VECTOR_UNITTEST(TestGenerateZipIterator);


void TestGenerateTuple(void)
{
    typedef int T;
    typedef thrust::tuple<T,T> Tuple;

    thrust::host_vector<Tuple>   h(3, Tuple(0,0));
    thrust::device_vector<Tuple> d(3, Tuple(0,0));

    thrust::generate(h.begin(), h.end(), return_value<Tuple>(Tuple(4,7)));
    thrust::generate(d.begin(), d.end(), return_value<Tuple>(Tuple(4,7)));

    ASSERT_EQUAL_QUIET(h, d);
};
DECLARE_UNITTEST(TestGenerateTuple);

__THRUST_DISABLE_MSVC_POSSIBLE_LOSS_OF_DATA_WARNING_END
