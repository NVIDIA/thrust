#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/set_operations.h>
#include <thrust/functional.h>
#include <thrust/sort.h>


template<typename InputIterator1,
         typename InputIterator2,
         typename InputIterator3,
         typename InputIterator4,
         typename OutputIterator1,
         typename OutputIterator2>
thrust::pair<OutputIterator1,OutputIterator2>
  set_union_by_key(my_system &system,
                   InputIterator1,
                   InputIterator1,
                   InputIterator2,
                   InputIterator2,
                   InputIterator3,
                   InputIterator4,
                   OutputIterator1 keys_result,
                   OutputIterator2 values_result)
{
  system.validate_dispatch();
  return thrust::make_pair(keys_result, values_result);
}

void TestSetUnionByKeyDispatchExplicit()
{
  thrust::device_vector<int> vec(1);

  my_system sys(0);
  thrust::set_union_by_key(sys,
                           vec.begin(),
                           vec.begin(),
                           vec.begin(),
                           vec.begin(),
                           vec.begin(),
                           vec.begin(),
                           vec.begin(),
                           vec.begin());

  ASSERT_EQUAL(true, sys.is_valid());
}
DECLARE_UNITTEST(TestSetUnionByKeyDispatchExplicit);


template<typename InputIterator1,
         typename InputIterator2,
         typename InputIterator3,
         typename InputIterator4,
         typename OutputIterator1,
         typename OutputIterator2>
thrust::pair<OutputIterator1,OutputIterator2>
  set_union_by_key(my_tag,
                   InputIterator1,
                   InputIterator1,
                   InputIterator2,
                   InputIterator2,
                   InputIterator3,
                   InputIterator4,
                   OutputIterator1 keys_result,
                   OutputIterator2 values_result)
{
  *keys_result = 13;
  return thrust::make_pair(keys_result,values_result);
}

void TestSetUnionByKeyDispatchImplicit()
{
  thrust::device_vector<int> vec(1);

  thrust::set_union_by_key(thrust::retag<my_tag>(vec.begin()),
                           thrust::retag<my_tag>(vec.begin()),
                           thrust::retag<my_tag>(vec.begin()),
                           thrust::retag<my_tag>(vec.begin()),
                           thrust::retag<my_tag>(vec.begin()),
                           thrust::retag<my_tag>(vec.begin()),
                           thrust::retag<my_tag>(vec.begin()),
                           thrust::retag<my_tag>(vec.begin()));

  ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestSetUnionByKeyDispatchImplicit);


template<typename Vector>
void TestSetUnionByKeySimple(void)
{
  typedef typename Vector::iterator Iterator;

  Vector a_key(3), b_key(4);
  Vector a_val(3), b_val(4);

  a_key[0] = 0; a_key[1] = 2; a_key[2] = 4;
  a_val[0] = 0; a_val[1] = 0; a_val[2] = 0;

  b_key[0] = 0; b_key[1] = 3; b_key[2] = 3; b_key[3] = 4;
  b_val[0] = 1; b_val[1] = 1; b_val[2] = 1; b_val[3] = 1;

  Vector ref_key(5), ref_val(5);
  ref_key[0] = 0; ref_key[1] = 2; ref_key[2] = 3; ref_key[3] = 3; ref_key[4] = 4;
  ref_val[0] = 0; ref_val[1] = 0; ref_val[2] = 1; ref_val[3] = 1; ref_val[4] = 0;

  Vector result_key(5), result_val(5);

  thrust::pair<Iterator,Iterator> end =
    thrust::set_union_by_key(a_key.begin(), a_key.end(),
                             b_key.begin(), b_key.end(),
                             a_val.begin(),
                             b_val.begin(),
                             result_key.begin(),
                             result_val.begin());

  ASSERT_EQUAL_QUIET(result_key.end(), end.first);
  ASSERT_EQUAL_QUIET(result_val.end(), end.second);
  ASSERT_EQUAL(ref_key, result_key);
  ASSERT_EQUAL(ref_val, result_val);
}
DECLARE_VECTOR_UNITTEST(TestSetUnionByKeySimple);


template<typename T>
void TestSetUnionByKey(const size_t n)
{
  thrust::host_vector<T> random_keys = unittest::random_integers<unittest::int8_t>(n);
  thrust::host_vector<T> random_vals = unittest::random_integers<unittest::int8_t>(n);

  size_t denominators[] = {1, 2, 3, 4, 5, 6, 7, 8, 9};
  size_t num_denominators = sizeof(denominators) / sizeof(size_t);

  for(size_t i = 0; i < num_denominators; ++i)
  {
    size_t size_a = n / denominators[i];

    thrust::host_vector<T> h_a_keys(random_keys.begin(), random_keys.begin() + size_a);
    thrust::host_vector<T> h_b_keys(random_keys.begin() + size_a, random_keys.end());

    thrust::host_vector<T> h_a_vals(random_vals.begin(), random_vals.begin() + size_a);
    thrust::host_vector<T> h_b_vals(random_vals.begin() + size_a, random_vals.end());

    thrust::stable_sort(h_a_keys.begin(), h_a_keys.end());
    thrust::stable_sort(h_b_keys.begin(), h_b_keys.end());

    thrust::device_vector<T> d_a_keys = h_a_keys;
    thrust::device_vector<T> d_b_keys = h_b_keys;

    thrust::device_vector<T> d_a_vals = h_a_vals;
    thrust::device_vector<T> d_b_vals = h_b_vals;

    size_t max_size = h_a_keys.size() + h_b_keys.size();

    thrust::host_vector<T> h_result_keys(max_size);
    thrust::host_vector<T> h_result_vals(max_size);

    thrust::device_vector<T> d_result_keys(max_size);
    thrust::device_vector<T> d_result_vals(max_size);


    thrust::pair<
      typename thrust::host_vector<T>::iterator,
      typename thrust::host_vector<T>::iterator
    > h_end;

    thrust::pair<
      typename thrust::device_vector<T>::iterator,
      typename thrust::device_vector<T>::iterator
    > d_end;


    h_end = thrust::set_union_by_key(h_a_keys.begin(), h_a_keys.end(),
                                     h_b_keys.begin(), h_b_keys.end(),
                                     h_a_vals.begin(),
                                     h_b_vals.begin(),
                                     h_result_keys.begin(),
                                     h_result_vals.begin());
    h_result_keys.erase(h_end.first, h_result_keys.end());
    h_result_vals.erase(h_end.second, h_result_vals.end());

    d_end = thrust::set_union_by_key(d_a_keys.begin(), d_a_keys.end(),
                                     d_b_keys.begin(), d_b_keys.end(),
                                     d_a_vals.begin(),
                                     d_b_vals.begin(),
                                     d_result_keys.begin(),
                                     d_result_vals.begin());
    d_result_keys.erase(d_end.first, d_result_keys.end());
    d_result_vals.erase(d_end.second, d_result_vals.end());

    ASSERT_EQUAL(h_result_keys, d_result_keys);
    ASSERT_EQUAL(h_result_vals, d_result_vals);
  }
}
DECLARE_VARIABLE_UNITTEST(TestSetUnionByKey);


template<typename T>
void TestSetUnionByKeyEquivalentRanges(const size_t n)
{
  thrust::host_vector<T> temp = unittest::random_integers<T>(n);

  thrust::host_vector<T> h_a_key = temp;
  thrust::sort(h_a_key.begin(), h_a_key.end());
  thrust::host_vector<T> h_b_key = h_a_key;

  thrust::host_vector<T> h_a_val = unittest::random_integers<T>(n);
  thrust::host_vector<T> h_b_val = unittest::random_integers<T>(n);

  thrust::device_vector<T> d_a_key = h_a_key;
  thrust::device_vector<T> d_b_key = h_b_key;

  thrust::device_vector<T> d_a_val = h_a_val;
  thrust::device_vector<T> d_b_val = h_b_val;

  size_t max_size = h_a_key.size() + h_b_key.size();

  thrust::host_vector<T>   h_result_key(max_size), h_result_val(max_size);
  thrust::device_vector<T> d_result_key(max_size), d_result_val(max_size);

  thrust::pair<
    typename thrust::host_vector<T>::iterator,
    typename thrust::host_vector<T>::iterator
  > h_end;
  
  thrust::pair<
    typename thrust::device_vector<T>::iterator,
    typename thrust::device_vector<T>::iterator
  > d_end;
  
  h_end = thrust::set_union_by_key(h_a_key.begin(), h_a_key.end(),
                                   h_b_key.begin(), h_b_key.end(),
                                   h_a_val.begin(),
                                   h_b_val.begin(),
                                   h_result_key.begin(),
                                   h_result_val.begin());
  h_result_key.erase(h_end.first,  h_result_key.end());
  h_result_val.erase(h_end.second, h_result_val.end());

  d_end = thrust::set_union_by_key(d_a_key.begin(), d_a_key.end(),
                                   d_b_key.begin(), d_b_key.end(),
                                   d_a_val.begin(),
                                   d_b_val.begin(),
                                   d_result_key.begin(),
                                   d_result_val.begin());
  d_result_key.erase(d_end.first,  d_result_key.end());
  d_result_val.erase(d_end.second, d_result_val.end());

  ASSERT_EQUAL(h_result_key, d_result_key);
  ASSERT_EQUAL(h_result_val, d_result_val);
}
DECLARE_VARIABLE_UNITTEST(TestSetUnionByKeyEquivalentRanges);


template<typename T>
void TestSetUnionByKeyMultiset(const size_t n)
{
  thrust::host_vector<T> temp = unittest::random_integers<T>(2 * n);

  // restrict elements to [min,13)
  for(typename thrust::host_vector<T>::iterator i = temp.begin();
      i != temp.end();
      ++i)
  {
    int temp = static_cast<int>(*i);
    temp %= 13;
    *i = temp;
  }

  thrust::host_vector<T> h_a_key(temp.begin(), temp.begin() + n);
  thrust::host_vector<T> h_b_key(temp.begin() + n, temp.end());

  thrust::sort(h_a_key.begin(), h_a_key.end());
  thrust::sort(h_b_key.begin(), h_b_key.end());

  thrust::host_vector<T> h_a_val = unittest::random_integers<T>(n);
  thrust::host_vector<T> h_b_val = unittest::random_integers<T>(n);

  thrust::device_vector<T> d_a_key = h_a_key;
  thrust::device_vector<T> d_b_key = h_b_key;

  thrust::device_vector<T> d_a_val = h_a_val;
  thrust::device_vector<T> d_b_val = h_b_val;

  size_t max_size = h_a_key.size() + h_b_key.size();
  thrust::host_vector<T>   h_result_key(max_size), h_result_val(max_size);
  thrust::device_vector<T> d_result_key(max_size), d_result_val(max_size);

  thrust::pair<
    typename thrust::host_vector<T>::iterator,
    typename thrust::host_vector<T>::iterator
  > h_end;

  thrust::pair<
    typename thrust::device_vector<T>::iterator,
    typename thrust::device_vector<T>::iterator
  > d_end;
  
  h_end = thrust::set_union_by_key(h_a_key.begin(), h_a_key.end(),
                                   h_b_key.begin(), h_b_key.end(),
                                   h_a_val.begin(),
                                   h_b_val.begin(),
                                   h_result_key.begin(),
                                   h_result_val.begin());
  h_result_key.erase(h_end.first,  h_result_key.end());
  h_result_val.erase(h_end.second, h_result_val.end());

  d_end = thrust::set_union_by_key(d_a_key.begin(), d_a_key.end(),
                                   d_b_key.begin(), d_b_key.end(),
                                   d_a_val.begin(),
                                   d_b_val.begin(),
                                   d_result_key.begin(),
                                   d_result_val.begin());
  d_result_key.erase(d_end.first,  d_result_key.end());
  d_result_val.erase(d_end.second, d_result_val.end());

  ASSERT_EQUAL(h_result_key, d_result_key);
  ASSERT_EQUAL(h_result_val, d_result_val);
}
DECLARE_VARIABLE_UNITTEST(TestSetUnionByKeyMultiset);

