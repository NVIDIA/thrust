#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/transform.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/tuple.h>
#include <thrust/pair.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/retag.h>


template <class Vector>
void TestTransformUnarySimple(void)
{
    typedef typename Vector::value_type T;
    
    typename Vector::iterator iter;

    Vector input(3);
    Vector output(3);
    Vector result(3);
    input[0]  =  1; input[1]  = -2; input[2]  =  3;
    result[0] = -1; result[1] =  2; result[2] = -3;

    iter = thrust::transform(input.begin(), input.end(), output.begin(), thrust::negate<T>());
    
    ASSERT_EQUAL(iter - output.begin(), input.size());
    ASSERT_EQUAL(output, result);
}
DECLARE_VECTOR_UNITTEST(TestTransformUnarySimple);


template<typename InputIterator,
         typename OutputIterator,
         typename UnaryFunction>
OutputIterator transform(my_system &system, InputIterator, InputIterator, OutputIterator result, UnaryFunction)
{
    system.validate_dispatch();
    return result;
}

void TestTransformUnaryDispatchExplicit()
{
    thrust::device_vector<int> vec(1);

    my_system sys(0);
    thrust::transform(sys,
                      vec.begin(),
                      vec.begin(),
                      vec.begin(),
                      0);

    ASSERT_EQUAL(true, sys.is_valid());
}
DECLARE_UNITTEST(TestTransformUnaryDispatchExplicit);


template<typename InputIterator,
         typename OutputIterator,
         typename UnaryFunction>
OutputIterator transform(my_tag, InputIterator, InputIterator, OutputIterator result, UnaryFunction)
{
    *result = 13;
    return result;
}

void TestTransformUnaryDispatchImplicit()
{
    thrust::device_vector<int> vec(1);

    thrust::transform(thrust::retag<my_tag>(vec.begin()),
                      thrust::retag<my_tag>(vec.begin()),
                      thrust::retag<my_tag>(vec.begin()),
                      0);

    ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestTransformUnaryDispatchImplicit);


template <class Vector>
void TestTransformIfUnaryNoStencilSimple(void)
{
    typedef typename Vector::value_type T;
    
    typename Vector::iterator iter;

    Vector input(3);
    Vector output(3);
    Vector result(3);

    input[0]   =  0; input[1]   = -2; input[2]   =  0;
    output[0]  = -1; output[1]  = -2; output[2]  = -3; 
    result[0]  = -1; result[1]  =  2; result[2]  = -3;

    iter = thrust::transform_if(input.begin(), input.end(),
                                output.begin(),
                                thrust::negate<T>(),
                                thrust::identity<T>());
    
    ASSERT_EQUAL(iter - output.begin(), input.size());
    ASSERT_EQUAL(output, result);
}
DECLARE_VECTOR_UNITTEST(TestTransformIfUnaryNoStencilSimple);


template<typename InputIterator,
         typename ForwardIterator,
         typename UnaryFunction,
         typename Predicate>
ForwardIterator transform_if(my_system &system,
                             InputIterator,
                             InputIterator,
                             ForwardIterator result,
                             UnaryFunction,
                             Predicate)
{
    system.validate_dispatch();
    return result;
}

void TestTransformIfUnaryNoStencilDispatchExplicit()
{
    thrust::device_vector<int> vec(1);

    my_system sys(0);
    thrust::transform_if(sys,
                         vec.begin(),
                         vec.begin(),
                         vec.begin(),
                         vec.begin(),
                         0);

    ASSERT_EQUAL(true, sys.is_valid());
}
DECLARE_UNITTEST(TestTransformIfUnaryNoStencilDispatchExplicit);


template<typename InputIterator,
         typename ForwardIterator,
         typename UnaryFunction,
         typename Predicate>
ForwardIterator transform_if(my_tag,
                             InputIterator,
                             InputIterator,
                             ForwardIterator result,
                             UnaryFunction,
                             Predicate)
{
    *result = 13;
    return result;
}

void TestTransformIfUnaryNoStencilDispatchImplicit()
{
    thrust::device_vector<int> vec(1);

    thrust::transform_if(thrust::retag<my_tag>(vec.begin()),
                         thrust::retag<my_tag>(vec.begin()),
                         thrust::retag<my_tag>(vec.begin()),
                         thrust::retag<my_tag>(vec.begin()),
                         0);

    ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestTransformIfUnaryNoStencilDispatchImplicit);


template <class Vector>
void TestTransformIfUnarySimple(void)
{
    typedef typename Vector::value_type T;
    
    typename Vector::iterator iter;

    Vector input(3);
    Vector stencil(3);
    Vector output(3);
    Vector result(3);

    input[0]   =  1; input[1]   = -2; input[2]   =  3;
    output[0]  =  1; output[1]  =  2; output[2]  =  3; 
    stencil[0] =  1; stencil[1] =  0; stencil[2] =  1;
    result[0]  = -1; result[1]  =  2; result[2]  = -3;

    iter = thrust::transform_if(input.begin(), input.end(),
                                stencil.begin(),
                                output.begin(),
                                thrust::negate<T>(),
                                thrust::identity<T>());
    
    ASSERT_EQUAL(iter - output.begin(), input.size());
    ASSERT_EQUAL(output, result);
}
DECLARE_VECTOR_UNITTEST(TestTransformIfUnarySimple);


template<typename InputIterator1,
         typename InputIterator2,
         typename ForwardIterator,
         typename UnaryFunction,
         typename Predicate>
ForwardIterator transform_if(my_system &system,
                             InputIterator1,
                             InputIterator1,
                             ForwardIterator result,
                             UnaryFunction,
                             Predicate)
{
    system.validate_dispatch();
    return result;
}

void TestTransformIfUnaryDispatchExplicit()
{
    thrust::device_vector<int> vec(1);

    my_system sys(0);
    thrust::transform_if(sys,
                         vec.begin(),
                         vec.begin(),
                         vec.begin(),
                         0,
                         0);

    ASSERT_EQUAL(true, sys.is_valid());
}
DECLARE_UNITTEST(TestTransformIfUnaryDispatchExplicit);


template<typename InputIterator1,
         typename InputIterator2,
         typename ForwardIterator,
         typename UnaryFunction,
         typename Predicate>
ForwardIterator transform_if(my_tag,
                             InputIterator1,
                             InputIterator1,
                             ForwardIterator result,
                             UnaryFunction,
                             Predicate)
{
    *result = 13;
    return result;
}

void TestTransformIfUnaryDispatchImplicit()
{
    thrust::device_vector<int> vec(1);

    thrust::transform_if(thrust::retag<my_tag>(vec.begin()),
                         thrust::retag<my_tag>(vec.begin()),
                         thrust::retag<my_tag>(vec.begin()),
                         0,
                         0);

    ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestTransformIfUnaryDispatchImplicit);


template <class Vector>
void TestTransformBinarySimple(void)
{
    typedef typename Vector::value_type T;

    typename Vector::iterator iter;

    Vector input1(3);
    Vector input2(3);
    Vector output(3);
    Vector result(3);
    input1[0] =  1; input1[1] = -2; input1[2] =  3;
    input2[0] = -4; input2[1] =  5; input2[2] =  6;
    result[0] =  5; result[1] = -7; result[2] = -3;

    iter = thrust::transform(input1.begin(), input1.end(), input2.begin(), output.begin(), thrust::minus<T>());
    
    ASSERT_EQUAL(iter - output.begin(), input1.size());
    ASSERT_EQUAL(output, result);
}
DECLARE_VECTOR_UNITTEST(TestTransformBinarySimple);


template<typename InputIterator1,
         typename InputIterator2,
         typename OutputIterator,
         typename UnaryFunction>
OutputIterator transform(my_system &system, InputIterator1, InputIterator1, InputIterator2, OutputIterator result, UnaryFunction)
{
    system.validate_dispatch();
    return result;
}

void TestTransformBinaryDispatchExplicit()
{
    thrust::device_vector<int> vec(1);

    my_system sys(0);
    thrust::transform(sys,
                      vec.begin(),
                      vec.begin(),
                      vec.begin(),
                      vec.begin(),
                      0);

    ASSERT_EQUAL(true, sys.is_valid());
}
DECLARE_UNITTEST(TestTransformBinaryDispatchExplicit);


template<typename InputIterator1,
         typename InputIterator2,
         typename OutputIterator,
         typename UnaryFunction>
OutputIterator transform(my_tag, InputIterator1, InputIterator1, InputIterator2, OutputIterator result, UnaryFunction)
{
    *result = 13;
    return result;
}

void TestTransformBinaryDispatchImplicit()
{
    thrust::device_vector<int> vec(1);

    thrust::transform(thrust::retag<my_tag>(vec.begin()),
                      thrust::retag<my_tag>(vec.begin()),
                      thrust::retag<my_tag>(vec.begin()),
                      thrust::retag<my_tag>(vec.begin()),
                      0);

    ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestTransformBinaryDispatchImplicit);




template <class Vector>
void TestTransformIfBinarySimple(void)
{
    typedef typename Vector::value_type T;
    
    typename Vector::iterator iter;

    Vector input1(3);
    Vector input2(3);
    Vector stencil(3);
    Vector output(3);
    Vector result(3);

    input1[0]  =  1; input1[1]  = -2; input1[2]  =  3;
    input2[0]  = -4; input2[1]  =  5; input2[2]  =  6;
    stencil[0] =  0; stencil[1] =  1; stencil[2] =  0;
    output[0]  =  1; output[1]  =  2; output[2]  =  3;
    result[0]  =  5; result[1]  =  2; result[2]  = -3;

    thrust::identity<T> identity;

    iter = thrust::transform_if(input1.begin(), input1.end(),
                                input2.begin(),
                                stencil.begin(),
                                output.begin(),
                                thrust::minus<T>(),
                                thrust::not1(identity));
    
    ASSERT_EQUAL(iter - output.begin(), input1.size());
    ASSERT_EQUAL(output, result);
}
DECLARE_VECTOR_UNITTEST(TestTransformIfBinarySimple);


template<typename InputIterator1,
         typename InputIterator2,
         typename InputIterator3,
         typename ForwardIterator,
         typename BinaryFunction,
         typename Predicate>
ForwardIterator transform_if(my_system &system,
                             InputIterator1,
                             InputIterator1,
                             InputIterator2,
                             InputIterator3,
                             ForwardIterator result,
                             BinaryFunction,
                             Predicate)
{
    system.validate_dispatch();
    return result;
}

void TestTransformIfBinaryDispatchExplicit()
{
    thrust::device_vector<int> vec(1);

    my_system sys(0);
    thrust::transform_if(sys,
                         vec.begin(),
                         vec.begin(),
                         vec.begin(),
                         vec.begin(),
                         vec.begin(),
                         0,
                         0);

    ASSERT_EQUAL(true, sys.is_valid());
}
DECLARE_UNITTEST(TestTransformIfBinaryDispatchExplicit);


template<typename InputIterator1,
         typename InputIterator2,
         typename InputIterator3,
         typename ForwardIterator,
         typename BinaryFunction,
         typename Predicate>
ForwardIterator transform_if(my_tag,
                             InputIterator1,
                             InputIterator1,
                             InputIterator2,
                             InputIterator3,
                             ForwardIterator result,
                             BinaryFunction,
                             Predicate)
{
    *result = 13;
    return result;
}

void TestTransformIfBinaryDispatchImplicit()
{
    thrust::device_vector<int> vec(1);

    thrust::transform_if(thrust::retag<my_tag>(vec.begin()),
                         thrust::retag<my_tag>(vec.begin()),
                         thrust::retag<my_tag>(vec.begin()),
                         thrust::retag<my_tag>(vec.begin()),
                         thrust::retag<my_tag>(vec.begin()),
                         0,
                         0);

    ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestTransformIfBinaryDispatchImplicit);


template <typename T>
void TestTransformUnary(const size_t n)
{
    thrust::host_vector<T>   h_input = unittest::random_integers<T>(n);
    thrust::device_vector<T> d_input = h_input;

    thrust::host_vector<T>   h_output(n);
    thrust::device_vector<T> d_output(n);

    thrust::transform(h_input.begin(), h_input.end(), h_output.begin(), thrust::negate<T>());
    thrust::transform(d_input.begin(), d_input.end(), d_output.begin(), thrust::negate<T>());
    
    ASSERT_EQUAL(h_output, d_output);
}
DECLARE_VARIABLE_UNITTEST(TestTransformUnary);


template <typename T>
void TestTransformUnaryToDiscardIterator(const size_t n)
{
    thrust::host_vector<T>   h_input = unittest::random_integers<T>(n);
    thrust::device_vector<T> d_input = h_input;

    thrust::discard_iterator<> h_result =
      thrust::transform(h_input.begin(), h_input.end(), thrust::make_discard_iterator(), thrust::negate<T>());

    thrust::discard_iterator<> d_result =
      thrust::transform(d_input.begin(), d_input.end(), thrust::make_discard_iterator(), thrust::negate<T>());

    thrust::discard_iterator<> reference(n);
    
    ASSERT_EQUAL_QUIET(reference, h_result);
    ASSERT_EQUAL_QUIET(reference, d_result);
}
DECLARE_VARIABLE_UNITTEST(TestTransformUnaryToDiscardIterator);


struct repeat2
{
  template<typename T>
  __host__ __device__
  thrust::pair<T,T> operator()(T x)
  {
    return thrust::make_pair(x,x);
  }
};


template<typename T>
void TestTransformUnaryToDiscardIteratorZipped(const size_t n)
{
    thrust::host_vector<T>   h_input = unittest::random_integers<T>(n);
    thrust::device_vector<T> d_input = h_input;

    thrust::host_vector<T>   h_output(n);
    thrust::device_vector<T> d_output(n);

    typedef typename thrust::host_vector<T>::iterator Iterator1;
    typedef typename thrust::device_vector<T>::iterator Iterator2;

    typedef thrust::tuple<Iterator1,thrust::discard_iterator<> > Tuple1;
    typedef thrust::tuple<Iterator2,thrust::discard_iterator<> > Tuple2;

    typedef thrust::zip_iterator<Tuple1> ZipIterator1;
    typedef thrust::zip_iterator<Tuple2> ZipIterator2;

    ZipIterator1 z1(thrust::make_tuple(h_output.begin(), thrust::make_discard_iterator()));
    ZipIterator2 z2(thrust::make_tuple(d_output.begin(), thrust::make_discard_iterator()));

    ZipIterator1 h_result =
      thrust::transform(h_input.begin(), h_input.end(), z1, repeat2());

    ZipIterator2 d_result =
      thrust::transform(d_input.begin(), d_input.end(), z2, repeat2());

    thrust::discard_iterator<> reference(n);

    ASSERT_EQUAL(h_output, d_output);
    
    ASSERT_EQUAL_QUIET(reference, thrust::get<1>(h_result.get_iterator_tuple()));
    ASSERT_EQUAL_QUIET(reference, thrust::get<1>(d_result.get_iterator_tuple()));
}
DECLARE_VARIABLE_UNITTEST(TestTransformUnaryToDiscardIteratorZipped);

struct is_positive
{
  template<typename T>
  __host__ __device__
  bool operator()(T &x)
  {
    return x > 0;
  } // end operator()()
}; // end is_positive


template <typename T>
void TestTransformIfUnaryNoStencil(const size_t n)
{
    thrust::host_vector<T>   h_input   = unittest::random_integers<T>(n);
    thrust::host_vector<T>   h_output  = unittest::random_integers<T>(n);

    thrust::device_vector<T> d_input   = h_input;
    thrust::device_vector<T> d_output  = h_output;

    thrust::transform_if(h_input.begin(), h_input.end(),
                         h_output.begin(),
                         thrust::negate<T>(), is_positive());

    thrust::transform_if(d_input.begin(), d_input.end(),
                         d_output.begin(),
                         thrust::negate<T>(), is_positive());
    
    ASSERT_EQUAL(h_output, d_output);
}
DECLARE_VARIABLE_UNITTEST(TestTransformIfUnaryNoStencil);


template <typename T>
void TestTransformIfUnary(const size_t n)
{
    thrust::host_vector<T>   h_input   = unittest::random_integers<T>(n);
    thrust::host_vector<T>   h_stencil = unittest::random_integers<T>(n);
    thrust::host_vector<T>   h_output  = unittest::random_integers<T>(n);

    thrust::device_vector<T> d_input   = h_input;
    thrust::device_vector<T> d_stencil = h_stencil;
    thrust::device_vector<T> d_output  = h_output;

    thrust::transform_if(h_input.begin(), h_input.end(),
                          h_stencil.begin(),
                          h_output.begin(),
                          thrust::negate<T>(), is_positive());

    thrust::transform_if(d_input.begin(), d_input.end(),
                          d_stencil.begin(),
                          d_output.begin(),
                          thrust::negate<T>(), is_positive());
    
    ASSERT_EQUAL(h_output, d_output);
}
DECLARE_VARIABLE_UNITTEST(TestTransformIfUnary);


template <typename T>
void TestTransformIfUnaryToDiscardIterator(const size_t n)
{
    thrust::host_vector<T>   h_input   = unittest::random_integers<T>(n);
    thrust::host_vector<T>   h_stencil = unittest::random_integers<T>(n);

    thrust::device_vector<T> d_input   = h_input;
    thrust::device_vector<T> d_stencil = h_stencil;

    thrust::discard_iterator<> h_result =
      thrust::transform_if(h_input.begin(), h_input.end(),
                           h_stencil.begin(),
                           thrust::make_discard_iterator(),
                           thrust::negate<T>(), is_positive());

    thrust::discard_iterator<> d_result =
      thrust::transform_if(d_input.begin(), d_input.end(),
                           d_stencil.begin(),
                           thrust::make_discard_iterator(),
                           thrust::negate<T>(), is_positive());

    thrust::discard_iterator<> reference(n);
    
    ASSERT_EQUAL_QUIET(reference, h_result);
    ASSERT_EQUAL_QUIET(reference, d_result);
}
DECLARE_VARIABLE_UNITTEST(TestTransformIfUnaryToDiscardIterator);


template <typename T>
void TestTransformBinary(const size_t n)
{
    thrust::host_vector<T>   h_input1 = unittest::random_integers<T>(n);
    thrust::host_vector<T>   h_input2 = unittest::random_integers<T>(n);
    thrust::device_vector<T> d_input1 = h_input1;
    thrust::device_vector<T> d_input2 = h_input2;

    thrust::host_vector<T>   h_output(n);
    thrust::device_vector<T> d_output(n);

    thrust::transform(h_input1.begin(), h_input1.end(), h_input2.begin(), h_output.begin(), thrust::minus<T>());
    thrust::transform(d_input1.begin(), d_input1.end(), d_input2.begin(), d_output.begin(), thrust::minus<T>());
    
    ASSERT_EQUAL(h_output, d_output);
    
    thrust::transform(h_input1.begin(), h_input1.end(), h_input2.begin(), h_output.begin(), thrust::multiplies<T>());
    thrust::transform(d_input1.begin(), d_input1.end(), d_input2.begin(), d_output.begin(), thrust::multiplies<T>());
    
    ASSERT_EQUAL(h_output, d_output);
}
DECLARE_VARIABLE_UNITTEST(TestTransformBinary);


template <typename T>
void TestTransformBinaryToDiscardIterator(const size_t n)
{
    thrust::host_vector<T>   h_input1 = unittest::random_integers<T>(n);
    thrust::host_vector<T>   h_input2 = unittest::random_integers<T>(n);
    thrust::device_vector<T> d_input1 = h_input1;
    thrust::device_vector<T> d_input2 = h_input2;

    thrust::discard_iterator<> h_result =
      thrust::transform(h_input1.begin(), h_input1.end(), h_input2.begin(), thrust::make_discard_iterator(), thrust::minus<T>());
    thrust::discard_iterator<> d_result =
      thrust::transform(d_input1.begin(), d_input1.end(), d_input2.begin(), thrust::make_discard_iterator(), thrust::minus<T>());

    thrust::discard_iterator<> reference(n);
    
    ASSERT_EQUAL_QUIET(reference, h_result);
    ASSERT_EQUAL_QUIET(reference, d_result);
}
DECLARE_VARIABLE_UNITTEST(TestTransformBinaryToDiscardIterator);


template <typename T>
void TestTransformIfBinary(const size_t n)
{
    thrust::host_vector<T>   h_input1  = unittest::random_integers<T>(n);
    thrust::host_vector<T>   h_input2  = unittest::random_integers<T>(n);
    thrust::host_vector<T>   h_stencil = unittest::random_integers<T>(n);
    thrust::host_vector<T>   h_output  = unittest::random_integers<T>(n);

    thrust::device_vector<T> d_input1  = h_input1;
    thrust::device_vector<T> d_input2  = h_input2;
    thrust::device_vector<T> d_stencil = h_stencil;
    thrust::device_vector<T> d_output  = h_output;

    thrust::transform_if(h_input1.begin(), h_input1.end(),
                         h_input2.begin(),
                         h_stencil.begin(),
                         h_output.begin(),
                         thrust::minus<T>(), is_positive());

    thrust::transform_if(d_input1.begin(), d_input1.end(),
                         d_input2.begin(),
                         d_stencil.begin(),
                         d_output.begin(),
                         thrust::minus<T>(), is_positive());
    
    ASSERT_EQUAL(h_output, d_output);

    h_stencil = unittest::random_integers<T>(n);
    d_stencil = h_stencil;
    
    thrust::transform_if(h_input1.begin(), h_input1.end(),
                         h_input2.begin(),
                         h_stencil.begin(),
                         h_output.begin(),
                         thrust::multiplies<T>(), is_positive());

    thrust::transform_if(d_input1.begin(), d_input1.end(),
                         d_input2.begin(),
                         d_stencil.begin(),
                         d_output.begin(),
                         thrust::multiplies<T>(), is_positive());
    
    ASSERT_EQUAL(h_output, d_output);
}
DECLARE_VARIABLE_UNITTEST(TestTransformIfBinary);


template <typename T>
void TestTransformIfBinaryToDiscardIterator(const size_t n)
{
    thrust::host_vector<T>   h_input1  = unittest::random_integers<T>(n);
    thrust::host_vector<T>   h_input2  = unittest::random_integers<T>(n);
    thrust::host_vector<T>   h_stencil = unittest::random_integers<T>(n);

    thrust::device_vector<T> d_input1  = h_input1;
    thrust::device_vector<T> d_input2  = h_input2;
    thrust::device_vector<T> d_stencil = h_stencil;

    thrust::discard_iterator<> h_result =
      thrust::transform_if(h_input1.begin(), h_input1.end(),
                           h_input2.begin(),
                           h_stencil.begin(),
                           thrust::make_discard_iterator(),
                           thrust::minus<T>(), is_positive());

    thrust::discard_iterator<> d_result =
      thrust::transform_if(d_input1.begin(), d_input1.end(),
                           d_input2.begin(),
                           d_stencil.begin(),
                           thrust::make_discard_iterator(),
                           thrust::minus<T>(), is_positive());

    thrust::discard_iterator<> reference(n);
    
    ASSERT_EQUAL_QUIET(reference, h_result);
    ASSERT_EQUAL_QUIET(reference, d_result);
}
DECLARE_VARIABLE_UNITTEST(TestTransformIfBinaryToDiscardIterator);


template <class T>
  void TestTransformUnaryCountingIterator(size_t n)
{
    // GCC 4.4.x has a known failure with auto-vectorization (due to -O3 or -ftree-vectorize) of this test
    // See https://gcc.gnu.org/bugzilla/show_bug.cgi?id=43251
#if (__GNUC__ * 10000 + __GNUC_MINOR__ * 100) == 40400
    KNOWN_FAILURE;
#else
    // be careful not to generate a range larger than we can represent
    n = thrust::min<size_t>(n, std::numeric_limits<T>::max());

    thrust::counting_iterator<T, thrust::host_system_tag>   h_first = thrust::make_counting_iterator<T>(0);
    thrust::counting_iterator<T, thrust::device_system_tag> d_first = thrust::make_counting_iterator<T>(0);

    thrust::host_vector<T>   h_result(n);
    thrust::device_vector<T> d_result(n);

    thrust::transform(h_first, h_first + n, h_result.begin(), thrust::identity<T>());
    thrust::transform(d_first, d_first + n, d_result.begin(), thrust::identity<T>());

    ASSERT_EQUAL(h_result, d_result);
#endif
}
DECLARE_VARIABLE_UNITTEST(TestTransformUnaryCountingIterator);

template <typename T>
  void TestTransformBinaryCountingIterator(size_t n)
{
    // GCC 4.4.x has a known failure with auto-vectorization (due to -O3 or -ftree-vectorize) of this test
    // See https://gcc.gnu.org/bugzilla/show_bug.cgi?id=43251
#if (__GNUC__ * 10000 + __GNUC_MINOR__ * 100) == 40400
    KNOWN_FAILURE;
#else
    // be careful not to generate a range larger than we can represent
    n = thrust::min<size_t>(n, std::numeric_limits<T>::max());

    thrust::counting_iterator<T, thrust::host_system_tag>   h_first = thrust::make_counting_iterator<T>(0);
    thrust::counting_iterator<T, thrust::device_system_tag> d_first = thrust::make_counting_iterator<T>(0);

    thrust::host_vector<T>   h_result(n);
    thrust::device_vector<T> d_result(n);

    thrust::transform(h_first, h_first + n, h_first, h_result.begin(), thrust::plus<T>());
    thrust::transform(d_first, d_first + n, d_first, d_result.begin(), thrust::plus<T>());

    ASSERT_EQUAL(h_result, d_result);
#endif
}
DECLARE_VARIABLE_UNITTEST(TestTransformBinaryCountingIterator);


template <typename T>
struct plus_mod3
{
    T * table;

    plus_mod3(T * table) : table(table) {}

    __host__ __device__
    T operator()(T a, T b)
    {
        return table[(int) (a + b)];
    }
};

template <typename Vector>
void TestTransformWithIndirection(void)
{
    // add numbers modulo 3 with external lookup table
    typedef typename Vector::value_type T;

    Vector input1(7);
    Vector input2(7);
    Vector output(7, 0);
    input1[0] = 0;  input2[0] = 2; 
    input1[1] = 1;  input2[1] = 2;
    input1[2] = 2;  input2[2] = 2;
    input1[3] = 1;  input2[3] = 0;
    input1[4] = 2;  input2[4] = 2;
    input1[5] = 0;  input2[5] = 1;
    input1[6] = 1;  input2[6] = 0;

    Vector table(6);
    table[0] = 0;
    table[1] = 1;
    table[2] = 2;
    table[3] = 0;
    table[4] = 1;
    table[5] = 2;

    thrust::transform(input1.begin(), input1.end(),
                      input2.begin(), 
                      output.begin(),
                      plus_mod3<T>(thrust::raw_pointer_cast(&table[0])));
    
    ASSERT_EQUAL(output[0], T(2));
    ASSERT_EQUAL(output[1], T(0));
    ASSERT_EQUAL(output[2], T(1));
    ASSERT_EQUAL(output[3], T(1));
    ASSERT_EQUAL(output[4], T(1));
    ASSERT_EQUAL(output[5], T(1));
    ASSERT_EQUAL(output[6], T(1));
}
DECLARE_VECTOR_UNITTEST(TestTransformWithIndirection);

