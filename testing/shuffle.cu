#include "hip/hip_runtime.h"
#include <thrust/detail/config.h>

#if THRUST_CPP_DIALECT >= 2011
#include <thrust/random.h>
#include <thrust/sequence.h>
#include <thrust/shuffle.h>
#include <thrust/sort.h>
#include <unittest/unittest.h>
#include <map>

template <typename Vector>
void TestShuffleSimple() {
  Vector data(5);
  data[0] = 0;
  data[1] = 1;
  data[2] = 2;
  data[3] = 3;
  data[4] = 4;
  Vector shuffled(data.begin(), data.end());
  thrust::default_random_engine g(2);
  thrust::shuffle(shuffled.begin(), shuffled.end(), g);
  thrust::sort(shuffled.begin(), shuffled.end());
  // Check all of our data is present
  // This only tests for strange conditions like duplicated elements
  ASSERT_EQUAL(shuffled, data);
}
DECLARE_VECTOR_UNITTEST(TestShuffleSimple);

template <typename Vector>
void TestShuffleCopySimple() {
  Vector data(5);
  data[0] = 0;
  data[1] = 1;
  data[2] = 2;
  data[3] = 3;
  data[4] = 4;
  Vector shuffled(5);
  thrust::default_random_engine g(2);
  thrust::shuffle_copy(data.begin(), data.end(), shuffled.begin(), g);
  g.seed(2);
  thrust::shuffle(data.begin(), data.end(), g);
  ASSERT_EQUAL(shuffled, data);
}
DECLARE_VECTOR_UNITTEST(TestShuffleCopySimple);

template <typename T>
void TestHostDeviceIdentical(size_t m) {
  thrust::host_vector<T> host_result(m);
  thrust::host_vector<T> device_result(m);
  thrust::sequence(host_result.begin(), host_result.end(), 0llu);
  thrust::sequence(device_result.begin(), device_result.end(), 0llu);

  thrust::default_random_engine host_g(183);
  thrust::default_random_engine device_g(183);

  thrust::shuffle(host_result.begin(), host_result.end(), host_g);
  thrust::shuffle(device_result.begin(), device_result.end(), device_g);

  ASSERT_EQUAL(device_result, host_result);
}
DECLARE_VARIABLE_UNITTEST(TestHostDeviceIdentical);

// Individual input keys should be permuted to output locations with uniform
// probability. Perform chi-squared test with confidence 99.9%.
template <typename Vector>
void TestShuffleKeyPosition() {
  typedef typename Vector::value_type T;
  size_t m = 20;
  size_t num_samples = 100;
  thrust::host_vector<size_t> index_sum(m, 0);
  thrust::host_vector<T> sequence(m);
  thrust::sequence(sequence.begin(), sequence.end(), T(0));

  for (size_t i = 0; i < num_samples; i++) {
    Vector shuffled(sequence.begin(), sequence.end());
    thrust::default_random_engine g(i);
    thrust::shuffle(shuffled.begin(), shuffled.end(), g);
    thrust::host_vector<T> tmp(shuffled.begin(), shuffled.end());

    for (auto j = 0ull; j < m; j++) {
      index_sum[tmp[j]] += j;
    }
  }
  double expected_average_position = static_cast<double>(m - 1) / 2;
  double chi_squared = 0.0;
  for (auto j = 0ull; j < m; j++) {
    double average_position = static_cast<double>(index_sum[j]) / num_samples;
    chi_squared += std::pow(expected_average_position - average_position, 2) /
                   expected_average_position;
  }
  // Tabulated chi-squared critical value for m-1=19 degrees of freedom
  // and 99.9% confidence
  double confidence_threshold = 43.82;
  ASSERT_LESS(chi_squared, confidence_threshold);
}
DECLARE_INTEGRAL_VECTOR_UNITTEST(TestShuffleKeyPosition);

struct vector_compare {
  template <typename VectorT>
  bool operator()(const VectorT& a, const VectorT& b) const {
    for (auto i = 0ull; i < a.size(); i++) {
      if (a[i] < b[i]) return true;
      if (a[i] > b[i]) return false;
    }
    return false;
  }
};

// Brute force check permutations are uniformly distributed on small input
// Uses a chi-squared test indicating 99% confidence the output is uniformly
// random
template <typename Vector>
void TestShuffleUniformPermutation() {
  typedef typename Vector::value_type T;

  size_t m = 5;
  size_t num_samples = 1000;
  size_t total_permutations = 1 * 2 * 3 * 4 * 5;
  std::map<thrust::host_vector<T>, size_t, vector_compare> permutation_counts;
  Vector sequence(m);
  thrust::sequence(sequence.begin(), sequence.end(), T(0));
  thrust::default_random_engine g(17);
  for (auto i = 0ull; i < num_samples; i++) {
    thrust::shuffle(sequence.begin(), sequence.end(), g);
    thrust::host_vector<T> tmp(sequence.begin(), sequence.end());
    permutation_counts[tmp]++;
  }

  ASSERT_EQUAL(permutation_counts.size(), total_permutations);

  double chi_squared = 0.0;
  double expected_count = static_cast<double>(num_samples) / total_permutations;
  for (auto kv : permutation_counts) {
    chi_squared += std::pow(expected_count - kv.second, 2) / expected_count;
  }
  // Tabulated chi-squared critical value for 119 degrees of freedom (5! - 1)
  // and 99% confidence
  double confidence_threshold = 157.8;
  ASSERT_LESS(chi_squared, confidence_threshold);
}
DECLARE_VECTOR_UNITTEST(TestShuffleUniformPermutation);
#endif
