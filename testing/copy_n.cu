#include <unittest/unittest.h>
#include <thrust/copy.h>

#include <list>
#include <iterator>
#include <thrust/sequence.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/discard_iterator.h>

void TestCopyNFromConstIterator(void)
{
    typedef int T;

    std::vector<T> v(5);
    v[0] = 0; v[1] = 1; v[2] = 2; v[3] = 3; v[4] = 4;

    std::vector<int>::const_iterator begin = v.begin();

    // copy to host_vector
    thrust::host_vector<T> h(5, (T) 10);
    thrust::host_vector<T>::iterator h_result = thrust::copy_n(begin, h.size(), h.begin());
    ASSERT_EQUAL(h[0], 0);
    ASSERT_EQUAL(h[1], 1);
    ASSERT_EQUAL(h[2], 2);
    ASSERT_EQUAL(h[3], 3);
    ASSERT_EQUAL(h[4], 4);
    ASSERT_EQUAL_QUIET(h_result, h.end());

    // copy to device_vector
    thrust::device_vector<T> d(5, (T) 10);
    thrust::device_vector<T>::iterator d_result = thrust::copy_n(begin, d.size(), d.begin());
    ASSERT_EQUAL(d[0], 0);
    ASSERT_EQUAL(d[1], 1);
    ASSERT_EQUAL(d[2], 2);
    ASSERT_EQUAL(d[3], 3);
    ASSERT_EQUAL(d[4], 4);
    ASSERT_EQUAL_QUIET(d_result, d.end());
}
DECLARE_UNITTEST(TestCopyNFromConstIterator);

void TestCopyNToDiscardIterator(void)
{
    typedef int T;

    thrust::host_vector<T> h_input(5, 1);
    thrust::device_vector<T> d_input = h_input;

    // copy from host_vector
    thrust::discard_iterator<> h_result =
      thrust::copy_n(h_input.begin(), h_input.size(), thrust::make_discard_iterator());

    // copy from device_vector
    thrust::discard_iterator<> d_result =
      thrust::copy_n(d_input.begin(), d_input.size(), thrust::make_discard_iterator());

    thrust::discard_iterator<> reference(5);

    ASSERT_EQUAL_QUIET(reference, h_result);
    ASSERT_EQUAL_QUIET(reference, d_result);
}
DECLARE_UNITTEST(TestCopyNToDiscardIterator);

template <class Vector>
void TestCopyNMatchingTypes(void)
{
    typedef typename Vector::value_type T;

    Vector v(5);
    v[0] = 0; v[1] = 1; v[2] = 2; v[3] = 3; v[4] = 4;

    // copy to host_vector
    thrust::host_vector<T> h(5, (T) 10);
    typename thrust::host_vector<T>::iterator h_result = thrust::copy_n(v.begin(), v.size(), h.begin());
    ASSERT_EQUAL(h[0], 0);
    ASSERT_EQUAL(h[1], 1);
    ASSERT_EQUAL(h[2], 2);
    ASSERT_EQUAL(h[3], 3);
    ASSERT_EQUAL(h[4], 4);
    ASSERT_EQUAL_QUIET(h_result, h.end());

    // copy to device_vector
    thrust::device_vector<T> d(5, (T) 10);
    typename thrust::device_vector<T>::iterator d_result = thrust::copy_n(v.begin(), v.size(), d.begin());
    ASSERT_EQUAL(d[0], 0);
    ASSERT_EQUAL(d[1], 1);
    ASSERT_EQUAL(d[2], 2);
    ASSERT_EQUAL(d[3], 3);
    ASSERT_EQUAL(d[4], 4);
    ASSERT_EQUAL_QUIET(d_result, d.end());
}
DECLARE_VECTOR_UNITTEST(TestCopyNMatchingTypes);

template <class Vector>
void TestCopyNMixedTypes(void)
{
    typedef typename Vector::value_type T;

    Vector v(5);
    v[0] = 0; v[1] = 1; v[2] = 2; v[3] = 3; v[4] = 4;

    // copy to host_vector with different type
    thrust::host_vector<float> h(5, (float) 10);
    typename thrust::host_vector<float>::iterator h_result = thrust::copy_n(v.begin(), v.size(), h.begin());

    ASSERT_EQUAL(h[0], 0);
    ASSERT_EQUAL(h[1], 1);
    ASSERT_EQUAL(h[2], 2);
    ASSERT_EQUAL(h[3], 3);
    ASSERT_EQUAL(h[4], 4);
    ASSERT_EQUAL_QUIET(h_result, h.end());

    // copy to device_vector with different type
    thrust::device_vector<float> d(5, (float) 10);
    typename thrust::device_vector<float>::iterator d_result = thrust::copy_n(v.begin(), v.size(), d.begin());
    ASSERT_EQUAL(d[0], 0);
    ASSERT_EQUAL(d[1], 1);
    ASSERT_EQUAL(d[2], 2);
    ASSERT_EQUAL(d[3], 3);
    ASSERT_EQUAL(d[4], 4);
    ASSERT_EQUAL_QUIET(d_result, d.end());
}
DECLARE_VECTOR_UNITTEST(TestCopyNMixedTypes);


void TestCopyNVectorBool(void)
{
    std::vector<bool> v(3);
    v[0] = true; v[1] = false; v[2] = true;

    thrust::host_vector<bool> h(3);
    thrust::device_vector<bool> d(3);
    
    thrust::copy_n(v.begin(), v.size(), h.begin());
    thrust::copy_n(v.begin(), v.size(), d.begin());

    ASSERT_EQUAL(h[0], true);
    ASSERT_EQUAL(h[1], false);
    ASSERT_EQUAL(h[2], true);

    ASSERT_EQUAL(d[0], true);
    ASSERT_EQUAL(d[1], false);
    ASSERT_EQUAL(d[2], true);
}
DECLARE_UNITTEST(TestCopyNVectorBool);


template <class Vector>
void TestCopyNListTo(void)
{
    typedef typename Vector::value_type T;

    // copy from list to Vector
    std::list<T> l;
    l.push_back(0);
    l.push_back(1);
    l.push_back(2);
    l.push_back(3);
    l.push_back(4);
   
    Vector v(l.size());

    typename Vector::iterator v_result = thrust::copy_n(l.begin(), l.size(), v.begin());

    ASSERT_EQUAL(v[0], 0);
    ASSERT_EQUAL(v[1], 1);
    ASSERT_EQUAL(v[2], 2);
    ASSERT_EQUAL(v[3], 3);
    ASSERT_EQUAL(v[4], 4);
    ASSERT_EQUAL_QUIET(v_result, v.end());

    l.clear();

    thrust::copy_n(v.begin(), v.size(), std::back_insert_iterator< std::list<T> >(l));

    ASSERT_EQUAL(l.size(), 5);

    typename std::list<T>::const_iterator iter = l.begin();
    ASSERT_EQUAL(*iter, 0);  iter++;
    ASSERT_EQUAL(*iter, 1);  iter++;
    ASSERT_EQUAL(*iter, 2);  iter++;
    ASSERT_EQUAL(*iter, 3);  iter++;
    ASSERT_EQUAL(*iter, 4);  iter++;
}
DECLARE_VECTOR_UNITTEST(TestCopyNListTo);


template <typename Vector>
void TestCopyNCountingIterator(void)
{
    typedef typename Vector::value_type T;

    thrust::counting_iterator<T> iter(1);

    Vector vec(4);

    thrust::copy_n(iter, 4, vec.begin());

    ASSERT_EQUAL(vec[0], 1);
    ASSERT_EQUAL(vec[1], 2);
    ASSERT_EQUAL(vec[2], 3);
    ASSERT_EQUAL(vec[3], 4);
}
DECLARE_VECTOR_UNITTEST(TestCopyNCountingIterator);

template <typename Vector>
void TestCopyNZipIterator(void)
{
    typedef typename Vector::value_type T;

    Vector v1(3); v1[0] = 1; v1[1] = 2; v1[2] = 3;
    Vector v2(3); v2[0] = 4; v2[1] = 5; v2[2] = 6; 
    Vector v3(3, T(0));
    Vector v4(3, T(0));

    thrust::copy_n(thrust::make_zip_iterator(thrust::make_tuple(v1.begin(),v2.begin())),
                   3,
                   thrust::make_zip_iterator(thrust::make_tuple(v3.begin(),v4.begin())));

    ASSERT_EQUAL(v1, v3);
    ASSERT_EQUAL(v2, v4);
};
DECLARE_VECTOR_UNITTEST(TestCopyNZipIterator);

template <typename Vector>
void TestCopyNConstantIteratorToZipIterator(void)
{
    typedef typename Vector::value_type T;

    Vector v1(3,T(0));
    Vector v2(3,T(0));

    thrust::copy_n(thrust::make_constant_iterator(thrust::tuple<T,T>(4,7)),
                   v1.size(),
                   thrust::make_zip_iterator(thrust::make_tuple(v1.begin(),v2.begin())));

    ASSERT_EQUAL(v1[0], 4);
    ASSERT_EQUAL(v1[1], 4);
    ASSERT_EQUAL(v1[2], 4);
    ASSERT_EQUAL(v2[0], 7);
    ASSERT_EQUAL(v2[1], 7);
    ASSERT_EQUAL(v2[2], 7);
};
DECLARE_VECTOR_UNITTEST(TestCopyNConstantIteratorToZipIterator);

struct my_system : thrust::device_system<my_system> {};

template<typename InputIterator, typename Size, typename OutputIterator>
OutputIterator copy_n(my_system, InputIterator, Size, OutputIterator result)
{
    *result = 13;

    return result;
}

void TestCopyNDispatchExplicit()
{
    thrust::device_vector<int> vec(1);

    my_system sys;
    thrust::copy_n(sys,
                   vec.begin(),
                   1,
                   vec.begin());

    ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestCopyNDispatchExplicit);

void TestCopyNDispatchImplicit()
{
    thrust::device_vector<int> vec(1);

    thrust::copy_n(thrust::retag<my_system>(vec.begin()),
                   1,
                   thrust::retag<my_system>(vec.begin()));

    ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestCopyNDispatchImplicit);

