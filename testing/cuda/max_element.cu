#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/extrema.h>
#include <thrust/execution_policy.h>


template<typename ExecutionPolicy, typename Iterator, typename Iterator2>
__global__
void max_element_kernel(ExecutionPolicy exec, Iterator first, Iterator last, Iterator2 result)
{
  *result = thrust::max_element(exec, first, last);
}


template<typename ExecutionPolicy, typename Iterator, typename BinaryPredicate, typename Iterator2>
__global__
void max_element_kernel(ExecutionPolicy exec, Iterator first, Iterator last, BinaryPredicate pred, Iterator2 result)
{
  *result = thrust::max_element(exec, first, last, pred);
}


template<typename ExecutionPolicy>
void TestMaxElementDevice(ExecutionPolicy exec)
{
  size_t n = 1000;
  thrust::host_vector<int> h_data = unittest::random_samples<int>(n);
  thrust::device_vector<int> d_data = h_data;

  typedef typename thrust::device_vector<int>::iterator iter_type;

  thrust::device_vector<iter_type> d_result(1);
  
  typename thrust::host_vector<int>::iterator   h_max = thrust::max_element(h_data.begin(), h_data.end());

  max_element_kernel<<<1,1>>>(exec, d_data.begin(), d_data.end(), d_result.begin());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  ASSERT_EQUAL(h_max - h_data.begin(), (iter_type)d_result[0] - d_data.begin());

  
  typename thrust::host_vector<int>::iterator   h_min = thrust::max_element(h_data.begin(), h_data.end(), thrust::greater<int>());

  max_element_kernel<<<1,1>>>(exec, d_data.begin(), d_data.end(), thrust::greater<int>(), d_result.begin());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  ASSERT_EQUAL(h_min - h_data.begin(), (iter_type)d_result[0] - d_data.begin());
}


void TestMaxElementDeviceSeq()
{
  TestMaxElementDevice(thrust::seq);
}
DECLARE_UNITTEST(TestMaxElementDeviceSeq);


void TestMaxElementDeviceDevice()
{
  TestMaxElementDevice(thrust::device);
}
DECLARE_UNITTEST(TestMaxElementDeviceDevice);


void TestMaxElementDeviceNoSync()
{
  TestMaxElementDevice(thrust::cuda::par_nosync);
}
DECLARE_UNITTEST(TestMaxElementDeviceNoSync);


template<typename ExecutionPolicy>
void TestMaxElementCudaStreams(ExecutionPolicy policy)
{
  typedef thrust::device_vector<int> Vector;
  typedef Vector::value_type T;

  Vector data(6);
  data[0] = 3;
  data[1] = 5;
  data[2] = 1;
  data[3] = 2;
  data[4] = 5;
  data[5] = 1;

  hipStream_t s;
  hipStreamCreate(&s);

  auto streampolicy = policy.on(s);

  ASSERT_EQUAL( *thrust::max_element(streampolicy, data.begin(), data.end()), 5);
  ASSERT_EQUAL( thrust::max_element(streampolicy, data.begin(), data.end()) - data.begin(), 1);
  
  ASSERT_EQUAL( *thrust::max_element(streampolicy, data.begin(), data.end(), thrust::greater<T>()), 1);
  ASSERT_EQUAL( thrust::max_element(streampolicy, data.begin(), data.end(), thrust::greater<T>()) - data.begin(), 2);

  hipStreamDestroy(s);
}

void TestMaxElementCudaStreamsSync(){
  TestMaxElementCudaStreams(thrust::cuda::par);
}
DECLARE_UNITTEST(TestMaxElementCudaStreamsSync);


void TestMaxElementCudaStreamsNoSync(){
  TestMaxElementCudaStreams(thrust::cuda::par_nosync);
}
DECLARE_UNITTEST(TestMaxElementCudaStreamsNoSync);


void TestMaxElementDevicePointer()
{
  typedef thrust::device_vector<int> Vector;
  typedef Vector::value_type T;

  Vector data(6);
  data[0] = 3;
  data[1] = 5;
  data[2] = 1;
  data[3] = 2;
  data[4] = 5;
  data[5] = 1;

  T* raw_ptr = thrust::raw_pointer_cast(data.data());
  size_t n = data.size();
  ASSERT_EQUAL( thrust::max_element(thrust::device, raw_ptr, raw_ptr+n) - raw_ptr, 1);
  ASSERT_EQUAL( thrust::max_element(thrust::device, raw_ptr, raw_ptr+n, thrust::greater<T>()) - raw_ptr, 2);
}
DECLARE_UNITTEST(TestMaxElementDevicePointer);
