#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/set_operations.h>
#include <thrust/execution_policy.h>


#ifdef THRUST_TEST_DEVICE_SIDE
template<typename ExecutionPolicy, typename Iterator1, typename Iterator2, typename Iterator3, typename Iterator4>
__global__
void set_difference_kernel(ExecutionPolicy exec, Iterator1 first1, Iterator1 last1, Iterator2 first2, Iterator2 last2, Iterator3 result1, Iterator4 result2)
{
  *result2 = thrust::set_difference(exec, first1, last1, first2, last2, result1);
}


template<typename ExecutionPolicy>
void TestSetDifferenceDevice(ExecutionPolicy exec)
{
  typedef thrust::device_vector<int> Vector;
  typedef typename Vector::iterator Iterator;

  Vector a(4), b(5);

  a[0] = 0; a[1] = 2; a[2] = 4; a[3] = 5;
  b[0] = 0; b[1] = 3; b[2] = 3; b[3] = 4; b[4] = 6;

  Vector ref(2);
  ref[0] = 2; ref[1] = 5;

  Vector result(2);

  thrust::device_vector<Iterator> end_vec(1);

  set_difference_kernel<<<1,1>>>(exec, a.begin(), a.end(), b.begin(), b.end(), result.begin(), end_vec.begin());
  hipError_t const err = hipDeviceSynchronize();
  ASSERT_EQUAL(hipSuccess, err);

  Iterator end = end_vec.front();

  ASSERT_EQUAL_QUIET(result.end(), end);
  ASSERT_EQUAL(ref, result);
}


void TestSetDifferenceDeviceSeq()
{
  TestSetDifferenceDevice(thrust::seq);
}
DECLARE_UNITTEST(TestSetDifferenceDeviceSeq);


void TestSetDifferenceDeviceDevice()
{
  TestSetDifferenceDevice(thrust::device);
}
DECLARE_UNITTEST(TestSetDifferenceDeviceDevice);
#endif


void TestSetDifferenceCudaStreams()
{
  typedef thrust::device_vector<int> Vector;
  typedef Vector::iterator Iterator;

  Vector a(4), b(5);

  a[0] = 0; a[1] = 2; a[2] = 4; a[3] = 5;
  b[0] = 0; b[1] = 3; b[2] = 3; b[3] = 4; b[4] = 6;

  Vector ref(2);
  ref[0] = 2; ref[1] = 5;

  Vector result(2);

  hipStream_t s;
  hipStreamCreate(&s);

  Iterator end = thrust::set_difference(thrust::cuda::par.on(s), a.begin(), a.end(), b.begin(), b.end(), result.begin());
  hipStreamSynchronize(s);

  ASSERT_EQUAL_QUIET(result.end(), end);
  ASSERT_EQUAL(ref, result);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestSetDifferenceCudaStreams);

