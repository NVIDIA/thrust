#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/generate.h>
#include <thrust/execution_policy.h>


template<typename T>
struct return_value
{
  T val;
  
  return_value(void){}
  return_value(T v):val(v){}
  
  __host__ __device__
  T operator()(void){ return val; }
};


#ifdef THRUST_TEST_DEVICE_SIDE
template<typename ExecutionPolicy, typename Iterator, typename Function>
__global__
void generate_kernel(ExecutionPolicy exec, Iterator first, Iterator last, Function f)
{
  thrust::generate(exec, first, last, f);
}


template<typename T, typename ExecutionPolicy>
void TestGenerateDevice(ExecutionPolicy exec, const size_t n)
{
  thrust::host_vector<T> h_result(n);
  thrust::device_vector<T> d_result(n);
  
  T value = 13;
  return_value<T> f(value);
  
  thrust::generate(h_result.begin(), h_result.end(), f);

  generate_kernel<<<1,1>>>(exec, d_result.begin(), d_result.end(), f);
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }
  
  ASSERT_EQUAL(h_result, d_result);
}


template<typename T>
void TestGenerateDeviceSeq(const size_t n)
{
  TestGenerateDevice<T>(thrust::seq, n);
}
DECLARE_VARIABLE_UNITTEST(TestGenerateDeviceSeq);


template<typename T>
void TestGenerateDeviceDevice(const size_t n)
{
  TestGenerateDevice<T>(thrust::device, n);
}
DECLARE_VARIABLE_UNITTEST(TestGenerateDeviceDevice);
#endif


void TestGenerateCudaStreams()
{
  thrust::device_vector<int> result(5);
  
  int value = 13;
  
  return_value<int> f(value);

  hipStream_t s;
  hipStreamCreate(&s);
  
  thrust::generate(thrust::cuda::par.on(s), result.begin(), result.end(), f);
  hipStreamSynchronize(s);
  
  ASSERT_EQUAL(result[0], value);
  ASSERT_EQUAL(result[1], value);
  ASSERT_EQUAL(result[2], value);
  ASSERT_EQUAL(result[3], value);
  ASSERT_EQUAL(result[4], value);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestGenerateCudaStreams);


#ifdef THRUST_TEST_DEVICE_SIDE
template<typename ExecutionPolicy, typename Iterator, typename Size, typename Function>
__global__
void generate_n_kernel(ExecutionPolicy exec, Iterator first, Size n, Function f)
{
  thrust::generate_n(exec, first, n, f);
}


template<typename T, typename ExecutionPolicy>
void TestGenerateNDevice(ExecutionPolicy exec, const size_t n)
{
  thrust::host_vector<T> h_result(n);
  thrust::device_vector<T> d_result(n);
  
  T value = 13;
  return_value<T> f(value);
  
  thrust::generate_n(h_result.begin(), h_result.size(), f);

  generate_n_kernel<<<1,1>>>(exec, d_result.begin(), d_result.size(), f);
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }
  
  ASSERT_EQUAL(h_result, d_result);
}


template<typename T>
void TestGenerateNDeviceSeq(const size_t n)
{
  TestGenerateNDevice<T>(thrust::seq, n);
}
DECLARE_VARIABLE_UNITTEST(TestGenerateNDeviceSeq);


template<typename T>
void TestGenerateNDeviceDevice(const size_t n)
{
  TestGenerateNDevice<T>(thrust::device, n);
}
DECLARE_VARIABLE_UNITTEST(TestGenerateNDeviceDevice);
#endif


void TestGenerateNCudaStreams()
{
  thrust::device_vector<int> result(5);
  
  int value = 13;
  
  return_value<int> f(value);

  hipStream_t s;
  hipStreamCreate(&s);
  
  thrust::generate_n(thrust::cuda::par.on(s), result.begin(), result.size(), f);
  hipStreamSynchronize(s);
  
  ASSERT_EQUAL(result[0], value);
  ASSERT_EQUAL(result[1], value);
  ASSERT_EQUAL(result[2], value);
  ASSERT_EQUAL(result[3], value);
  ASSERT_EQUAL(result[4], value);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestGenerateNCudaStreams);

