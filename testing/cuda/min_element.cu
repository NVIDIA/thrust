#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/extrema.h>
#include <thrust/execution_policy.h>


#ifdef THRUST_TEST_DEVICE_SIDE
template<typename ExecutionPolicy, typename Iterator, typename Iterator2>
__global__
void min_element_kernel(ExecutionPolicy exec, Iterator first, Iterator last, Iterator2 result)
{
  *result = thrust::min_element(exec, first, last);
}


template<typename ExecutionPolicy, typename Iterator, typename BinaryPredicate, typename Iterator2>
__global__
void min_element_kernel(ExecutionPolicy exec, Iterator first, Iterator last, BinaryPredicate pred, Iterator2 result)
{
  *result = thrust::min_element(exec, first, last, pred);
}


template<typename ExecutionPolicy>
void TestMinElementDevice(ExecutionPolicy exec)
{
  size_t n = 1000;
  thrust::host_vector<int> h_data = unittest::random_samples<int>(n);
  thrust::device_vector<int> d_data = h_data;

  typedef typename thrust::device_vector<int>::iterator iter_type;

  thrust::device_vector<iter_type> d_result(1);
  
  typename thrust::host_vector<int>::iterator   h_min = thrust::min_element(h_data.begin(), h_data.end());

  min_element_kernel<<<1,1>>>(exec, d_data.begin(), d_data.end(), d_result.begin());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  ASSERT_EQUAL(h_min - h_data.begin(), (iter_type)d_result[0] - d_data.begin());

  typename thrust::host_vector<int>::iterator   h_max = thrust::min_element(h_data.begin(), h_data.end(), thrust::greater<int>());

  min_element_kernel<<<1,1>>>(exec, d_data.begin(), d_data.end(), thrust::greater<int>(), d_result.begin());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  ASSERT_EQUAL(h_max - h_data.begin(), (iter_type)d_result[0] - d_data.begin());
}


void TestMinElementDeviceSeq()
{
  TestMinElementDevice(thrust::seq);
}
DECLARE_UNITTEST(TestMinElementDeviceSeq);


void TestMinElementDeviceDevice()
{
  TestMinElementDevice(thrust::device);
}
DECLARE_UNITTEST(TestMinElementDeviceDevice);
#endif


void TestMinElementCudaStreams()
{
  typedef thrust::device_vector<int> Vector;
  typedef Vector::value_type T;

  Vector data(6);
  data[0] = 3;
  data[1] = 5;
  data[2] = 1;
  data[3] = 2;
  data[4] = 5;
  data[5] = 1;

  hipStream_t s;
  hipStreamCreate(&s);

  ASSERT_EQUAL( *thrust::min_element(thrust::cuda::par.on(s), data.begin(), data.end()), 1);
  ASSERT_EQUAL( thrust::min_element(thrust::cuda::par.on(s), data.begin(), data.end()) - data.begin(), 2);
  
  ASSERT_EQUAL( *thrust::min_element(thrust::cuda::par.on(s), data.begin(), data.end(), thrust::greater<T>()), 5);
  ASSERT_EQUAL( thrust::min_element(thrust::cuda::par.on(s), data.begin(), data.end(), thrust::greater<T>()) - data.begin(), 1);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestMinElementCudaStreams);

void TestMinElementDevicePointer()
{
  typedef thrust::device_vector<int> Vector;
  typedef Vector::value_type T;

  Vector data(6);
  data[0] = 3;
  data[1] = 5;
  data[2] = 1;
  data[3] = 2;
  data[4] = 5;
  data[5] = 1;

  T* raw_ptr = thrust::raw_pointer_cast(data.data());
  size_t n = data.size();
  ASSERT_EQUAL( thrust::min_element(thrust::device, raw_ptr, raw_ptr+n) - raw_ptr, 2);
  ASSERT_EQUAL( thrust::min_element(thrust::device, raw_ptr, raw_ptr+n, thrust::greater<T>()) - raw_ptr, 1);
}
DECLARE_UNITTEST(TestMinElementDevicePointer);
