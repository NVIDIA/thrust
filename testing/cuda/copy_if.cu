#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/copy.h>
#include <thrust/sequence.h>
#include <thrust/execution_policy.h>


template<typename T>
struct is_even
{
  __host__ __device__
  bool operator()(T x) { return (static_cast<unsigned int>(x) & 1) == 0; }
};


template<typename T>
struct mod_3
{
  __host__ __device__
  unsigned int operator()(T x) { return static_cast<unsigned int>(x) % 3; }
};


template<typename ExecutionPolicy, typename Iterator1, typename Iterator2, typename Predicate, typename Iterator3>
__global__ void copy_if_kernel(ExecutionPolicy exec, Iterator1 first, Iterator1 last, Iterator2 result1, Predicate pred, Iterator3 result2)
{
  *result2 = thrust::copy_if(exec, first, last, result1, pred);
}


template<typename ExecutionPolicy>
void TestCopyIfDevice(ExecutionPolicy exec)
{
  size_t n = 1000;
  thrust::host_vector<int>   h_data = unittest::random_integers<int>(n);
  thrust::device_vector<int> d_data = h_data;
  
  typename thrust::host_vector<int>::iterator   h_new_end;
  typename thrust::device_vector<int>::iterator d_new_end;

  thrust::device_vector<
    typename thrust::device_vector<int>::iterator
  > d_new_end_vec(1);
  
  // test with Predicate that returns a bool
  {
    thrust::host_vector<int>   h_result(n);
    thrust::device_vector<int> d_result(n);
    
    h_new_end = thrust::copy_if(h_data.begin(), h_data.end(), h_result.begin(), is_even<int>());

    copy_if_kernel<<<1,1>>>(exec, d_data.begin(), d_data.end(), d_result.begin(), is_even<int>(), d_new_end_vec.begin());
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);

    d_new_end = d_new_end_vec[0];
    
    h_result.resize(h_new_end - h_result.begin());
    d_result.resize(d_new_end - d_result.begin());
    
    ASSERT_EQUAL(h_result, d_result);
  }
  
  // test with Predicate that returns a non-bool
  {
    thrust::host_vector<int>   h_result(n);
    thrust::device_vector<int> d_result(n);
    
    h_new_end = thrust::copy_if(h_data.begin(), h_data.end(), h_result.begin(), mod_3<int>());

    copy_if_kernel<<<1,1>>>(exec, d_data.begin(), d_data.end(), d_result.begin(), mod_3<int>(), d_new_end_vec.begin());
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);

    d_new_end = d_new_end_vec[0];
    
    h_result.resize(h_new_end - h_result.begin());
    d_result.resize(d_new_end - d_result.begin());
    
    ASSERT_EQUAL(h_result, d_result);
  }
}


void TestCopyIfDeviceSeq()
{
  TestCopyIfDevice(thrust::seq);
}
DECLARE_UNITTEST(TestCopyIfDeviceSeq);


void TestCopyIfDeviceDevice()
{
  TestCopyIfDevice(thrust::device);
}
DECLARE_UNITTEST(TestCopyIfDeviceDevice);


void TestCopyIfDeviceNoSync()
{
  TestCopyIfDevice(thrust::cuda::par_nosync);
}
DECLARE_UNITTEST(TestCopyIfDeviceNoSync);

template<typename ExecutionPolicy>
void TestCopyIfCudaStreams(ExecutionPolicy policy)
{
  typedef thrust::device_vector<int> Vector;

  Vector data(5);
  data[0] =  1; 
  data[1] =  2; 
  data[2] =  1;
  data[3] =  3; 
  data[4] =  2; 

  Vector result(5);

  hipStream_t s;
  hipStreamCreate(&s);

  Vector::iterator end = thrust::copy_if(policy.on(s),
                                         data.begin(), 
                                         data.end(), 
                                         result.begin(),
                                         is_even<int>());

  ASSERT_EQUAL(end - result.begin(), 2);

  ASSERT_EQUAL(result[0], 2);
  ASSERT_EQUAL(result[1], 2);

  hipStreamDestroy(s);
}

void TestCopyIfCudaStreamsSync(){
  TestCopyIfCudaStreams(thrust::cuda::par);
}
DECLARE_UNITTEST(TestCopyIfCudaStreamsSync);

void TestCopyIfCudaStreamsNoSync(){
  TestCopyIfCudaStreams(thrust::cuda::par_nosync);
}
DECLARE_UNITTEST(TestCopyIfCudaStreamsNoSync);


template<typename ExecutionPolicy, typename Iterator1, typename Iterator2, typename Iterator3, typename Predicate, typename Iterator4>
__global__ void copy_if_kernel(ExecutionPolicy exec, Iterator1 first, Iterator1 last, Iterator2 stencil_first, Iterator3 result1, Predicate pred, Iterator4 result2)
{
  *result2 = thrust::copy_if(exec, first, last, stencil_first, result1, pred);
}


template<typename ExecutionPolicy>
void TestCopyIfStencilDevice(ExecutionPolicy exec)
{
  size_t n = 1000;
  thrust::host_vector<int>   h_data(n); thrust::sequence(h_data.begin(), h_data.end());
  thrust::device_vector<int> d_data(n); thrust::sequence(d_data.begin(), d_data.end()); 
  
  thrust::host_vector<int>   h_stencil = unittest::random_integers<int>(n);
  thrust::device_vector<int> d_stencil = unittest::random_integers<int>(n);
  
  typename thrust::host_vector<int>::iterator   h_new_end;
  typename thrust::device_vector<int>::iterator d_new_end;

  thrust::device_vector<
    typename thrust::device_vector<int>::iterator
  > d_new_end_vec(1);
  
  // test with Predicate that returns a bool
  {
    thrust::host_vector<int>   h_result(n);
    thrust::device_vector<int> d_result(n);
    
    h_new_end = thrust::copy_if(h_data.begin(), h_data.end(), h_result.begin(), is_even<int>());

    copy_if_kernel<<<1,1>>>(exec, d_data.begin(), d_data.end(), d_result.begin(), is_even<int>(), d_new_end_vec.begin());
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);

    d_new_end = d_new_end_vec[0];
    
    h_result.resize(h_new_end - h_result.begin());
    d_result.resize(d_new_end - d_result.begin());
    
    ASSERT_EQUAL(h_result, d_result);
  }
  
  // test with Predicate that returns a non-bool
  {
    thrust::host_vector<int>   h_result(n);
    thrust::device_vector<int> d_result(n);
    
    h_new_end = thrust::copy_if(h_data.begin(), h_data.end(), h_result.begin(), mod_3<int>());

    copy_if_kernel<<<1,1>>>(exec, d_data.begin(), d_data.end(), d_result.begin(), mod_3<int>(), d_new_end_vec.begin());
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);

    d_new_end = d_new_end_vec[0];
    
    h_result.resize(h_new_end - h_result.begin());
    d_result.resize(d_new_end - d_result.begin());
    
    ASSERT_EQUAL(h_result, d_result);
  }
}


void TestCopyIfStencilDeviceSeq()
{
  TestCopyIfStencilDevice(thrust::seq);
}
DECLARE_UNITTEST(TestCopyIfStencilDeviceSeq);


void TestCopyIfStencilDeviceDevice()
{
  TestCopyIfStencilDevice(thrust::device);
}
DECLARE_UNITTEST(TestCopyIfStencilDeviceDevice);


void TestCopyIfStencilDeviceNoSync()
{
  TestCopyIfStencilDevice(thrust::cuda::par_nosync);
}
DECLARE_UNITTEST(TestCopyIfStencilDeviceNoSync);


template<typename ExecutionPolicy>
void TestCopyIfStencilCudaStreams(ExecutionPolicy policy)
{
  typedef thrust::device_vector<int> Vector;
  typedef Vector::value_type T;

  Vector data(5);
  data[0] =  1; 
  data[1] =  2; 
  data[2] =  1;
  data[3] =  3; 
  data[4] =  2; 

  Vector result(5);

  Vector stencil(5);
  stencil[0] = 0;
  stencil[1] = 1;
  stencil[2] = 0;
  stencil[3] = 0;
  stencil[4] = 1;

  hipStream_t s;
  hipStreamCreate(&s);

  Vector::iterator end = thrust::copy_if(policy.on(s),
                                         data.begin(), 
                                         data.end(),
                                         stencil.begin(),
                                         result.begin(),
                                         thrust::identity<T>());

  ASSERT_EQUAL(end - result.begin(), 2);

  ASSERT_EQUAL(result[0], 2);
  ASSERT_EQUAL(result[1], 2);

  hipStreamDestroy(s);
}

void TestCopyIfStencilCudaStreamsSync()
{
  TestCopyIfStencilCudaStreams(thrust::cuda::par);
}
DECLARE_UNITTEST(TestCopyIfStencilCudaStreamsSync);


void TestCopyIfStencilCudaStreamsNoSync()
{
  TestCopyIfStencilCudaStreams(thrust::cuda::par_nosync);
}
DECLARE_UNITTEST(TestCopyIfStencilCudaStreamsNoSync);

