#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/reverse.h>
#include <thrust/execution_policy.h>


#ifdef THRUST_TEST_DEVICE_SIDE
template<typename ExecutionPolicy, typename Iterator>
__global__
void reverse_kernel(ExecutionPolicy exec, Iterator first, Iterator last)
{
  thrust::reverse(exec, first, last);
}


template<typename ExecutionPolicy>
void TestReverseDevice(ExecutionPolicy exec)
{
  size_t n = 1000;
  thrust::host_vector<int> h_data = unittest::random_integers<int>(n);
  thrust::device_vector<int> d_data = h_data;
  
  thrust::reverse(h_data.begin(), h_data.end());

  reverse_kernel<<<1,1>>>(exec, raw_pointer_cast(d_data.data()), raw_pointer_cast(d_data.data() + d_data.size()));
  hipError_t const err = hipDeviceSynchronize();
  ASSERT_EQUAL(hipSuccess, err);
  
  ASSERT_EQUAL(h_data, d_data);
};


void TestReverseDeviceSeq()
{
  TestReverseDevice(thrust::seq);
}
DECLARE_UNITTEST(TestReverseDeviceSeq);


void TestReverseDeviceDevice()
{
  TestReverseDevice(thrust::device);
}
DECLARE_UNITTEST(TestReverseDeviceDevice);


template<typename ExecutionPolicy, typename Iterator1, typename Iterator2>
__global__
void reverse_copy_kernel(ExecutionPolicy exec, Iterator1 first, Iterator1 last, Iterator2 result)
{
  thrust::reverse_copy(exec, first, last, result);
}


template<typename ExecutionPolicy>
void TestReverseCopyDevice(ExecutionPolicy exec)
{
  size_t n = 1000;
  thrust::host_vector<int> h_data = unittest::random_integers<int>(n);
  thrust::device_vector<int> d_data = h_data;

  thrust::host_vector<int> h_result(n);
  thrust::device_vector<int> d_result(n);

  thrust::reverse_copy(h_data.begin(), h_data.end(), h_result.begin());

  reverse_copy_kernel<<<1,1>>>(exec, d_data.begin(), d_data.end(), d_result.begin());
  hipError_t const err = hipDeviceSynchronize();
  ASSERT_EQUAL(hipSuccess, err);

  ASSERT_EQUAL(h_result, d_result);
};


void TestReverseCopyDeviceSeq()
{
  TestReverseCopyDevice(thrust::seq);
}
DECLARE_UNITTEST(TestReverseCopyDeviceSeq);


void TestReverseCopyDeviceDevice()
{
  TestReverseCopyDevice(thrust::device);
}
DECLARE_UNITTEST(TestReverseCopyDeviceDevice);
#endif


void TestReverseCudaStreams()
{
  typedef thrust::device_vector<int> Vector;
  Vector data(5);
  data[0] = 1;
  data[1] = 2;
  data[2] = 3;
  data[3] = 4;
  data[4] = 5;

  hipStream_t s;
  hipStreamCreate(&s);

  thrust::reverse(thrust::cuda::par.on(s), data.begin(), data.end());

  hipStreamSynchronize(s);

  Vector ref(5);
  ref[0] = 5;
  ref[1] = 4;
  ref[2] = 3;
  ref[3] = 2;
  ref[4] = 1;

  ASSERT_EQUAL(ref, data);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestReverseCudaStreams);


void TestReverseCopyCudaStreams()
{
  typedef thrust::device_vector<int> Vector;
  Vector data(5);
  data[0] = 1;
  data[1] = 2;
  data[2] = 3;
  data[3] = 4;
  data[4] = 5;

  Vector result(5);

  hipStream_t s;
  hipStreamCreate(&s);

  thrust::reverse_copy(thrust::cuda::par.on(s), data.begin(), data.end(), result.begin());

  hipStreamSynchronize(s);

  Vector ref(5);
  ref[0] = 5;
  ref[1] = 4;
  ref[2] = 3;
  ref[3] = 2;
  ref[4] = 1;

  ASSERT_EQUAL(ref, result);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestReverseCopyCudaStreams);

