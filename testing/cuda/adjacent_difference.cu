#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/adjacent_difference.h>
#include <thrust/execution_policy.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>


template<typename ExecutionPolicy, typename Iterator1, typename Iterator2>
__global__ void adjacent_difference_kernel(ExecutionPolicy exec, Iterator1 first, Iterator1 last, Iterator2 result)
{
  thrust::adjacent_difference(exec, first, last, result);
}


template<typename ExecutionPolicy, typename Iterator1, typename Iterator2, typename BinaryFunction>
__global__ void adjacent_difference_kernel(ExecutionPolicy exec, Iterator1 first, Iterator1 last, Iterator2 result, BinaryFunction f)
{
  thrust::adjacent_difference(exec, first, last, result, f);
}


template<typename T, typename ExecutionPolicy>
void TestAdjacentDifferenceDevice(ExecutionPolicy exec, const size_t n)
{
  thrust::host_vector<T>   h_input = unittest::random_samples<T>(n);
  thrust::device_vector<T> d_input = h_input;

  thrust::host_vector<T>   h_output(n);
  thrust::device_vector<T> d_output(n);

  thrust::adjacent_difference(h_input.begin(), h_input.end(), h_output.begin());
  adjacent_difference_kernel<<<1,1>>>(exec, d_input.begin(), d_input.end(), d_output.begin());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  ASSERT_EQUAL(h_output, d_output);

  thrust::adjacent_difference(h_input.begin(), h_input.end(), h_output.begin(), thrust::plus<T>());
  adjacent_difference_kernel<<<1,1>>>(exec, d_input.begin(), d_input.end(), d_output.begin(), thrust::plus<T>());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  ASSERT_EQUAL(h_output, d_output);

  // in-place operation
  thrust::adjacent_difference(h_input.begin(), h_input.end(), h_input.begin(), thrust::plus<T>());
  adjacent_difference_kernel<<<1,1>>>(exec, d_input.begin(), d_input.end(), d_input.begin(), thrust::plus<T>());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  ASSERT_EQUAL(h_input, h_output); //computed previously
  ASSERT_EQUAL(d_input, d_output); //computed previously
}


template<typename T>
void TestAdjacentDifferenceDeviceSeq(const size_t n)
{
  TestAdjacentDifferenceDevice<T>(thrust::seq, n);
}
DECLARE_VARIABLE_UNITTEST(TestAdjacentDifferenceDeviceSeq);


template<typename T>
void TestAdjacentDifferenceDeviceDevice(const size_t n)
{
  TestAdjacentDifferenceDevice<T>(thrust::device, n);
}
DECLARE_VARIABLE_UNITTEST(TestAdjacentDifferenceDeviceDevice);


void TestAdjacentDifferenceCudaStreams()
{
  hipStream_t s;
  hipStreamCreate(&s);

  thrust::device_vector<int> input(3);
  thrust::device_vector<int> output(3);
  input[0] = 1; input[1] = 4; input[2] = 6;

  thrust::adjacent_difference(thrust::cuda::par.on(s), input.begin(), input.end(), output.begin());

  hipStreamSynchronize(s);

  ASSERT_EQUAL(output[0], 1);
  ASSERT_EQUAL(output[1], 3);
  ASSERT_EQUAL(output[2], 2);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestAdjacentDifferenceCudaStreams);

struct detect_wrong_difference
{
    using difference_type = void;
    using value_type = void;
    using pointer = void;
    using reference = void;
    using iterator_category = std::output_iterator_tag;

    bool * flag;

    __host__ __device__ detect_wrong_difference operator++() const { return *this; }
    __host__ __device__ detect_wrong_difference operator*() const { return *this; }
    template<typename Difference>
    __host__ __device__ detect_wrong_difference operator+(Difference) const { return *this; }
    template<typename Index>
    __host__ __device__ detect_wrong_difference operator[](Index) const { return *this; }

    __device__
    void operator=(long long difference) const
    {
        if (difference != 1)
        {
            *flag = false;
        }
    }
};

void TestAdjacentDifferenceWithBigIndexesHelper(int magnitude)
{
    thrust::counting_iterator<long long> begin(1);
    thrust::counting_iterator<long long> end = begin + (1ll << magnitude);
    ASSERT_EQUAL(thrust::distance(begin, end), 1ll << magnitude);

    thrust::device_ptr<bool> all_differences_correct = thrust::device_malloc<bool>(1);
    *all_differences_correct = true;

    detect_wrong_difference out = { thrust::raw_pointer_cast(all_differences_correct) };

    thrust::adjacent_difference(thrust::device, begin, end, out);

    bool all_differences_correct_h = *all_differences_correct;
    thrust::device_free(all_differences_correct);

    ASSERT_EQUAL(all_differences_correct_h, true);
}

void TestAdjacentDifferenceWithBigIndexes()
{
    TestAdjacentDifferenceWithBigIndexesHelper(30);
    TestAdjacentDifferenceWithBigIndexesHelper(31);
    TestAdjacentDifferenceWithBigIndexesHelper(32);
    TestAdjacentDifferenceWithBigIndexesHelper(33);
}
DECLARE_UNITTEST(TestAdjacentDifferenceWithBigIndexes);
