#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/inner_product.h>
#include <thrust/execution_policy.h>


#ifdef THRUST_TEST_DEVICE_SIDE
template<typename ExecutionPolicy, typename Iterator1, typename Iterator2, typename T, typename Iterator3>
__global__
void inner_product_kernel(ExecutionPolicy exec, Iterator1 first1, Iterator1 last1, Iterator2 first2, T init, Iterator3 result)
{
  *result = thrust::inner_product(exec, first1, last1, first2, init);
}


template<typename ExecutionPolicy>
void TestInnerProductDevice(ExecutionPolicy exec)
{
  size_t n = 1000;

  thrust::host_vector<int> h_v1 = unittest::random_integers<int>(n);
  thrust::host_vector<int> h_v2 = unittest::random_integers<int>(n);
  
  thrust::device_vector<int> d_v1 = h_v1;
  thrust::device_vector<int> d_v2 = h_v2;
  
  thrust::device_vector<int> result(1);
  
  int init = 13;
  
  int expected = thrust::inner_product(h_v1.begin(), h_v1.end(), h_v2.begin(), init);

  inner_product_kernel<<<1,1>>>(exec, d_v1.begin(), d_v1.end(), d_v2.begin(), init, result.begin());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }
  
  ASSERT_EQUAL(expected, result[0]);
}


void TestInnerProductDeviceSeq()
{
  TestInnerProductDevice(thrust::seq);
};
DECLARE_UNITTEST(TestInnerProductDeviceSeq);


void TestInnerProductDeviceDevice()
{
  TestInnerProductDevice(thrust::device);
};
DECLARE_UNITTEST(TestInnerProductDeviceDevice);
#endif


void TestInnerProductCudaStreams()
{
  thrust::device_vector<int> v1(3);
  thrust::device_vector<int> v2(3);
  v1[0] =  1; v1[1] = -2; v1[2] =  3;
  v2[0] = -4; v2[1] =  5; v2[2] =  6;

  hipStream_t s;
  hipStreamCreate(&s);
  
  int init = 3;
  int result = thrust::inner_product(thrust::cuda::par.on(s), v1.begin(), v1.end(), v2.begin(), init);
  ASSERT_EQUAL(result, 7);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestInnerProductCudaStreams);

