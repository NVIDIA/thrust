#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/swap.h>
#include <thrust/execution_policy.h>


#ifdef THRUST_TEST_DEVICE_SIDE
template<typename ExecutionPolicy, typename Iterator1, typename Iterator2>
__global__
void swap_ranges_kernel(ExecutionPolicy exec, Iterator1 first1, Iterator1 last1, Iterator2 first2)
{
  thrust::swap_ranges(exec, first1, last1, first2);
}


template<typename ExecutionPolicy>
void TestSwapRangesDevice(ExecutionPolicy exec)
{
  typedef thrust::device_vector<int> Vector;

  Vector v1(5);
  v1[0] = 0; v1[1] = 1; v1[2] = 2; v1[3] = 3; v1[4] = 4;

  Vector v2(5);
  v2[0] = 5; v2[1] = 6; v2[2] = 7; v2[3] = 8; v2[4] = 9;

  swap_ranges_kernel<<<1,1>>>(exec, v1.begin(), v1.end(), v2.begin());
  hipError_t const err = hipDeviceSynchronize();
  ASSERT_EQUAL(hipSuccess, err);

  ASSERT_EQUAL(v1[0], 5);
  ASSERT_EQUAL(v1[1], 6);
  ASSERT_EQUAL(v1[2], 7);
  ASSERT_EQUAL(v1[3], 8);
  ASSERT_EQUAL(v1[4], 9);
  
  ASSERT_EQUAL(v2[0], 0);
  ASSERT_EQUAL(v2[1], 1);
  ASSERT_EQUAL(v2[2], 2);
  ASSERT_EQUAL(v2[3], 3);
  ASSERT_EQUAL(v2[4], 4);
}

void TestSwapRangesDeviceSeq()
{
  TestSwapRangesDevice(thrust::seq);
}
DECLARE_UNITTEST(TestSwapRangesDeviceSeq);

void TestSwapRangesDeviceDevice()
{
  TestSwapRangesDevice(thrust::device);
}
DECLARE_UNITTEST(TestSwapRangesDeviceDevice);
#endif

void TestSwapRangesCudaStreams()
{
  typedef thrust::device_vector<int> Vector;

  Vector v1(5);
  v1[0] = 0; v1[1] = 1; v1[2] = 2; v1[3] = 3; v1[4] = 4;

  Vector v2(5);
  v2[0] = 5; v2[1] = 6; v2[2] = 7; v2[3] = 8; v2[4] = 9;

  hipStream_t s;
  hipStreamCreate(&s);

  thrust::swap_ranges(thrust::cuda::par.on(s), v1.begin(), v1.end(), v2.begin());
  hipStreamSynchronize(s);

  ASSERT_EQUAL(v1[0], 5);
  ASSERT_EQUAL(v1[1], 6);
  ASSERT_EQUAL(v1[2], 7);
  ASSERT_EQUAL(v1[3], 8);
  ASSERT_EQUAL(v1[4], 9);
  
  ASSERT_EQUAL(v2[0], 0);
  ASSERT_EQUAL(v2[1], 1);
  ASSERT_EQUAL(v2[2], 2);
  ASSERT_EQUAL(v2[3], 3);
  ASSERT_EQUAL(v2[4], 4);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestSwapRangesCudaStreams);

