#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>


template<typename ExecutionPolicy, typename Iterator1, typename Iterator2, typename Compare, typename Iterator3>
__global__
void sort_by_key_kernel(ExecutionPolicy exec, Iterator1 keys_first, Iterator1 keys_last, Iterator2 values_first, Compare comp, Iterator3 is_supported)
{
  *is_supported = true;
  thrust::sort_by_key(exec, keys_first, keys_last, values_first, comp);
}


template<typename T>
struct my_less
{
  __host__ __device__
  bool operator()(const T& lhs, const T& rhs) const
  {
    return lhs < rhs;
  }
};


template<typename T, typename ExecutionPolicy, typename Compare>
void TestComparisonSortByKeyDevice(ExecutionPolicy exec, const size_t n, Compare comp)
{
  thrust::host_vector<T>   h_keys = unittest::random_integers<T>(n);
  thrust::device_vector<T> d_keys = h_keys;

  thrust::host_vector<T>   h_values = h_keys;
  thrust::device_vector<T> d_values = d_keys;
  
  thrust::device_vector<bool> is_supported(1);
  sort_by_key_kernel<<<1,1>>>(exec, d_keys.begin(), d_keys.end(), d_values.begin(), comp, is_supported.begin());
  hipError_t const err = hipDeviceSynchronize();
  ASSERT_EQUAL(hipSuccess, err);

  if(is_supported[0])
  {
    thrust::sort_by_key(h_keys.begin(), h_keys.end(), h_values.begin(), comp);
    
    ASSERT_EQUAL(h_keys, d_keys);
    ASSERT_EQUAL(h_values, d_values);
  }
};


template<typename T>
  struct TestComparisonSortByKeyDeviceSeq
{
  void operator()(const size_t n)
  {
    TestComparisonSortByKeyDevice<T>(thrust::seq, n, my_less<T>());
  }
};
VariableUnitTest<
  TestComparisonSortByKeyDeviceSeq,
  unittest::type_list<unittest::int8_t,unittest::int32_t>
> TestComparisonSortByKeyDeviceSeqInstance;


template<typename T>
  struct TestComparisonSortByKeyDeviceDevice
{
  void operator()(const size_t n)
  {
    TestComparisonSortByKeyDevice<T>(thrust::device, n, my_less<T>());
  }
};
VariableUnitTest<
  TestComparisonSortByKeyDeviceDevice,
  unittest::type_list<unittest::int8_t,unittest::int32_t>
> TestComparisonSortByKeyDeviceDeviceDeviceInstance;


template<typename T, typename ExecutionPolicy>
void TestSortByKeyDevice(ExecutionPolicy exec, const size_t n)
{
  TestComparisonSortByKeyDevice<T>(exec, n, thrust::less<T>());
};


template<typename T>
  struct TestSortByKeyDeviceSeq
{
  void operator()(const size_t n)
  {
    TestSortByKeyDevice<T>(thrust::seq, n);
  }
};
VariableUnitTest<
  TestSortByKeyDeviceSeq,
  unittest::type_list<unittest::int8_t,unittest::int32_t>
> TestSortByKeyDeviceSeqInstance;


template<typename T>
  struct TestSortByKeyDeviceDevice
{
  void operator()(const size_t n)
  {
    TestSortByKeyDevice<T>(thrust::device, n);
  }
};
VariableUnitTest<
  TestSortByKeyDeviceDevice,
  unittest::type_list<unittest::int8_t,unittest::int32_t>
> TestSortByKeyDeviceDeviceInstance;


void TestComparisonSortByKeyCudaStreams()
{
  thrust::device_vector<int> keys(10);
  thrust::device_vector<int> vals(10);

  keys[0] = 9; vals[0] = 9;
  keys[1] = 3; vals[1] = 3;
  keys[2] = 2; vals[2] = 2;
  keys[3] = 0; vals[3] = 0;
  keys[4] = 4; vals[4] = 4;
  keys[5] = 7; vals[5] = 7;
  keys[6] = 8; vals[6] = 8;
  keys[7] = 1; vals[7] = 1;
  keys[8] = 5; vals[8] = 5;
  keys[9] = 6; vals[9] = 6;

  hipStream_t s;
  hipStreamCreate(&s);

  thrust::sort_by_key(thrust::cuda::par.on(s), keys.begin(), keys.end(), vals.begin(), my_less<int>());
  hipStreamSynchronize(s);

  ASSERT_EQUAL(true, thrust::is_sorted(keys.begin(), keys.end()));
  ASSERT_EQUAL(true, thrust::is_sorted(vals.begin(), vals.end()));
                      
  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestComparisonSortByKeyCudaStreams);


void TestSortByKeyCudaStreams()
{
  thrust::device_vector<int> keys(10);
  thrust::device_vector<int> vals(10);

  keys[0] = 9; vals[0] = 9;
  keys[1] = 3; vals[1] = 3;
  keys[2] = 2; vals[2] = 2;
  keys[3] = 0; vals[3] = 0;
  keys[4] = 4; vals[4] = 4;
  keys[5] = 7; vals[5] = 7;
  keys[6] = 8; vals[6] = 8;
  keys[7] = 1; vals[7] = 1;
  keys[8] = 5; vals[8] = 5;
  keys[9] = 6; vals[9] = 6;

  hipStream_t s;
  hipStreamCreate(&s);

  thrust::sort_by_key(thrust::cuda::par.on(s), keys.begin(), keys.end(), vals.begin());
  hipStreamSynchronize(s);

  ASSERT_EQUAL(true, thrust::is_sorted(keys.begin(), keys.end()));
  ASSERT_EQUAL(true, thrust::is_sorted(vals.begin(), vals.end()));
                      
  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestSortByKeyCudaStreams);

