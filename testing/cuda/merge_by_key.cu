#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/merge.h>
#include <thrust/functional.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>


#ifdef THRUST_TEST_DEVICE_SIDE
template<typename ExecutionPolicy,
         typename Iterator1,
         typename Iterator2,
         typename Iterator3,
         typename Iterator4,
         typename Iterator5,
         typename Iterator6,
         typename Iterator7>
__global__
void merge_by_key_kernel(ExecutionPolicy exec,
                         Iterator1 keys_first1, Iterator1 keys_last1,
                         Iterator2 keys_first2, Iterator2 keys_last2,
                         Iterator3 values_first1,
                         Iterator4 values_first2,
                         Iterator5 keys_result,
                         Iterator6 values_result,
                         Iterator7 result)
{
  *result = thrust::merge_by_key(exec, keys_first1, keys_last1, keys_first2, keys_last2, values_first1, values_first2, keys_result, values_result);
}


template<typename ExecutionPolicy>
void TestMergeByKeyDevice(ExecutionPolicy exec)
{
  thrust::device_vector<int> a_key(3), a_val(3), b_key(4), b_val(4);

  a_key[0] = 0;  a_key[1] = 2; a_key[2] = 4;
  a_val[0] = 13; a_val[1] = 7; a_val[2] = 42;

  b_key[0] = 0 ; b_key[1] = 3;  b_key[2] = 3; b_key[3] = 4;
  b_val[0] = 42; b_val[1] = 42; b_val[2] = 7; b_val[3] = 13;

  thrust::device_vector<int> ref_key(7), ref_val(7);
  ref_key[0] = 0; ref_val[0] = 13;
  ref_key[1] = 0; ref_val[1] = 42;
  ref_key[2] = 2; ref_val[2] = 7;
  ref_key[3] = 3; ref_val[3] = 42;
  ref_key[4] = 3; ref_val[4] = 7;
  ref_key[5] = 4; ref_val[5] = 42;
  ref_key[6] = 4; ref_val[6] = 13;

  thrust::device_vector<int> result_key(7), result_val(7);

  typedef typename thrust::device_vector<int>::iterator Iterator;

  thrust::device_vector<thrust::pair<Iterator,Iterator> > result_ends(1);

  merge_by_key_kernel<<<1,1>>>(exec,
                               a_key.begin(), a_key.end(),
                               b_key.begin(), b_key.end(),
                               a_val.begin(), b_val.begin(),
                               result_key.begin(),
                               result_val.begin(),
                               result_ends.begin());
  hipError_t const err = hipDeviceSynchronize();
  ASSERT_EQUAL(hipSuccess, err);

  thrust::pair<Iterator,Iterator> ends = result_ends[0];

  ASSERT_EQUAL_QUIET(result_key.end(), ends.first);
  ASSERT_EQUAL_QUIET(result_val.end(), ends.second);
  ASSERT_EQUAL(ref_key, result_key);
  ASSERT_EQUAL(ref_val, result_val);
}


void TestMergeByKeyDeviceSeq()
{
  TestMergeByKeyDevice(thrust::seq);
}
DECLARE_UNITTEST(TestMergeByKeyDeviceSeq);


void TestMergeByKeyDeviceDevice()
{
  TestMergeByKeyDevice(thrust::device);
}
DECLARE_UNITTEST(TestMergeByKeyDeviceDevice);
#endif


void TestMergeByKeyCudaStreams()
{
  typedef thrust::device_vector<int> Vector;
  typedef Vector::iterator Iterator;

  Vector a_key(3), a_val(3), b_key(4), b_val(4);

  a_key[0] = 0;  a_key[1] = 2; a_key[2] = 4;
  a_val[0] = 13; a_val[1] = 7; a_val[2] = 42;

  b_key[0] = 0 ; b_key[1] = 3;  b_key[2] = 3; b_key[3] = 4;
  b_val[0] = 42; b_val[1] = 42; b_val[2] = 7; b_val[3] = 13;

  Vector ref_key(7), ref_val(7);
  ref_key[0] = 0; ref_val[0] = 13;
  ref_key[1] = 0; ref_val[1] = 42;
  ref_key[2] = 2; ref_val[2] = 7;
  ref_key[3] = 3; ref_val[3] = 42;
  ref_key[4] = 3; ref_val[4] = 7;
  ref_key[5] = 4; ref_val[5] = 42;
  ref_key[6] = 4; ref_val[6] = 13;

  Vector result_key(7), result_val(7);

  hipStream_t s;
  hipStreamCreate(&s);

  thrust::pair<Iterator,Iterator> ends =
    thrust::merge_by_key(thrust::cuda::par.on(s),
                         a_key.begin(), a_key.end(),
                         b_key.begin(), b_key.end(),
                         a_val.begin(), b_val.begin(),
                         result_key.begin(),
                         result_val.begin());

  hipStreamSynchronize(s);

  ASSERT_EQUAL_QUIET(result_key.end(), ends.first);
  ASSERT_EQUAL_QUIET(result_val.end(), ends.second);
  ASSERT_EQUAL(ref_key, result_key);
  ASSERT_EQUAL(ref_val, result_val);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestMergeByKeyCudaStreams);

