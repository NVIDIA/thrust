#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/find.h>
#include <thrust/execution_policy.h>


template<typename T>
struct equal_to_value_pred
{
    T value;

    equal_to_value_pred(T value) : value(value) {}

    __host__ __device__
    bool operator()(T v) const { return v == value; }
};


template<typename T>
struct not_equal_to_value_pred
{
    T value;

    not_equal_to_value_pred(T value) : value(value) {}

    __host__ __device__
    bool operator()(T v) const { return v != value; }
};


template<typename T>
struct less_than_value_pred
{
    T value;

    less_than_value_pred(T value) : value(value) {}

    __host__ __device__
    bool operator()(T v) const { return v < value; }
};


#ifdef THRUST_TEST_DEVICE_SIDE
template<typename ExecutionPolicy, typename Iterator, typename T, typename Iterator2>
__global__ void find_kernel(ExecutionPolicy exec, Iterator first, Iterator last, T value, Iterator2 result)
{
  *result = thrust::find(exec, first, last, value);
}


template<typename ExecutionPolicy>
void TestFindDevice(ExecutionPolicy exec)
{
  size_t n = 100;

  thrust::host_vector<int>   h_data = unittest::random_integers<int>(n);
  thrust::device_vector<int> d_data = h_data;
  
  typename thrust::host_vector<int>::iterator   h_iter;
  
  typedef typename thrust::device_vector<int>::iterator iter_type;
  thrust::device_vector<iter_type> d_result(1);
  
  h_iter = thrust::find(h_data.begin(), h_data.end(), int(0));

  find_kernel<<<1,1>>>(exec, d_data.begin(), d_data.end(), int(0), d_result.begin());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }
  
  ASSERT_EQUAL(h_iter - h_data.begin(), (iter_type)d_result[0] - d_data.begin());
  
  for(size_t i = 1; i < n; i *= 2)
  {
    int sample = h_data[i];

    h_iter = thrust::find(h_data.begin(), h_data.end(), sample);

    find_kernel<<<1,1>>>(exec, d_data.begin(), d_data.end(), sample, d_result.begin());
    {
      hipError_t const err = hipDeviceSynchronize();
      ASSERT_EQUAL(hipSuccess, err);
    }

    ASSERT_EQUAL(h_iter - h_data.begin(), (iter_type)d_result[0] - d_data.begin());
  }
}


void TestFindDeviceSeq()
{
  TestFindDevice(thrust::seq);
};
DECLARE_UNITTEST(TestFindDeviceSeq);


void TestFindDeviceDevice()
{
  TestFindDevice(thrust::device);
};
DECLARE_UNITTEST(TestFindDeviceDevice);


template<typename ExecutionPolicy, typename Iterator, typename Predicate, typename Iterator2>
__global__ void find_if_kernel(ExecutionPolicy exec, Iterator first, Iterator last, Predicate pred, Iterator2 result)
{
  *result = thrust::find_if(exec, first, last, pred);
}


template<typename ExecutionPolicy>
void TestFindIfDevice(ExecutionPolicy exec)
{
  size_t n = 100;

  thrust::host_vector<int>   h_data = unittest::random_integers<int>(n);
  thrust::device_vector<int> d_data = h_data;
  
  typename thrust::host_vector<int>::iterator   h_iter;
  
  typedef typename thrust::device_vector<int>::iterator iter_type;
  thrust::device_vector<iter_type> d_result(1);
  
  h_iter = thrust::find_if(h_data.begin(), h_data.end(), equal_to_value_pred<int>(0));

  find_if_kernel<<<1,1>>>(exec, d_data.begin(), d_data.end(), equal_to_value_pred<int>(0), d_result.begin());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  ASSERT_EQUAL(h_iter - h_data.begin(), (iter_type)d_result[0] - d_data.begin());
  
  for (size_t i = 1; i < n; i *= 2)
  {
    int sample = h_data[i];

    h_iter = thrust::find_if(h_data.begin(), h_data.end(), equal_to_value_pred<int>(sample));

    find_if_kernel<<<1,1>>>(exec, d_data.begin(), d_data.end(), equal_to_value_pred<int>(sample), d_result.begin());
    {
      hipError_t const err = hipDeviceSynchronize();
      ASSERT_EQUAL(hipSuccess, err);
    }

    ASSERT_EQUAL(h_iter - h_data.begin(), (iter_type)d_result[0] - d_data.begin());
  }
}


void TestFindIfDeviceSeq()
{
  TestFindIfDevice(thrust::seq);
};
DECLARE_UNITTEST(TestFindIfDeviceSeq);


void TestFindIfDeviceDevice()
{
  TestFindIfDevice(thrust::device);
};
DECLARE_UNITTEST(TestFindIfDeviceDevice);


template<typename ExecutionPolicy, typename Iterator, typename Predicate, typename Iterator2>
__global__ void find_if_not_kernel(ExecutionPolicy exec, Iterator first, Iterator last, Predicate pred, Iterator2 result)
{
  *result = thrust::find_if_not(exec, first, last, pred);
}


template<typename ExecutionPolicy>
void TestFindIfNotDevice(ExecutionPolicy exec)
{
  size_t n = 100;
  thrust::host_vector<int>   h_data = unittest::random_integers<int>(n);
  thrust::device_vector<int> d_data = h_data;
  
  typename thrust::host_vector<int>::iterator   h_iter;
  
  typedef typename thrust::device_vector<int>::iterator iter_type;
  thrust::device_vector<iter_type> d_result(1);
  
  h_iter = thrust::find_if_not(h_data.begin(), h_data.end(), not_equal_to_value_pred<int>(0));

  find_if_not_kernel<<<1,1>>>(exec, d_data.begin(), d_data.end(), not_equal_to_value_pred<int>(0), d_result.begin());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  ASSERT_EQUAL(h_iter - h_data.begin(), (iter_type)d_result[0] - d_data.begin());
  
  for(size_t i = 1; i < n; i *= 2)
  {
    int sample = h_data[i];

    h_iter = thrust::find_if_not(h_data.begin(), h_data.end(), not_equal_to_value_pred<int>(sample));

    find_if_not_kernel<<<1,1>>>(exec, d_data.begin(), d_data.end(), not_equal_to_value_pred<int>(sample), d_result.begin());
    {
      hipError_t const err = hipDeviceSynchronize();
      ASSERT_EQUAL(hipSuccess, err);
    }

    ASSERT_EQUAL(h_iter - h_data.begin(), (iter_type)d_result[0] - d_data.begin());
  }
}


void TestFindIfNotDeviceSeq()
{
  TestFindIfNotDevice(thrust::seq);
};
DECLARE_UNITTEST(TestFindIfNotDeviceSeq);


void TestFindIfNotDeviceDevice()
{
  TestFindIfNotDevice(thrust::device);
};
DECLARE_UNITTEST(TestFindIfNotDeviceDevice);
#endif


void TestFindCudaStreams()
{
  thrust::device_vector<int> vec(5);
  vec[0] = 1;
  vec[1] = 2;
  vec[2] = 3;
  vec[3] = 3;
  vec[4] = 5;

  hipStream_t s;
  hipStreamCreate(&s);
  
  ASSERT_EQUAL(thrust::find(thrust::cuda::par.on(s), vec.begin(), vec.end(), 0) - vec.begin(), 5);
  ASSERT_EQUAL(thrust::find(thrust::cuda::par.on(s), vec.begin(), vec.end(), 1) - vec.begin(), 0);
  ASSERT_EQUAL(thrust::find(thrust::cuda::par.on(s), vec.begin(), vec.end(), 2) - vec.begin(), 1);
  ASSERT_EQUAL(thrust::find(thrust::cuda::par.on(s), vec.begin(), vec.end(), 3) - vec.begin(), 2);
  ASSERT_EQUAL(thrust::find(thrust::cuda::par.on(s), vec.begin(), vec.end(), 4) - vec.begin(), 5);
  ASSERT_EQUAL(thrust::find(thrust::cuda::par.on(s), vec.begin(), vec.end(), 5) - vec.begin(), 4);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestFindCudaStreams);

