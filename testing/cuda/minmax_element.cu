#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/extrema.h>


#ifdef THRUST_TEST_DEVICE_SIDE
template<typename ExecutionPolicy, typename Iterator1, typename Iterator2>
__global__
void minmax_element_kernel(ExecutionPolicy exec, Iterator1 first, Iterator1 last, Iterator2 result)
{
  *result = thrust::minmax_element(exec, first, last);
}


template<typename ExecutionPolicy, typename Iterator1, typename Iterator2, typename BinaryPredicate>
__global__
void minmax_element_kernel(ExecutionPolicy exec, Iterator1 first, Iterator1 last, BinaryPredicate pred, Iterator2 result)
{
  *result = thrust::minmax_element(exec, first, last, pred);
}


template<typename ExecutionPolicy>
void TestMinMaxElementDevice(ExecutionPolicy exec)
{
  size_t n = 1000;

  thrust::host_vector<int>   h_data = unittest::random_samples<int>(n);
  thrust::device_vector<int> d_data = h_data;
  
  typename thrust::host_vector<int>::iterator   h_min;
  typename thrust::host_vector<int>::iterator   h_max;
  typename thrust::device_vector<int>::iterator d_min;
  typename thrust::device_vector<int>::iterator d_max;

  typedef thrust::pair<
    typename thrust::device_vector<int>::iterator,
    typename thrust::device_vector<int>::iterator
  > pair_type;

  thrust::device_vector<pair_type> d_result(1);
  
  h_min = thrust::minmax_element(h_data.begin(), h_data.end()).first;
  h_max = thrust::minmax_element(h_data.begin(), h_data.end()).second;

  d_min = thrust::minmax_element(d_data.begin(), d_data.end()).first;
  d_max = thrust::minmax_element(d_data.begin(), d_data.end()).second;

  minmax_element_kernel<<<1,1>>>(exec, d_data.begin(), d_data.end(), d_result.begin());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  d_min = ((pair_type)d_result[0]).first;
  d_max = ((pair_type)d_result[0]).second;
  
  ASSERT_EQUAL(h_min - h_data.begin(), d_min - d_data.begin());
  ASSERT_EQUAL(h_max - h_data.begin(), d_max - d_data.begin());
  
  h_max = thrust::minmax_element(h_data.begin(), h_data.end(), thrust::greater<int>()).first;
  h_min = thrust::minmax_element(h_data.begin(), h_data.end(), thrust::greater<int>()).second;

  minmax_element_kernel<<<1,1>>>(exec, d_data.begin(), d_data.end(), thrust::greater<int>(), d_result.begin());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  d_max = ((pair_type)d_result[0]).first;
  d_min = ((pair_type)d_result[0]).second;
  
  ASSERT_EQUAL(h_min - h_data.begin(), d_min - d_data.begin());
  ASSERT_EQUAL(h_max - h_data.begin(), d_max - d_data.begin());
}


void TestMinMaxElementDeviceSeq()
{
  TestMinMaxElementDevice(thrust::seq);
}
DECLARE_UNITTEST(TestMinMaxElementDeviceSeq);


void TestMinMaxElementDeviceDevice()
{
  TestMinMaxElementDevice(thrust::device);
}
DECLARE_UNITTEST(TestMinMaxElementDeviceDevice);
#endif


void TestMinMaxElementCudaStreams()
{
  typedef thrust::device_vector<int> Vector;

  Vector data(6);
  data[0] = 3;
  data[1] = 5;
  data[2] = 1;
  data[3] = 2;
  data[4] = 5;
  data[5] = 1;

  hipStream_t s;
  hipStreamCreate(&s);

  ASSERT_EQUAL( *thrust::minmax_element(thrust::cuda::par.on(s), data.begin(), data.end()).first,  1);
  ASSERT_EQUAL( *thrust::minmax_element(thrust::cuda::par.on(s), data.begin(), data.end()).second, 5);
  ASSERT_EQUAL(  thrust::minmax_element(thrust::cuda::par.on(s), data.begin(), data.end()).first  - data.begin(), 2);
  ASSERT_EQUAL(  thrust::minmax_element(thrust::cuda::par.on(s), data.begin(), data.end()).second - data.begin(), 1);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestMinMaxElementCudaStreams);

void TestMinMaxElementDevicePointer()
{
  typedef thrust::device_vector<int> Vector;
  typedef Vector::value_type T;

  Vector data(6);
  data[0] = 3;
  data[1] = 5;
  data[2] = 1;
  data[3] = 2;
  data[4] = 5;
  data[5] = 1;

  T* raw_ptr = thrust::raw_pointer_cast(data.data());
  size_t n = data.size();
  ASSERT_EQUAL( thrust::minmax_element(thrust::device, raw_ptr, raw_ptr+n).first - raw_ptr,  2);
  ASSERT_EQUAL( thrust::minmax_element(thrust::device, raw_ptr, raw_ptr+n).second - raw_ptr, 1);
}
DECLARE_UNITTEST(TestMinMaxElementDevicePointer);

