#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/tabulate.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>


#ifdef THRUST_TEST_DEVICE_SIDE
template<typename ExecutionPolicy, typename Iterator, typename Function>
__global__
void tabulate_kernel(ExecutionPolicy exec, Iterator first, Iterator last, Function f)
{
  thrust::tabulate(exec, first, last, f);
}


template<typename ExecutionPolicy>
void TestTabulateDevice(ExecutionPolicy exec)
{
  typedef thrust::device_vector<int> Vector;
  using namespace thrust::placeholders;
  typedef typename Vector::value_type T;
  
  Vector v(5);

  tabulate_kernel<<<1,1>>>(exec, v.begin(), v.end(), thrust::identity<T>());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  ASSERT_EQUAL(v[0], 0);
  ASSERT_EQUAL(v[1], 1);
  ASSERT_EQUAL(v[2], 2);
  ASSERT_EQUAL(v[3], 3);
  ASSERT_EQUAL(v[4], 4);

  tabulate_kernel<<<1,1>>>(exec, v.begin(), v.end(), -_1);
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  ASSERT_EQUAL(v[0],  0);
  ASSERT_EQUAL(v[1], -1);
  ASSERT_EQUAL(v[2], -2);
  ASSERT_EQUAL(v[3], -3);
  ASSERT_EQUAL(v[4], -4);
  
  tabulate_kernel<<<1,1>>>(exec, v.begin(), v.end(), _1 * _1 * _1);
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  ASSERT_EQUAL(v[0], 0);
  ASSERT_EQUAL(v[1], 1);
  ASSERT_EQUAL(v[2], 8);
  ASSERT_EQUAL(v[3], 27);
  ASSERT_EQUAL(v[4], 64);
}

void TestTabulateDeviceSeq()
{
  TestTabulateDevice(thrust::seq);
}
DECLARE_UNITTEST(TestTabulateDeviceSeq);

void TestTabulateDeviceDevice()
{
  TestTabulateDevice(thrust::device);
}
DECLARE_UNITTEST(TestTabulateDeviceDevice);
#endif

void TestTabulateCudaStreams()
{
  using namespace thrust::placeholders;
  typedef thrust::device_vector<int> Vector;
  typedef Vector::value_type T;
  
  Vector v(5);

  hipStream_t s;
  hipStreamCreate(&s);

  thrust::tabulate(thrust::cuda::par.on(s), v.begin(), v.end(), thrust::identity<T>());
  hipStreamSynchronize(s);

  ASSERT_EQUAL(v[0], 0);
  ASSERT_EQUAL(v[1], 1);
  ASSERT_EQUAL(v[2], 2);
  ASSERT_EQUAL(v[3], 3);
  ASSERT_EQUAL(v[4], 4);

  thrust::tabulate(thrust::cuda::par.on(s), v.begin(), v.end(), -_1);
  hipStreamSynchronize(s);

  ASSERT_EQUAL(v[0],  0);
  ASSERT_EQUAL(v[1], -1);
  ASSERT_EQUAL(v[2], -2);
  ASSERT_EQUAL(v[3], -3);
  ASSERT_EQUAL(v[4], -4);
  
  thrust::tabulate(thrust::cuda::par.on(s), v.begin(), v.end(), _1 * _1 * _1);
  hipStreamSynchronize(s);

  ASSERT_EQUAL(v[0], 0);
  ASSERT_EQUAL(v[1], 1);
  ASSERT_EQUAL(v[2], 8);
  ASSERT_EQUAL(v[3], 27);
  ASSERT_EQUAL(v[4], 64);

  hipStreamSynchronize(s);
}
DECLARE_UNITTEST(TestTabulateCudaStreams);

