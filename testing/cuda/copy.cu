#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/copy.h>
#include <thrust/execution_policy.h>


#ifdef THRUST_TEST_DEVICE_SIDE
template<typename ExecutionPolicy, typename Iterator1, typename Iterator2>
__global__
void copy_kernel(ExecutionPolicy exec, Iterator1 first, Iterator1 last, Iterator2 result)
{
  thrust::copy(exec, first, last, result);
}


template<typename T, typename ExecutionPolicy>
void TestCopyDevice(ExecutionPolicy exec, size_t n)
{
  thrust::host_vector<T>   h_src = unittest::random_integers<T>(n);
  thrust::host_vector<T>   h_dst(n);

  thrust::device_vector<T> d_src = h_src;
  thrust::device_vector<T> d_dst(n);
  
  thrust::copy(h_src.begin(), h_src.end(), h_dst.begin());
  copy_kernel<<<1,1>>>(exec, d_src.begin(), d_src.end(), d_dst.begin());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }
  
  ASSERT_EQUAL(h_dst, d_dst);
}


template<typename T>
void TestCopyDeviceSeq(size_t n)
{
  TestCopyDevice<T>(thrust::seq, n);
}
DECLARE_VARIABLE_UNITTEST(TestCopyDeviceSeq);


template<typename T>
void TestCopyDeviceDevice(size_t n)
{
  TestCopyDevice<T>(thrust::device, n);
}
DECLARE_VARIABLE_UNITTEST(TestCopyDeviceDevice);


template<typename ExecutionPolicy, typename Iterator1, typename Size, typename Iterator2>
__global__
void copy_n_kernel(ExecutionPolicy exec, Iterator1 first, Size n, Iterator2 result)
{
  thrust::copy_n(exec, first, n, result);
}


template<typename T, typename ExecutionPolicy>
void TestCopyNDevice(ExecutionPolicy exec, size_t n)
{
  thrust::host_vector<T>   h_src = unittest::random_integers<T>(n);
  thrust::host_vector<T>   h_dst(n);

  thrust::device_vector<T> d_src = h_src;
  thrust::device_vector<T> d_dst(n);
  
  thrust::copy_n(h_src.begin(), h_src.size(), h_dst.begin());
  copy_n_kernel<<<1,1>>>(exec, d_src.begin(), d_src.size(), d_dst.begin());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }
  
  ASSERT_EQUAL(h_dst, d_dst);
}


template<typename T>
void TestCopyNDeviceSeq(size_t n)
{
  TestCopyNDevice<T>(thrust::seq, n);
}
DECLARE_VARIABLE_UNITTEST(TestCopyNDeviceSeq);


template<typename T>
void TestCopyNDeviceDevice(size_t n)
{
  TestCopyNDevice<T>(thrust::device, n);
}
DECLARE_VARIABLE_UNITTEST(TestCopyNDeviceDevice);
#endif

