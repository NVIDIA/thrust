#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/unique.h>
#include <thrust/execution_policy.h>


template<typename ExecutionPolicy, typename Iterator1, typename Iterator2>
__global__
void unique_kernel(ExecutionPolicy exec, Iterator1 first, Iterator1 last, Iterator2 result)
{
  *result = thrust::unique(exec, first, last);
}


template<typename ExecutionPolicy, typename Iterator1, typename BinaryPredicate, typename Iterator2>
__global__
void unique_kernel(ExecutionPolicy exec, Iterator1 first, Iterator1 last, BinaryPredicate pred, Iterator2 result)
{
  *result = thrust::unique(exec, first, last, pred);
}


template<typename T>
struct is_equal_div_10_unique
{
  __host__ __device__
  bool operator()(const T x, const T& y) const { return ((int) x / 10) == ((int) y / 10); }
};


template<typename ExecutionPolicy>
void TestUniqueDevice(ExecutionPolicy exec)
{
  typedef thrust::device_vector<int> Vector;
  typedef Vector::value_type T;

  Vector data(10);
  data[0] = 11; 
  data[1] = 11; 
  data[2] = 12;
  data[3] = 20; 
  data[4] = 29; 
  data[5] = 21; 
  data[6] = 21; 
  data[7] = 31; 
  data[8] = 31; 
  data[9] = 37; 

  thrust::device_vector<Vector::iterator> new_last_vec(1);
  Vector::iterator new_last;
  
  unique_kernel<<<1,1>>>(exec, data.begin(), data.end(), new_last_vec.begin());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  new_last = new_last_vec[0];

  ASSERT_EQUAL(new_last - data.begin(), 7);
  ASSERT_EQUAL(data[0], 11);
  ASSERT_EQUAL(data[1], 12);
  ASSERT_EQUAL(data[2], 20);
  ASSERT_EQUAL(data[3], 29);
  ASSERT_EQUAL(data[4], 21);
  ASSERT_EQUAL(data[5], 31);
  ASSERT_EQUAL(data[6], 37);

  unique_kernel<<<1,1>>>(exec, data.begin(), new_last, is_equal_div_10_unique<T>(), new_last_vec.begin());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  new_last = new_last_vec[0];

  ASSERT_EQUAL(new_last - data.begin(), 3);
  ASSERT_EQUAL(data[0], 11);
  ASSERT_EQUAL(data[1], 20);
  ASSERT_EQUAL(data[2], 31);
}


void TestUniqueDeviceSeq()
{
  TestUniqueDevice(thrust::seq);
}
DECLARE_UNITTEST(TestUniqueDeviceSeq);


void TestUniqueDeviceDevice()
{
  TestUniqueDevice(thrust::device);
}
DECLARE_UNITTEST(TestUniqueDeviceDevice);


void TestUniqueDeviceNoSync()
{
  TestUniqueDevice(thrust::cuda::par_nosync);
}
DECLARE_UNITTEST(TestUniqueDeviceNoSync);


template<typename ExecutionPolicy>
void TestUniqueCudaStreams(ExecutionPolicy policy)
{
  typedef thrust::device_vector<int> Vector;
  typedef Vector::value_type T;

  Vector data(10);
  data[0] = 11; 
  data[1] = 11; 
  data[2] = 12;
  data[3] = 20; 
  data[4] = 29; 
  data[5] = 21; 
  data[6] = 21; 
  data[7] = 31; 
  data[8] = 31; 
  data[9] = 37; 

  thrust::device_vector<Vector::iterator> new_last_vec(1);
  Vector::iterator new_last;

  hipStream_t s;
  hipStreamCreate(&s);

  auto streampolicy = policy.on(s);
  
  new_last = thrust::unique(streampolicy, data.begin(), data.end());
  hipStreamSynchronize(s);

  ASSERT_EQUAL(new_last - data.begin(), 7);
  ASSERT_EQUAL(data[0], 11);
  ASSERT_EQUAL(data[1], 12);
  ASSERT_EQUAL(data[2], 20);
  ASSERT_EQUAL(data[3], 29);
  ASSERT_EQUAL(data[4], 21);
  ASSERT_EQUAL(data[5], 31);
  ASSERT_EQUAL(data[6], 37);

  new_last = thrust::unique(streampolicy, data.begin(), new_last, is_equal_div_10_unique<T>());
  hipStreamSynchronize(s);

  ASSERT_EQUAL(new_last - data.begin(), 3);
  ASSERT_EQUAL(data[0], 11);
  ASSERT_EQUAL(data[1], 20);
  ASSERT_EQUAL(data[2], 31);

  hipStreamDestroy(s);
}

void TestUniqueCudaStreamsSync()
{
  TestUniqueCudaStreams(thrust::cuda::par);
}
DECLARE_UNITTEST(TestUniqueCudaStreamsSync);


void TestUniqueCudaStreamsNoSync()
{
  TestUniqueCudaStreams(thrust::cuda::par_nosync);
}
DECLARE_UNITTEST(TestUniqueCudaStreamsNoSync);


template<typename ExecutionPolicy, typename Iterator1, typename Iterator2, typename Iterator3>
__global__
void unique_copy_kernel(ExecutionPolicy exec, Iterator1 first, Iterator1 last, Iterator2 result1, Iterator3 result2)
{
  *result2 = thrust::unique_copy(exec, first, last, result1);
}


template<typename ExecutionPolicy, typename Iterator1, typename Iterator2, typename BinaryPredicate, typename Iterator3>
__global__
void unique_copy_kernel(ExecutionPolicy exec, Iterator1 first, Iterator1 last, Iterator2 result1, BinaryPredicate pred, Iterator3 result2)
{
  *result2 = thrust::unique_copy(exec, first, last, result1, pred);
}


template<typename ExecutionPolicy>
void TestUniqueCopyDevice(ExecutionPolicy exec)
{
  typedef thrust::device_vector<int> Vector;
  typedef Vector::value_type T;

  Vector data(10);
  data[0] = 11; 
  data[1] = 11; 
  data[2] = 12;
  data[3] = 20; 
  data[4] = 29; 
  data[5] = 21; 
  data[6] = 21; 
  data[7] = 31; 
  data[8] = 31; 
  data[9] = 37; 
  
  Vector output(10, -1);

  thrust::device_vector<Vector::iterator> new_last_vec(1);
  Vector::iterator new_last;
  
  unique_copy_kernel<<<1,1>>>(exec, data.begin(), data.end(), output.begin(), new_last_vec.begin());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  new_last = new_last_vec[0];

  ASSERT_EQUAL(new_last - output.begin(), 7);
  ASSERT_EQUAL(output[0], 11);
  ASSERT_EQUAL(output[1], 12);
  ASSERT_EQUAL(output[2], 20);
  ASSERT_EQUAL(output[3], 29);
  ASSERT_EQUAL(output[4], 21);
  ASSERT_EQUAL(output[5], 31);
  ASSERT_EQUAL(output[6], 37);

  unique_copy_kernel<<<1,1>>>(exec, output.begin(), new_last, data.begin(), is_equal_div_10_unique<T>(), new_last_vec.begin());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  new_last = new_last_vec[0];

  ASSERT_EQUAL(new_last - data.begin(), 3);
  ASSERT_EQUAL(data[0], 11);
  ASSERT_EQUAL(data[1], 20);
  ASSERT_EQUAL(data[2], 31);
}


void TestUniqueCopyDeviceSeq()
{
  TestUniqueCopyDevice(thrust::seq);
}
DECLARE_UNITTEST(TestUniqueCopyDeviceSeq);


void TestUniqueCopyDeviceDevice()
{
  TestUniqueCopyDevice(thrust::device);
}
DECLARE_UNITTEST(TestUniqueCopyDeviceDevice);


void TestUniqueCopyDeviceNoSync()
{
  TestUniqueCopyDevice(thrust::cuda::par_nosync);
}
DECLARE_UNITTEST(TestUniqueCopyDeviceNoSync);


template<typename ExecutionPolicy>
void TestUniqueCopyCudaStreams(ExecutionPolicy policy)
{
  typedef thrust::device_vector<int> Vector;
  typedef Vector::value_type T;

  Vector data(10);
  data[0] = 11; 
  data[1] = 11; 
  data[2] = 12;
  data[3] = 20; 
  data[4] = 29; 
  data[5] = 21; 
  data[6] = 21; 
  data[7] = 31; 
  data[8] = 31; 
  data[9] = 37; 
  
  Vector output(10, -1);

  thrust::device_vector<Vector::iterator> new_last_vec(1);
  Vector::iterator new_last;

  hipStream_t s;
  hipStreamCreate(&s);

  auto streampolicy = policy.on(s);
  
  new_last = thrust::unique_copy(streampolicy, data.begin(), data.end(), output.begin());
  hipStreamSynchronize(s);

  ASSERT_EQUAL(new_last - output.begin(), 7);
  ASSERT_EQUAL(output[0], 11);
  ASSERT_EQUAL(output[1], 12);
  ASSERT_EQUAL(output[2], 20);
  ASSERT_EQUAL(output[3], 29);
  ASSERT_EQUAL(output[4], 21);
  ASSERT_EQUAL(output[5], 31);
  ASSERT_EQUAL(output[6], 37);

  new_last = thrust::unique_copy(streampolicy, output.begin(), new_last, data.begin(), is_equal_div_10_unique<T>());
  hipStreamSynchronize(s);

  ASSERT_EQUAL(new_last - data.begin(), 3);
  ASSERT_EQUAL(data[0], 11);
  ASSERT_EQUAL(data[1], 20);
  ASSERT_EQUAL(data[2], 31);

  hipStreamDestroy(s);
}

void TestUniqueCopyCudaStreamsSync()
{
  TestUniqueCopyCudaStreams(thrust::cuda::par);
}
DECLARE_UNITTEST(TestUniqueCopyCudaStreamsSync);


void TestUniqueCopyCudaStreamsNoSync()
{
  TestUniqueCopyCudaStreams(thrust::cuda::par_nosync);
}
DECLARE_UNITTEST(TestUniqueCopyCudaStreamsNoSync);


template<typename ExecutionPolicy, typename Iterator1, typename Iterator2>
__global__
void unique_count_kernel(ExecutionPolicy exec, Iterator1 first, Iterator1 last, Iterator2 result)
{
  *result = thrust::unique_count(exec, first, last);
}


template<typename ExecutionPolicy, typename Iterator1, typename BinaryPredicate, typename Iterator2>
__global__
void unique_count_kernel(ExecutionPolicy exec, Iterator1 first, Iterator1 last, BinaryPredicate pred, Iterator2 result)
{
  *result = thrust::unique_count(exec, first, last, pred);
}


template<typename ExecutionPolicy>
void TestUniqueCountDevice(ExecutionPolicy exec)
{
  typedef thrust::device_vector<int> Vector;
  typedef Vector::value_type T;

  Vector data(10);
  data[0] = 11; 
  data[1] = 11; 
  data[2] = 12;
  data[3] = 20; 
  data[4] = 29; 
  data[5] = 21; 
  data[6] = 21; 
  data[7] = 31; 
  data[8] = 31; 
  data[9] = 37; 
  
  Vector output(1, -1);
  
  unique_count_kernel<<<1,1>>>(exec, data.begin(), data.end(), output.begin());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  ASSERT_EQUAL(output[0], 7);

  unique_count_kernel<<<1,1>>>(exec, data.begin(), data.end(), is_equal_div_10_unique<T>(), output.begin());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  ASSERT_EQUAL(output[0], 3);
}


void TestUniqueCountDeviceSeq()
{
  TestUniqueCountDevice(thrust::seq);
}
DECLARE_UNITTEST(TestUniqueCountDeviceSeq);


void TestUniqueCountDeviceDevice()
{
  TestUniqueCountDevice(thrust::device);
}
DECLARE_UNITTEST(TestUniqueCountDeviceDevice);


void TestUniqueCountDeviceNoSync()
{
  TestUniqueCountDevice(thrust::cuda::par_nosync);
}
DECLARE_UNITTEST(TestUniqueCountDeviceNoSync);


template<typename ExecutionPolicy>
void TestUniqueCountCudaStreams(ExecutionPolicy policy)
{
  typedef thrust::device_vector<int> Vector;
  typedef Vector::value_type T;

  Vector data(10);
  data[0] = 11; 
  data[1] = 11; 
  data[2] = 12;
  data[3] = 20; 
  data[4] = 29; 
  data[5] = 21; 
  data[6] = 21; 
  data[7] = 31; 
  data[8] = 31; 
  data[9] = 37;

  hipStream_t s;
  hipStreamCreate(&s);

  auto streampolicy = policy.on(s);
  
  int result = thrust::unique_count(streampolicy, data.begin(), data.end());
  hipStreamSynchronize(s);

  ASSERT_EQUAL(result, 7);

  result = thrust::unique_count(streampolicy, data.begin(), data.end(), is_equal_div_10_unique<T>());
  hipStreamSynchronize(s);

  ASSERT_EQUAL(result, 3);

  hipStreamDestroy(s);
}

void TestUniqueCountCudaStreamsSync()
{
  TestUniqueCountCudaStreams(thrust::cuda::par);
}
DECLARE_UNITTEST(TestUniqueCountCudaStreamsSync);


void TestUniqueCountCudaStreamsNoSync()
{
  TestUniqueCountCudaStreams(thrust::cuda::par_nosync);
}
DECLARE_UNITTEST(TestUniqueCountCudaStreamsNoSync);

