#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/sequence.h>
#include <thrust/execution_policy.h>


#ifdef THRUST_TEST_DEVICE_SIDE
template<typename ExecutionPolicy, typename Iterator>
__global__
void sequence_kernel(ExecutionPolicy exec, Iterator first, Iterator last)
{
  thrust::sequence(exec, first, last);
}


template<typename ExecutionPolicy, typename Iterator, typename T>
__global__
void sequence_kernel(ExecutionPolicy exec, Iterator first, Iterator last, T init)
{
  thrust::sequence(exec, first, last, init);
}


template<typename ExecutionPolicy, typename Iterator, typename T>
__global__
void sequence_kernel(ExecutionPolicy exec, Iterator first, Iterator last, T init, T step)
{
  thrust::sequence(exec, first, last, init, step);
}


template<typename ExecutionPolicy>
void TestSequenceDevice(ExecutionPolicy exec)
{
  thrust::device_vector<int> v(5);
  
  sequence_kernel<<<1,1>>>(exec, v.begin(), v.end());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }
 
  ASSERT_EQUAL(v[0], 0);
  ASSERT_EQUAL(v[1], 1);
  ASSERT_EQUAL(v[2], 2);
  ASSERT_EQUAL(v[3], 3);
  ASSERT_EQUAL(v[4], 4);
  
  sequence_kernel<<<1,1>>>(exec, v.begin(), v.end(), 10);
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }
  
  ASSERT_EQUAL(v[0], 10);
  ASSERT_EQUAL(v[1], 11);
  ASSERT_EQUAL(v[2], 12);
  ASSERT_EQUAL(v[3], 13);
  ASSERT_EQUAL(v[4], 14);
  
  sequence_kernel<<<1,1>>>(exec, v.begin(), v.end(), 10, 2);
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }
  
  ASSERT_EQUAL(v[0], 10);
  ASSERT_EQUAL(v[1], 12);
  ASSERT_EQUAL(v[2], 14);
  ASSERT_EQUAL(v[3], 16);
  ASSERT_EQUAL(v[4], 18);
}

void TestSequenceDeviceSeq()
{
  TestSequenceDevice(thrust::seq);
}
DECLARE_UNITTEST(TestSequenceDeviceSeq);

void TestSequenceDeviceDevice()
{
  TestSequenceDevice(thrust::device);
}
DECLARE_UNITTEST(TestSequenceDeviceDevice);
#endif

void TestSequenceCudaStreams()
{
  typedef thrust::device_vector<int> Vector;
  
  Vector v(5);

  hipStream_t s;
  hipStreamCreate(&s);

  thrust::sequence(thrust::cuda::par.on(s), v.begin(), v.end());
  hipStreamSynchronize(s);

  ASSERT_EQUAL(v[0], 0);
  ASSERT_EQUAL(v[1], 1);
  ASSERT_EQUAL(v[2], 2);
  ASSERT_EQUAL(v[3], 3);
  ASSERT_EQUAL(v[4], 4);

  thrust::sequence(thrust::cuda::par.on(s), v.begin(), v.end(), 10);
  hipStreamSynchronize(s);

  ASSERT_EQUAL(v[0], 10);
  ASSERT_EQUAL(v[1], 11);
  ASSERT_EQUAL(v[2], 12);
  ASSERT_EQUAL(v[3], 13);
  ASSERT_EQUAL(v[4], 14);
  
  thrust::sequence(thrust::cuda::par.on(s), v.begin(), v.end(), 10, 2);
  hipStreamSynchronize(s);

  ASSERT_EQUAL(v[0], 10);
  ASSERT_EQUAL(v[1], 12);
  ASSERT_EQUAL(v[2], 14);
  ASSERT_EQUAL(v[3], 16);
  ASSERT_EQUAL(v[4], 18);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestSequenceCudaStreams);

