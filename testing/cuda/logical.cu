#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/logical.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>


#ifdef THRUST_TEST_DEVICE_SIDE
template<typename ExecutionPolicy, typename Iterator, typename Function, typename Iterator2>
__global__
void all_of_kernel(ExecutionPolicy exec, Iterator first, Iterator last, Function f, Iterator2 result)
{
  *result = thrust::all_of(exec, first, last, f);
}


template<typename ExecutionPolicy>
void TestAllOfDevice(ExecutionPolicy exec)
{
  typedef int T;
  thrust::device_vector<T> v(3, 1);
  thrust::device_vector<bool> result(1);
  
  all_of_kernel<<<1,1>>>(exec, v.begin(), v.end(), thrust::identity<T>(), result.begin());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  ASSERT_EQUAL(true, result[0]);
  
  v[1] = 0;
  
  all_of_kernel<<<1,1>>>(exec, v.begin(), v.end(), thrust::identity<T>(), result.begin());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  ASSERT_EQUAL(false, result[0]);
  
  all_of_kernel<<<1,1>>>(exec, v.begin() + 0, v.begin() + 0, thrust::identity<T>(), result.begin());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  ASSERT_EQUAL(true, result[0]);

  all_of_kernel<<<1,1>>>(exec, v.begin() + 0, v.begin() + 1, thrust::identity<T>(), result.begin());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  ASSERT_EQUAL(true, result[0]);

  all_of_kernel<<<1,1>>>(exec, v.begin() + 0, v.begin() + 2, thrust::identity<T>(), result.begin());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  ASSERT_EQUAL(false, result[0]);

  all_of_kernel<<<1,1>>>(exec, v.begin() + 1, v.begin() + 2, thrust::identity<T>(), result.begin());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  ASSERT_EQUAL(false, result[0]);
}


void TestAllOfDeviceSeq()
{
  TestAllOfDevice(thrust::seq);
}
DECLARE_UNITTEST(TestAllOfDeviceSeq);


void TestAllOfDeviceDevice()
{
  TestAllOfDevice(thrust::device);
}
DECLARE_UNITTEST(TestAllOfDeviceDevice);
#endif


void TestAllOfCudaStreams()
{
  typedef thrust::device_vector<int> Vector;
  typedef Vector::value_type T;
  
  Vector v(3, 1);

  hipStream_t s;
  hipStreamCreate(&s);
  
  ASSERT_EQUAL(thrust::all_of(thrust::cuda::par.on(s), v.begin(), v.end(), thrust::identity<T>()), true);
  
  v[1] = 0;
  
  ASSERT_EQUAL(thrust::all_of(thrust::cuda::par.on(s), v.begin(), v.end(), thrust::identity<T>()), false);
  
  ASSERT_EQUAL(thrust::all_of(thrust::cuda::par.on(s), v.begin() + 0, v.begin() + 0, thrust::identity<T>()), true);
  ASSERT_EQUAL(thrust::all_of(thrust::cuda::par.on(s), v.begin() + 0, v.begin() + 1, thrust::identity<T>()), true);
  ASSERT_EQUAL(thrust::all_of(thrust::cuda::par.on(s), v.begin() + 0, v.begin() + 2, thrust::identity<T>()), false);
  ASSERT_EQUAL(thrust::all_of(thrust::cuda::par.on(s), v.begin() + 1, v.begin() + 2, thrust::identity<T>()), false);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestAllOfCudaStreams);


#ifdef THRUST_TEST_DEVICE_SIDE
template<typename ExecutionPolicy, typename Iterator, typename Function, typename Iterator2>
__global__
void any_of_kernel(ExecutionPolicy exec, Iterator first, Iterator last, Function f, Iterator2 result)
{
  *result = thrust::any_of(exec, first, last, f);
}


template<typename ExecutionPolicy>
void TestAnyOfDevice(ExecutionPolicy exec)
{
  typedef int T;
  
  thrust::device_vector<T> v(3, 1);
  thrust::device_vector<bool> result(1);
  
  any_of_kernel<<<1,1>>>(exec, v.begin(), v.end(), thrust::identity<T>(), result.begin());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  ASSERT_EQUAL(true, result[0]);
  
  v[1] = 0;
  
  any_of_kernel<<<1,1>>>(exec, v.begin(), v.end(), thrust::identity<T>(), result.begin());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  ASSERT_EQUAL(true, result[0]);
  
  any_of_kernel<<<1,1>>>(exec, v.begin() + 0, v.begin() + 0, thrust::identity<T>(), result.begin());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  ASSERT_EQUAL(false, result[0]);

  any_of_kernel<<<1,1>>>(exec, v.begin() + 0, v.begin() + 1, thrust::identity<T>(), result.begin());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  ASSERT_EQUAL(true, result[0]);

  any_of_kernel<<<1,1>>>(exec, v.begin() + 0, v.begin() + 2, thrust::identity<T>(), result.begin());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  ASSERT_EQUAL(true, result[0]);

  any_of_kernel<<<1,1>>>(exec, v.begin() + 1, v.begin() + 2, thrust::identity<T>(), result.begin());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  ASSERT_EQUAL(false, result[0]);
}


void TestAnyOfDeviceSeq()
{
  TestAnyOfDevice(thrust::seq);
}
DECLARE_UNITTEST(TestAnyOfDeviceSeq);


void TestAnyOfDeviceDevice()
{
  TestAnyOfDevice(thrust::device);
}
DECLARE_UNITTEST(TestAnyOfDeviceDevice);
#endif


void TestAnyOfCudaStreams()
{
  typedef thrust::device_vector<int> Vector;
  typedef Vector::value_type T;

  Vector v(3, 1);

  hipStream_t s;
  hipStreamCreate(&s);

  ASSERT_EQUAL(thrust::any_of(thrust::cuda::par.on(s), v.begin(), v.end(), thrust::identity<T>()), true);

  v[1] = 0;
  
  ASSERT_EQUAL(thrust::any_of(thrust::cuda::par.on(s), v.begin(), v.end(), thrust::identity<T>()), true);

  ASSERT_EQUAL(thrust::any_of(thrust::cuda::par.on(s), v.begin() + 0, v.begin() + 0, thrust::identity<T>()), false);
  ASSERT_EQUAL(thrust::any_of(thrust::cuda::par.on(s), v.begin() + 0, v.begin() + 1, thrust::identity<T>()), true);
  ASSERT_EQUAL(thrust::any_of(thrust::cuda::par.on(s), v.begin() + 0, v.begin() + 2, thrust::identity<T>()), true);
  ASSERT_EQUAL(thrust::any_of(thrust::cuda::par.on(s), v.begin() + 1, v.begin() + 2, thrust::identity<T>()), false);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestAnyOfCudaStreams);


#ifdef THRUST_TEST_DEVICE_SIDE
template<typename ExecutionPolicy, typename Iterator, typename Function, typename Iterator2>
__global__
void none_of_kernel(ExecutionPolicy exec, Iterator first, Iterator last, Function f, Iterator2 result)
{
  *result = thrust::none_of(exec, first, last, f);
}


template<typename ExecutionPolicy>
void TestNoneOfDevice(ExecutionPolicy exec)
{
  typedef int T;
  
  thrust::device_vector<T> v(3, 1);
  thrust::device_vector<bool> result(1);
  
  none_of_kernel<<<1,1>>>(exec, v.begin(), v.end(), thrust::identity<T>(), result.begin());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  ASSERT_EQUAL(false, result[0]);
  
  v[1] = 0;
  
  none_of_kernel<<<1,1>>>(exec, v.begin(), v.end(), thrust::identity<T>(), result.begin());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  ASSERT_EQUAL(false, result[0]);
  
  none_of_kernel<<<1,1>>>(exec, v.begin() + 0, v.begin() + 0, thrust::identity<T>(), result.begin());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  ASSERT_EQUAL(true, result[0]);

  none_of_kernel<<<1,1>>>(exec, v.begin() + 0, v.begin() + 1, thrust::identity<T>(), result.begin());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  ASSERT_EQUAL(false, result[0]);

  none_of_kernel<<<1,1>>>(exec, v.begin() + 0, v.begin() + 2, thrust::identity<T>(), result.begin());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  ASSERT_EQUAL(false, result[0]);

  none_of_kernel<<<1,1>>>(exec, v.begin() + 1, v.begin() + 2, thrust::identity<T>(), result.begin());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  ASSERT_EQUAL(true, result[0]);
}


void TestNoneOfDeviceSeq()
{
  TestNoneOfDevice(thrust::seq);
}
DECLARE_UNITTEST(TestNoneOfDeviceSeq);


void TestNoneOfDeviceDevice()
{
  TestNoneOfDevice(thrust::device);
}
DECLARE_UNITTEST(TestNoneOfDeviceDevice);
#endif


void TestNoneOfCudaStreams()
{
  typedef thrust::device_vector<int> Vector;
  typedef Vector::value_type T;

  Vector v(3, 1);

  hipStream_t s;
  hipStreamCreate(&s);

  ASSERT_EQUAL(thrust::none_of(thrust::cuda::par.on(s), v.begin(), v.end(), thrust::identity<T>()), false);

  v[1] = 0;
  
  ASSERT_EQUAL(thrust::none_of(thrust::cuda::par.on(s), v.begin(), v.end(), thrust::identity<T>()), false);

  ASSERT_EQUAL(thrust::none_of(thrust::cuda::par.on(s), v.begin() + 0, v.begin() + 0, thrust::identity<T>()), true);
  ASSERT_EQUAL(thrust::none_of(thrust::cuda::par.on(s), v.begin() + 0, v.begin() + 1, thrust::identity<T>()), false);
  ASSERT_EQUAL(thrust::none_of(thrust::cuda::par.on(s), v.begin() + 0, v.begin() + 2, thrust::identity<T>()), false);
  ASSERT_EQUAL(thrust::none_of(thrust::cuda::par.on(s), v.begin() + 1, v.begin() + 2, thrust::identity<T>()), true);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestNoneOfCudaStreams);

