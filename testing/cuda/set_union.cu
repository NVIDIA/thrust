#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/set_operations.h>
#include <thrust/execution_policy.h>


#ifdef THRUST_TEST_DEVICE_SIDE
template<typename ExecutionPolicy, typename Iterator1, typename Iterator2, typename Iterator3, typename Iterator4>
__global__
void set_union_kernel(ExecutionPolicy exec,
                      Iterator1 first1, Iterator1 last1,
                      Iterator2 first2, Iterator2 last2,
                      Iterator3 result1,
                      Iterator4 result2)
{
  *result2 = thrust::set_union(exec, first1, last1, first2, last2, result1);
}


template<typename ExecutionPolicy>
void TestSetUnionDevice(ExecutionPolicy exec)
{
  typedef thrust::device_vector<int> Vector;
  typedef typename Vector::iterator Iterator;

  Vector a(3), b(4);

  a[0] = 0; a[1] = 2; a[2] = 4;
  b[0] = 0; b[1] = 3; b[2] = 3; b[3] = 4;

  Vector ref(5);
  ref[0] = 0; ref[1] = 2; ref[2] = 3; ref[3] = 3; ref[4] = 4;

  Vector result(5);
  thrust::device_vector<Iterator> end_vec(1);

  set_union_kernel<<<1,1>>>(exec,
                            a.begin(), a.end(),
                            b.begin(), b.end(),
                            result.begin(),
                            end_vec.begin());
  hipError_t const err = hipDeviceSynchronize();
  ASSERT_EQUAL(hipSuccess, err);

  Iterator end = end_vec[0];

  ASSERT_EQUAL_QUIET(result.end(), end);
  ASSERT_EQUAL(ref, result);
}


void TestSetUnionDeviceSeq()
{
  TestSetUnionDevice(thrust::seq);
}
DECLARE_UNITTEST(TestSetUnionDeviceSeq);


void TestSetUnionDeviceDevice()
{
  TestSetUnionDevice(thrust::device);
}
DECLARE_UNITTEST(TestSetUnionDeviceDevice);
#endif


void TestSetUnionCudaStreams()
{
  typedef thrust::device_vector<int> Vector;
  typedef Vector::iterator Iterator;

  Vector a(3), b(4);

  a[0] = 0; a[1] = 2; a[2] = 4;
  b[0] = 0; b[1] = 3; b[2] = 3; b[3] = 4;

  Vector ref(5);
  ref[0] = 0; ref[1] = 2; ref[2] = 3; ref[3] = 3; ref[4] = 4;

  Vector result(5);

  hipStream_t s;
  hipStreamCreate(&s);

  Iterator end = thrust::set_union(thrust::cuda::par.on(s),
                                   a.begin(), a.end(),
                                   b.begin(), b.end(),
                                   result.begin());
  hipStreamSynchronize(s);

  ASSERT_EQUAL_QUIET(result.end(), end);
  ASSERT_EQUAL(ref, result);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestSetUnionCudaStreams);

