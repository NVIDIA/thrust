#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/sort.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>


template<typename ExecutionPolicy, typename Iterator, typename Compare, typename Iterator2>
__global__
void sort_kernel(ExecutionPolicy exec, Iterator first, Iterator last, Compare comp, Iterator2 is_supported)
{
  *is_supported = true;
  thrust::sort(exec, first, last, comp);
}


template<typename T>
struct my_less
{
  __host__ __device__
  bool operator()(const T& lhs, const T& rhs) const
  {
    return lhs < rhs;
  }
};


template<typename T, typename ExecutionPolicy, typename Compare>
void TestComparisonSortDevice(ExecutionPolicy exec, const size_t n, Compare comp)
{
  thrust::host_vector<T>   h_data = unittest::random_integers<T>(n);
  thrust::device_vector<T> d_data = h_data;
  
  thrust::device_vector<bool> is_supported(1);

  sort_kernel<<<1,1>>>(exec, d_data.begin(), d_data.end(), comp, is_supported.begin());
  hipError_t const err = hipDeviceSynchronize();
  ASSERT_EQUAL(hipSuccess, err);


  if(is_supported[0])
  {
    thrust::sort(h_data.begin(), h_data.end(), comp);
    
    ASSERT_EQUAL(h_data, d_data);
  }
};


template<typename T>
  struct TestComparisonSortDeviceSeq
{
  void operator()(const size_t n)
  {
    TestComparisonSortDevice<T>(thrust::seq, n, my_less<T>());
  }
};
VariableUnitTest<
  TestComparisonSortDeviceSeq,
  unittest::type_list<unittest::int8_t,unittest::int32_t>
> TestComparisonSortDeviceSeqInstance;


template<typename T>
  struct TestComparisonSortDeviceDevice
{
  void operator()(const size_t n)
  {
    TestComparisonSortDevice<T>(thrust::device, n, my_less<T>());
  }
};
VariableUnitTest<
  TestComparisonSortDeviceDevice,
  unittest::type_list<unittest::int8_t,unittest::int32_t>
> TestComparisonSortDeviceDeviceDeviceInstance;


template<typename T, typename ExecutionPolicy>
void TestSortDevice(ExecutionPolicy exec, const size_t n)
{
  TestComparisonSortDevice<T>(exec, n, thrust::less<T>());
};


template<typename T>
  struct TestSortDeviceSeq
{
  void operator()(const size_t n)
  {
    TestSortDevice<T>(thrust::seq, n);
  }
};
VariableUnitTest<
  TestSortDeviceSeq,
  unittest::type_list<unittest::int8_t,unittest::int32_t>
> TestSortDeviceSeqInstance;


template<typename T>
  struct TestSortDeviceDevice
{
  void operator()(const size_t n)
  {
    TestSortDevice<T>(thrust::device, n);
  }
};
VariableUnitTest<
  TestSortDeviceDevice,
  unittest::type_list<unittest::int8_t,unittest::int32_t>
> TestSortDeviceDeviceInstance;


void TestSortCudaStreams()
{
  thrust::device_vector<int> keys(10);

  keys[0] = 9;
  keys[1] = 3;
  keys[2] = 2;
  keys[3] = 0;
  keys[4] = 4;
  keys[5] = 7;
  keys[6] = 8;
  keys[7] = 1;
  keys[8] = 5;
  keys[9] = 6;

  hipStream_t s;
  hipStreamCreate(&s);

  thrust::sort(thrust::cuda::par.on(s), keys.begin(), keys.end());
  hipStreamSynchronize(s);

  ASSERT_EQUAL(true, thrust::is_sorted(keys.begin(), keys.end()));
                      
  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestSortCudaStreams);


void TestComparisonSortCudaStreams()
{
  thrust::device_vector<int> keys(10);

  keys[0] = 9;
  keys[1] = 3;
  keys[2] = 2;
  keys[3] = 0;
  keys[4] = 4;
  keys[5] = 7;
  keys[6] = 8;
  keys[7] = 1;
  keys[8] = 5;
  keys[9] = 6;

  hipStream_t s;
  hipStreamCreate(&s);

  thrust::sort(thrust::cuda::par.on(s), keys.begin(), keys.end(), my_less<int>());
  hipStreamSynchronize(s);

  ASSERT_EQUAL(true, thrust::is_sorted(keys.begin(), keys.end(), my_less<int>()));
                      
  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestComparisonSortCudaStreams);

