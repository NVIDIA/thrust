#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/remove.h>
#include <thrust/execution_policy.h>


#ifdef THRUST_TEST_DEVICE_SIDE
template<typename ExecutionPolicy, typename Iterator, typename T, typename Iterator2>
__global__
void remove_kernel(ExecutionPolicy exec, Iterator first, Iterator last, T val, Iterator2 result)
{
  *result = thrust::remove(exec, first, last, val);
}


template<typename ExecutionPolicy, typename Iterator, typename Predicate, typename Iterator2>
__global__
void remove_if_kernel(ExecutionPolicy exec, Iterator first, Iterator last, Predicate pred, Iterator2 result)
{
  *result = thrust::remove_if(exec, first, last, pred);
}


template<typename ExecutionPolicy, typename Iterator1, typename Iterator2, typename Predicate, typename Iterator3>
__global__
void remove_if_kernel(ExecutionPolicy exec, Iterator1 first, Iterator1 last, Iterator2 stencil_first, Predicate pred, Iterator3 result)
{
  *result = thrust::remove_if(exec, first, last, stencil_first, pred);
}


template<typename ExecutionPolicy, typename Iterator1, typename Iterator2, typename T, typename Iterator3>
__global__
void remove_copy_kernel(ExecutionPolicy exec, Iterator1 first, Iterator1 last, Iterator2 result1, T val, Iterator3 result2)
{
  *result2 = thrust::remove_copy(exec, first, last, result1, val);
}


template<typename ExecutionPolicy, typename Iterator1, typename Iterator2, typename Predicate, typename Iterator3>
__global__
void remove_copy_if_kernel(ExecutionPolicy exec, Iterator1 first, Iterator1 last, Iterator2 result, Predicate pred, Iterator3 result_end)
{
  *result_end = thrust::remove_copy_if(exec, first, last, result, pred);
}


template<typename ExecutionPolicy, typename Iterator1, typename Iterator2, typename Iterator3, typename Predicate, typename Iterator4>
__global__
void remove_copy_if_kernel(ExecutionPolicy exec, Iterator1 first, Iterator1 last, Iterator2 stencil_first, Iterator3 result, Predicate pred, Iterator4 result_end)
{
  *result_end = thrust::remove_copy_if(exec, first, last, stencil_first, result, pred);
}
#endif


template<typename T>
struct is_even
  : thrust::unary_function<T,bool>
{
  __host__ __device__
  bool operator()(T x) { return (static_cast<unsigned int>(x) & 1) == 0; }
};


template<typename T>
struct is_true
  : thrust::unary_function<T,bool>
{
  __host__ __device__
  bool operator()(T x) { return x ? true : false; }
};


#ifdef THRUST_TEST_DEVICE_SIDE
template<typename ExecutionPolicy>
void TestRemoveDevice(ExecutionPolicy exec)
{
  size_t n = 1000;
  thrust::host_vector<int>   h_data = unittest::random_samples<int>(n);
  thrust::device_vector<int> d_data = h_data;

  typedef typename thrust::device_vector<int>::iterator iterator;
  thrust::device_vector<iterator> d_result(1);
  
  size_t h_size = thrust::remove(h_data.begin(), h_data.end(), 0) - h_data.begin();

  remove_kernel<<<1,1>>>(exec, d_data.begin(), d_data.end(), 0, d_result.begin());
  hipError_t const err = hipDeviceSynchronize();
  ASSERT_EQUAL(hipSuccess, err);

  size_t d_size = (iterator)d_result[0] - d_data.begin();
  
  ASSERT_EQUAL(h_size, d_size);
  
  h_data.resize(h_size);
  d_data.resize(d_size);
  
  ASSERT_EQUAL(h_data, d_data);
}


void TestRemoveDeviceSeq()
{
  TestRemoveDevice(thrust::seq);
}
DECLARE_UNITTEST(TestRemoveDeviceSeq);


void TestRemoveDeviceDevice()
{
  TestRemoveDevice(thrust::device);
}
DECLARE_UNITTEST(TestRemoveDeviceDevice);


template<typename ExecutionPolicy>
void TestRemoveIfDevice(ExecutionPolicy exec)
{
  size_t n = 1000;
  thrust::host_vector<int>   h_data = unittest::random_samples<int>(n);
  thrust::device_vector<int> d_data = h_data;

  typedef typename thrust::device_vector<int>::iterator iterator;
  thrust::device_vector<iterator> d_result(1);
  
  size_t h_size = thrust::remove_if(h_data.begin(), h_data.end(), is_true<int>()) - h_data.begin();

  remove_if_kernel<<<1,1>>>(exec, d_data.begin(), d_data.end(), is_true<int>(), d_result.begin());
  hipError_t const err = hipDeviceSynchronize();
  ASSERT_EQUAL(hipSuccess, err);

  size_t d_size = (iterator)d_result[0] - d_data.begin();
  
  ASSERT_EQUAL(h_size, d_size);
  
  h_data.resize(h_size);
  d_data.resize(d_size);
  
  ASSERT_EQUAL(h_data, d_data);
}


void TestRemoveIfDeviceSeq()
{
  TestRemoveIfDevice(thrust::seq);
}
DECLARE_UNITTEST(TestRemoveIfDeviceSeq);


void TestRemoveIfDeviceDevice()
{
  TestRemoveIfDevice(thrust::device);
}
DECLARE_UNITTEST(TestRemoveIfDeviceDevice);


template<typename ExecutionPolicy>
void TestRemoveIfStencilDevice(ExecutionPolicy exec)
{
  size_t n = 1000;
  thrust::host_vector<int>   h_data = unittest::random_samples<int>(n);
  thrust::device_vector<int> d_data = h_data;

  typedef typename thrust::device_vector<int>::iterator iterator;
  thrust::device_vector<iterator> d_result(1);
  
  thrust::host_vector<bool>   h_stencil = unittest::random_integers<bool>(n);
  thrust::device_vector<bool> d_stencil = h_stencil;
  
  size_t h_size = thrust::remove_if(h_data.begin(), h_data.end(), h_stencil.begin(), is_true<int>()) - h_data.begin();

  remove_if_kernel<<<1,1>>>(exec, d_data.begin(), d_data.end(), d_stencil.begin(), is_true<int>(), d_result.begin());
  hipError_t const err = hipDeviceSynchronize();
  ASSERT_EQUAL(hipSuccess, err);

  size_t d_size = (iterator)d_result[0] - d_data.begin();
  
  ASSERT_EQUAL(h_size, d_size);
  
  h_data.resize(h_size);
  d_data.resize(d_size);
  
  ASSERT_EQUAL(h_data, d_data);
}


void TestRemoveIfStencilDeviceSeq()
{
  TestRemoveIfStencilDevice(thrust::seq);
}
DECLARE_UNITTEST(TestRemoveIfStencilDeviceSeq);


void TestRemoveIfStencilDeviceDevice()
{
  TestRemoveIfStencilDevice(thrust::device);
}
DECLARE_UNITTEST(TestRemoveIfStencilDeviceDevice);


template<typename ExecutionPolicy>
void TestRemoveCopyDevice(ExecutionPolicy exec)
{
  size_t n = 1000;
  thrust::host_vector<int>   h_data = unittest::random_samples<int>(n);
  thrust::device_vector<int> d_data = h_data;
  
  thrust::host_vector<int>   h_result(n);
  thrust::device_vector<int> d_result(n);

  typedef typename thrust::device_vector<int>::iterator iterator;
  thrust::device_vector<iterator> d_new_end(1);
  
  size_t h_size = thrust::remove_copy(h_data.begin(), h_data.end(), h_result.begin(), 0) - h_result.begin();

  remove_copy_kernel<<<1,1>>>(exec, d_data.begin(), d_data.end(), d_result.begin(), 0, d_new_end.begin());
  hipError_t const err = hipDeviceSynchronize();
  ASSERT_EQUAL(hipSuccess, err);

  size_t d_size = (iterator)d_new_end[0] - d_result.begin();
  
  ASSERT_EQUAL(h_size, d_size);
  
  h_result.resize(h_size);
  d_result.resize(d_size);
  
  ASSERT_EQUAL(h_result, d_result);
}


void TestRemoveCopyDeviceSeq()
{
  TestRemoveCopyDevice(thrust::seq);
}
DECLARE_UNITTEST(TestRemoveCopyDeviceSeq);


void TestRemoveCopyDeviceDevice()
{
  TestRemoveCopyDevice(thrust::device);
}
DECLARE_UNITTEST(TestRemoveCopyDeviceDevice);


template<typename ExecutionPolicy>
void TestRemoveCopyIfDevice(ExecutionPolicy exec)
{
  size_t n = 1000;
  thrust::host_vector<int>   h_data = unittest::random_samples<int>(n);
  thrust::device_vector<int> d_data = h_data;
  
  thrust::host_vector<int>   h_result(n);
  thrust::device_vector<int> d_result(n);

  typedef typename thrust::device_vector<int>::iterator iterator;
  thrust::device_vector<iterator> d_new_end(1);
  
  size_t h_size = thrust::remove_copy_if(h_data.begin(), h_data.end(), h_result.begin(), is_true<int>()) - h_result.begin();

  remove_copy_if_kernel<<<1,1>>>(exec, d_data.begin(), d_data.end(), d_result.begin(), is_true<int>(), d_new_end.begin());
  hipError_t const err = hipDeviceSynchronize();
  ASSERT_EQUAL(hipSuccess, err);

  size_t d_size = (iterator)d_new_end[0] - d_result.begin();
  
  ASSERT_EQUAL(h_size, d_size);
  
  h_result.resize(h_size);
  d_result.resize(d_size);
  
  ASSERT_EQUAL(h_result, d_result);
}


void TestRemoveCopyIfDeviceSeq()
{
  TestRemoveCopyIfDevice(thrust::seq);
}
DECLARE_UNITTEST(TestRemoveCopyIfDeviceSeq);


void TestRemoveCopyIfDeviceDevice()
{
  TestRemoveCopyIfDevice(thrust::device);
}
DECLARE_UNITTEST(TestRemoveCopyIfDeviceDevice);


template<typename ExecutionPolicy>
void TestRemoveCopyIfStencilDevice(ExecutionPolicy exec)
{
  size_t n = 1000;
  thrust::host_vector<int>   h_data = unittest::random_samples<int>(n);
  thrust::device_vector<int> d_data = h_data;
  
  thrust::host_vector<int>   h_result(n);
  thrust::device_vector<int> d_result(n);

  typedef typename thrust::device_vector<int>::iterator iterator;
  thrust::device_vector<iterator> d_new_end(1);

  thrust::host_vector<bool>   h_stencil = unittest::random_integers<bool>(n);
  thrust::device_vector<bool> d_stencil = h_stencil;
  
  size_t h_size = thrust::remove_copy_if(h_data.begin(), h_data.end(), h_stencil.begin(), h_result.begin(), is_true<int>()) - h_result.begin();

  remove_copy_if_kernel<<<1,1>>>(exec, d_data.begin(), d_data.end(), d_stencil.begin(), d_result.begin(), is_true<int>(), d_new_end.begin());
  hipError_t const err = hipDeviceSynchronize();
  ASSERT_EQUAL(hipSuccess, err);

  size_t d_size = (iterator)d_new_end[0] - d_result.begin();
  
  ASSERT_EQUAL(h_size, d_size);
  
  h_result.resize(h_size);
  d_result.resize(d_size);
  
  ASSERT_EQUAL(h_result, d_result);
}


void TestRemoveCopyIfStencilDeviceSeq()
{
  TestRemoveCopyIfStencilDevice(thrust::seq);
}
DECLARE_UNITTEST(TestRemoveCopyIfStencilDeviceSeq);


void TestRemoveCopyIfStencilDeviceDevice()
{
  TestRemoveCopyIfStencilDevice(thrust::device);
}
DECLARE_UNITTEST(TestRemoveCopyIfStencilDeviceDevice);
#endif


void TestRemoveCudaStreams()
{
  typedef thrust::device_vector<int> Vector;
  typedef Vector::value_type T;

  Vector data(5);
  data[0] =  1; 
  data[1] =  2; 
  data[2] =  1;
  data[3] =  3; 
  data[4] =  2; 

  hipStream_t s;
  hipStreamCreate(&s);

  Vector::iterator end = thrust::remove(thrust::cuda::par.on(s),
                                        data.begin(), 
                                        data.end(), 
                                        (T) 2);

  ASSERT_EQUAL(end - data.begin(), 3);

  ASSERT_EQUAL(data[0], 1);
  ASSERT_EQUAL(data[1], 1);
  ASSERT_EQUAL(data[2], 3);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestRemoveCudaStreams);


void TestRemoveCopyCudaStreams()
{
  typedef thrust::device_vector<int> Vector;
  typedef Vector::value_type T;

  Vector data(5);
  data[0] =  1; 
  data[1] =  2; 
  data[2] =  1;
  data[3] =  3; 
  data[4] =  2; 

  Vector result(5);

  hipStream_t s;
  hipStreamCreate(&s);

  Vector::iterator end = thrust::remove_copy(thrust::cuda::par.on(s),
                                             data.begin(), 
                                             data.end(), 
                                             result.begin(), 
                                             (T) 2);

  ASSERT_EQUAL(end - result.begin(), 3);

  ASSERT_EQUAL(result[0], 1);
  ASSERT_EQUAL(result[1], 1);
  ASSERT_EQUAL(result[2], 3);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestRemoveCopyCudaStreams);


void TestRemoveIfCudaStreams()
{
  typedef thrust::device_vector<int> Vector;
  typedef Vector::value_type T;

  Vector data(5);
  data[0] =  1; 
  data[1] =  2; 
  data[2] =  1;
  data[3] =  3; 
  data[4] =  2; 

  hipStream_t s;
  hipStreamCreate(&s);

  Vector::iterator end = thrust::remove_if(thrust::cuda::par.on(s),
                                           data.begin(), 
                                           data.end(), 
                                           is_even<T>());

  ASSERT_EQUAL(end - data.begin(), 3);

  ASSERT_EQUAL(data[0], 1);
  ASSERT_EQUAL(data[1], 1);
  ASSERT_EQUAL(data[2], 3);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestRemoveIfCudaStreams);


void TestRemoveIfStencilCudaStreams()
{
  typedef thrust::device_vector<int> Vector;
  typedef Vector::value_type T;

  Vector data(5);
  data[0] =  1; 
  data[1] =  2; 
  data[2] =  1;
  data[3] =  3; 
  data[4] =  2; 

  Vector stencil(5);
  stencil[0] = 0;
  stencil[1] = 1;
  stencil[2] = 0;
  stencil[3] = 0;
  stencil[4] = 1;

  hipStream_t s;
  hipStreamCreate(&s);

  Vector::iterator end = thrust::remove_if(thrust::cuda::par.on(s),
                                           data.begin(), 
                                           data.end(),
                                           stencil.begin(),
                                           thrust::identity<T>());

  ASSERT_EQUAL(end - data.begin(), 3);

  ASSERT_EQUAL(data[0], 1);
  ASSERT_EQUAL(data[1], 1);
  ASSERT_EQUAL(data[2], 3);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestRemoveIfStencilCudaStreams);


void TestRemoveCopyIfCudaStreams()
{
  typedef thrust::device_vector<int> Vector;
  typedef Vector::value_type T;

  Vector data(5);
  data[0] =  1; 
  data[1] =  2; 
  data[2] =  1;
  data[3] =  3; 
  data[4] =  2; 

  Vector result(5);

  hipStream_t s;
  hipStreamCreate(&s);

  Vector::iterator end = thrust::remove_copy_if(thrust::cuda::par.on(s),
                                                data.begin(), 
                                                data.end(), 
                                                result.begin(), 
                                                is_even<T>());

  ASSERT_EQUAL(end - result.begin(), 3);

  ASSERT_EQUAL(result[0], 1);
  ASSERT_EQUAL(result[1], 1);
  ASSERT_EQUAL(result[2], 3);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestRemoveCopyIfCudaStreams);


void TestRemoveCopyIfStencilCudaStreams()
{
  typedef thrust::device_vector<int> Vector;
  typedef Vector::value_type T;

  Vector data(5);
  data[0] =  1; 
  data[1] =  2; 
  data[2] =  1;
  data[3] =  3; 
  data[4] =  2; 

  Vector stencil(5);
  stencil[0] = 0;
  stencil[1] = 1;
  stencil[2] = 0;
  stencil[3] = 0;
  stencil[4] = 1;

  Vector result(5);

  hipStream_t s;
  hipStreamCreate(&s);

  Vector::iterator end = thrust::remove_copy_if(thrust::cuda::par.on(s),
                                                data.begin(), 
                                                data.end(), 
                                                stencil.begin(),
                                                result.begin(), 
                                                thrust::identity<T>());

  ASSERT_EQUAL(end - result.begin(), 3);

  ASSERT_EQUAL(result[0], 1);
  ASSERT_EQUAL(result[1], 1);
  ASSERT_EQUAL(result[2], 3);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestRemoveCopyIfStencilCudaStreams);

