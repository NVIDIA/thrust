#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/equal.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>


#ifdef THRUST_TEST_DEVICE_SIDE
template<typename ExecutionPolicy, typename Iterator1, typename Iterator2, typename Iterator3>
__global__
void equal_kernel(ExecutionPolicy exec, Iterator1 first1, Iterator1 last1, Iterator2 first2, Iterator3 result)
{
  *result = thrust::equal(exec, first1, last1, first2);
}


template<typename ExecutionPolicy, typename Iterator1, typename Iterator2, typename BinaryPredicate, typename Iterator3>
__global__
void equal_kernel(ExecutionPolicy exec, Iterator1 first1, Iterator1 last1, Iterator2 first2, BinaryPredicate pred, Iterator3 result)
{
  *result = thrust::equal(exec, first1, last1, first2, pred);
}


template<typename T, typename ExecutionPolicy>
void TestEqualDevice(ExecutionPolicy exec, const size_t n)
{
  thrust::device_vector<T> d_data1 = unittest::random_samples<T>(n);
  thrust::device_vector<T> d_data2 = unittest::random_samples<T>(n);
  thrust::device_vector<bool> d_result(1, false);
  
  //empty ranges
  equal_kernel<<<1,1>>>(exec, d_data1.begin(), d_data1.begin(), d_data1.begin(), d_result.begin());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  ASSERT_EQUAL(d_result[0], true);
  
  //symmetric cases
  equal_kernel<<<1,1>>>(exec, d_data1.begin(), d_data1.end(), d_data1.begin(), d_result.begin());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  ASSERT_EQUAL(d_result[0], true);
  
  if(n > 0)
  {
    d_data1[0] = 0; d_data2[0] = 1;
    
    //different vectors
    equal_kernel<<<1,1>>>(exec, d_data1.begin(), d_data1.end(), d_data2.begin(), d_result.begin());
    {
      hipError_t const err = hipDeviceSynchronize();
      ASSERT_EQUAL(hipSuccess, err);
    }

    ASSERT_EQUAL(d_result[0], false);
    
    //different predicates
    equal_kernel<<<1,1>>>(exec, d_data1.begin(), d_data1.begin() + 1, d_data2.begin(), thrust::less<T>(), d_result.begin());
    {
      hipError_t const err = hipDeviceSynchronize();
      ASSERT_EQUAL(hipSuccess, err);
    }

    ASSERT_EQUAL(d_result[0], true);

    equal_kernel<<<1,1>>>(exec, d_data1.begin(), d_data1.begin() + 1, d_data2.begin(), thrust::greater<T>(), d_result.begin());
    {
      hipError_t const err = hipDeviceSynchronize();
      ASSERT_EQUAL(hipSuccess, err);
    }

    ASSERT_EQUAL(d_result[0], false);
  }
}


template<typename T>
void TestEqualDeviceSeq(const size_t n)
{
  TestEqualDevice<T>(thrust::seq, n);
}
DECLARE_VARIABLE_UNITTEST(TestEqualDeviceSeq);


template<typename T>
void TestEqualDeviceDevice(const size_t n)
{
  TestEqualDevice<T>(thrust::device, n);
}
DECLARE_VARIABLE_UNITTEST(TestEqualDeviceDevice);
#endif


void TestEqualCudaStreams()
{
  thrust::device_vector<int> v1(5);
  thrust::device_vector<int> v2(5);
  v1[0] = 5; v1[1] = 2; v1[2] = 0; v1[3] = 0; v1[4] = 0;
  v2[0] = 5; v2[1] = 2; v2[2] = 0; v2[3] = 6; v2[4] = 1;

  hipStream_t s;
  hipStreamCreate(&s);
  
  ASSERT_EQUAL(thrust::equal(thrust::cuda::par.on(s), v1.begin(), v1.end(), v1.begin()), true);
  ASSERT_EQUAL(thrust::equal(thrust::cuda::par.on(s), v1.begin(), v1.end(), v2.begin()), false);
  ASSERT_EQUAL(thrust::equal(thrust::cuda::par.on(s), v2.begin(), v2.end(), v2.begin()), true);
  
  ASSERT_EQUAL(thrust::equal(thrust::cuda::par.on(s), v1.begin(), v1.begin() + 0, v1.begin()), true);
  ASSERT_EQUAL(thrust::equal(thrust::cuda::par.on(s), v1.begin(), v1.begin() + 1, v1.begin()), true);
  ASSERT_EQUAL(thrust::equal(thrust::cuda::par.on(s), v1.begin(), v1.begin() + 3, v2.begin()), true);
  ASSERT_EQUAL(thrust::equal(thrust::cuda::par.on(s), v1.begin(), v1.begin() + 4, v2.begin()), false);
  
  ASSERT_EQUAL(thrust::equal(thrust::cuda::par.on(s), v1.begin(), v1.end(), v2.begin(), thrust::less_equal<int>()), true);
  ASSERT_EQUAL(thrust::equal(thrust::cuda::par.on(s), v1.begin(), v1.end(), v2.begin(), thrust::greater<int>()),    false);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestEqualCudaStreams);

