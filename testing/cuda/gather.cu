#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/gather.h>
#include <thrust/execution_policy.h>
#include <algorithm>

#ifdef THRUST_TEST_DEVICE_SIDE
template<typename ExecutionPolicy, typename Iterator1, typename Iterator2, typename Iterator3>
__global__
void gather_kernel(ExecutionPolicy exec, Iterator1 map_first, Iterator1 map_last, Iterator2 elements_first, Iterator3 result)
{
  thrust::gather(exec, map_first, map_last, elements_first, result);
}


template<typename T, typename ExecutionPolicy>
void TestGatherDevice(ExecutionPolicy exec, const size_t n)
{
  const size_t source_size = std::min((size_t) 10, 2 * n);
  
  // source vectors to gather from
  thrust::host_vector<T>   h_source = unittest::random_samples<T>(source_size);
  thrust::device_vector<T> d_source = h_source;
  
  // gather indices
  thrust::host_vector<unsigned int> h_map = unittest::random_integers<unsigned int>(n);
  
  for(size_t i = 0; i < n; i++)
    h_map[i] =  h_map[i] % source_size;
  
  thrust::device_vector<unsigned int> d_map = h_map;
  
  // gather destination
  thrust::host_vector<T>   h_output(n);
  thrust::device_vector<T> d_output(n);
  
  thrust::gather(h_map.begin(), h_map.end(), h_source.begin(), h_output.begin());

  gather_kernel<<<1,1>>>(exec, d_map.begin(), d_map.end(), d_source.begin(), d_output.begin());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }
  
  ASSERT_EQUAL(h_output, d_output);
}

template<typename T>
void TestGatherDeviceSeq(const size_t n)
{
  TestGatherDevice<T>(thrust::seq, n);
}
DECLARE_VARIABLE_UNITTEST(TestGatherDeviceSeq);

template<typename T>
void TestGatherDeviceDevice(const size_t n)
{
  TestGatherDevice<T>(thrust::device, n);
}
DECLARE_VARIABLE_UNITTEST(TestGatherDeviceDevice);
#endif


void TestGatherCudaStreams()
{
  thrust::device_vector<int> map(5);  // gather indices
  thrust::device_vector<int> src(8);  // source vector
  thrust::device_vector<int> dst(5);  // destination vector
  
  map[0] = 6; map[1] = 2; map[2] = 1; map[3] = 7; map[4] = 2;
  src[0] = 0; src[1] = 1; src[2] = 2; src[3] = 3; src[4] = 4; src[5] = 5; src[6] = 6; src[7] = 7;
  dst[0] = 0; dst[1] = 0; dst[2] = 0; dst[3] = 0; dst[4] = 0;

  hipStream_t s;
  hipStreamCreate(&s);
  
  thrust::gather(thrust::cuda::par.on(s), map.begin(), map.end(), src.begin(), dst.begin());
  hipStreamSynchronize(s);
  
  ASSERT_EQUAL(dst[0], 6);
  ASSERT_EQUAL(dst[1], 2);
  ASSERT_EQUAL(dst[2], 1);
  ASSERT_EQUAL(dst[3], 7);
  ASSERT_EQUAL(dst[4], 2);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestGatherCudaStreams);


#ifdef THRUST_TEST_DEVICE_SIDE
template<typename ExecutionPolicy, typename Iterator1, typename Iterator2, typename Iterator3, typename Iterator4, typename Predicate>
__global__
void gather_if_kernel(ExecutionPolicy exec, Iterator1 map_first, Iterator1 map_last, Iterator2 stencil_first, Iterator3 elements_first, Iterator4 result, Predicate pred)
{
  thrust::gather_if(exec, map_first, map_last, stencil_first, elements_first, result, pred);
}


template<typename T>
struct is_even_gather_if
{
  __host__ __device__
  bool operator()(const T i) const
  { 
    return (i % 2) == 0;
  }
};


template<typename T, typename ExecutionPolicy>
void TestGatherIfDevice(ExecutionPolicy exec, const size_t n)
{
  const size_t source_size = std::min((size_t) 10, 2 * n);
  
  // source vectors to gather from
  thrust::host_vector<T>   h_source = unittest::random_samples<T>(source_size);
  thrust::device_vector<T> d_source = h_source;
  
  // gather indices
  thrust::host_vector<unsigned int> h_map = unittest::random_integers<unsigned int>(n);
  
  for(size_t i = 0; i < n; i++)
      h_map[i] = h_map[i] % source_size;
  
  thrust::device_vector<unsigned int> d_map = h_map;
  
  // gather stencil
  thrust::host_vector<unsigned int> h_stencil = unittest::random_integers<unsigned int>(n);
  
  for(size_t i = 0; i < n; i++)
    h_stencil[i] = h_stencil[i] % 2;
  
  thrust::device_vector<unsigned int> d_stencil = h_stencil;
  
  // gather destination
  thrust::host_vector<T>   h_output(n);
  thrust::device_vector<T> d_output(n);
  
  thrust::gather_if(h_map.begin(), h_map.end(), h_stencil.begin(), h_source.begin(), h_output.begin(), is_even_gather_if<unsigned int>());

  gather_if_kernel<<<1,1>>>(exec, d_map.begin(), d_map.end(), d_stencil.begin(), d_source.begin(), d_output.begin(), is_even_gather_if<unsigned int>());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }
  
  ASSERT_EQUAL(h_output, d_output);
}

template<typename T>
void TestGatherIfDeviceSeq(const size_t n)
{
  TestGatherIfDevice<T>(thrust::seq, n);
}
DECLARE_VARIABLE_UNITTEST(TestGatherIfDeviceSeq);

template<typename T>
void TestGatherIfDeviceDevice(const size_t n)
{
  TestGatherIfDevice<T>(thrust::device, n);
}
DECLARE_VARIABLE_UNITTEST(TestGatherIfDeviceDevice);
#endif

void TestGatherIfCudaStreams(void)
{
  thrust::device_vector<int> flg(5);  // predicate array
  thrust::device_vector<int> map(5);  // gather indices
  thrust::device_vector<int> src(8);  // source vector
  thrust::device_vector<int> dst(5);  // destination vector
  
  flg[0] = 0; flg[1] = 1; flg[2] = 0; flg[3] = 1; flg[4] = 0;
  map[0] = 6; map[1] = 2; map[2] = 1; map[3] = 7; map[4] = 2;
  src[0] = 0; src[1] = 1; src[2] = 2; src[3] = 3; src[4] = 4; src[5] = 5; src[6] = 6; src[7] = 7;
  dst[0] = 0; dst[1] = 0; dst[2] = 0; dst[3] = 0; dst[4] = 0;

  hipStream_t s;
  hipStreamCreate(&s);
  
  thrust::gather_if(thrust::cuda::par.on(s), map.begin(), map.end(), flg.begin(), src.begin(), dst.begin());
  hipStreamSynchronize(s);
  
  ASSERT_EQUAL(dst[0], 0);
  ASSERT_EQUAL(dst[1], 2);
  ASSERT_EQUAL(dst[2], 0);
  ASSERT_EQUAL(dst[3], 7);
  ASSERT_EQUAL(dst[4], 0);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestGatherIfCudaStreams);

