#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/uninitialized_copy.h>
#include <thrust/execution_policy.h>


#ifdef THRUST_TEST_DEVICE_SIDE
template<typename ExecutionPolicy, typename Iterator1, typename Iterator2>
__global__
void uninitialized_copy_kernel(ExecutionPolicy exec, Iterator1 first, Iterator1 last, Iterator2 result)
{
  thrust::uninitialized_copy(exec, first, last, result);
}


template<typename ExecutionPolicy>
void TestUninitializedCopyDevice(ExecutionPolicy exec)
{
  typedef thrust::device_vector<int> Vector;
  
  Vector v1(5);
  v1[0] = 0; v1[1] = 1; v1[2] = 2; v1[3] = 3; v1[4] = 4;
  
  // copy to Vector
  Vector v2(5);
  uninitialized_copy_kernel<<<1,1>>>(exec, v1.begin(), v1.end(), v2.begin());
  hipError_t const err = hipDeviceSynchronize();
  ASSERT_EQUAL(hipSuccess, err);

  ASSERT_EQUAL(v2[0], 0);
  ASSERT_EQUAL(v2[1], 1);
  ASSERT_EQUAL(v2[2], 2);
  ASSERT_EQUAL(v2[3], 3);
  ASSERT_EQUAL(v2[4], 4);
}


void TestUninitializedCopyDeviceSeq()
{
  TestUninitializedCopyDevice(thrust::seq);
}
DECLARE_UNITTEST(TestUninitializedCopyDeviceSeq);


void TestUninitializedCopyDeviceDevice()
{
  TestUninitializedCopyDevice(thrust::device);
}
DECLARE_UNITTEST(TestUninitializedCopyDeviceDevice);
#endif


void TestUninitializedCopyCudaStreams()
{
  typedef thrust::device_vector<int> Vector;
  
  Vector v1(5);
  v1[0] = 0; v1[1] = 1; v1[2] = 2; v1[3] = 3; v1[4] = 4;
  
  // copy to Vector
  Vector v2(5);

  hipStream_t s;
  hipStreamCreate(&s);

  thrust::uninitialized_copy(thrust::cuda::par.on(s), v1.begin(), v1.end(), v2.begin());
  hipStreamSynchronize(s);

  ASSERT_EQUAL(v2[0], 0);
  ASSERT_EQUAL(v2[1], 1);
  ASSERT_EQUAL(v2[2], 2);
  ASSERT_EQUAL(v2[3], 3);
  ASSERT_EQUAL(v2[4], 4);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestUninitializedCopyCudaStreams);


#ifdef THRUST_TEST_DEVICE_SIDE
template<typename ExecutionPolicy, typename Iterator1, typename Size, typename Iterator2>
__global__
void uninitialized_copy_n_kernel(ExecutionPolicy exec, Iterator1 first, Size n, Iterator2 result)
{
  thrust::uninitialized_copy_n(exec, first, n, result);
}


template<typename ExecutionPolicy>
void TestUninitializedCopyNDevice(ExecutionPolicy exec)
{
  typedef thrust::device_vector<int> Vector;
  
  Vector v1(5);
  v1[0] = 0; v1[1] = 1; v1[2] = 2; v1[3] = 3; v1[4] = 4;
  
  // copy to Vector
  Vector v2(5);
  uninitialized_copy_n_kernel<<<1,1>>>(exec, v1.begin(), v1.size(), v2.begin());
  hipError_t const err = hipDeviceSynchronize();
  ASSERT_EQUAL(hipSuccess, err);

  ASSERT_EQUAL(v2[0], 0);
  ASSERT_EQUAL(v2[1], 1);
  ASSERT_EQUAL(v2[2], 2);
  ASSERT_EQUAL(v2[3], 3);
  ASSERT_EQUAL(v2[4], 4);
}


void TestUninitializedCopyNDeviceSeq()
{
  TestUninitializedCopyNDevice(thrust::seq);
}
DECLARE_UNITTEST(TestUninitializedCopyNDeviceSeq);


void TestUninitializedCopyNDeviceDevice()
{
  TestUninitializedCopyNDevice(thrust::device);
}
DECLARE_UNITTEST(TestUninitializedCopyNDeviceDevice);
#endif


void TestUninitializedCopyNCudaStreams()
{
  typedef thrust::device_vector<int> Vector;
  
  Vector v1(5);
  v1[0] = 0; v1[1] = 1; v1[2] = 2; v1[3] = 3; v1[4] = 4;
  
  // copy to Vector
  Vector v2(5);

  hipStream_t s;
  hipStreamCreate(&s);

  thrust::uninitialized_copy_n(thrust::cuda::par.on(s), v1.begin(), v1.size(), v2.begin());
  hipStreamSynchronize(s);

  ASSERT_EQUAL(v2[0], 0);
  ASSERT_EQUAL(v2[1], 1);
  ASSERT_EQUAL(v2[2], 2);
  ASSERT_EQUAL(v2[3], 3);
  ASSERT_EQUAL(v2[4], 4);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestUninitializedCopyNCudaStreams);

