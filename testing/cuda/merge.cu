#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/merge.h>
#include <thrust/functional.h>
#include <thrust/sort.h>
#include <thrust/extrema.h>
#include <thrust/execution_policy.h>


#ifdef THRUST_TEST_DEVICE_SIDE
template<typename ExecutionPolicy, typename Iterator1, typename Iterator2, typename Iterator3, typename Iterator4>
__global__
void merge_kernel(ExecutionPolicy exec,
                  Iterator1 first1, Iterator1 last1,
                  Iterator2 first2, Iterator2 last2,
                  Iterator3 result1,
                  Iterator4 result2)
{
  *result2 = thrust::merge(exec, first1, last1, first2, last2, result1);
}


template<typename ExecutionPolicy>
void TestMergeDevice(ExecutionPolicy exec)
{
  size_t n = 10000;
  size_t sizes[]   = {0, 1, n / 2, n, n + 1, 2 * n};
  size_t num_sizes = sizeof(sizes) / sizeof(size_t);

  thrust::host_vector<int> random = unittest::random_integers<unittest::int8_t>(n + *thrust::max_element(sizes, sizes + num_sizes));

  thrust::host_vector<int> h_a(random.begin(), random.begin() + n);
  thrust::host_vector<int> h_b(random.begin() + n, random.end());

  thrust::stable_sort(h_a.begin(), h_a.end());
  thrust::stable_sort(h_b.begin(), h_b.end());
  
  thrust::device_vector<int> d_a = h_a;
  thrust::device_vector<int> d_b = h_b;

  for(size_t i = 0; i < num_sizes; i++)
  {
    size_t size = sizes[i];
    
    thrust::host_vector<int>   h_result(n + size);
    thrust::device_vector<int> d_result(n + size);

    typename thrust::host_vector<int>::iterator   h_end;

    typedef typename thrust::device_vector<int>::iterator iter_type;
    thrust::device_vector<iter_type> d_end(1);
    
    h_end = thrust::merge(h_a.begin(), h_a.end(),
                          h_b.begin(), h_b.begin() + size,
                          h_result.begin());
    h_result.resize(h_end - h_result.begin());

    merge_kernel<<<1,1>>>(exec,
                          d_a.begin(), d_a.end(),
                          d_b.begin(), d_b.begin() + size,
                          d_result.begin(),
                          d_end.begin());
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);

    d_result.resize((iter_type)d_end[0] - d_result.begin());

    ASSERT_EQUAL(h_result, d_result);
  }
}


void TestMergeDeviceSeq()
{
  TestMergeDevice(thrust::seq);
}
DECLARE_UNITTEST(TestMergeDeviceSeq);


void TestMergeDeviceDevice()
{
  TestMergeDevice(thrust::device);
}
DECLARE_UNITTEST(TestMergeDeviceDevice);
#endif


void TestMergeCudaStreams()
{
  typedef thrust::device_vector<int> Vector;
  typedef Vector::iterator Iterator;

  Vector a(3), b(4);

  a[0] = 0; a[1] = 2; a[2] = 4;
  b[0] = 0; b[1] = 3; b[2] = 3; b[3] = 4;

  Vector ref(7);
  ref[0] = 0;
  ref[1] = 0;
  ref[2] = 2;
  ref[3] = 3;
  ref[4] = 3;
  ref[5] = 4;
  ref[6] = 4;

  Vector result(7);

  hipStream_t s;
  hipStreamCreate(&s);

  Iterator end = thrust::merge(thrust::cuda::par.on(s),
                               a.begin(), a.end(),
                               b.begin(), b.end(),
                               result.begin());

  ASSERT_EQUAL_QUIET(result.end(), end);
  ASSERT_EQUAL(ref, result);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestMergeCudaStreams);

