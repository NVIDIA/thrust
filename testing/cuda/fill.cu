#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/fill.h>
#include <thrust/execution_policy.h>
#include <algorithm>

#ifdef THRUST_TEST_DEVICE_SIDE
template<typename ExecutionPolicy, typename Iterator, typename T>
__global__
void fill_kernel(ExecutionPolicy exec, Iterator first, Iterator last, T value)
{
  thrust::fill(exec, first, last, value);
}


template<typename T, typename ExecutionPolicy>
void TestFillDevice(ExecutionPolicy exec, size_t n)
{
  thrust::host_vector<T>   h_data = unittest::random_integers<T>(n);
  thrust::device_vector<T> d_data = h_data;
  
  thrust::fill(h_data.begin() + std::min((size_t)1, n), h_data.begin() + std::min((size_t)3, n), (T) 0);

  fill_kernel<<<1,1>>>(exec, d_data.begin() + std::min((size_t)1, n), d_data.begin() + std::min((size_t)3, n), (T) 0);
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }
  
  ASSERT_EQUAL(h_data, d_data);
  
  thrust::fill(h_data.begin() + std::min((size_t)117, n), h_data.begin() + std::min((size_t)367, n), (T) 1);

  fill_kernel<<<1,1>>>(exec, d_data.begin() + std::min((size_t)117, n), d_data.begin() + std::min((size_t)367, n), (T) 1);
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }
  
  ASSERT_EQUAL(h_data, d_data);
  
  thrust::fill(h_data.begin() + std::min((size_t)8, n), h_data.begin() + std::min((size_t)259, n), (T) 2);

  fill_kernel<<<1,1>>>(exec, d_data.begin() + std::min((size_t)8, n), d_data.begin() + std::min((size_t)259, n), (T) 2);
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }
  
  ASSERT_EQUAL(h_data, d_data);
  
  thrust::fill(h_data.begin() + std::min((size_t)3, n), h_data.end(), (T) 3);

  fill_kernel<<<1,1>>>(exec, d_data.begin() + std::min((size_t)3, n), d_data.end(), (T) 3);
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }
  
  ASSERT_EQUAL(h_data, d_data);
  
  thrust::fill(h_data.begin(), h_data.end(), (T) 4);

  fill_kernel<<<1,1>>>(exec, d_data.begin(), d_data.end(), (T) 4);
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }
  
  ASSERT_EQUAL(h_data, d_data);
}

template<typename T>
void TestFillDeviceSeq(size_t n)
{
  TestFillDevice<T>(thrust::seq, n);
}
DECLARE_VARIABLE_UNITTEST(TestFillDeviceSeq);

template<typename T>
void TestFillDeviceDevice(size_t n)
{
  TestFillDevice<T>(thrust::device, n);
}
DECLARE_VARIABLE_UNITTEST(TestFillDeviceDevice);


template<typename ExecutionPolicy, typename Iterator, typename Size, typename T>
__global__
void fill_n_kernel(ExecutionPolicy exec, Iterator first, Size n, T value)
{
  thrust::fill_n(exec, first, n, value);
}


template<typename T, typename ExecutionPolicy>
void TestFillNDevice(ExecutionPolicy exec, size_t n)
{
  thrust::host_vector<T>   h_data = unittest::random_integers<T>(n);
  thrust::device_vector<T> d_data = h_data;
  
  size_t begin_offset = std::min<size_t>(1,n);

  thrust::fill_n(h_data.begin() + begin_offset, std::min((size_t)3, n) - begin_offset, (T) 0);

  fill_n_kernel<<<1,1>>>(exec, d_data.begin() + begin_offset, std::min((size_t)3, n) - begin_offset, (T) 0);
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }
 
  ASSERT_EQUAL(h_data, d_data);
  
  begin_offset = std::min<size_t>(117, n);

  thrust::fill_n(h_data.begin() + begin_offset, std::min((size_t)367, n) - begin_offset, (T) 1);

  fill_n_kernel<<<1,1>>>(exec, d_data.begin() + begin_offset, std::min((size_t)367, n) - begin_offset, (T) 1);
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }
  
  ASSERT_EQUAL(h_data, d_data);
  
  begin_offset = std::min<size_t>(8, n);

  thrust::fill_n(h_data.begin() + begin_offset, std::min((size_t)259, n) - begin_offset, (T) 2);

  fill_n_kernel<<<1,1>>>(exec, d_data.begin() + begin_offset, std::min((size_t)259, n) - begin_offset, (T) 2);
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }
  
  ASSERT_EQUAL(h_data, d_data);
  
  begin_offset = std::min<size_t>(3, n);

  thrust::fill_n(h_data.begin() + begin_offset, h_data.size() - begin_offset, (T) 3);

  fill_n_kernel<<<1,1>>>(exec, d_data.begin() + begin_offset, d_data.size() - begin_offset, (T) 3);
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }
  
  ASSERT_EQUAL(h_data, d_data);
  
  thrust::fill_n(h_data.begin(), h_data.size(), (T) 4);

  fill_n_kernel<<<1,1>>>(exec, d_data.begin(), d_data.size(), (T) 4);
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }
  
  ASSERT_EQUAL(h_data, d_data);
}

template<typename T>
void TestFillNDeviceSeq(size_t n)
{
  TestFillNDevice<T>(thrust::seq, n);
}
DECLARE_VARIABLE_UNITTEST(TestFillNDeviceSeq);

template<typename T>
void TestFillNDeviceDevice(size_t n)
{
  TestFillNDevice<T>(thrust::device, n);
}
DECLARE_VARIABLE_UNITTEST(TestFillNDeviceDevice);
#endif

void TestFillCudaStreams()
{
  thrust::device_vector<int> v(5);
  v[0] = 0; v[1] = 1; v[2] = 2; v[3] = 3; v[4] = 4;

  hipStream_t s;
  hipStreamCreate(&s);
  
  thrust::fill(thrust::cuda::par.on(s), v.begin() + 1, v.begin() + 4, 7);
  hipStreamSynchronize(s);
  
  ASSERT_EQUAL(v[0], 0);
  ASSERT_EQUAL(v[1], 7);
  ASSERT_EQUAL(v[2], 7);
  ASSERT_EQUAL(v[3], 7);
  ASSERT_EQUAL(v[4], 4);
  
  thrust::fill(thrust::cuda::par.on(s), v.begin() + 0, v.begin() + 3, 8);
  hipStreamSynchronize(s);
  
  ASSERT_EQUAL(v[0], 8);
  ASSERT_EQUAL(v[1], 8);
  ASSERT_EQUAL(v[2], 8);
  ASSERT_EQUAL(v[3], 7);
  ASSERT_EQUAL(v[4], 4);
  
  thrust::fill(thrust::cuda::par.on(s), v.begin() + 2, v.end(), 9);
  hipStreamSynchronize(s);
  
  ASSERT_EQUAL(v[0], 8);
  ASSERT_EQUAL(v[1], 8);
  ASSERT_EQUAL(v[2], 9);
  ASSERT_EQUAL(v[3], 9);
  ASSERT_EQUAL(v[4], 9);
  
  thrust::fill(thrust::cuda::par.on(s), v.begin(), v.end(), 1);
  hipStreamSynchronize(s);
  
  ASSERT_EQUAL(v[0], 1);
  ASSERT_EQUAL(v[1], 1);
  ASSERT_EQUAL(v[2], 1);
  ASSERT_EQUAL(v[3], 1);
  ASSERT_EQUAL(v[4], 1);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestFillCudaStreams);

