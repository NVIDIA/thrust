#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/uninitialized_fill.h>
#include <thrust/execution_policy.h>


#ifdef THRUST_TEST_DEVICE_SIDE
template<typename ExecutionPolicy, typename Iterator, typename T>
__global__
void uninitialized_fill_kernel(ExecutionPolicy exec, Iterator first, Iterator last, T val)
{
  thrust::uninitialized_fill(exec, first, last, val);
}


template<typename ExecutionPolicy>
void TestUninitializedFillDevice(ExecutionPolicy exec)
{
  typedef thrust::device_vector<int> Vector;
  typedef Vector::value_type T;
  
  Vector v(5);
  v[0] = 0; v[1] = 1; v[2] = 2; v[3] = 3; v[4] = 4;
  
  T exemplar(7);
  
  uninitialized_fill_kernel<<<1,1>>>(exec, v.begin() + 1, v.begin() + 4, exemplar);
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }
  
  ASSERT_EQUAL(v[0], 0);
  ASSERT_EQUAL(v[1], exemplar);
  ASSERT_EQUAL(v[2], exemplar);
  ASSERT_EQUAL(v[3], exemplar);
  ASSERT_EQUAL(v[4], 4);
  
  exemplar = 8;
  
  uninitialized_fill_kernel<<<1,1>>>(exec, v.begin() + 0, v.begin() + 3, exemplar);
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }
  
  ASSERT_EQUAL(v[0], exemplar);
  ASSERT_EQUAL(v[1], exemplar);
  ASSERT_EQUAL(v[2], exemplar);
  ASSERT_EQUAL(v[3], 7);
  ASSERT_EQUAL(v[4], 4);
  
  exemplar = 9;
  
  uninitialized_fill_kernel<<<1,1>>>(exec, v.begin() + 2, v.end(), exemplar);
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }
  
  ASSERT_EQUAL(v[0], 8);
  ASSERT_EQUAL(v[1], 8);
  ASSERT_EQUAL(v[2], exemplar);
  ASSERT_EQUAL(v[3], exemplar);
  ASSERT_EQUAL(v[4], 9);
  
  exemplar = 1;
  
  uninitialized_fill_kernel<<<1,1>>>(exec, v.begin(), v.end(), exemplar);
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }
  
  ASSERT_EQUAL(v[0], exemplar);
  ASSERT_EQUAL(v[1], exemplar);
  ASSERT_EQUAL(v[2], exemplar);
  ASSERT_EQUAL(v[3], exemplar);
  ASSERT_EQUAL(v[4], exemplar);
}


void TestUninitializedFillDeviceSeq()
{
  TestUninitializedFillDevice(thrust::seq);
}
DECLARE_UNITTEST(TestUninitializedFillDeviceSeq);


void TestUninitializedFillDeviceDevice()
{
  TestUninitializedFillDevice(thrust::device);
}
DECLARE_UNITTEST(TestUninitializedFillDeviceDevice);
#endif


void TestUninitializedFillCudaStreams()
{
  typedef thrust::device_vector<int> Vector;
  typedef Vector::value_type T;
  
  Vector v(5);
  v[0] = 0; v[1] = 1; v[2] = 2; v[3] = 3; v[4] = 4;
  
  T exemplar(7);

  hipStream_t s;
  hipStreamCreate(&s);
  
  thrust::uninitialized_fill(thrust::cuda::par.on(s), v.begin(), v.end(), exemplar);
  hipStreamSynchronize(s);
  
  ASSERT_EQUAL(v[0], exemplar);
  ASSERT_EQUAL(v[1], exemplar);
  ASSERT_EQUAL(v[2], exemplar);
  ASSERT_EQUAL(v[3], exemplar);
  ASSERT_EQUAL(v[4], exemplar);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestUninitializedFillCudaStreams);


#ifdef THRUST_TEST_DEVICE_SIDE
template<typename ExecutionPolicy, typename Iterator1, typename Size, typename T, typename Iterator2>
__global__
void uninitialized_fill_n_kernel(ExecutionPolicy exec, Iterator1 first, Size n, T val, Iterator2 result)
{
  *result = thrust::uninitialized_fill_n(exec, first, n, val);
}


template<typename ExecutionPolicy>
void TestUninitializedFillNDevice(ExecutionPolicy exec)
{
  typedef thrust::device_vector<int> Vector;
  typedef Vector::value_type T;
  
  Vector v(5);
  v[0] = 0; v[1] = 1; v[2] = 2; v[3] = 3; v[4] = 4;
  
  T exemplar(7);

  thrust::device_vector<Vector::iterator> iter_vec(1);
  
  uninitialized_fill_n_kernel<<<1,1>>>(exec, v.begin() + 1, 3, exemplar, iter_vec.begin());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  Vector::iterator iter = iter_vec[0];
  
  ASSERT_EQUAL(v[0], 0);
  ASSERT_EQUAL(v[1], exemplar);
  ASSERT_EQUAL(v[2], exemplar);
  ASSERT_EQUAL(v[3], exemplar);
  ASSERT_EQUAL(v[4], 4);
  ASSERT_EQUAL_QUIET(v.begin() + 4, iter);
  
  exemplar = 8;
  
  uninitialized_fill_n_kernel<<<1,1>>>(exec, v.begin() + 0, 3, exemplar, iter_vec.begin());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  iter = iter_vec[0];
  
  ASSERT_EQUAL(v[0], exemplar);
  ASSERT_EQUAL(v[1], exemplar);
  ASSERT_EQUAL(v[2], exemplar);
  ASSERT_EQUAL(v[3], 7);
  ASSERT_EQUAL(v[4], 4);
  ASSERT_EQUAL_QUIET(v.begin() + 3, iter);
  
  exemplar = 9;
  
  uninitialized_fill_n_kernel<<<1,1>>>(exec, v.begin() + 2, 3, exemplar, iter_vec.begin());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  iter = iter_vec[0];
  
  ASSERT_EQUAL(v[0], 8);
  ASSERT_EQUAL(v[1], 8);
  ASSERT_EQUAL(v[2], exemplar);
  ASSERT_EQUAL(v[3], exemplar);
  ASSERT_EQUAL(v[4], 9);
  ASSERT_EQUAL_QUIET(v.end(), iter);
  
  exemplar = 1;
  
  uninitialized_fill_n_kernel<<<1,1>>>(exec, v.begin(), v.size(), exemplar, iter_vec.begin());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  iter = iter_vec[0];
  
  ASSERT_EQUAL(v[0], exemplar);
  ASSERT_EQUAL(v[1], exemplar);
  ASSERT_EQUAL(v[2], exemplar);
  ASSERT_EQUAL(v[3], exemplar);
  ASSERT_EQUAL(v[4], exemplar);
  ASSERT_EQUAL_QUIET(v.end(), iter);
}


void TestUninitializedFillNDeviceSeq()
{
  TestUninitializedFillNDevice(thrust::seq);
}
DECLARE_UNITTEST(TestUninitializedFillNDeviceSeq);


void TestUninitializedFillNDeviceDevice()
{
  TestUninitializedFillNDevice(thrust::device);
}
DECLARE_UNITTEST(TestUninitializedFillNDeviceDevice);
#endif


void TestUninitializedFillNCudaStreams()
{
  typedef thrust::device_vector<int> Vector;
  typedef Vector::value_type T;
  
  Vector v(5);
  v[0] = 0; v[1] = 1; v[2] = 2; v[3] = 3; v[4] = 4;
  
  T exemplar(7);

  hipStream_t s;
  hipStreamCreate(&s);
  
  thrust::uninitialized_fill_n(thrust::cuda::par.on(s), v.begin(), v.size(), exemplar);
  hipStreamSynchronize(s);
  
  ASSERT_EQUAL(v[0], exemplar);
  ASSERT_EQUAL(v[1], exemplar);
  ASSERT_EQUAL(v[2], exemplar);
  ASSERT_EQUAL(v[3], exemplar);
  ASSERT_EQUAL(v[4], exemplar);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestUninitializedFillNCudaStreams);

