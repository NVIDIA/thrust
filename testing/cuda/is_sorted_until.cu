#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>


#ifdef THRUST_TEST_DEVICE_SIDE
template<typename ExecutionPolicy, typename Iterator1, typename Iterator2>
__global__
void is_sorted_until_kernel(ExecutionPolicy exec, Iterator1 first, Iterator1 last, Iterator2 result)
{
  *result = thrust::is_sorted_until(exec, first, last);
}


template<typename ExecutionPolicy>
void TestIsSortedUntilDevice(ExecutionPolicy exec)
{
  size_t n = 1000;

  thrust::device_vector<int> v = unittest::random_integers<int>(n);

  typedef typename thrust::device_vector<int>::iterator iter_type;

  thrust::device_vector<iter_type> result(1);

  v[0] = 1;
  v[1] = 0;
  
  is_sorted_until_kernel<<<1,1>>>(exec, v.begin(), v.end(), result.begin());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  ASSERT_EQUAL_QUIET(v.begin() + 1, (iter_type)result[0]);
  
  thrust::sort(v.begin(), v.end());
  
  is_sorted_until_kernel<<<1,1>>>(exec, v.begin(), v.end(), result.begin());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  ASSERT_EQUAL_QUIET(v.end(), (iter_type)result[0]);
}


void TestIsSortedUntilDeviceSeq()
{
  TestIsSortedUntilDevice(thrust::seq);
}
DECLARE_UNITTEST(TestIsSortedUntilDeviceSeq);


void TestIsSortedUntilDeviceDevice()
{
  TestIsSortedUntilDevice(thrust::device);
}
DECLARE_UNITTEST(TestIsSortedUntilDeviceDevice);
#endif


void TestIsSortedUntilCudaStreams()
{
  typedef thrust::device_vector<int> Vector;

  typedef Vector::value_type T;
  typedef Vector::iterator Iterator;

  hipStream_t s;
  hipStreamCreate(&s);

  Vector v(4);
  v[0] = 0; v[1] = 5; v[2] = 8; v[3] = 0;

  Iterator first = v.begin();

  Iterator last  = v.begin() + 0;
  Iterator ref = last;
  ASSERT_EQUAL_QUIET(ref, thrust::is_sorted_until(thrust::cuda::par.on(s), first, last));

  last = v.begin() + 1;
  ref = last;
  ASSERT_EQUAL_QUIET(ref, thrust::is_sorted_until(thrust::cuda::par.on(s), first, last));

  last = v.begin() + 2;
  ref = last;
  ASSERT_EQUAL_QUIET(ref, thrust::is_sorted_until(thrust::cuda::par.on(s), first, last));

  last = v.begin() + 3;
  ref = v.begin() + 3;
  ASSERT_EQUAL_QUIET(ref, thrust::is_sorted_until(thrust::cuda::par.on(s), first, last));

  last = v.begin() + 4;
  ref = v.begin() + 3;
  ASSERT_EQUAL_QUIET(ref, thrust::is_sorted_until(thrust::cuda::par.on(s), first, last));

  last = v.begin() + 3;
  ref = v.begin() + 3;
  ASSERT_EQUAL_QUIET(ref, thrust::is_sorted_until(thrust::cuda::par.on(s), first, last, thrust::less<T>()));

  last = v.begin() + 4;
  ref = v.begin() + 3;
  ASSERT_EQUAL_QUIET(ref, thrust::is_sorted_until(thrust::cuda::par.on(s), first, last, thrust::less<T>()));

  last = v.begin() + 1;
  ref = v.begin() + 1;
  ASSERT_EQUAL_QUIET(ref, thrust::is_sorted_until(thrust::cuda::par.on(s), first, last, thrust::greater<T>()));

  last = v.begin() + 4;
  ref = v.begin() + 1;
  ASSERT_EQUAL_QUIET(ref, thrust::is_sorted_until(thrust::cuda::par.on(s), first, last, thrust::greater<T>()));

  first = v.begin() + 2;
  last = v.begin() + 4;
  ref = v.begin() + 4;
  ASSERT_EQUAL_QUIET(ref, thrust::is_sorted_until(thrust::cuda::par.on(s), first, last, thrust::greater<T>()));

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestIsSortedUntilCudaStreams);

