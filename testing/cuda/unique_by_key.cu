#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/unique.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>


template<typename T>
struct is_equal_div_10_unique
{
  __host__ __device__
  bool operator()(const T x, const T& y) const { return ((int) x / 10) == ((int) y / 10); }
};


template<typename Vector>
void initialize_keys(Vector& keys)
{
  keys.resize(9);
  keys[0] = 11;
  keys[1] = 11;
  keys[2] = 21;
  keys[3] = 20;
  keys[4] = 21;
  keys[5] = 21;
  keys[6] = 21;
  keys[7] = 37;
  keys[8] = 37;
}


template<typename Vector>
void initialize_values(Vector& values)
{
  values.resize(9);
  values[0] = 0; 
  values[1] = 1;
  values[2] = 2;
  values[3] = 3;
  values[4] = 4;
  values[5] = 5;
  values[6] = 6;
  values[7] = 7;
  values[8] = 8;
}


template<typename ExecutionPolicy, typename Iterator1, typename Iterator2, typename Iterator3>
__global__
void unique_by_key_kernel(ExecutionPolicy exec, Iterator1 keys_first, Iterator1 keys_last, Iterator2 values_first, Iterator3 result)
{
  *result = thrust::unique_by_key(exec, keys_first, keys_last, values_first);
}


template<typename ExecutionPolicy, typename Iterator1, typename Iterator2, typename BinaryPredicate, typename Iterator3>
__global__
void unique_by_key_kernel(ExecutionPolicy exec, Iterator1 keys_first, Iterator1 keys_last, Iterator2 values_first, BinaryPredicate pred, Iterator3 result)
{
  *result = thrust::unique_by_key(exec, keys_first, keys_last, values_first, pred);
}


template<typename ExecutionPolicy>
void TestUniqueByKeyDevice(ExecutionPolicy exec)
{
  typedef thrust::device_vector<int> Vector;
  typedef Vector::value_type T;
  
  Vector keys;
  Vector values;
  
  typedef thrust::pair<typename Vector::iterator, typename Vector::iterator> iter_pair;
  thrust::device_vector<iter_pair> new_last_vec(1);
  iter_pair new_last;
  
  // basic test
  initialize_keys(keys);  initialize_values(values);
  
  unique_by_key_kernel<<<1,1>>>(exec, keys.begin(), keys.end(), values.begin(), new_last_vec.begin());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  new_last = new_last_vec[0];
  
  ASSERT_EQUAL(new_last.first  - keys.begin(),   5);
  ASSERT_EQUAL(new_last.second - values.begin(), 5);
  ASSERT_EQUAL(keys[0], 11);
  ASSERT_EQUAL(keys[1], 21);
  ASSERT_EQUAL(keys[2], 20);
  ASSERT_EQUAL(keys[3], 21);
  ASSERT_EQUAL(keys[4], 37);
  
  ASSERT_EQUAL(values[0], 0);
  ASSERT_EQUAL(values[1], 2);
  ASSERT_EQUAL(values[2], 3);
  ASSERT_EQUAL(values[3], 4);
  ASSERT_EQUAL(values[4], 7);
  
  // test BinaryPredicate
  initialize_keys(keys);  initialize_values(values);
  
  unique_by_key_kernel<<<1,1>>>(exec, keys.begin(), keys.end(), values.begin(), is_equal_div_10_unique<T>(), new_last_vec.begin());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  new_last = new_last_vec[0];
  
  ASSERT_EQUAL(new_last.first  - keys.begin(),   3);
  ASSERT_EQUAL(new_last.second - values.begin(), 3);
  ASSERT_EQUAL(keys[0], 11);
  ASSERT_EQUAL(keys[1], 21);
  ASSERT_EQUAL(keys[2], 37);
  
  ASSERT_EQUAL(values[0], 0);
  ASSERT_EQUAL(values[1], 2);
  ASSERT_EQUAL(values[2], 7);
}

void TestUniqueByKeyDeviceSeq()
{
  TestUniqueByKeyDevice(thrust::seq);
}
DECLARE_UNITTEST(TestUniqueByKeyDeviceSeq);


void TestUniqueByKeyDeviceDevice()
{
  TestUniqueByKeyDevice(thrust::device);
}
DECLARE_UNITTEST(TestUniqueByKeyDeviceDevice);


void TestUniqueByKeyDeviceNoSync()
{
  TestUniqueByKeyDevice(thrust::cuda::par_nosync);
}
DECLARE_UNITTEST(TestUniqueByKeyDeviceNoSync);


template<typename ExecutionPolicy>
void TestUniqueByKeyCudaStreams(ExecutionPolicy policy)
{
  typedef thrust::device_vector<int> Vector;
  typedef Vector::value_type T;
  
  Vector keys;
  Vector values;
  
  typedef thrust::pair<Vector::iterator, Vector::iterator> iter_pair;
  iter_pair new_last;
  
  // basic test
  initialize_keys(keys);  initialize_values(values);

  hipStream_t s;
  hipStreamCreate(&s);

  auto streampolicy = policy.on(s);
  
  new_last = thrust::unique_by_key(streampolicy, keys.begin(), keys.end(), values.begin());
  hipStreamSynchronize(s);
  
  ASSERT_EQUAL(new_last.first  - keys.begin(),   5);
  ASSERT_EQUAL(new_last.second - values.begin(), 5);
  ASSERT_EQUAL(keys[0], 11);
  ASSERT_EQUAL(keys[1], 21);
  ASSERT_EQUAL(keys[2], 20);
  ASSERT_EQUAL(keys[3], 21);
  ASSERT_EQUAL(keys[4], 37);
  
  ASSERT_EQUAL(values[0], 0);
  ASSERT_EQUAL(values[1], 2);
  ASSERT_EQUAL(values[2], 3);
  ASSERT_EQUAL(values[3], 4);
  ASSERT_EQUAL(values[4], 7);
  
  // test BinaryPredicate
  initialize_keys(keys);  initialize_values(values);
  
  new_last = thrust::unique_by_key(streampolicy, keys.begin(), keys.end(), values.begin(), is_equal_div_10_unique<T>());
  
  ASSERT_EQUAL(new_last.first  - keys.begin(),   3);
  ASSERT_EQUAL(new_last.second - values.begin(), 3);
  ASSERT_EQUAL(keys[0], 11);
  ASSERT_EQUAL(keys[1], 21);
  ASSERT_EQUAL(keys[2], 37);
  
  ASSERT_EQUAL(values[0], 0);
  ASSERT_EQUAL(values[1], 2);
  ASSERT_EQUAL(values[2], 7);

  hipStreamDestroy(s);
}

void TestUniqueByKeyCudaStreamsSync()
{
  TestUniqueByKeyCudaStreams(thrust::cuda::par);
}
DECLARE_UNITTEST(TestUniqueByKeyCudaStreamsSync);


void TestUniqueByKeyCudaStreamsNoSync()
{
  TestUniqueByKeyCudaStreams(thrust::cuda::par_nosync);
}
DECLARE_UNITTEST(TestUniqueByKeyCudaStreamsNoSync);


template<typename ExecutionPolicy, typename Iterator1, typename Iterator2, typename Iterator3, typename Iterator4, typename Iterator5>
__global__
void unique_by_key_copy_kernel(ExecutionPolicy exec, Iterator1 keys_first, Iterator1 keys_last, Iterator2 values_first, Iterator3 keys_result, Iterator4 values_result, Iterator5 result)
{
  *result = thrust::unique_by_key_copy(exec, keys_first, keys_last, values_first, keys_result, values_result);
}


template<typename ExecutionPolicy, typename Iterator1, typename Iterator2, typename Iterator3, typename Iterator4, typename BinaryPredicate, typename Iterator5>
__global__
void unique_by_key_copy_kernel(ExecutionPolicy exec, Iterator1 keys_first, Iterator1 keys_last, Iterator2 values_first, Iterator3 keys_result, Iterator4 values_result, BinaryPredicate pred, Iterator5 result)
{
  *result = thrust::unique_by_key_copy(exec, keys_first, keys_last, values_first, keys_result, values_result, pred);
}


template<typename ExecutionPolicy>
void TestUniqueCopyByKeyDevice(ExecutionPolicy exec)
{
  typedef thrust::device_vector<int> Vector;
  typedef Vector::value_type T;

  Vector keys;
  Vector values;

  typedef thrust::pair<typename Vector::iterator, typename Vector::iterator> iter_pair;
  thrust::device_vector<iter_pair> new_last_vec(1);
  iter_pair new_last;

  // basic test
  initialize_keys(keys);  initialize_values(values);

  Vector output_keys(keys.size());
  Vector output_values(values.size());

  unique_by_key_copy_kernel<<<1,1>>>(exec, keys.begin(), keys.end(), values.begin(), output_keys.begin(), output_values.begin(), new_last_vec.begin());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  new_last = new_last_vec[0];

  ASSERT_EQUAL(new_last.first  - output_keys.begin(),   5);
  ASSERT_EQUAL(new_last.second - output_values.begin(), 5);
  ASSERT_EQUAL(output_keys[0], 11);
  ASSERT_EQUAL(output_keys[1], 21);
  ASSERT_EQUAL(output_keys[2], 20);
  ASSERT_EQUAL(output_keys[3], 21);
  ASSERT_EQUAL(output_keys[4], 37);
  
  ASSERT_EQUAL(output_values[0], 0);
  ASSERT_EQUAL(output_values[1], 2);
  ASSERT_EQUAL(output_values[2], 3);
  ASSERT_EQUAL(output_values[3], 4);
  ASSERT_EQUAL(output_values[4], 7);

  // test BinaryPredicate
  initialize_keys(keys);  initialize_values(values);
  
  unique_by_key_copy_kernel<<<1,1>>>(exec, keys.begin(), keys.end(), values.begin(), output_keys.begin(), output_values.begin(), is_equal_div_10_unique<T>(), new_last_vec.begin());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  new_last = new_last_vec[0];

  ASSERT_EQUAL(new_last.first  - output_keys.begin(),   3);
  ASSERT_EQUAL(new_last.second - output_values.begin(), 3);
  ASSERT_EQUAL(output_keys[0], 11);
  ASSERT_EQUAL(output_keys[1], 21);
  ASSERT_EQUAL(output_keys[2], 37);
  
  ASSERT_EQUAL(output_values[0], 0);
  ASSERT_EQUAL(output_values[1], 2);
  ASSERT_EQUAL(output_values[2], 7);
}


void TestUniqueCopyByKeyDeviceSeq()
{
  TestUniqueCopyByKeyDevice(thrust::seq);
}
DECLARE_UNITTEST(TestUniqueCopyByKeyDeviceSeq);


void TestUniqueCopyByKeyDeviceDevice()
{
  TestUniqueCopyByKeyDevice(thrust::device);
}
DECLARE_UNITTEST(TestUniqueCopyByKeyDeviceDevice);


void TestUniqueCopyByKeyDeviceNoSync()
{
  TestUniqueCopyByKeyDevice(thrust::cuda::par_nosync);
}
DECLARE_UNITTEST(TestUniqueCopyByKeyDeviceNoSync);


template<typename ExecutionPolicy>
void TestUniqueCopyByKeyCudaStreams(ExecutionPolicy policy)
{
  typedef thrust::device_vector<int> Vector;
  typedef Vector::value_type T;

  Vector keys;
  Vector values;

  typedef thrust::pair<Vector::iterator, Vector::iterator> iter_pair;
  iter_pair new_last;

  // basic test
  initialize_keys(keys);  initialize_values(values);

  Vector output_keys(keys.size());
  Vector output_values(values.size());

  hipStream_t s;
  hipStreamCreate(&s);

  auto streampolicy = policy.on(s);

  new_last = thrust::unique_by_key_copy(streampolicy, keys.begin(), keys.end(), values.begin(), output_keys.begin(), output_values.begin());
  hipStreamSynchronize(s);

  ASSERT_EQUAL(new_last.first  - output_keys.begin(),   5);
  ASSERT_EQUAL(new_last.second - output_values.begin(), 5);
  ASSERT_EQUAL(output_keys[0], 11);
  ASSERT_EQUAL(output_keys[1], 21);
  ASSERT_EQUAL(output_keys[2], 20);
  ASSERT_EQUAL(output_keys[3], 21);
  ASSERT_EQUAL(output_keys[4], 37);
  
  ASSERT_EQUAL(output_values[0], 0);
  ASSERT_EQUAL(output_values[1], 2);
  ASSERT_EQUAL(output_values[2], 3);
  ASSERT_EQUAL(output_values[3], 4);
  ASSERT_EQUAL(output_values[4], 7);

  // test BinaryPredicate
  initialize_keys(keys);  initialize_values(values);
  
  new_last = thrust::unique_by_key_copy(streampolicy, keys.begin(), keys.end(), values.begin(), output_keys.begin(), output_values.begin(), is_equal_div_10_unique<T>());
  hipStreamSynchronize(s);

  ASSERT_EQUAL(new_last.first  - output_keys.begin(),   3);
  ASSERT_EQUAL(new_last.second - output_values.begin(), 3);
  ASSERT_EQUAL(output_keys[0], 11);
  ASSERT_EQUAL(output_keys[1], 21);
  ASSERT_EQUAL(output_keys[2], 37);
  
  ASSERT_EQUAL(output_values[0], 0);
  ASSERT_EQUAL(output_values[1], 2);
  ASSERT_EQUAL(output_values[2], 7);

  hipStreamDestroy(s);
}

void TestUniqueCopyByKeyCudaStreamsSync()
{
  TestUniqueCopyByKeyCudaStreams(thrust::cuda::par);
}
DECLARE_UNITTEST(TestUniqueCopyByKeyCudaStreamsSync);


void TestUniqueCopyByKeyCudaStreamsNoSync()
{
  TestUniqueCopyByKeyCudaStreams(thrust::cuda::par_nosync);
}
DECLARE_UNITTEST(TestUniqueCopyByKeyCudaStreamsNoSync);

