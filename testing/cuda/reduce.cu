#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>


template<typename ExecutionPolicy, typename Iterator, typename T, typename Iterator2>
__global__
void reduce_kernel(ExecutionPolicy exec, Iterator first, Iterator last, T init, Iterator2 result)
{
  *result = thrust::reduce(exec, first, last, init);
}


template<typename T, typename ExecutionPolicy>
void TestReduceDevice(ExecutionPolicy exec, const size_t n)
{
  thrust::host_vector<T>   h_data = unittest::random_integers<T>(n);
  thrust::device_vector<T> d_data = h_data;
  
  thrust::device_vector<T> d_result(1);
  
  T init = 13;
  
  T h_result = thrust::reduce(h_data.begin(), h_data.end(), init);
  
  reduce_kernel<<<1,1>>>(exec, d_data.begin(), d_data.end(), init, d_result.begin());
  hipError_t const err = hipDeviceSynchronize();
  ASSERT_EQUAL(hipSuccess, err);
  
  ASSERT_EQUAL(h_result, d_result[0]);
}


template<typename T>
struct TestReduceDeviceSeq
{
  void operator()(const size_t n)
  {
    TestReduceDevice<T>(thrust::seq, n);
  }
};
VariableUnitTest<TestReduceDeviceSeq, IntegralTypes> TestReduceDeviceSeqInstance;


template<typename T>
struct TestReduceDeviceDevice
{
  void operator()(const size_t n)
  {
    TestReduceDevice<T>(thrust::device, n);
  }
};
VariableUnitTest<TestReduceDeviceDevice, IntegralTypes> TestReduceDeviceDeviceInstance;


template<typename T>
struct TestReduceDeviceNoSync
{
  void operator()(const size_t n)
  {
    TestReduceDevice<T>(thrust::cuda::par_nosync, n);
  }
};
VariableUnitTest<TestReduceDeviceNoSync, IntegralTypes> TestReduceDeviceNoSyncInstance;


template<typename ExecutionPolicy>
void TestReduceCudaStreams(ExecutionPolicy policy)
{
  typedef thrust::device_vector<int> Vector;

  Vector v(3);
  v[0] = 1; v[1] = -2; v[2] = 3;

  hipStream_t s;
  hipStreamCreate(&s);

  auto streampolicy = policy.on(s);

  // no initializer
  ASSERT_EQUAL(thrust::reduce(streampolicy, v.begin(), v.end()), 2);

  // with initializer
  ASSERT_EQUAL(thrust::reduce(streampolicy, v.begin(), v.end(), 10), 12);

  hipStreamDestroy(s);
}

void TestReduceCudaStreamsSync()
{
  TestReduceCudaStreams(thrust::cuda::par);
}
DECLARE_UNITTEST(TestReduceCudaStreamsSync);


void TestReduceCudaStreamsNoSync()
{
  TestReduceCudaStreams(thrust::cuda::par_nosync);
}
DECLARE_UNITTEST(TestReduceCudaStreamsNoSync);

