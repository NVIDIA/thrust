#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>


#ifdef THRUST_TEST_DEVICE_SIDE
template<typename ExecutionPolicy, typename Iterator, typename Iterator2>
__global__
void is_sorted_kernel(ExecutionPolicy exec, Iterator first, Iterator last, Iterator2 result)
{
  *result = thrust::is_sorted(exec, first, last);
}


template<typename ExecutionPolicy>
void TestIsSortedDevice(ExecutionPolicy exec)
{
  size_t n = 1000;

  thrust::device_vector<int> v = unittest::random_integers<int>(n);

  thrust::device_vector<bool> result(1);

  v[0] = 1;
  v[1] = 0;

  is_sorted_kernel<<<1,1>>>(exec, v.begin(), v.end(), result.begin());

  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  ASSERT_EQUAL(false, result[0]);

  thrust::sort(v.begin(), v.end());

  is_sorted_kernel<<<1,1>>>(exec, v.begin(), v.end(), result.begin());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  ASSERT_EQUAL(true, result[0]);
}

void TestIsSortedDeviceSeq()
{
  TestIsSortedDevice(thrust::seq);
}
DECLARE_UNITTEST(TestIsSortedDeviceSeq);


void TestIsSortedDeviceDevice()
{
  TestIsSortedDevice(thrust::device);
}
DECLARE_UNITTEST(TestIsSortedDeviceDevice);
#endif


void TestIsSortedCudaStreams()
{
  thrust::device_vector<int> v(4);
  v[0] = 0; v[1] = 5; v[2] = 8; v[3] = 0;

  hipStream_t s;
  hipStreamCreate(&s);
  
  ASSERT_EQUAL(thrust::is_sorted(thrust::cuda::par.on(s), v.begin(), v.begin() + 0), true);
  ASSERT_EQUAL(thrust::is_sorted(thrust::cuda::par.on(s), v.begin(), v.begin() + 1), true);
  
  // the following line crashes gcc 4.3
#if (__GNUC__ == 4) && (__GNUC_MINOR__ == 3)
  // do nothing
#else
  // compile this line on other compilers
  ASSERT_EQUAL(thrust::is_sorted(thrust::cuda::par.on(s), v.begin(), v.begin() + 2), true);
#endif // GCC

  ASSERT_EQUAL(thrust::is_sorted(thrust::cuda::par.on(s), v.begin(), v.begin() + 3), true);
  ASSERT_EQUAL(thrust::is_sorted(thrust::cuda::par.on(s), v.begin(), v.begin() + 4), false);
  
  ASSERT_EQUAL(thrust::is_sorted(thrust::cuda::par.on(s), v.begin(), v.begin() + 3, thrust::less<int>()),    true);
  
  ASSERT_EQUAL(thrust::is_sorted(thrust::cuda::par.on(s), v.begin(), v.begin() + 1, thrust::greater<int>()), true);
  ASSERT_EQUAL(thrust::is_sorted(thrust::cuda::par.on(s), v.begin(), v.begin() + 4, thrust::greater<int>()), false);
  
  ASSERT_EQUAL(thrust::is_sorted(thrust::cuda::par.on(s), v.begin(), v.end()), false);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestIsSortedCudaStreams);

