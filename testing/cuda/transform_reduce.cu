#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/transform_reduce.h>
#include <thrust/execution_policy.h>


#ifdef THRUST_TEST_DEVICE_SIDE
template<typename ExecutionPolicy, typename Iterator1, typename Function1, typename T, typename Function2, typename Iterator2>
__global__
void transform_reduce_kernel(ExecutionPolicy exec, Iterator1 first, Iterator1 last, Function1 f1, T init, Function2 f2, Iterator2 result)
{
  *result = thrust::transform_reduce(exec, first, last, f1, init, f2);
}


template<typename ExecutionPolicy>
void TestTransformReduceDevice(ExecutionPolicy exec)
{
  typedef thrust::device_vector<int> Vector;
  typedef typename Vector::value_type T;
  
  Vector data(3);
  data[0] = 1; data[1] = -2; data[2] = 3;
  
  T init = 10;

  thrust::device_vector<T> result(1);

  transform_reduce_kernel<<<1,1>>>(exec, data.begin(), data.end(), thrust::negate<T>(), init, thrust::plus<T>(), result.begin());
  hipError_t const err = hipDeviceSynchronize();
  ASSERT_EQUAL(hipSuccess, err);
  
  ASSERT_EQUAL(8, (T)result[0]);
}


void TestTransformReduceDeviceSeq()
{
  TestTransformReduceDevice(thrust::seq);
}
DECLARE_UNITTEST(TestTransformReduceDeviceSeq);


void TestTransformReduceDeviceDevice()
{
  TestTransformReduceDevice(thrust::device);
}
DECLARE_UNITTEST(TestTransformReduceDeviceDevice);
#endif


void TestTransformReduceCudaStreams()
{
  typedef thrust::device_vector<int> Vector;
  typedef Vector::value_type T;
  
  Vector data(3);
  data[0] = 1; data[1] = -2; data[2] = 3;
  
  T init = 10;

  hipStream_t s;
  hipStreamCreate(&s);

  T result = thrust::transform_reduce(thrust::cuda::par.on(s), data.begin(), data.end(), thrust::negate<T>(), init, thrust::plus<T>());
  hipStreamSynchronize(s);
  
  ASSERT_EQUAL(8, result);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestTransformReduceCudaStreams);

