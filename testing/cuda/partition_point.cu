#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/partition.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>


#ifdef THRUST_TEST_DEVICE_SIDE
template<typename ExecutionPolicy, typename Iterator1, typename Predicate, typename Iterator2>
__global__
void partition_point_kernel(ExecutionPolicy exec, Iterator1 first, Iterator1 last, Predicate pred, Iterator2 result)
{
  *result = thrust::partition_point(exec, first, last, pred);
}


template<typename T>
struct is_even
{
  __host__ __device__
  bool operator()(T x) const { return ((int) x % 2) == 0; }
};


template<typename ExecutionPolicy>
void TestPartitionPointDevice(ExecutionPolicy exec)
{
  size_t n = 1000;
  thrust::device_vector<int> v = unittest::random_integers<int>(n);
  typedef typename thrust::device_vector<int>::iterator iterator;

  iterator ref = thrust::stable_partition(v.begin(), v.end(), is_even<int>());

  thrust::device_vector<iterator> result(1);
  partition_point_kernel<<<1,1>>>(exec, v.begin(), v.end(), is_even<int>(), result.begin());
  hipError_t const err = hipDeviceSynchronize();
  ASSERT_EQUAL(hipSuccess, err);

  ASSERT_EQUAL(ref - v.begin(), (iterator)result[0] - v.begin());
}


void TestPartitionPointDeviceSeq()
{
  TestPartitionPointDevice(thrust::seq);
}
DECLARE_UNITTEST(TestPartitionPointDeviceSeq);


void TestPartitionPointDeviceDevice()
{
  TestPartitionPointDevice(thrust::device);
}
DECLARE_UNITTEST(TestPartitionPointDeviceDevice);
#endif


void TestPartitionPointCudaStreams()
{
  typedef thrust::device_vector<int> Vector;
  typedef Vector::value_type T;
  typedef Vector::iterator Iterator;

  Vector v(4);
  v[0] = 1; v[1] = 1; v[2] = 1; v[3] = 0;

  Iterator first = v.begin();

  Iterator last = v.begin() + 4;
  Iterator ref = first + 3;

  hipStream_t s;
  hipStreamCreate(&s);

  ASSERT_EQUAL_QUIET(ref, thrust::partition_point(thrust::cuda::par.on(s), first, last, thrust::identity<T>()));

  last = v.begin() + 3;
  ref = last;
  ASSERT_EQUAL_QUIET(ref, thrust::partition_point(thrust::cuda::par.on(s), first, last, thrust::identity<T>()));

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestPartitionPointCudaStreams);

