#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/set_operations.h>
#include <thrust/functional.h>
#include <thrust/sort.h>
#include <thrust/extrema.h>
#include <thrust/iterator/discard_iterator.h>


template<typename ExecutionPolicy, typename Iterator1, typename Iterator2, typename Iterator3, typename Iterator4>
__global__
void set_intersection_kernel(ExecutionPolicy exec, Iterator1 first1, Iterator1 last1,
                             Iterator2 first2, Iterator2 last2,
                             Iterator3 result1,
                             Iterator4 result2)
{
  *result2 = thrust::set_intersection(exec, first1, last1, first2, last2, result1);
}


template<typename ExecutionPolicy>
void TestSetIntersectionDevice(ExecutionPolicy exec)
{
  typedef thrust::device_vector<int> Vector;
  typedef Vector::iterator Iterator;

  Vector a(3), b(4);

  a[0] = 0; a[1] = 2; a[2] = 4;
  b[0] = 0; b[1] = 3; b[2] = 3; b[3] = 4;

  Vector ref(2);
  ref[0] = 0; ref[1] = 4;

  Vector result(2);
  thrust::device_vector<Iterator> end_vec(1);

  set_intersection_kernel<<<1,1>>>(exec, a.begin(), a.end(), b.begin(), b.end(), result.begin(), end_vec.begin());
  hipError_t const err = hipDeviceSynchronize();
  ASSERT_EQUAL(hipSuccess, err);

  Iterator end = end_vec.front();

  ASSERT_EQUAL_QUIET(result.end(), end);
  ASSERT_EQUAL(ref, result);
}


void TestSetIntersectionDeviceSeq()
{
  TestSetIntersectionDevice(thrust::seq);
}
DECLARE_UNITTEST(TestSetIntersectionDeviceSeq);


void TestSetIntersectionDeviceDevice()
{
  TestSetIntersectionDevice(thrust::device);
}
DECLARE_UNITTEST(TestSetIntersectionDeviceDevice);


void TestSetIntersectionDeviceNoSync()
{
  TestSetIntersectionDevice(thrust::cuda::par_nosync);
}
DECLARE_UNITTEST(TestSetIntersectionDeviceNoSync);


template<typename ExecutionPolicy>
void TestSetIntersectionCudaStreams(ExecutionPolicy policy)
{
  typedef thrust::device_vector<int> Vector;
  typedef Vector::iterator Iterator;

  Vector a(3), b(4);

  a[0] = 0; a[1] = 2; a[2] = 4;
  b[0] = 0; b[1] = 3; b[2] = 3; b[3] = 4;

  Vector ref(2);
  ref[0] = 0; ref[1] = 4;

  Vector result(2);

  hipStream_t s;
  hipStreamCreate(&s);

  auto streampolicy = policy.on(s);

  Iterator end = thrust::set_intersection(streampolicy,
                                          a.begin(), a.end(),
                                          b.begin(), b.end(),
                                          result.begin());
  hipStreamSynchronize(s);

  ASSERT_EQUAL_QUIET(result.end(), end);
  ASSERT_EQUAL(ref, result);

  hipStreamDestroy(s);
}

void TestSetIntersectionCudaStreamsSync()
{
  TestSetIntersectionCudaStreams(thrust::cuda::par);
}
DECLARE_UNITTEST(TestSetIntersectionCudaStreamsSync);


void TestSetIntersectionCudaStreamsNoSync()
{
  TestSetIntersectionCudaStreams(thrust::cuda::par_nosync);
}
DECLARE_UNITTEST(TestSetIntersectionCudaStreamsNoSync);

