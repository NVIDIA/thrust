#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/set_operations.h>
#include <thrust/functional.h>
#include <thrust/sort.h>
#include <thrust/iterator/discard_iterator.h>


template<typename InputIterator1,
         typename InputIterator2,
         typename OutputIterator>
OutputIterator set_intersection(my_system &system,
                                InputIterator1,
                                InputIterator1,
                                InputIterator2,
                                InputIterator2,
                                OutputIterator result)
{
  system.validate_dispatch();
  return result;
}

void TestSetIntersectionDispatchExplicit()
{
  thrust::device_vector<int> vec(1);

  my_system sys(0);
  thrust::set_intersection(sys,
                           vec.begin(),
                           vec.begin(),
                           vec.begin(),
                           vec.begin(),
                           vec.begin());

  ASSERT_EQUAL(true, sys.is_valid());
}
DECLARE_UNITTEST(TestSetIntersectionDispatchExplicit);


template<typename InputIterator1,
         typename InputIterator2,
         typename OutputIterator>
OutputIterator set_intersection(my_tag,
                                InputIterator1,
                                InputIterator1,
                                InputIterator2,
                                InputIterator2,
                                OutputIterator result)
{
  *result = 13;
  return result;
}

void TestSetIntersectionDispatchImplicit()
{
  thrust::device_vector<int> vec(1);

  thrust::set_intersection(thrust::retag<my_tag>(vec.begin()),
                           thrust::retag<my_tag>(vec.begin()),
                           thrust::retag<my_tag>(vec.begin()),
                           thrust::retag<my_tag>(vec.begin()),
                           thrust::retag<my_tag>(vec.begin()));

  ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestSetIntersectionDispatchImplicit);


template<typename Vector>
void TestSetIntersectionSimple(void)
{
  typedef typename Vector::iterator Iterator;

  Vector a(3), b(4);

  a[0] = 0; a[1] = 2; a[2] = 4;
  b[0] = 0; b[1] = 3; b[2] = 3; b[3] = 4;

  Vector ref(2);
  ref[0] = 0; ref[1] = 4;

  Vector result(2);

  Iterator end = thrust::set_intersection(a.begin(), a.end(),
                                          b.begin(), b.end(),
                                          result.begin());

  ASSERT_EQUAL_QUIET(result.end(), end);
  ASSERT_EQUAL(ref, result);
}
DECLARE_VECTOR_UNITTEST(TestSetIntersectionSimple);


template<typename T>
void TestSetIntersection(const size_t n)
{
  size_t sizes[]   = {0, 1, n / 2, n, n + 1, 2 * n};
  size_t num_sizes = sizeof(sizes) / sizeof(size_t);

  thrust::host_vector<T> random = unittest::random_integers<unittest::int8_t>(n + *thrust::max_element(sizes, sizes + num_sizes));

  thrust::host_vector<T> h_a(random.begin(), random.begin() + n);
  thrust::host_vector<T> h_b(random.begin() + n, random.end());

  thrust::stable_sort(h_a.begin(), h_a.end());
  thrust::stable_sort(h_b.begin(), h_b.end());
  
  thrust::device_vector<T> d_a = h_a;
  thrust::device_vector<T> d_b = h_b;

  for (size_t i = 0; i < num_sizes; i++)
  {
    size_t size = sizes[i];
    
    thrust::host_vector<T>   h_result(n + size);
    thrust::device_vector<T> d_result(n + size);

    typename thrust::host_vector<T>::iterator   h_end;
    typename thrust::device_vector<T>::iterator d_end;
    
    h_end = thrust::set_intersection(h_a.begin(), h_a.end(),
                                     h_b.begin(), h_b.begin() + size,
                                     h_result.begin());
    h_result.resize(h_end - h_result.begin());

    d_end = thrust::set_intersection(d_a.begin(), d_a.end(),
                                     d_b.begin(), d_b.begin() + size,
                                     d_result.begin());
    d_result.resize(d_end - d_result.begin());

    ASSERT_EQUAL(h_result, d_result);
  }
}
DECLARE_VARIABLE_UNITTEST(TestSetIntersection);


template<typename T>
void TestSetIntersectionToDiscardIterator(const size_t n)
{
  thrust::host_vector<T> temp = unittest::random_integers<T>(2 * n);
  thrust::host_vector<T> h_a(temp.begin(), temp.begin() + n);
  thrust::host_vector<T> h_b(temp.begin() + n, temp.end());

  thrust::sort(h_a.begin(), h_a.end());
  thrust::sort(h_b.begin(), h_b.end());

  thrust::device_vector<T> d_a = h_a;
  thrust::device_vector<T> d_b = h_b;

  thrust::discard_iterator<> h_result;
  thrust::discard_iterator<> d_result;

  thrust::host_vector<T> h_reference(n);
  typename thrust::host_vector<T>::iterator h_end = 
    thrust::set_intersection(h_a.begin(), h_a.end(),
                             h_b.begin(), h_b.end(),
                             h_reference.begin());
  h_reference.erase(h_end, h_reference.end());
  
  h_result = thrust::set_intersection(h_a.begin(), h_a.end(),
                                      h_b.begin(), h_b.end(),
                                      thrust::make_discard_iterator());

  d_result = thrust::set_intersection(d_a.begin(), d_a.end(),
                                      d_b.begin(), d_b.end(),
                                      thrust::make_discard_iterator());

  thrust::discard_iterator<> reference(h_reference.size());

  ASSERT_EQUAL_QUIET(reference, h_result);
  ASSERT_EQUAL_QUIET(reference, d_result);
}
DECLARE_VARIABLE_UNITTEST(TestSetIntersectionToDiscardIterator);


template<typename T>
void TestSetIntersectionEquivalentRanges(const size_t n)
{
  thrust::host_vector<T> temp = unittest::random_integers<T>(n);
  thrust::host_vector<T> h_a = temp; thrust::sort(h_a.begin(), h_a.end());
  thrust::host_vector<T> h_b = h_a;

  thrust::device_vector<T> d_a = h_a;
  thrust::device_vector<T> d_b = h_b;

  thrust::host_vector<T>   h_result(n);
  thrust::device_vector<T> d_result(n);

  typename thrust::host_vector<T>::iterator   h_end;
  typename thrust::device_vector<T>::iterator d_end;
  
  h_end = thrust::set_intersection(h_a.begin(), h_a.end(),
                                   h_b.begin(), h_b.end(),
                                   h_result.begin());
  h_result.resize(h_end - h_result.begin());

  d_end = thrust::set_intersection(d_a.begin(), d_a.end(),
                                   d_b.begin(), d_b.end(),
                                   d_result.begin());

  d_result.resize(d_end - d_result.begin());

  ASSERT_EQUAL(h_result, d_result);
}
DECLARE_VARIABLE_UNITTEST(TestSetIntersectionEquivalentRanges);


template<typename T>
void TestSetIntersectionMultiset(const size_t n)
{
  thrust::host_vector<T> temp = unittest::random_integers<T>(2 * n);

  // restrict elements to [min,13)
  for(typename thrust::host_vector<T>::iterator i = temp.begin();
      i != temp.end();
      ++i)
  {
    int temp = static_cast<int>(*i);
    temp %= 13;
    *i = temp;
  }

  thrust::host_vector<T> h_a(temp.begin(), temp.begin() + n);
  thrust::host_vector<T> h_b(temp.begin() + n, temp.end());

  thrust::sort(h_a.begin(), h_a.end());
  thrust::sort(h_b.begin(), h_b.end());

  thrust::device_vector<T> d_a = h_a;
  thrust::device_vector<T> d_b = h_b;

  thrust::host_vector<T> h_result(n);
  thrust::device_vector<T> d_result(n);

  typename thrust::host_vector<T>::iterator h_end;
  typename thrust::device_vector<T>::iterator d_end;
  
  h_end = thrust::set_intersection(h_a.begin(), h_a.end(),
                                   h_b.begin(), h_b.end(),
                                   h_result.begin());
  h_result.resize(h_end - h_result.begin());

  d_end = thrust::set_intersection(d_a.begin(), d_a.end(),
                                   d_b.begin(), d_b.end(),
                                   d_result.begin());

  d_result.resize(d_end - d_result.begin());

  ASSERT_EQUAL(h_result, d_result);
}
DECLARE_VARIABLE_UNITTEST(TestSetIntersectionMultiset);

