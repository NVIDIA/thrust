#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/host_vector.h>
#include <thrust/complex.h>
#include <thrust/transform.h>
#include <iostream>

#if THRUST_DEVICE_SYSTEM == THRUST_DEVICE_SYSTEM_CUDA
#include <unittest/cuda/testframework.h>
#endif

struct basic_arithmetic_functor
{
  template<typename T>
  __host__ __device__
  thrust::complex<T> operator()(const thrust::complex<T> &x,
				const thrust::complex<T> &y)
  {
    // exercise unary and binary arithmetic operators
    // Should return approximately 1
    return (+x + +y) + (x * y) / (y * x) + (-y + -x);
  } // end operator()()
}; // end make_pair_functor

struct complex_plane_functor
{
  template<typename T>
  __host__ __device__
  thrust::complex<T> operator()(const thrust::complex<T> &x)
  {
    // Should return a proximately 1
    return thrust::proj( (thrust::polar(abs(x),arg(x)) * conj(x))/norm(x));
  } // end operator()()
}; // end make_pair_functor

struct pow_functor
{
  template<typename T>
  __host__ __device__
  thrust::complex<T> operator()(const thrust::complex<T> &x,
				const thrust::complex<T> &y)
  {
    // exercise power functions
    return pow(x,y);
  } // end operator()()
}; // end make_pair_functor

struct sqrt_functor
{
  template<typename T>
  __host__ __device__
  thrust::complex<T> operator()(const thrust::complex<T> &x)
  {
    // exercise power functions
    return sqrt(x);
  } // end operator()()
}; // end make_pair_functor

struct log_functor
{
  template<typename T>
  __host__ __device__
  thrust::complex<T> operator()(const thrust::complex<T> &x)
  {
    return log(x);
  } // end operator()()
}; // end make_pair_functor

struct exp_functor
{
  template<typename T>
  __host__ __device__
  thrust::complex<T> operator()(const thrust::complex<T> &x)
  {
    return exp(x);
  } // end operator()()
}; // end make_pair_functor

struct log10_functor
{
  template<typename T>
  __host__ __device__
  thrust::complex<T> operator()(const thrust::complex<T> &x)
  {
    return log10(x);
  } // end operator()()
}; // end make_pair_functor


struct cos_functor
{
  template<typename T>
  __host__ __device__
  thrust::complex<T> operator()(const thrust::complex<T> &x)
  {
    return cos(x);
  } 
}; 

struct sin_functor
{
  template<typename T>
  __host__ __device__
  thrust::complex<T> operator()(const thrust::complex<T> &x)
  {
    return sin(x);
  } 
}; 

struct tan_functor
{
  template<typename T>
  __host__ __device__
  thrust::complex<T> operator()(const thrust::complex<T> &x)
  {
    return tan(x);
  } 
}; 



struct cosh_functor
{
  template<typename T>
  __host__ __device__
  thrust::complex<T> operator()(const thrust::complex<T> &x)
  {
    return cosh(x);
  } 
}; 

struct sinh_functor
{
  template<typename T>
  __host__ __device__
  thrust::complex<T> operator()(const thrust::complex<T> &x)
  {
    return sinh(x);
  } 
}; 

struct tanh_functor
{
  template<typename T>
  __host__ __device__
  thrust::complex<T> operator()(const thrust::complex<T> &x)
  {
    return tanh(x);
  } 
}; 


struct acos_functor
{
  template<typename T>
  __host__ __device__
  thrust::complex<T> operator()(const thrust::complex<T> &x)
  {
    return acos(x);
  } 
}; 

struct asin_functor
{
  template<typename T>
  __host__ __device__
  thrust::complex<T> operator()(const thrust::complex<T> &x)
  {
    return asin(x);
  } 
}; 

struct atan_functor
{
  template<typename T>
  __host__ __device__
  thrust::complex<T> operator()(const thrust::complex<T> &x)
  {
    return atan(x);
  } 
}; 


struct acosh_functor
{
  template<typename T>
  __host__ __device__
  thrust::complex<T> operator()(const thrust::complex<T> &x)
  {
    return acosh(x);
  } 
}; 

struct asinh_functor
{
  template<typename T>
  __host__ __device__
  thrust::complex<T> operator()(const thrust::complex<T> &x)
  {
    return asinh(x);
  } 
}; 

struct atanh_functor
{
  template<typename T>
  __host__ __device__
  thrust::complex<T> operator()(const thrust::complex<T> &x)
  {
    return atanh(x);
  } 
}; 


template <typename T>
thrust::host_vector<thrust::complex<T> > random_complex_samples(size_t n){
  thrust::host_vector<T> real = unittest::random_samples<T>(2*n);
  thrust::host_vector<thrust::complex<T> > h_p1(n);
  for(size_t i = 0; i<n; i++){
    h_p1[i].real(real[i]);
    h_p1[i].imag(real[2*i]);
  }
  return h_p1;
}

template <typename T>
struct TestComplexArithmeticTransform
{
  void operator()(const size_t n)
  {
    typedef thrust::complex<T> type;
    thrust::host_vector<type> h_p1 = random_complex_samples<T>(n);
    thrust::host_vector<type> h_p2 = random_complex_samples<T>(n);
    thrust::host_vector<type>   h_result(n);

    thrust::device_vector<type> d_p1 = h_p1;
    thrust::device_vector<type> d_p2 = h_p2;
    thrust::device_vector<type> d_result(n);

    thrust::transform(h_p1.begin(), h_p1.end(), h_p2.begin(), h_result.begin(), basic_arithmetic_functor());
    thrust::transform(d_p1.begin(), d_p1.end(), d_p2.begin(), d_result.begin(), basic_arithmetic_functor());    
    ASSERT_ALMOST_EQUAL(h_result, d_result);
  }
};
VariableUnitTest<TestComplexArithmeticTransform, FloatingPointTypes> TestComplexArithmeticTransformInstance;

template <typename T>
struct TestComplexPlaneTransform
{
  void operator()(const size_t n)
  {
    typedef thrust::complex<T> type;
    thrust::host_vector<type> h_p1 = random_complex_samples<T>(n);
    thrust::host_vector<type>   h_result(n);

    thrust::device_vector<type> d_p1 = h_p1;
    thrust::device_vector<type> d_result(n);

    thrust::transform(h_p1.begin(), h_p1.end(), h_result.begin(), complex_plane_functor());
    thrust::transform(d_p1.begin(), d_p1.end(), d_result.begin(), complex_plane_functor());    
    ASSERT_ALMOST_EQUAL(h_result, d_result);
  }
};
VariableUnitTest<TestComplexPlaneTransform, FloatingPointTypes> TestComplexPlaneTransformInstance;


template <typename T>
struct TestComplexPowerTransform
{
  void operator()(const size_t n)
  {
    typedef thrust::complex<T> type;
    thrust::host_vector<type> h_p1 = random_complex_samples<T>(n);
    thrust::host_vector<type> h_p2 = random_complex_samples<T>(n);
    thrust::host_vector<type>   h_result(n);

    thrust::device_vector<type> d_p1 = h_p1;
    thrust::device_vector<type> d_p2 = h_p2;
    thrust::device_vector<type> d_result(n);

    thrust::transform(h_p1.begin(), h_p1.end(), h_p2.begin(), h_result.begin(), pow_functor());
    thrust::transform(d_p1.begin(), d_p1.end(), d_p2.begin(), d_result.begin(), pow_functor());    
    // pow can be very innacurate there's no point trying to check for equality
    // Currently just checking for compilation
    //    ASSERT_ALMOST_EQUAL(h_result, d_result);

    thrust::transform(h_p1.begin(), h_p1.end(), h_result.begin(), sqrt_functor());
    thrust::transform(d_p1.begin(), d_p1.end(), d_result.begin(), sqrt_functor());    
    ASSERT_ALMOST_EQUAL(h_result, d_result);
  }
};
VariableUnitTest<TestComplexPowerTransform, FloatingPointTypes> TestComplexPowerTransformInstance;

template <typename T>
struct TestComplexExponentialTransform
{
  void operator()(const size_t n)
  {
    typedef thrust::complex<T> type;
    thrust::host_vector<type> h_p1 = random_complex_samples<T>(n);
    thrust::host_vector<type>   h_result(n);

    thrust::device_vector<type> d_p1 = h_p1;
    thrust::device_vector<type> d_result(n);

    thrust::transform(h_p1.begin(), h_p1.end(), h_result.begin(), exp_functor());
    thrust::transform(d_p1.begin(), d_p1.end(), d_result.begin(), exp_functor());    
    ASSERT_ALMOST_EQUAL(h_result, d_result);

    thrust::transform(h_p1.begin(), h_p1.end(), h_result.begin(), log_functor());
    thrust::transform(d_p1.begin(), d_p1.end(), d_result.begin(), log_functor());    
    ASSERT_ALMOST_EQUAL(h_result, d_result);

    thrust::transform(h_p1.begin(), h_p1.end(), h_result.begin(), log10_functor());
    thrust::transform(d_p1.begin(), d_p1.end(), d_result.begin(), log10_functor());    
    ASSERT_ALMOST_EQUAL(h_result, d_result);
  }
};
VariableUnitTest<TestComplexExponentialTransform, FloatingPointTypes> TestComplexExponentialTransformInstance;

template <typename T>
struct TestComplexTrigonometricTransform
{
  void operator()(const size_t n)
  {
    typedef thrust::complex<T> type;
    thrust::host_vector<type> h_p1 = random_complex_samples<T>(n);
    thrust::host_vector<type>   h_result(n);

    thrust::device_vector<type> d_p1 = h_p1;
    thrust::device_vector<type> d_result(n);


    thrust::transform(h_p1.begin(), h_p1.end(), h_result.begin(), sin_functor());
    thrust::transform(d_p1.begin(), d_p1.end(), d_result.begin(), sin_functor());    
    ASSERT_ALMOST_EQUAL(h_result, d_result);

    thrust::transform(h_p1.begin(), h_p1.end(), h_result.begin(), cos_functor());
    thrust::transform(d_p1.begin(), d_p1.end(), d_result.begin(), cos_functor());    
    ASSERT_ALMOST_EQUAL(h_result, d_result);

    thrust::transform(h_p1.begin(), h_p1.end(), h_result.begin(), tan_functor());
    thrust::transform(d_p1.begin(), d_p1.end(), d_result.begin(), tan_functor());    
    ASSERT_ALMOST_EQUAL(h_result, d_result);


    thrust::transform(h_p1.begin(), h_p1.end(), h_result.begin(), sinh_functor());
    thrust::transform(d_p1.begin(), d_p1.end(), d_result.begin(), sinh_functor());    
    ASSERT_ALMOST_EQUAL(h_result, d_result);

    thrust::transform(h_p1.begin(), h_p1.end(), h_result.begin(), cosh_functor());
    thrust::transform(d_p1.begin(), d_p1.end(), d_result.begin(), cosh_functor());    
    ASSERT_ALMOST_EQUAL(h_result, d_result);

    thrust::transform(h_p1.begin(), h_p1.end(), h_result.begin(), tanh_functor());
    thrust::transform(d_p1.begin(), d_p1.end(), d_result.begin(), tanh_functor());    
    ASSERT_ALMOST_EQUAL(h_result, d_result);


    thrust::transform(h_p1.begin(), h_p1.end(), h_result.begin(), asin_functor());
    thrust::transform(d_p1.begin(), d_p1.end(), d_result.begin(), asin_functor());    
    ASSERT_ALMOST_EQUAL(h_result, d_result);

    thrust::transform(h_p1.begin(), h_p1.end(), h_result.begin(), acos_functor());
    thrust::transform(d_p1.begin(), d_p1.end(), d_result.begin(), acos_functor());    
    ASSERT_ALMOST_EQUAL(h_result, d_result);

    thrust::transform(h_p1.begin(), h_p1.end(), h_result.begin(), atan_functor());
    thrust::transform(d_p1.begin(), d_p1.end(), d_result.begin(), atan_functor());    
    ASSERT_ALMOST_EQUAL(h_result, d_result);


    thrust::transform(h_p1.begin(), h_p1.end(), h_result.begin(), asinh_functor());
    thrust::transform(d_p1.begin(), d_p1.end(), d_result.begin(), asinh_functor());    
    ASSERT_ALMOST_EQUAL(h_result, d_result);

    thrust::transform(h_p1.begin(), h_p1.end(), h_result.begin(), acosh_functor());
    thrust::transform(d_p1.begin(), d_p1.end(), d_result.begin(), acosh_functor());    
    ASSERT_ALMOST_EQUAL(h_result, d_result);

    thrust::transform(h_p1.begin(), h_p1.end(), h_result.begin(), atanh_functor());
    thrust::transform(d_p1.begin(), d_p1.end(), d_result.begin(), atanh_functor());    
    ASSERT_ALMOST_EQUAL(h_result, d_result);

  }
};
VariableUnitTest<TestComplexTrigonometricTransform, FloatingPointTypes> TestComplexTrigonometricTransformInstance;

