#include <unittest/unittest.h>
#include <thrust/replace.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/retag.h>


template <class Vector>
void TestReplaceSimple(void)
{
    typedef typename Vector::value_type T;

    Vector data(5);
    data[0] =  1; 
    data[1] =  2; 
    data[2] =  1;
    data[3] =  3; 
    data[4] =  2; 

    thrust::replace(data.begin(), data.end(), (T) 1, (T) 4);
    thrust::replace(data.begin(), data.end(), (T) 2, (T) 5);

    Vector result(5);
    result[0] =  4; 
    result[1] =  5; 
    result[2] =  4;
    result[3] =  3; 
    result[4] =  5; 

    ASSERT_EQUAL(data, result);
}
DECLARE_VECTOR_UNITTEST(TestReplaceSimple);


template<typename ForwardIterator, typename T>
void replace(my_system &system,
             ForwardIterator, ForwardIterator, const T &,
             const T &)
{
    system.validate_dispatch();
}

void TestReplaceDispatchExplicit()
{
    thrust::device_vector<int> vec(1);

    my_system sys(0);
    thrust::replace(sys,
                    vec.begin(),
                    vec.begin(),
                    0,
                    0);

    ASSERT_EQUAL(true, sys.is_valid());
}
DECLARE_UNITTEST(TestReplaceDispatchExplicit);


template<typename ForwardIterator, typename T>
void replace(my_tag,
             ForwardIterator first, ForwardIterator, const T &,
             const T &)
{
    *first = 13;
}

void TestReplaceDispatchImplicit()
{
    thrust::device_vector<int> vec(1);

    thrust::replace(thrust::retag<my_tag>(vec.begin()),
                    thrust::retag<my_tag>(vec.begin()),
                    0,
                    0);

    ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestReplaceDispatchImplicit);


template <typename T>
void TestReplace(const size_t n)
{
    thrust::host_vector<T>   h_data = unittest::random_samples<T>(n);
    thrust::device_vector<T> d_data = h_data;

    T old_value = 0;
    T new_value = 1;

    thrust::replace(h_data.begin(), h_data.end(), old_value, new_value);
    thrust::replace(d_data.begin(), d_data.end(), old_value, new_value);

    ASSERT_ALMOST_EQUAL(h_data, d_data);
}
DECLARE_VARIABLE_UNITTEST(TestReplace);


template <class Vector>
void TestReplaceCopySimple(void)
{
    typedef typename Vector::value_type T;

    Vector data(5);
    data[0] = 1; 
    data[1] = 2; 
    data[2] = 1;
    data[3] = 3; 
    data[4] = 2; 

    Vector dest(5);

    thrust::replace_copy(data.begin(), data.end(), dest.begin(), (T) 1, (T) 4);
    thrust::replace_copy(dest.begin(), dest.end(), dest.begin(), (T) 2, (T) 5);

    Vector result(5);
    result[0] = 4; 
    result[1] = 5; 
    result[2] = 4;
    result[3] = 3; 
    result[4] = 5; 

    ASSERT_EQUAL(dest, result);
}
DECLARE_VECTOR_UNITTEST(TestReplaceCopySimple);


template<typename InputIterator, typename OutputIterator, typename T>
OutputIterator replace_copy(my_system &system,
                            InputIterator, InputIterator,
                            OutputIterator result,
                            const T &,
                            const T &)
{
    system.validate_dispatch();
    return result;
}

void TestReplaceCopyDispatchExplicit()
{
    thrust::device_vector<int> vec(1);

    my_system sys(0);
    thrust::replace_copy(sys,
                         vec.begin(),
                         vec.begin(),
                         vec.begin(),
                         0,
                         0);

    ASSERT_EQUAL(true, sys.is_valid());
}
DECLARE_UNITTEST(TestReplaceCopyDispatchExplicit);


template<typename InputIterator, typename OutputIterator, typename T>
OutputIterator replace_copy(my_tag,
                            InputIterator, InputIterator,
                            OutputIterator result,
                            const T &,
                            const T &)
{
    *result = 13;
    return result;
}

void TestReplaceCopyDispatchImplicit()
{
    thrust::device_vector<int> vec(1);

    thrust::replace_copy(thrust::retag<my_tag>(vec.begin()),
                         thrust::retag<my_tag>(vec.begin()),
                         thrust::retag<my_tag>(vec.begin()),
                         0,
                         0);

    ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestReplaceCopyDispatchImplicit);


template <typename T>
void TestReplaceCopy(const size_t n)
{
    thrust::host_vector<T>   h_data = unittest::random_samples<T>(n);
    thrust::device_vector<T> d_data = h_data;
    
    T old_value = 0;
    T new_value = 1;
    
    thrust::host_vector<T>   h_dest(n);
    thrust::device_vector<T> d_dest(n);

    thrust::replace_copy(h_data.begin(), h_data.end(), h_dest.begin(), old_value, new_value);
    thrust::replace_copy(d_data.begin(), d_data.end(), d_dest.begin(), old_value, new_value);

    ASSERT_ALMOST_EQUAL(h_data, d_data);
    ASSERT_ALMOST_EQUAL(h_dest, d_dest);
}
DECLARE_VARIABLE_UNITTEST(TestReplaceCopy);


template <typename T>
void TestReplaceCopyToDiscardIterator(const size_t n)
{
    thrust::host_vector<T>   h_data = unittest::random_samples<T>(n);
    thrust::device_vector<T> d_data = h_data;
    
    T old_value = 0;
    T new_value = 1;

    thrust::discard_iterator<> h_result =
      thrust::replace_copy(h_data.begin(), h_data.end(), thrust::make_discard_iterator(), old_value, new_value);

    thrust::discard_iterator<> d_result =
      thrust::replace_copy(d_data.begin(), d_data.end(), thrust::make_discard_iterator(), old_value, new_value);

    thrust::discard_iterator<> reference(n);

    ASSERT_EQUAL_QUIET(reference, h_result);
    ASSERT_EQUAL_QUIET(reference, d_result);
}
DECLARE_VARIABLE_UNITTEST(TestReplaceCopyToDiscardIterator);



template <typename T>
struct less_than_five
{
  __host__ __device__ bool operator()(const T &val) const {return val < 5;}
};

template <class Vector>
void TestReplaceIfSimple(void)
{
    typedef typename Vector::value_type T;

    Vector data(5);
    data[0] =  1; 
    data[1] =  3; 
    data[2] =  4;
    data[3] =  6; 
    data[4] =  5; 

    thrust::replace_if(data.begin(), data.end(), less_than_five<T>(), (T) 0);

    Vector result(5);
    result[0] =  0; 
    result[1] =  0; 
    result[2] =  0;
    result[3] =  6; 
    result[4] =  5; 

    ASSERT_EQUAL(data, result);
}
DECLARE_VECTOR_UNITTEST(TestReplaceIfSimple);


template<typename ForwardIterator, typename Predicate, typename T>
void replace_if(my_system &system,
                ForwardIterator, ForwardIterator,
                Predicate,
                const T &)
{
    system.validate_dispatch();
}

void TestReplaceIfDispatchExplicit()
{
    thrust::device_vector<int> vec(1);

    my_system sys(0);
    thrust::replace_if(sys,
                       vec.begin(),
                       vec.begin(),
                       0,
                       0);

    ASSERT_EQUAL(true, sys.is_valid());
}
DECLARE_UNITTEST(TestReplaceIfDispatchExplicit);


template<typename ForwardIterator, typename Predicate, typename T>
void replace_if(my_tag,
                ForwardIterator first, ForwardIterator,
                Predicate,
                const T &)
{
    *first = 13;
}

void TestReplaceIfDispatchImplicit()
{
    thrust::device_vector<int> vec(1);

    thrust::replace_if(thrust::retag<my_tag>(vec.begin()),
                       thrust::retag<my_tag>(vec.begin()),
                       0,
                       0);

    ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestReplaceIfDispatchImplicit);


template <class Vector>
void TestReplaceIfStencilSimple(void)
{
    typedef typename Vector::value_type T;

    Vector data(5);
    data[0] =  1; 
    data[1] =  3; 
    data[2] =  4;
    data[3] =  6; 
    data[4] =  5; 

    Vector stencil(5);
    stencil[0] = 5;
    stencil[1] = 4;
    stencil[2] = 6;
    stencil[3] = 3;
    stencil[4] = 7;

    thrust::replace_if(data.begin(), data.end(), stencil.begin(), less_than_five<T>(), (T) 0);

    Vector result(5);
    result[0] =  1; 
    result[1] =  0; 
    result[2] =  4;
    result[3] =  0; 
    result[4] =  5; 

    ASSERT_EQUAL(data, result);
}
DECLARE_VECTOR_UNITTEST(TestReplaceIfStencilSimple);


template<typename ForwardIterator, typename InputIterator, typename Predicate, typename T>
void replace_if(my_system &system,
                ForwardIterator, ForwardIterator,
                InputIterator,
                Predicate,
                const T &)
{
    system.validate_dispatch();
}

void TestReplaceIfStencilDispatchExplicit()
{
    thrust::device_vector<int> vec(1);

    my_system sys(0);
    thrust::replace_if(sys,
                       vec.begin(),
                       vec.begin(),
                       vec.begin(),
                       0,
                       0);

    ASSERT_EQUAL(true, sys.is_valid());
}
DECLARE_UNITTEST(TestReplaceIfStencilDispatchExplicit);


template<typename ForwardIterator, typename InputIterator, typename Predicate, typename T>
void replace_if(my_tag,
                ForwardIterator first, ForwardIterator,
                InputIterator,
                Predicate,
                const T &)
{
    *first = 13;
}

void TestReplaceIfStencilDispatchImplicit()
{
    thrust::device_vector<int> vec(1);

    thrust::replace_if(thrust::retag<my_tag>(vec.begin()),
                       thrust::retag<my_tag>(vec.begin()),
                       thrust::retag<my_tag>(vec.begin()),
                       0,
                       0);

    ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestReplaceIfStencilDispatchImplicit);


template <typename T>
void TestReplaceIf(const size_t n)
{
    thrust::host_vector<T>   h_data = unittest::random_samples<T>(n);
    thrust::device_vector<T> d_data = h_data;

    thrust::replace_if(h_data.begin(), h_data.end(), less_than_five<T>(), (T) 0);
    thrust::replace_if(d_data.begin(), d_data.end(), less_than_five<T>(), (T) 0);

    ASSERT_ALMOST_EQUAL(h_data, d_data);
}
DECLARE_VARIABLE_UNITTEST(TestReplaceIf);


template <typename T>
void TestReplaceIfStencil(const size_t n)
{
    thrust::host_vector<T>   h_data = unittest::random_samples<T>(n);
    thrust::device_vector<T> d_data = h_data;

    thrust::host_vector<T>   h_stencil = unittest::random_samples<T>(n);
    thrust::device_vector<T> d_stencil = h_stencil;

    thrust::replace_if(h_data.begin(), h_data.end(), h_stencil.begin(), less_than_five<T>(), (T) 0);
    thrust::replace_if(d_data.begin(), d_data.end(), d_stencil.begin(), less_than_five<T>(), (T) 0);

    ASSERT_ALMOST_EQUAL(h_data, d_data);
}
DECLARE_VARIABLE_UNITTEST(TestReplaceIfStencil);


template <class Vector>
void TestReplaceCopyIfSimple(void)
{
    typedef typename Vector::value_type T;
    
    Vector data(5);
    data[0] =  1; 
    data[1] =  3; 
    data[2] =  4;
    data[3] =  6; 
    data[4] =  5; 

    Vector dest(5);

    thrust::replace_copy_if(data.begin(), data.end(), dest.begin(), less_than_five<T>(), (T) 0);

    Vector result(5);
    result[0] =  0; 
    result[1] =  0; 
    result[2] =  0;
    result[3] =  6; 
    result[4] =  5; 

    ASSERT_EQUAL(dest, result);
}
DECLARE_VECTOR_UNITTEST(TestReplaceCopyIfSimple);


template<typename InputIterator, typename OutputIterator, typename Predicate, typename T>
OutputIterator replace_copy_if(my_system &system,
                               InputIterator, InputIterator,
                               OutputIterator result,
                               Predicate,
                               const T &)
{
    system.validate_dispatch();
    return result;
}

void TestReplaceCopyIfDispatchExplicit()
{
    thrust::device_vector<int> vec(1);

    my_system sys(0);
    thrust::replace_copy_if(sys,
                            vec.begin(),
                            vec.begin(),
                            vec.begin(),
                            0,
                            0);

    ASSERT_EQUAL(true, sys.is_valid());
}
DECLARE_UNITTEST(TestReplaceCopyIfDispatchExplicit);


template<typename InputIterator, typename OutputIterator, typename Predicate, typename T>
OutputIterator replace_copy_if(my_tag,
                               InputIterator, InputIterator,
                               OutputIterator result,
                               Predicate,
                               const T &)
{
    *result = 13;
    return result;
}

void TestReplaceCopyIfDispatchImplicit()
{
    thrust::device_vector<int> vec(1);

    thrust::replace_copy_if(thrust::retag<my_tag>(vec.begin()),
                            thrust::retag<my_tag>(vec.begin()),
                            thrust::retag<my_tag>(vec.begin()),
                            0,
                            0);

    ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestReplaceCopyIfDispatchImplicit);


template <class Vector>
void TestReplaceCopyIfStencilSimple(void)
{
    typedef typename Vector::value_type T;
    
    Vector data(5);
    data[0] =  1; 
    data[1] =  3; 
    data[2] =  4;
    data[3] =  6; 
    data[4] =  5; 

    Vector stencil(5);
    stencil[0] = 1;
    stencil[1] = 5;
    stencil[2] = 4;
    stencil[3] = 7;
    stencil[4] = 8;

    Vector dest(5);

    thrust::replace_copy_if(data.begin(), data.end(), stencil.begin(), dest.begin(), less_than_five<T>(), (T) 0);

    Vector result(5);
    result[0] =  0; 
    result[1] =  3; 
    result[2] =  0;
    result[3] =  6; 
    result[4] =  5; 

    ASSERT_EQUAL(dest, result);
}
DECLARE_VECTOR_UNITTEST(TestReplaceCopyIfStencilSimple);


template<typename InputIterator1, typename InputIterator2, typename OutputIterator, typename Predicate, typename T>
OutputIterator replace_copy_if(my_system &system,
                               InputIterator1, InputIterator1,
                               InputIterator2,
                               OutputIterator result,
                               Predicate,
                               const T &)
{
    system.validate_dispatch();
    return result;
}


void TestReplaceCopyIfStencilDispatchExplicit()
{
    thrust::device_vector<int> vec(1);

    my_system sys(0);
    thrust::replace_copy_if(sys,
                            vec.begin(),
                            vec.begin(),
                            vec.begin(),
                            vec.begin(),
                            0,
                            0);

    ASSERT_EQUAL(true, sys.is_valid());
}
DECLARE_UNITTEST(TestReplaceCopyIfStencilDispatchExplicit);


template<typename InputIterator1, typename InputIterator2, typename OutputIterator, typename Predicate, typename T>
OutputIterator replace_copy_if(my_tag,
                               InputIterator1, InputIterator1,
                               InputIterator2,
                               OutputIterator result,
                               Predicate,
                               const T &)
{
    *result = 13;
    return result;
}

void TestReplaceCopyIfStencilDispatchImplicit()
{
    thrust::device_vector<int> vec(1);

    thrust::replace_copy_if(thrust::retag<my_tag>(vec.begin()),
                            thrust::retag<my_tag>(vec.begin()),
                            thrust::retag<my_tag>(vec.begin()),
                            thrust::retag<my_tag>(vec.begin()),
                            0,
                            0);

    ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestReplaceCopyIfStencilDispatchImplicit);


template <typename T>
void TestReplaceCopyIf(const size_t n)
{
    thrust::host_vector<T>   h_data = unittest::random_samples<T>(n);
    thrust::device_vector<T> d_data = h_data;

    thrust::host_vector<T>   h_dest(n);
    thrust::device_vector<T> d_dest(n);

    thrust::replace_copy_if(h_data.begin(), h_data.end(), h_dest.begin(), less_than_five<T>(), 0);
    thrust::replace_copy_if(d_data.begin(), d_data.end(), d_dest.begin(), less_than_five<T>(), 0);

    ASSERT_ALMOST_EQUAL(h_data, d_data);
    ASSERT_ALMOST_EQUAL(h_dest, d_dest);
}
DECLARE_VARIABLE_UNITTEST(TestReplaceCopyIf);


template <typename T>
void TestReplaceCopyIfToDiscardIterator(const size_t n)
{
    thrust::host_vector<T>   h_data = unittest::random_samples<T>(n);
    thrust::device_vector<T> d_data = h_data;

    thrust::discard_iterator<> h_result =
      thrust::replace_copy_if(h_data.begin(), h_data.end(), thrust::make_discard_iterator(), less_than_five<T>(), 0);

    thrust::discard_iterator<> d_result =
      thrust::replace_copy_if(d_data.begin(), d_data.end(), thrust::make_discard_iterator(), less_than_five<T>(), 0);

    thrust::discard_iterator<> reference(n);

    ASSERT_EQUAL_QUIET(reference, h_result);
    ASSERT_EQUAL_QUIET(reference, d_result);
}
DECLARE_VARIABLE_UNITTEST(TestReplaceCopyIfToDiscardIterator);

template <typename T>
void TestReplaceCopyIfStencil(const size_t n)
{
    thrust::host_vector<T>   h_data = unittest::random_samples<T>(n);
    thrust::device_vector<T> d_data = h_data;

    thrust::host_vector<T>   h_stencil = unittest::random_samples<T>(n);
    thrust::device_vector<T> d_stencil = h_stencil;

    thrust::host_vector<T>   h_dest(n);
    thrust::device_vector<T> d_dest(n);

    thrust::replace_copy_if(h_data.begin(), h_data.end(), h_stencil.begin(), h_dest.begin(), less_than_five<T>(), 0);
    thrust::replace_copy_if(d_data.begin(), d_data.end(), d_stencil.begin(), d_dest.begin(), less_than_five<T>(), 0);

    ASSERT_ALMOST_EQUAL(h_data, d_data);
    ASSERT_ALMOST_EQUAL(h_dest, d_dest);
}
DECLARE_VARIABLE_UNITTEST(TestReplaceCopyIfStencil);

template <typename T>
void TestReplaceCopyIfStencilToDiscardIterator(const size_t n)
{
    thrust::host_vector<T>   h_data = unittest::random_samples<T>(n);
    thrust::device_vector<T> d_data = h_data;

    thrust::host_vector<T>   h_stencil = unittest::random_samples<T>(n);
    thrust::device_vector<T> d_stencil = h_stencil;

    thrust::discard_iterator<> h_result =
      thrust::replace_copy_if(h_data.begin(), h_data.end(), h_stencil.begin(), thrust::make_discard_iterator(), less_than_five<T>(), 0);

    thrust::discard_iterator<> d_result =
      thrust::replace_copy_if(d_data.begin(), d_data.end(), d_stencil.begin(), thrust::make_discard_iterator(), less_than_five<T>(), 0);

    thrust::discard_iterator<> reference(n);

    ASSERT_EQUAL_QUIET(reference, h_result);
    ASSERT_EQUAL_QUIET(reference, d_result);
}
DECLARE_VARIABLE_UNITTEST(TestReplaceCopyIfStencilToDiscardIterator);

