#include <unittest/unittest.h>
#include <thrust/sort.h>

template <class Vector>
void TestIsSortedSimple(void)
{
    typedef typename Vector::value_type T;

    Vector v(4);
    v[0] = 0; v[1] = 5; v[2] = 8; v[3] = 0;

    ASSERT_EQUAL(thrust::is_sorted(v.begin(), v.begin() + 0), true);
    ASSERT_EQUAL(thrust::is_sorted(v.begin(), v.begin() + 1), true);

    // the following line crashes gcc 4.3
#if (__GNUC__ == 4) && (__GNUC_MINOR__ == 3)
    // do nothing
#else
    // compile this line on other compilers
    ASSERT_EQUAL(thrust::is_sorted(v.begin(), v.begin() + 2), true);
#endif // GCC

    ASSERT_EQUAL(thrust::is_sorted(v.begin(), v.begin() + 3), true);
    ASSERT_EQUAL(thrust::is_sorted(v.begin(), v.begin() + 4), false);

    ASSERT_EQUAL(thrust::is_sorted(v.begin(), v.begin() + 3, thrust::less<T>()),    true);

    ASSERT_EQUAL(thrust::is_sorted(v.begin(), v.begin() + 1, thrust::greater<T>()), true);
    ASSERT_EQUAL(thrust::is_sorted(v.begin(), v.begin() + 4, thrust::greater<T>()), false);

    ASSERT_EQUAL(thrust::is_sorted(v.begin(), v.end()), false);
}
DECLARE_VECTOR_UNITTEST(TestIsSortedSimple);

template <class Vector>
void TestIsSortedRepeatedElements(void)
{
  Vector v(10);

  v[0] = 0;
  v[1] = 1;
  v[2] = 1;
  v[3] = 2;
  v[4] = 3;
  v[5] = 4;
  v[6] = 5;
  v[7] = 5;
  v[8] = 5;
  v[9] = 6;

  ASSERT_EQUAL(true, thrust::is_sorted(v.begin(), v.end()));
}
DECLARE_VECTOR_UNITTEST(TestIsSortedRepeatedElements);


template <class Vector>
void TestIsSorted(void)
{
    typedef typename Vector::value_type T;

    const size_t n = (1 << 16) + 13;

    Vector v = unittest::random_integers<T>(n);

    v[0] = 1;
    v[1] = 0;

    ASSERT_EQUAL(thrust::is_sorted(v.begin(), v.end()), false);

    thrust::sort(v.begin(), v.end());

    ASSERT_EQUAL(thrust::is_sorted(v.begin(), v.end()), true);
}
DECLARE_VECTOR_UNITTEST(TestIsSorted);

struct my_system : thrust::device_system<my_system> {};

template<typename InputIterator>
bool is_sorted(my_system, InputIterator first, InputIterator)
{
  *first = 13;
  return false;
}

void TestIsSortedDispatchExplicit()
{
  thrust::device_vector<int> vec(1);

  my_system sys;
  thrust::is_sorted(sys,
                    vec.begin(),
                    vec.end());

  ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestIsSortedDispatchExplicit);

void TestIsSortedDispatchImplicit()
{
  thrust::device_vector<int> vec(1);

  thrust::is_sorted(thrust::retag<my_system>(vec.begin()),
                    thrust::retag<my_system>(vec.end()));

  ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestIsSortedDispatchImplicit);

