#include <unittest/unittest.h>
#include <thrust/functional.h>
#include <thrust/transform.h>

#include <thrust/detail/allocator/allocator_traits.h>

static const size_t num_samples = 10000;

template<typename Vector, typename U> struct rebind_vector;

template<typename T, typename U, typename Allocator>
  struct rebind_vector<thrust::host_vector<T, Allocator>, U>
{
    typedef typename thrust::detail::allocator_traits<Allocator> alloc_traits;
    typedef typename alloc_traits::template rebind_alloc<U> new_alloc;
    typedef thrust::host_vector<U, new_alloc> type;
};

template<typename T, typename U, typename Allocator>
  struct rebind_vector<thrust::device_vector<T, Allocator>, U>
{
  typedef thrust::device_vector<U,
    typename Allocator::template rebind<U>::other> type;
};

#define BINARY_FUNCTIONAL_PLACEHOLDERS_TEST(name, reference_operator, functor) \
template<typename Vector> \
  void TestFunctionalPlaceholdersBinary##name(void) \
{ \
  typedef typename Vector::value_type T; \
  typedef typename rebind_vector<Vector,bool>::type bool_vector; \
  Vector lhs = unittest::random_samples<T>(num_samples); \
  Vector rhs = unittest::random_samples<T>(num_samples); \
\
  bool_vector reference(lhs.size()); \
  thrust::transform(lhs.begin(), lhs.end(), rhs.begin(), reference.begin(), functor<T>()); \
\
  using namespace thrust::placeholders; \
  bool_vector result(lhs.size()); \
  thrust::transform(lhs.begin(), lhs.end(), rhs.begin(), result.begin(), _1 reference_operator _2); \
\
  ASSERT_EQUAL(reference, result); \
} \
DECLARE_VECTOR_UNITTEST(TestFunctionalPlaceholdersBinary##name);

BINARY_FUNCTIONAL_PLACEHOLDERS_TEST(EqualTo,      ==, thrust::equal_to);
BINARY_FUNCTIONAL_PLACEHOLDERS_TEST(NotEqualTo,   !=, thrust::not_equal_to);
BINARY_FUNCTIONAL_PLACEHOLDERS_TEST(Greater,       >, thrust::greater);
BINARY_FUNCTIONAL_PLACEHOLDERS_TEST(Less,          <, thrust::less);
BINARY_FUNCTIONAL_PLACEHOLDERS_TEST(GreaterEqual, >=, thrust::greater_equal);
BINARY_FUNCTIONAL_PLACEHOLDERS_TEST(LessEqual,    <=, thrust::less_equal);

