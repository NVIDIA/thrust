#include <unittest/unittest.h>
#include <thrust/functional.h>
#include <thrust/transform.h>

#include <thrust/detail/allocator/allocator_traits.h>

static const size_t num_samples = 10000;

template<typename Vector, typename U> struct rebind_vector;

template<typename T, typename U, typename Allocator>
  struct rebind_vector<thrust::host_vector<T, Allocator>, U>
{
  typedef typename thrust::detail::allocator_traits<Allocator> alloc_traits;
  typedef typename alloc_traits::template rebind_alloc<U> new_alloc;
  typedef thrust::host_vector<U, new_alloc> type;
};

template<typename T, typename U, typename Allocator>
  struct rebind_vector<thrust::device_vector<T, Allocator>, U>
{
  typedef thrust::device_vector<U,
    typename Allocator::template rebind<U>::other> type;
};

#define BINARY_FUNCTIONAL_PLACEHOLDERS_TEST(name, reference_operator, functor) \
template<typename Vector> \
  void TestFunctionalPlaceholders##name(void) \
{ \
  typedef typename Vector::value_type T; \
  typedef typename rebind_vector<Vector,bool>::type bool_vector; \
  Vector lhs = unittest::random_samples<T>(num_samples); \
  Vector rhs = unittest::random_samples<T>(num_samples); \
\
  bool_vector reference(lhs.size()); \
  thrust::transform(lhs.begin(), lhs.end(), rhs.begin(), reference.begin(), functor<T>()); \
\
  using namespace thrust::placeholders; \
  bool_vector result(lhs.size()); \
  thrust::transform(lhs.begin(), lhs.end(), rhs.begin(), result.begin(), _1 reference_operator _2); \
\
  ASSERT_EQUAL(reference, result); \
} \
DECLARE_VECTOR_UNITTEST(TestFunctionalPlaceholders##name);

BINARY_FUNCTIONAL_PLACEHOLDERS_TEST(LogicalAnd, &&, thrust::logical_and);
BINARY_FUNCTIONAL_PLACEHOLDERS_TEST(LogicalOr,  ||, thrust::logical_or);

template<typename Vector>
  void TestFunctionalPlaceholdersLogicalNot(void)
{
  typedef typename Vector::value_type T;
  typedef typename rebind_vector<Vector,bool>::type bool_vector;
  Vector input = unittest::random_samples<T>(num_samples);

  if(input.size() > 0)
  {
    // produce at least one true in the output
    input[0] = T(0);
  } // end if

  bool_vector reference(input.size());
  thrust::transform(input.begin(), input.end(), reference.begin(), thrust::logical_not<T>());

  using namespace thrust::placeholders;
  bool_vector result(input.size());
  thrust::transform(input.begin(), input.end(), result.begin(), !_1);

  ASSERT_EQUAL(reference, result);
}
DECLARE_INTEGRAL_VECTOR_UNITTEST(TestFunctionalPlaceholdersLogicalNot);

