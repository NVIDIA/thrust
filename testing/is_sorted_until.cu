#include <unittest/unittest.h>
#include <thrust/sort.h>

template<typename Vector>
void TestIsSortedUntilSimple(void)
{
    typedef typename Vector::value_type T;
    typedef typename Vector::iterator Iterator;

    Vector v(4);
    v[0] = 0; v[1] = 5; v[2] = 8; v[3] = 0;

    Iterator first = v.begin();

    Iterator last  = v.begin() + 0;
    Iterator ref = last;
    ASSERT_EQUAL_QUIET(ref, thrust::is_sorted_until(first, last));

    last = v.begin() + 1;
    ref = last;
    ASSERT_EQUAL_QUIET(ref, thrust::is_sorted_until(first, last));

    last = v.begin() + 2;
    ref = last;
    ASSERT_EQUAL_QUIET(ref, thrust::is_sorted_until(first, last));

    last = v.begin() + 3;
    ref = v.begin() + 3;
    ASSERT_EQUAL_QUIET(ref, thrust::is_sorted_until(first, last));

    last = v.begin() + 4;
    ref = v.begin() + 3;
    ASSERT_EQUAL_QUIET(ref, thrust::is_sorted_until(first, last));

    last = v.begin() + 3;
    ref = v.begin() + 3;
    ASSERT_EQUAL_QUIET(ref, thrust::is_sorted_until(first, last, thrust::less<T>()));

    last = v.begin() + 4;
    ref = v.begin() + 3;
    ASSERT_EQUAL_QUIET(ref, thrust::is_sorted_until(first, last, thrust::less<T>()));

    last = v.begin() + 1;
    ref = v.begin() + 1;
    ASSERT_EQUAL_QUIET(ref, thrust::is_sorted_until(first, last, thrust::greater<T>()));

    last = v.begin() + 4;
    ref = v.begin() + 1;
    ASSERT_EQUAL_QUIET(ref, thrust::is_sorted_until(first, last, thrust::greater<T>()));

    first = v.begin() + 2;
    last = v.begin() + 4;
    ref = v.begin() + 4;
    ASSERT_EQUAL_QUIET(ref, thrust::is_sorted_until(first, last, thrust::greater<T>()));
}
DECLARE_VECTOR_UNITTEST(TestIsSortedUntilSimple);

template<typename Vector>
void TestIsSortedUntilRepeatedElements(void)
{
  Vector v(10);

  v[0] = 0;
  v[1] = 1;
  v[2] = 1;
  v[3] = 2;
  v[4] = 3;
  v[5] = 4;
  v[6] = 5;
  v[7] = 5;
  v[8] = 5;
  v[9] = 6;

  ASSERT_EQUAL_QUIET(v.end(), thrust::is_sorted_until(v.begin(), v.end()));
}
DECLARE_VECTOR_UNITTEST(TestIsSortedUntilRepeatedElements);

template <class Vector>
void TestIsSortedUntil(void)
{
    typedef typename Vector::value_type T;

    const size_t n = (1 << 16) + 13;

    Vector v = unittest::random_integers<T>(n);

    v[0] = 1;
    v[1] = 0;

    ASSERT_EQUAL_QUIET(v.begin() + 1, thrust::is_sorted_until(v.begin(), v.end()));

    thrust::sort(v.begin(), v.end());

    ASSERT_EQUAL_QUIET(v.end(), thrust::is_sorted_until(v.begin(), v.end()));
}
DECLARE_VECTOR_UNITTEST(TestIsSortedUntil);

struct my_tag : thrust::device_system_tag {};

template<typename ForwardIterator>
ForwardIterator is_sorted_until(my_tag, ForwardIterator first, ForwardIterator)
{
    *first = 13;
    return first;
}

void TestIsSortedUntil()
{
    thrust::device_vector<int> vec(1);

    thrust::is_sorted_until(thrust::retag<my_tag>(vec.begin()),
                            thrust::retag<my_tag>(vec.end()));

    ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestIsSortedUntil);

