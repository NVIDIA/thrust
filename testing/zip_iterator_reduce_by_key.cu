#include <unittest/unittest.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/reduce.h>

#if THRUST_DEVICE_SYSTEM == THRUST_DEVICE_SYSTEM_CUDA
#include <backend/cuda/testframework.h>
#endif

using namespace unittest;

template<typename Tuple>
struct TuplePlus
{
  __host__ __device__
  Tuple operator()(Tuple x, Tuple y) const
  {
    using namespace thrust;
    return make_tuple(get<0>(x) + get<0>(y),
                      get<1>(x) + get<1>(y));
  }
}; // end TuplePlus


template <typename T>
struct TestZipIteratorReduceByKey
{
  void operator()(const size_t n)
  {
    using namespace thrust;

    host_vector<T> h_data0 = unittest::random_integers<bool>(n);
    host_vector<T> h_data1 = unittest::random_integers<T>(n);
    host_vector<T> h_data2 = unittest::random_integers<T>(n);

    device_vector<T> d_data0 = h_data0;
    device_vector<T> d_data1 = h_data1;
    device_vector<T> d_data2 = h_data2;

    typedef tuple<T,T> Tuple;

    // integer key, tuple value
    {
      host_vector<T> h_data3(n,0);
      host_vector<T> h_data4(n,0);
      host_vector<T> h_data5(n,0);
      device_vector<T> d_data3(n,0);
      device_vector<T> d_data4(n,0);
      device_vector<T> d_data5(n,0);

      // run on host
      reduce_by_key
          ( h_data0.begin(), h_data0.end(),
            make_zip_iterator(make_tuple(h_data1.begin(), h_data2.begin())),
            h_data3.begin(),
            make_zip_iterator(make_tuple(h_data4.begin(), h_data5.begin())),
            equal_to<T>(),
            TuplePlus<Tuple>());

      // run on device
      reduce_by_key
          ( d_data0.begin(), d_data0.end(),
            make_zip_iterator(make_tuple(d_data1.begin(), d_data2.begin())),
            d_data3.begin(),
            make_zip_iterator(make_tuple(d_data4.begin(), d_data5.begin())),
            equal_to<T>(),
            TuplePlus<Tuple>());

      ASSERT_EQUAL(h_data3, d_data3);
      ASSERT_EQUAL(h_data4, d_data4);
      ASSERT_EQUAL(h_data5, d_data5);
    }
    
    // The tests below get miscompiled on Tesla hw for 8b types

#if THRUST_DEVICE_SYSTEM == THRUST_DEVICE_SYSTEM_CUDA
    if(const CUDATestDriver *driver = dynamic_cast<const CUDATestDriver*>(&UnitTestDriver::s_driver()))
    {
      if(typeid(T) == typeid(unittest::uint8_t) && driver->current_device_architecture() < 200)
      {
        KNOWN_FAILURE;
      } // end if
    } // end if
#endif

    // tuple key, tuple value
    {
      host_vector<T> h_data3(n,0);
      host_vector<T> h_data4(n,0);
      host_vector<T> h_data5(n,0);
      host_vector<T> h_data6(n,0);
      device_vector<T> d_data3(n,0);
      device_vector<T> d_data4(n,0);
      device_vector<T> d_data5(n,0);
      device_vector<T> d_data6(n,0);

      // run on host
      reduce_by_key
          ( make_zip_iterator(make_tuple(h_data0.begin(), h_data0.begin())),
            make_zip_iterator(make_tuple(h_data0.end(),   h_data0.end())),
            make_zip_iterator(make_tuple(h_data1.begin(), h_data2.begin())),
            make_zip_iterator(make_tuple(h_data3.begin(), h_data4.begin())),
            make_zip_iterator(make_tuple(h_data5.begin(), h_data6.begin())),
            equal_to<Tuple>(),
            TuplePlus<Tuple>());

      // run on device
      reduce_by_key
          ( make_zip_iterator(make_tuple(d_data0.begin(), d_data0.begin())),
            make_zip_iterator(make_tuple(d_data0.end(),   d_data0.end())),
            make_zip_iterator(make_tuple(d_data1.begin(), d_data2.begin())),
            make_zip_iterator(make_tuple(d_data3.begin(), d_data4.begin())),
            make_zip_iterator(make_tuple(d_data5.begin(), d_data6.begin())),
            equal_to<Tuple>(),
            TuplePlus<Tuple>());

      ASSERT_EQUAL(h_data3, d_data3);
      ASSERT_EQUAL(h_data4, d_data4);
      ASSERT_EQUAL(h_data5, d_data5);
      ASSERT_EQUAL(h_data6, d_data6);
    }
  }
};
VariableUnitTest<TestZipIteratorReduceByKey, UnsignedIntegralTypes> TestZipIteratorReduceByKeyInstance;

