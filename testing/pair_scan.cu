#include <unittest/unittest.h>
#include <thrust/pair.h>
#include <thrust/transform.h>
#include <thrust/scan.h>

#if THRUST_DEVICE_SYSTEM == THRUST_DEVICE_SYSTEM_CUDA
#include <backend/cuda/testframework.h>
#endif

struct make_pair_functor
{
  template<typename T1, typename T2>
  __host__ __device__
    thrust::pair<T1,T2> operator()(const T1 &x, const T2 &y)
  {
    return thrust::make_pair(x,y);
  } // end operator()()
}; // end make_pair_functor


struct add_pairs
{
  template <typename Pair1, typename Pair2>
  __host__ __device__
    Pair1 operator()(const Pair1 &x, const Pair2 &y)
  {
    return thrust::make_pair(x.first + y.first, x.second + y.second);
  } // end operator()
}; // end add_pairs


template <typename T>
  struct TestPairScan
{
  void operator()(const size_t n)
  {
    typedef thrust::pair<T,T> P;

    thrust::host_vector<T>   h_p1 = unittest::random_integers<T>(n);
    thrust::host_vector<T>   h_p2 = unittest::random_integers<T>(n);
    thrust::host_vector<P>   h_pairs(n);
    thrust::host_vector<P>   h_output(n);

    // zip up pairs on the host
    thrust::transform(h_p1.begin(), h_p1.end(), h_p2.begin(), h_pairs.begin(), make_pair_functor());

    thrust::device_vector<T> d_p1 = h_p1;
    thrust::device_vector<T> d_p2 = h_p2;
    thrust::device_vector<P> d_pairs = h_pairs;
    thrust::device_vector<P> d_output(n);

    P init = thrust::make_pair(13,13);

    // scan with plus
    thrust::inclusive_scan(h_pairs.begin(), h_pairs.end(), h_output.begin(), add_pairs());
    thrust::inclusive_scan(d_pairs.begin(), d_pairs.end(), d_output.begin(), add_pairs());
    ASSERT_EQUAL_QUIET(h_output, d_output);

    // scan with maximum (thrust issue #69)
    thrust::inclusive_scan(h_pairs.begin(), h_pairs.end(), h_output.begin(), thrust::maximum<P>());
    thrust::inclusive_scan(d_pairs.begin(), d_pairs.end(), d_output.begin(), thrust::maximum<P>());
    ASSERT_EQUAL_QUIET(h_output, d_output);


    // The tests below get miscompiled on Tesla hw for 8b types

#if THRUST_DEVICE_SYSTEM == THRUST_DEVICE_SYSTEM_CUDA
    if(const CUDATestDriver *driver = dynamic_cast<const CUDATestDriver*>(&UnitTestDriver::s_driver()))
    {
      if(sizeof(T) == sizeof(unittest::uint8_t) && driver->current_device_architecture() < 200)
      {
        KNOWN_FAILURE;
      } // end if
    } // end if
#endif

    // scan with plus
    thrust::exclusive_scan(h_pairs.begin(), h_pairs.end(), h_output.begin(), init, add_pairs());
    thrust::exclusive_scan(d_pairs.begin(), d_pairs.end(), d_output.begin(), init, add_pairs());
    ASSERT_EQUAL_QUIET(h_output, d_output);
    
    // scan with maximum (thrust issue #69)
    thrust::exclusive_scan(h_pairs.begin(), h_pairs.end(), h_output.begin(), init, thrust::maximum<P>());
    thrust::exclusive_scan(d_pairs.begin(), d_pairs.end(), d_output.begin(), init, thrust::maximum<P>());
    ASSERT_EQUAL_QUIET(h_output, d_output);
  }
};
VariableUnitTest<TestPairScan, unittest::type_list<unittest::int8_t,unittest::int16_t,unittest::int32_t> > TestPairScanInstance;

