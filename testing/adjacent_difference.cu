#include <unittest/unittest.h>
#include <thrust/adjacent_difference.h>
#include <thrust/iterator/discard_iterator.h>

template <class Vector>
void TestAdjacentDifferenceSimple(void)
{
    typedef typename Vector::value_type T;

    Vector input(3);
    Vector output(3);
    input[0] = 1; input[1] = 4; input[2] = 6;

    typename Vector::iterator result;
    
    result = thrust::adjacent_difference(input.begin(), input.end(), output.begin());

    ASSERT_EQUAL(result - output.begin(), 3);
    ASSERT_EQUAL(output[0], T(1));
    ASSERT_EQUAL(output[1], T(3));
    ASSERT_EQUAL(output[2], T(2));
    
    result = thrust::adjacent_difference(input.begin(), input.end(), output.begin(), thrust::plus<T>());
    
    ASSERT_EQUAL(result - output.begin(), 3);
    ASSERT_EQUAL(output[0], T( 1));
    ASSERT_EQUAL(output[1], T( 5));
    ASSERT_EQUAL(output[2], T(10));
    
    // test in-place operation, result and first are permitted to be the same
    result = thrust::adjacent_difference(input.begin(), input.end(), input.begin());

    ASSERT_EQUAL(result - input.begin(), 3);
    ASSERT_EQUAL(input[0], T(1));
    ASSERT_EQUAL(input[1], T(3));
    ASSERT_EQUAL(input[2], T(2));
}
DECLARE_VECTOR_UNITTEST(TestAdjacentDifferenceSimple);


template <typename T>
void TestAdjacentDifference(const size_t n)
{
    thrust::host_vector<T>   h_input = unittest::random_samples<T>(n);
    thrust::device_vector<T> d_input = h_input;

    thrust::host_vector<T>   h_output(n);
    thrust::device_vector<T> d_output(n);

    typename thrust::host_vector<T>::iterator   h_result;
    typename thrust::device_vector<T>::iterator d_result;

    h_result = thrust::adjacent_difference(h_input.begin(), h_input.end(), h_output.begin());
    d_result = thrust::adjacent_difference(d_input.begin(), d_input.end(), d_output.begin());

    ASSERT_EQUAL(h_result - h_output.begin(), n);
    ASSERT_EQUAL(d_result - d_output.begin(), n);
    ASSERT_EQUAL(h_output, d_output);
    
    h_result = thrust::adjacent_difference(h_input.begin(), h_input.end(), h_output.begin(), thrust::plus<T>());
    d_result = thrust::adjacent_difference(d_input.begin(), d_input.end(), d_output.begin(), thrust::plus<T>());

    ASSERT_EQUAL(h_result - h_output.begin(), n);
    ASSERT_EQUAL(d_result - d_output.begin(), n);
    ASSERT_EQUAL(h_output, d_output);
    
    // in-place operation
    h_result = thrust::adjacent_difference(h_input.begin(), h_input.end(), h_input.begin(), thrust::plus<T>());
    d_result = thrust::adjacent_difference(d_input.begin(), d_input.end(), d_input.begin(), thrust::plus<T>());

    ASSERT_EQUAL(h_result - h_input.begin(), n);
    ASSERT_EQUAL(d_result - d_input.begin(), n);
    ASSERT_EQUAL(h_input, h_output); //computed previously
    ASSERT_EQUAL(d_input, d_output); //computed previously
}
DECLARE_VARIABLE_UNITTEST(TestAdjacentDifference);

template<typename T>
void TestAdjacentDifferenceInPlaceWithRelatedIteratorTypes(const size_t n)
{
    thrust::host_vector<T>   h_input = unittest::random_samples<T>(n);
    thrust::device_vector<T> d_input = h_input;

    thrust::host_vector<T>   h_output(n);
    thrust::device_vector<T> d_output(n);

    typename thrust::host_vector<T>::iterator   h_result;
    typename thrust::device_vector<T>::iterator d_result;

    h_result = thrust::adjacent_difference(h_input.begin(), h_input.end(), h_output.begin(), thrust::plus<T>());
    d_result = thrust::adjacent_difference(d_input.begin(), d_input.end(), d_output.begin(), thrust::plus<T>());
    
    // in-place operation with different iterator types
    h_result = thrust::adjacent_difference(h_input.cbegin(), h_input.cend(), h_input.begin(), thrust::plus<T>());
    d_result = thrust::adjacent_difference(d_input.cbegin(), d_input.cend(), d_input.begin(), thrust::plus<T>());

    ASSERT_EQUAL(h_result - h_input.begin(), n);
    ASSERT_EQUAL(d_result - d_input.begin(), n);
    ASSERT_EQUAL(h_output, h_input); // reference computed previously
    ASSERT_EQUAL(d_output, d_input); // reference computed previously
}
DECLARE_VARIABLE_UNITTEST(TestAdjacentDifferenceInPlaceWithRelatedIteratorTypes);

template <typename T>
void TestAdjacentDifferenceDiscardIterator(const size_t n)
{
    thrust::host_vector<T>   h_input = unittest::random_samples<T>(n);
    thrust::device_vector<T> d_input = h_input;

    thrust::discard_iterator<> h_result =
      thrust::adjacent_difference(h_input.begin(), h_input.end(), thrust::make_discard_iterator());
    thrust::discard_iterator<> d_result =
      thrust::adjacent_difference(d_input.begin(), d_input.end(), thrust::make_discard_iterator());

    thrust::discard_iterator<> reference(n);

    ASSERT_EQUAL_QUIET(reference, h_result);
    ASSERT_EQUAL_QUIET(reference, d_result);
}
DECLARE_VARIABLE_UNITTEST(TestAdjacentDifferenceDiscardIterator);

struct my_tag : thrust::device_system_tag {};

template<typename InputIterator, typename OutputIterator>
OutputIterator adjacent_difference(my_tag, InputIterator, InputIterator, OutputIterator result)
{
    *result = 13;
    return result;
}

void TestAdjacentDifferenceDispatch()
{
    thrust::device_vector<int> d_input(1);

    thrust::adjacent_difference(thrust::retag<my_tag>(d_input.begin()),
                                thrust::retag<my_tag>(d_input.end()),
                                thrust::retag<my_tag>(d_input.begin()));

    ASSERT_EQUAL(13, d_input.front());
}
DECLARE_UNITTEST(TestAdjacentDifferenceDispatch);

